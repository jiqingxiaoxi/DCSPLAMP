#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<sys/stat.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ char str2int(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;
                case 'T':
                        return 3;
        }
        return 4;
}

__device__ char str2int_rev(char c)
{
        switch (c)
        {
                case 'T':
                        return 0;
                case 'G':
                        return 1;
                case 'C':
                        return 2;                 
                case 'A':               
                        return 3;         
        }
        return 4;
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ double Ss(int i,int j,int k,int Initint[],char numSeq1[],char numSeq2[],double parameter[])
{
	if(k==2)
	{
		if(i>=j)
			return -1.0;
		if(i==Initint[0]||j==Initint[1]+1)
			return -1.0;

		if(i>Initint[0])
			i-=Initint[0];
		if(j>Initint[1])
			j-=Initint[1];
		return parameter[numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]];
	}
	else
		return parameter[numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
}

__device__ double Hs(int i,int j,int k,int Initint[],char numSeq1[],char numSeq2[],double parameter[])
{
	if(k==2)
	{
		if(i>= j)
			return 1.0*INFINITY;
		if(i==Initint[0]||j==Initint[1]+1)
			return 1.0*INFINITY;

		if(i>Initint[0])
			i-=Initint[0];
		if(j>Initint[1])
			j-=Initint[1];
		if(fabs(parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]])<999999999)
			return parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]];
		else
			return 1.0*INFINITY;
	}
	else
		return parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
}

__device__ int equal(double a,double b)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
		return 0;
	return fabs(a-b)<1e-5;
}

__device__ void initMatrix(int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[])
{
	int i,j;

	for(i=1;i<=Initint[0];++i)
	{
		for(j=1;j<=Initint[1];++j)
		{
			if(numSeq1[i]+numSeq2[j]!=3)
			{
				d_DPT[id*1250+(i-1)*Initint[2]+j-1]=1.0*INFINITY;
				d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=-1.0;
			}
			else
			{
				d_DPT[id*1250+(i-1)*Initint[2]+j-1]=0.0;
				d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,double *EntropyEnthalpy,double Initdouble[],int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[],double parameter[])
{
	double S1,H1,T1,S2,H2,T2;

	if(numSeq1[i]+numSeq2[j]!=3)
	{
		d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=-1.0;
		d_DPT[id*1250+(i-1)*Initint[2]+j-1]=1.0*INFINITY;
		return;
	}

	S1=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[4430+numSeq2[j]*125+numSeq2[j-1]*25+numSeq1[i]*5+numSeq1[i-1]];
	H1=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5055+numSeq2[j]*125+numSeq2[j-1]*25+numSeq1[i]*5+numSeq1[i-1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]])<999999999&&fabs(parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]]+parameter[2750+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]]+parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2750+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	S2=parameter[5680+numSeq1[i]*5+numSeq2[j]];
	H2=parameter[5705+numSeq1[i]*5+numSeq2[j]];
	T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			EntropyEnthalpy[0]=S2;
			EntropyEnthalpy[1]=H2;
		}
		else
		{
			EntropyEnthalpy[0]=S1;
			EntropyEnthalpy[1]=H1;
		}
	}
	else
	{
		EntropyEnthalpy[0]=S2;
		EntropyEnthalpy[1]=H2;
	}
	return;
}

__device__ void maxTM(int i,int j,double Initdouble[],int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[],double parameter[])
{
	double T0,T1,S0,S1,H0,H1;

	S0=d_DPT[id*1250+625+(i-1)*Initint[2]+j-1];
	H0=d_DPT[id*1250+(i-1)*Initint[2]+j-1];
	T0=(H0+Initdouble[0])/(S0+Initdouble[1]+Initdouble[2]); // at current position 
	if(fabs(d_DPT[id*1250+(i-2)*Initint[2]+j-2])<999999999&&fabs(Hs(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter))<999999999)
	{
		S1=(d_DPT[id*1250+625+(i-2)*Initint[2]+j-2]+Ss(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter));
		H1=(d_DPT[id*1250+(i-2)*Initint[2]+j-2]+Hs(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter));
	}
	else
	{
		S1=-1.0;
		H1=1.0*INFINITY;
	}
	T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);

	if(S1<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S1=-3224.0;
		H1=0.0;
	}
	if(S0<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S0=-3224.0;
		H0=0.0;
	}
	if((T1>T0)||(S0>0&&H0>0)) // T1 on suurem 
	{
		d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=S1;
		d_DPT[id*1250+(i-1)*Initint[2]+j-1]=H1;
	}
	else if(T0>=T1)
	{
		d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=S0;
		d_DPT[id*1250+(i-1)*Initint[2]+j-1]=H0;
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,double* EntropyEnthalpy,int traceback,double Initdouble[],int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[],double parameter[])
{
	int loopSize1,loopSize2,loopSize,N,N_loop;
	double T1,T2,S,H;

	S=-3224.0;
	H=0;
	loopSize1=ii-i-1;
	loopSize2=jj-j-1;
	if(ii<jj)
	{
		N=i;
		N_loop=N;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
	}
	else
	{
		N=j;
		N_loop=2*jj;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
		N_loop=(N_loop/2)-1;
	}

	loopSize=loopSize1+loopSize2-1;
	if((loopSize1==0&&loopSize2>0)||(loopSize2==0&&loopSize1>0))// only bulges have to be considered
	{
		if(loopSize2==1||loopSize1==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((loopSize2==1&&loopSize1==0)||(loopSize2==0&&loopSize1==1))
			{
				H=parameter[3150+loopSize]+parameter[625+numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
				S=parameter[3060+loopSize]+parameter[numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
			}
			H+=d_DPT[id*1250+(i-1)*Initint[2]+j-1];
			S+=d_DPT[id*1250+625+(i-1)*Initint[2]+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}

			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(d_DPT[id*1250+(ii-1)*Initint[2]+jj-1]+Initdouble[0])/((d_DPT[id*1250+625+(ii-1)*Initint[2]+jj-1])+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			H=parameter[3150+loopSize]+parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5705+numSeq1[ii]*5+numSeq2[jj]];
			H+=d_DPT[id*1250+(i-1)*Initint[2]+j-1];

			S=parameter[3060+loopSize]+parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[5680+numSeq1[ii]*5+numSeq2[jj]];
			S+=d_DPT[id*1250+625+(i-1)*Initint[2]+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(d_DPT[id*1250+(ii-1)*Initint[2]+jj-1]+Initdouble[0])/(d_DPT[id*1250+625+(ii-1)*Initint[2]+jj-1]+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
	}
	else if(loopSize1==1&&loopSize2==1)
	{
		S=parameter[1250+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[1250+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		S+=d_DPT[id*1250+625+(i-1)*Initint[2]+j-1];

		H=parameter[1875+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[1875+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		H+=d_DPT[id*1250+(i-1)*Initint[2]+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(d_DPT[id*1250+(ii-1)*Initint[2]+jj-1]+Initdouble[0])/(d_DPT[id*1250+625+(ii-1)*Initint[2]+jj-1]+Initdouble[1]+Initdouble[2]);
		if((T1-T2>=0.000001)||traceback==1)
		{
			if((T1>T2)||(traceback&&T1>=T2))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		return;
	}
	else // only internal loops
	{
		H=parameter[3120+loopSize]+parameter[3805+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[3805+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		H+=d_DPT[id*1250+(i-1)*Initint[2]+j-1];

		S=parameter[3030+loopSize]+parameter[3180+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[3180+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]]+(-300/310.15*abs(loopSize1-loopSize2));
		S+=d_DPT[id*1250+625+(i-1)*Initint[2]+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(d_DPT[id*1250+(ii-1)*Initint[2]+jj-1]+Initdouble[0])/((d_DPT[id*1250+625+(ii-1)*Initint[2]+jj-1])+Initdouble[1]+Initdouble[2]);
		if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
		{
			EntropyEnthalpy[0]=S;
			EntropyEnthalpy[1]=H;
		}
	}
	return;
}

__device__ void fillMatrix(double Initdouble[],int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[],double *parameter)
{
	int d,i,j,ii,jj;
	double SH[2];

	for(i=1;i<=Initint[0];++i)
	{
		for(j=1;j<=Initint[1];++j)
		{
			if(fabs(d_DPT[id*1250+(i-1)*Initint[2]+j-1])<999999999)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				LSH(i,j,SH,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);

				if(fabs(SH[1])<999999999)
				{
					d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=SH[0];
					d_DPT[id*1250+(i-1)*Initint[2]+j-1]=SH[1];
				}
				if(i>1&&j>1)
				{
					maxTM(i,j,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);
					for(d=3;d<=32;d++)
					{
						ii=i-1;
						jj=-ii-d+(j+i);
						if(jj<1)
						{
							ii-=abs(jj-1);
							jj=1;
						}
						for(;ii>0&&jj<j;--ii,++jj)
						{
							if(fabs(d_DPT[id*1250+(ii-1)*Initint[2]+jj-1])<999999999)
							{
								SH[0]=-1.0;
								SH[1]=1.0*INFINITY;
								calc_bulge_internal(ii,jj,i,j,SH,0,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);

								if(SH[0]<-2500.0)
								{
									SH[0] =-3224.0;
									SH[1] = 0.0;
								}
								if(fabs(SH[1])<999999999)
								{
									d_DPT[id*1250+(i-1)*Initint[2]+j-1]=SH[1];
									d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]=SH[0];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double EntropyEnthalpy[],double Initdouble[],char numSeq1[],char numSeq2[],double *parameter)
{
	double S1,S2,H1,H2,T1,T2;

	if(numSeq1[i]+numSeq2[j]!=3)
	{
		EntropyEnthalpy[0]=-1.0;
		EntropyEnthalpy[1]=1.0*INFINITY;
		return;
	}
	S1=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[4430+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
	H1=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5055+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
	if(fabs(parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]])<999999999&&fabs(parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]]+parameter[2750+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]]+parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2750+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	S2=parameter[5680+numSeq1[i]*5+numSeq2[j]];
	H2=parameter[5705+numSeq1[i]*5+numSeq2[j]];
	T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			EntropyEnthalpy[0]=S2;
			EntropyEnthalpy[1]=H2;
		}
		else
		{
			EntropyEnthalpy[0]=S1;
			EntropyEnthalpy[1]=H1;
		}
	}
	else
	{
		EntropyEnthalpy[0]=S2;
		EntropyEnthalpy[1]=H2;
	}
	return;
}

__device__ void traceback(int i,int j,int *d_ps,double Initdouble[],int Initint[],double *d_DPT,int id,char numSeq1[],char numSeq2[],double *parameter)
{
	int d,ii,jj,done;
	double SH[2];

	d_ps[id*50+i-1]=j;
	d_ps[id*50+25+j-1]=i;
	while(1)
	{
		SH[0]=-1.0;
		SH[1]=1.0*INFINITY;
		LSH(i,j,SH,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);
		if(equal(d_DPT[id*1250+625+(i-1)*Initint[2]+j-1],SH[0])&&equal(d_DPT[id*1250+(i-1)*Initint[2]+j-1],SH[1]))
			break;

		done = 0;
		if(i>1&&j>1&&equal(d_DPT[id*1250+625+(i-1)*Initint[2]+j-1],Ss(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter)+d_DPT[id*1250+625+(i-2)*Initint[2]+j-2]))
		{
			i=i-1;
			j=j-1;
			d_ps[id*50+i-1]=j;
			d_ps[id*50+25+j-1]=i;
			done=1;
		}
		for(d=3;!done&&d<=32;++d)
		{
			ii=i-1;
			jj=-ii-d+(j+i);
			if(jj<1)
			{
				ii-=abs(jj-1);
				jj=1;
			}
			for(;!done&&ii>0&&jj<j;--ii,++jj)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				calc_bulge_internal(ii,jj,i,j,SH,1,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);
				if(equal(d_DPT[id*1250+625+(i-1)*Initint[2]+j-1],SH[0])&&equal(d_DPT[1250*id+(i-1)*Initint[2]+j-1],SH[1]))
				{
					i=ii;
					j=jj;
					d_ps[id*50+i-1]=j;
					d_ps[id*50+25+j-1]=i;
					done=1;
					break;
				}
			}
		}
	}
}

__device__ double drawDimer(int *d_ps,int id,double H,double S,double Initdouble[],int Initint[])
{
        int i,N;

        if(fabs(Initdouble[3])>999999999)
                return (double)0.0;
        else
        {
                N=0;
                for(i=0;i<Initint[0];i++)
                {
                        if(d_ps[id*50+i]>0)
                                ++N;
                }
                for(i=0;i<Initint[1];i++)
                {
                        if(d_ps[id*50+25+i]>0)
                                ++N;
                }
                N=(N/2)-1;
                return (double)(H/(S+(N*-0.51986)+Initdouble[2])-273.15);
        }
}

__device__ int symmetry_thermo(char *d_seq,int start,int length )
{
	int i = 0;
	if(length%2==1)
		return 0;

	while(i<length/2)
	{
		if((d_seq[i+start]=='A'&&d_seq[start+length-1-i]!='T')||(d_seq[i+start]=='T'&&d_seq[start+length-1-i]!='A')||(d_seq[start+length-1-i]=='A'&&d_seq[i+start]!='T')||(d_seq[start+length-1-i]=='T'&&d_seq[i+start]!='A'))
			return 0;
		if((d_seq[i+start]=='C'&&d_seq[start+length-1-i]!='G')||(d_seq[i+start]=='G'&&d_seq[start+length-1-i]!='C')||(d_seq[start+length-1-i]=='C'&&d_seq[i+start]!='G')||(d_seq[start+length-1-i]=='G'&&d_seq[i+start]!='C'))
			return 0;
		i++;
	}
	return 1;
}

__device__ double thal(char *d_seq,int *d_primer,int one_turn,int two_turn,int one_flag,int two_flag,int type,double *parameter,double *d_DPT,int id,int *d_ps)
{
	double SH[2],Initdouble[4];//0 is dplx_init_H, 1 is dplx_init_S, 2 is RC, 3 is SHleft
	int Initint[5]; //0 is len1, 1 is len2, 2 is len3, 3 is bestI, 4 is bestJ
	int i, j;
	double T1,result_TH;
	char numSeq1[27],numSeq2[27];

/*** INIT values for unimolecular and bimolecular structures ***/
	Initdouble[0]= 200;
	Initdouble[1]= -5.7;
	if(symmetry_thermo(d_seq,d_primer[4*one_turn],d_primer[4*one_turn+1])&&symmetry_thermo(d_seq,d_primer[4*two_turn],d_primer[4*two_turn+1]))
		Initdouble[2]=1.9872* log(38/1000000000.0);
	else
		Initdouble[2]=1.9872* log(38/4000000000.0);
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		Initint[0]=d_primer[4*one_turn+1];
		Initint[1]=d_primer[4*two_turn+1];
		if(one_flag==0) //plus
		{
	 		for(i=1;i<=Initint[0];++i)
				numSeq1[i]=str2int(d_seq[d_primer[4*one_turn]+i-1]);
		}
		else
		{
			for(i=1;i<=Initint[0];++i)
				numSeq1[i]=str2int_rev(d_seq[d_primer[4*one_turn]+d_primer[4*one_turn+1]-i]);
		}

		if(two_flag==0)
		{
			for(i=1;i<=Initint[1];++i)
				numSeq2[i]=str2int(d_seq[d_primer[4*two_turn]+d_primer[4*two_turn+1]-i]);
		}
		else
		{
			for(i=1;i<=Initint[1];++i)
				numSeq2[i]=str2int_rev(d_seq[d_primer[4*two_turn]+i-1]);
		}
	}
	else if(type==3)
	{
		Initint[0]=d_primer[4*two_turn+1];
		Initint[1]=d_primer[4*one_turn+1];
		if(two_flag==0)
		{
			for(i=1;i<=Initint[0];++i)
				numSeq1[i]=str2int(d_seq[d_primer[4*two_turn]+i-1]);
		}
		else
		{
			for(i=1;i<=Initint[0];++i)
				numSeq1[i]=str2int_rev(d_seq[d_primer[4*two_turn]+d_primer[4*two_turn+1]-i]);
		}
		if(one_flag==0)
		{
			for(i=1;i<=Initint[1];++i)
				numSeq2[i]=str2int(d_seq[d_primer[4*one_turn]+d_primer[4*one_turn+1]-i]);
		}
		else
		{
			for(i=1;i<=Initint[1];++i)
				numSeq2[i]=str2int_rev(d_seq[d_primer[4*one_turn]+i-1]);
		}
	}
	numSeq1[0]=numSeq1[Initint[0]+1]=numSeq2[0]=numSeq2[Initint[1]+1]=4; /* mark as N-s */

	result_TH=0;
	Initint[2]=Initint[1];
	initMatrix(Initint,d_DPT,id,numSeq1,numSeq2);
	fillMatrix(Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);

	Initdouble[3]=-1.0*INFINITY;
/* calculate terminal basepairs */
	Initint[3]=Initint[4]=0;
	if(type==1)
		for (i=1;i<=Initint[0];i++)
		{
			for (j=1;j<=Initint[1];j++)
			{
				RSH(i,j,SH,Initdouble,numSeq1,numSeq2,parameter);
				SH[0]=SH[0]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
				SH[1]=SH[1]+0.000001;
				T1=((d_DPT[id*1250+(i-1)*Initint[2]+j-1]+ SH[1] +Initdouble[0]) / ((d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]) + SH[0] +Initdouble[1] + Initdouble[2])) -273.15;
				if(T1>Initdouble[3]&&((d_DPT[id*1250+625+(i-1)*Initint[2]+j-1]+SH[0])<0&&(SH[1]+d_DPT[id*1250+(i-1)*Initint[2]+j-1])<0))
				{
					Initdouble[3]=T1;
					Initint[3]=i;
					Initint[4]=j;
				}
			}
		}
	if(type==2||type==3)
	{
	 //THAL_END1
		Initint[4]=0;
		Initint[3]=Initint[0];
		i=Initint[0];
		Initdouble[3]=-1.0*INFINITY;
		for (j=1;j<=Initint[1];++j)
		{
			RSH(i,j,SH,Initdouble,numSeq1,numSeq2,parameter);
			SH[0]=SH[0]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
			SH[1]=SH[1]+0.000001;
			T1=((d_DPT[id*1250+(i-1)*Initint[2]+j-1]+SH[1]+Initdouble[0])/((d_DPT[id*1250+625+(i-1)*Initint[2]+j-1])+SH[0]+Initdouble[1]+Initdouble[2]))-273.15;
			if (T1>Initdouble[3]&&((SH[0]+d_DPT[id*1250+625+(i-1)*Initint[2]+j-1])<0&&(SH[1]+d_DPT[id*1250+(i-1)*Initint[2]+j-1])<0))
			{
				Initdouble[3]=T1;
				Initint[4]=j;
			}
		}
	}
	if(fabs(Initdouble[3])>999999999)
		Initint[3]=Initint[4]=1;
	RSH(Initint[3],Initint[4],SH,Initdouble,numSeq1,numSeq2,parameter);
 // tracebacking 
	for (i=0;i<Initint[0];++i)
		d_ps[id*50+i]=0;
	for (j=0;j<Initint[1];++j)
		d_ps[id*50+25+j] = 0;
	if(fabs(d_DPT[id*1250+(Initint[3]-1)*Initint[2]+Initint[4]-1])<999999999)
	{
		traceback(Initint[3],Initint[4],d_ps,Initdouble,Initint,d_DPT,id,numSeq1,numSeq2,parameter);
		result_TH=drawDimer(d_ps,id,(d_DPT[id*1250+(Initint[3]-1)*Initint[2]+Initint[4]-1]+SH[1]+Initdouble[0]),(d_DPT[id*1250+625+(Initint[3]-1)*Initint[2]+Initint[4]-1]+SH[0]+Initdouble[1]),Initdouble,Initint);
		result_TH=(int)(100*result_TH+0.5)/100.0;
	}
        return result_TH;
}

struct Node
{
	int pos;
	int gi;
	int plus;  //as a flag, 1 is OK, 0 is no
	int minus; //as a flag
	struct Node *next;
};

struct Primer
{
	int pos;
	int len;
	int plus;
	int minus;
	int total_common;
	int total_special;
	int total; //common number
	struct Primer *next;
	struct Node *common;
	struct Node *special;
};

struct INFO
{
        char name[301];
        int turn;
        struct INFO *next;
};

int check_add(int F3_pos,int *par,int have)
{
        int i,dis;

        for(i=0;i<have;i++)
        {
		if(par[i]==-1)
			return 1;
                dis=par[i]-F3_pos;
                if(abs(dis)<300)              
                        return 0;
        }
        return 1;        
}

void generate_primer(char *seq,char primer[],int start,int length,int flag)
{
        int i;
        if(flag==0)
        {
                for(i=0;i<length;i++)
                	primer[i]=seq[start+i];
        }
        else
        {
                for(i=0;i<length;i++)
                {
                        if(seq[start+length-1-i]=='A')
                                primer[i]='T';
                        else if(seq[start+length-1-i]=='T')
                                primer[i]='A';
                        else if(seq[start+length-1-i]=='C')
                                primer[i]='G';
                        else
                                primer[i]='C';
                }
        }
	primer[length]='\0';
}

__device__ int check_structure(char *d_seq,int *d_primer,int turn[],int *d_int,double *parameter,double *d_TH,int id,double *d_DPT,int *d_ps)
{
	double TH;
	int i,j;

	for(i=0;i<5;i++)
	{
		for(j=i+1;j<6;j++)
		{
		if(i!=2||j!=3)
			continue;
			TH=thal(d_seq,d_primer,turn[i],turn[j],d_int[11+i],d_int[11+j],1,parameter,d_DPT,id,d_ps);
			if(TH>44+5*d_int[9])
                                return 0;
		d_TH[id*2]=TH;
			TH=thal(d_seq,d_primer,turn[i],turn[j],d_int[11+i],d_int[11+j],2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
		d_TH[id*2+1]=TH;
			TH=thal(d_seq,d_primer,turn[i],turn[j],d_int[11+i],d_int[11+j],3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
			d_TH[id*2+1]=TH;
			TH=thal(d_seq,d_primer,turn[j],turn[i],(1-d_int[11+j]),(1-d_int[11+i]),2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
                        d_TH[id*2+1]=TH;
                        TH=thal(d_seq,d_primer,turn[j],turn[i],(1-d_int[11+j]),(1-d_int[11+i]),3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
                        d_TH[id*2+1]=TH;
		}
	}
	return 1;
}

void how_many(struct Primer *head,int common)
{
        struct Primer *p_primer;
        struct Node *p_node;
        int i,num,*list;

	list=(int *)malloc(common*sizeof(int));
        p_primer=head;
        while(p_primer)
        {
                p_node=p_primer->common;
		for(i=0;i<common;i++)
                {
                        list[i]=0;
                }
                i=0;
                while(p_node)
                {
                        i++;
			list[p_node->gi]=1;
                        p_node=p_node->next;
                }
		p_primer->total_common=i;

        //special
                p_node=p_primer->special;
                i=0;
                while(p_node)       
                {
                        i++;
                        p_node=p_node->next;
                }
		p_primer->total_special=i;

		num=0;
                for(i=0;i<common;i++)
                {
                        num=num+list[i];
                }
		p_primer->total=num;
                p_primer=p_primer->next;
        }
	free(list);
}

//get the file size
int file_size2(char* filename)
{
        struct stat statbuf;
        stat(filename,&statbuf);
        int size=statbuf.st_size;
        return size;
}

////function read primer informatin and align information 
struct Primer *read_par(char *path,int common_flag,int special_flag)
{
        char *in;
        int pos,len,gi,position,plus,minus,size,i,flag;
        struct Primer *new_primer,*p_primer,*head;
        struct Node *new_node,*p_node;
        FILE *fp;

///read the  primer file
        if(access(path,0)==-1)
        {
                printf("Error! Don't have the %s file!\n",path);
                exit(1);
        }
        fp=fopen(path,"r");
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",path);
                exit(1);
        }
        
        size=sizeof(struct Primer);
        i=0;
        while(fscanf(fp,"pos:%d\tlength:%d\t+:%d\t-:%d\n",&pos,&len,&plus,&minus)!=EOF)
        {
                new_primer=(struct Primer *)malloc(size);
                new_primer->pos=pos;
                new_primer->len=len;
                new_primer->total=1;
		new_primer->total_common=0;
		new_primer->total_special=0;
                new_primer->plus=plus;
                new_primer->minus=minus;
                new_primer->next=NULL;
                new_primer->common=NULL;
                new_primer->special=NULL;

                if(i==0)
                {
                        head=new_primer;
                        p_primer=new_primer;
                        i++;
                }
                else
                {
                        p_primer->next=new_primer;
                        p_primer=new_primer;
                }
        }
        fclose(fp);
        if(i==0)
        {
                printf("Sorry! Don't have any candidate single primers in %s!\n",path);
                exit(1);
        }

//parameter of common
        if(common_flag==1)
        {
                i=strlen(path);
                in=(char *)malloc(i+20);
                memset(in,'\0',i+20);
                strcpy(in,path);
                strcat(in,"-common.txt"); //suffix of parameter
                if(access(in,0)==-1)
                {
                        printf("Error! Don't have the %s file!\n",in);
                        exit(1);
                }

                fp=fopen(in,"r");
                if(fp==NULL)
                {
                        printf("Error: can't open the %s file!\n",in);
                        exit(1);
                }

                p_primer=head;
                size=sizeof(struct Node);
                while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
                {
                        new_node=(struct Node *)malloc(size);
                        new_node->pos=position;
                        new_node->gi=gi;
                        new_node->plus=plus;
                        new_node->minus=minus;

        //find the primer
                        flag=0;
                        while((p_primer->pos!=pos||p_primer->len!=len)&&flag<2)
                        {
                                if((p_primer->next==NULL)||(p_primer->pos>pos))
                                {
                                        flag++;
                                        p_primer=head;
                                }
                                else
                                {
                                        p_primer=p_primer->next;
                                }
                        }
                        if(flag==2)
                        {
                                printf("Warning: the single primer(pos is %d, length is %d) is not in %s!\n",pos,len,path);
                                free(new_node);
                                continue;
                        } 
                        p_node=p_primer->common;
                        p_primer->common=new_node;
			p_primer->total_common++;
                        new_node->next=p_node;
                }
                fclose(fp);
                free(in);
        }
//paramter for special
        if(special_flag==1)
        {
                i=strlen(path);
                in=(char *)malloc(i+20);
                memset(in,'\0',i+20);
                strcpy(in,path);
                strcat(in,"-special.txt"); //suffix of parameter
                if(access(in,0)==-1)
                {
                        printf("Error! Don't have the %s file!\n",in);
                        exit(1);
                }

                fp=fopen(in,"r");
                if(fp==NULL)
                {
                        printf("Error: can't open the %s file!\n",in);
                        exit(1);
                }
        
                p_primer=head;
                while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
                {
                        new_node=(struct Node *)malloc(size);
                        new_node->pos=position;
                        new_node->gi=gi;
                        new_node->plus=plus;
                        new_node->minus=minus;
        
                        //find the primer
                        flag=0;
                        while((p_primer->pos!=pos||p_primer->len!=len)&&flag<2)
                        {
                                if((p_primer->next==NULL)||(p_primer->pos>pos))
                                {
                                        flag++;
                                        p_primer=head;
                                }
                                else
                                        p_primer=p_primer->next;
                        }
                        if(flag==2)
                        {
                                printf("Warning: the single primer(pos is %d, length is %d) is not in %s!\n",pos,len,path);
                                free(new_node);
                                continue;
                        }
                        p_node=p_primer->special;
                        p_primer->special=new_node;
			p_primer->total_special++;
                        new_node->next=p_node;
                }
                fclose(fp);
                free(in);
        }
        return head;
}

//function: check how many GIs this primer can be used for
__device__ int check_common(int *d_primer,int *d_common,int *d_sc,int *d_ec,int turn[],int common,int *d_apply)
{
        int pos[6],i,dis;

        for(i=0;i<common;i++)
        {
                d_apply[common*turn[0]+i]=0;
        }
//plus
        for(pos[0]=d_sc[turn[0]];pos[0]<d_ec[turn[0]];pos[0]++)
        {
                if(d_common[4*pos[0]+2]!=1)
                        continue;
		i=d_common[4*pos[0]];
                if(d_apply[common*turn[0]+i]==1)
                        continue;
                for(pos[1]=d_sc[turn[1]];pos[1]<d_ec[turn[1]];pos[1]++)
                {
                        if(d_common[4*pos[1]]!=i)
                                continue;
                        if(d_common[4*pos[1]+2]!=1)
                                continue;
                        for(pos[2]=d_sc[turn[2]];pos[2]<d_ec[turn[2]];pos[2]++)
                        {
                                if(d_common[4*pos[2]]!=i)
                                        continue;
                                if(d_common[4*pos[2]+3]!=1)
                                        continue;
                                for(pos[3]=d_sc[turn[3]];pos[3]<d_ec[turn[3]];pos[3]++)
                                {
                                        if(d_common[4*pos[3]]!=i)
                                                continue;
                                        if(d_common[4*pos[3]+2]!=1)
                                                continue;
                                        for(pos[4]=d_sc[turn[4]];pos[4]<d_ec[turn[4]];pos[4]++)
                                        {
                                                if(d_common[4*pos[4]]!=i)
                                                        continue;
                                                if(d_common[4*pos[4]+3]!=1)
                                                        continue;
                                                for(pos[5]=d_sc[turn[5]];pos[5]<d_ec[turn[5]];pos[5]++)
                                                {
                                                        if(d_common[4*pos[5]]!=i)
                                                                continue;
                                                        if(d_common[4*pos[5]+3]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_common[4*pos[1]+1]-(d_common[4*pos[0]+1]+d_primer[4*turn[0]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=d_common[4*pos[2]+1]-d_common[4*pos[1]+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_common[4*pos[3]+1]-(d_common[4*pos[2]+1]+d_primer[4*turn[2]+1]);
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=(d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]-1)-(d_common[4*pos[3]+1]+d_primer[4*turn[3]+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]-1-d_common[4*pos[1]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_common[4*pos[5]+1]-(d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                        d_apply[common*turn[0]+i]=1;
                                                }
                                        }
                                }
                        }
                }
        }
//minus
        for(pos[0]=d_sc[turn[0]];pos[0]<d_ec[turn[0]];pos[0]++)
        {
                if(d_common[4*pos[0]+3]!=1)
                        continue;
		i=d_common[4*pos[0]];
                if(d_apply[turn[0]*common+i]==1)
                        continue;  //this GI can common

                for(pos[1]=d_sc[turn[1]];pos[1]<d_ec[turn[1]];pos[1]++)
                {
                        if(d_common[4*pos[1]]!=i)
                                continue;
                        if(d_common[4*pos[1]+3]!=1)
                                continue;
                        for(pos[2]=d_sc[turn[2]];pos[2]<d_ec[turn[2]];pos[2]++)
                        {
                                if(d_common[4*pos[2]]!=i)
                                        continue;
                                if(d_common[4*pos[2]+2]!=1)
                                        continue;
                                for(pos[3]=d_sc[turn[3]];pos[3]<d_ec[turn[3]];pos[3]++)
                                {
                                        if(d_common[4*pos[3]]!=i)
                                                continue;
                                        if(d_common[4*pos[3]+3]!=1)
                                                continue;
                                        for(pos[4]=d_sc[turn[4]];pos[4]<d_ec[turn[4]];pos[4]++)
                                        {
                                                if(d_common[4*pos[4]]!=i)
                                                        continue;
                                                if(d_common[4*pos[4]+2]!=1)
                                                        continue;
                                                for(pos[5]=d_sc[turn[5]];pos[5]<d_ec[turn[5]];pos[5]++)
                                                {
                                                        if(d_common[4*pos[5]]!=i)
                                                                continue;
                                                        if(d_common[4*pos[5]+2]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_common[4*pos[0]+1]-(d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=(d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]-1)-(d_common[4*pos[2]+1]+d_primer[4*turn[2]+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_common[4*pos[2]+1]-(d_common[4*pos[3]+1]+d_primer[4*turn[3]+1]);
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=d_common[4*pos[3]+1]-d_common[4*pos[4]+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]-1-d_common[4*pos[4]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_common[4*pos[4]+1]-(d_common[4*pos[5]+1]+d_primer[4*turn[5]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                        d_apply[common*turn[0]+i]=1;
                                                }
                                        }
                                }
                        }
                }
        }
        dis=0;
        for(i=0;i<common;i++)
        {
                dis=dis+d_apply[common*turn[0]+i];
        }
        return dis;
}

//check this LAMP primers are uniq or not
//return=0: stop and return=1: go on
__device__ int check_uniq(int *d_primer,int *d_special,int *d_ss,int *d_es,int turn[])
{
        int pos[6],gi;

//plus
        for(pos[0]=d_ss[turn[0]];pos[0]<d_es[turn[0]];pos[0]++)
        {
                if(d_special[4*pos[0]+2]!=1)
                        continue;
		gi=d_special[4*pos[0]];
                for(pos[1]=d_ss[turn[1]];pos[1]<d_es[turn[1]];pos[1]++)
                {
			if(d_special[4*pos[1]]!=gi)
                                continue;
                        if(d_special[4*pos[1]+2]!=1)
				continue;
                        for(pos[2]=d_ss[turn[2]];pos[2]<d_es[turn[2]];pos[2]++) //F1c
                        {
                                if(d_special[4*pos[2]]!=gi)
                                        continue;
                                if(d_special[4*pos[2]+3]!=1)
                                        continue;
                                for(pos[3]=d_ss[turn[3]];pos[3]<d_es[turn[3]];pos[3]++) //B1c
                                {
                                        if(d_special[pos[3]*4]!=gi)
                                                continue;
                                        if(d_special[4*pos[3]+2]!=1)
                                                continue;
                                        for(pos[4]=d_ss[turn[4]];pos[4]<d_es[turn[4]];pos[4]++) //B2
                                        {
                                                if(d_special[4*pos[4]]!=gi)
                                                        continue;
                                                if(d_special[4*pos[4]+3]!=1)
                                                        continue;
                                                for(pos[5]=d_ss[turn[5]];pos[5]<d_es[turn[5]];pos[5]++)
                                                {
                                                        if(d_special[4*pos[5]]!=gi)
                                                                continue;
                                                        if(d_special[4*pos[5]+3]!=1)
                                                                continue;
                                                //F3-F2 
                                                        if(d_special[4*pos[1]+1]<d_special[4*pos[0]+1])
                                                                continue;
                                                //F2-F1c
                                                        if(d_special[4*pos[2]+1]<d_special[4*pos[1]+1]+d_primer[4*turn[1]+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(d_special[4*pos[3]+1]<d_special[4*pos[2]+1]+d_primer[4*turn[2]+1])
                                                                continue;
                                                //B1c-B2
                                                        if(d_special[4*pos[4]+1]<d_special[4*pos[3]+1]+d_primer[4*turn[3]+1])
                                                                continue;
                                                //B2-B3
                                                        if(d_special[4*pos[5]+1]<d_special[4*pos[4]+1])
                                                                continue;
                                                //whole
                                                        if(d_special[4*pos[5]+1]-d_special[4*pos[0]+1]>1000)
                                                                continue;
                                                        return 0;
                                                }//B3
                                        }
                                }//B1c
                        }
                }//F2
        }

//minus
        for(pos[0]=d_ss[turn[0]];pos[0]<d_es[turn[0]];pos[0]++)
        {
                if(d_special[4*pos[0]+3]!=1)
                        continue;
		gi=d_special[4*pos[0]];
                for(pos[1]=d_ss[turn[1]];pos[1]<d_es[turn[1]];pos[1]++)
                {
                        if(d_special[4*pos[1]]!=gi)
                                continue;
                        if(d_special[4*pos[1]+3]!=1)
                                continue;
                        for(pos[2]=d_ss[turn[2]];pos[2]<d_es[turn[2]];pos[2]++)
                        {
                                if(d_special[4*pos[2]]!=gi)
                                        continue;
                                if(d_special[4*pos[2]+2]!=1)
                                        continue;
                                for(pos[3]=d_ss[turn[3]];pos[3]<d_es[turn[3]];pos[3]++)
                                {
                                        if(d_special[4*pos[3]]!=gi)
                                                continue;
                                        if(d_special[4*pos[3]+3]!=1)
                                                continue;
                                        for(pos[4]=d_ss[turn[4]];pos[4]<d_es[turn[4]];pos[4]++)
                                        {
                                                if(d_special[4*pos[4]]!=gi)
                                                        continue;
                                                if(d_special[4*pos[4]+2]!=1)
                                                        continue;
                                                for(pos[5]=d_ss[turn[5]];pos[5]<d_es[turn[5]];pos[5]++)
                                                {
                                                        if(d_special[4*pos[5]]!=gi)
                                                                continue;
                                                        if(d_special[4*pos[5]+2]!=1)
                                                                continue;
                                                //F3-F2 
                                                        if(d_special[4*pos[0]+1]<d_special[4*pos[1]+1])
                                                                continue;
                                                //F2-F1c
                                                        if(d_special[4*pos[1]+1]<d_special[4*pos[2]+1]+d_primer[4*turn[2]+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(d_special[4*pos[2]+1]<d_special[4*pos[3]+1]+d_primer[4*turn[3]+1])
                                                                continue;
                                                //B1c-B2
                                                        if(d_special[4*pos[3]+1]<d_special[4*pos[4]+1]+d_primer[4*turn[4]+1])
                                                                continue;
                                                //B2-B3
                                                        if(d_special[4*pos[4]+1]<d_special[4*pos[5]+1])
                                                                continue;
                                                //whole
                                                        if(d_special[4*pos[0]+1]-d_special[4*pos[5]+1]>1000)
                                                                continue;
                                                        return 0;
                                                }
                                        }
                                }
                        }
                }
        }
        return 1;
}

//from first to second
__global__ void next_one(int *d_primer,int *next,int one_start,int one_end,int two_start,int two_end)
{
        int id=blockDim.x*blockIdx.x+threadIdx.x;
	int i;

	while(one_start+id<one_end)
	{
		i=id+two_start;
		if(i>=two_end)
		{
			i=two_end-1;
		}
		if(d_primer[4*i]>=d_primer[(id+one_start)*4]+d_primer[(id+one_start)*4+1])
		{
			while((i>=two_start)&&(d_primer[4*i]>=d_primer[(id+one_start)*4]+d_primer[(id+one_start)*4+1]))
			{
				next[id]=i;
				i--;
			}
		}
		else
		{
			while((i<two_end)&&(d_primer[4*i]<d_primer[(id+one_start)*4]+d_primer[(id+one_start)*4+1]))
				i++;
			next[id]=i;
			if(i==two_end)
				next[id]=-1;
		}
		id=id+blockDim.x*gridDim.x;
	}
	__syncthreads();
}

__device__ int check_gc(char *d_seq,int start,int end,int flag)
{
        int i,total=0;
        float gc;

        for(i=start;i<end;i++)
        {
                if(d_seq[i]=='C'||d_seq[i]=='G')
                        total++;
        }
        gc=total*100.0/(end-start);
        if(flag==1&&gc>=45)
                return 1;
        if(flag==0&&gc<=45)
                return 1;
        return 0;
}

__device__ int check_common_loop(int *d_primer,int *d_common,int *d_sc,int *d_ec,int turn[],int LF,int LB,int *d_int,int *d_apply)
{
        int dis,i,pos[7];
//plus
        for(pos[0]=d_sc[turn[0]];pos[0]<d_ec[turn[0]];pos[0]++)
        {
                if(d_common[4*pos[0]+2]!=1)
                        continue;
		i=d_common[4*pos[0]];
                if(d_apply[turn[0]*d_int[7]+i]==0||d_apply[turn[0]*d_int[7]+i]==2)
                        continue;
                for(pos[1]=d_sc[turn[1]];pos[1]<d_ec[turn[1]];pos[1]++)
                {
                        if(d_common[4*pos[1]]!=i)
                                continue;
                        if(d_common[4*pos[1]+2]!=1)
                                continue;
                        for(pos[2]=d_sc[turn[2]];pos[2]<d_ec[turn[2]];pos[2]++)
                        {
                                if(d_common[4*pos[2]]!=i)
                                        continue;
                                if(d_common[4*pos[2]+3]!=1)
                                        continue;
                                for(pos[3]=d_sc[turn[3]];pos[3]<d_ec[turn[3]];pos[3]++)
                                {
                                        if(d_common[4*pos[3]]!=i)
                                                continue;
                                        if(d_common[4*pos[3]+2]!=1)
                                                continue;
                                        for(pos[4]=d_sc[turn[4]];pos[4]<d_ec[turn[4]];pos[4]++)
                                        {
                                                if(d_common[4*pos[4]]!=i)
                                                        continue;
                                                if(d_common[4*pos[4]+3]!=1)
                                                        continue;
                                                for(pos[5]=d_sc[turn[5]];pos[5]<d_ec[turn[5]];pos[5]++)
                                                {
                                                        if(d_common[4*pos[5]]!=i)
                                                                continue;
                                                        if(d_common[4*pos[5]+3]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_common[4*pos[1]+1]-(d_common[4*pos[0]+1]+d_primer[4*turn[0]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=d_common[4*pos[2]+1]-d_common[4*pos[1]+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_common[4*pos[3]+1]-(d_common[4*pos[2]+1]+d_primer[4*turn[2]+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=(d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]-1)-(d_common[4*pos[3]+1]+d_primer[4*turn[3]+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]-1-d_common[4*pos[1]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_common[4*pos[5]+1]-(d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //LF
                                                        if(LF!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_sc[LF];pos[6]<d_ec[LF];pos[6]++)
                                                                {
                                                                        if(d_common[4*pos[6]]!=i)
                                                                                continue;
                                                                        if(d_common[4*pos[6]+3]!=1)
                                                                                continue;
                                                                        if(d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]>d_common[4*pos[6]+1])
                                                                                continue;
                                                                        if(d_common[4*pos[6]+1]+d_primer[4*LF+1]>d_common[4*pos[2]+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                //LB
                                                        if(LB!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_sc[LB];pos[6]=d_ec[LB];pos[6]++)
                                                                {
                                                                        if(d_common[4*pos[6]]!=i)
                                                                                continue;
                                                                        if(d_common[4*pos[6]+2]!=1)
                                                                                continue;
                                                                        if(d_common[4*pos[3]+1]+d_primer[4*turn[3]+1]>d_common[4*pos[6]+1])
                                                                                continue;
                                                                        if(d_common[4*pos[6]+1]+d_primer[4*LB+1]>d_common[4*pos[4]+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                        d_apply[turn[0]*d_int[7]+i]=2;
                                                }
                                        }
                                }
                        }
                }
        }
//minus
	for(pos[0]=d_sc[turn[0]];pos[0]<d_ec[turn[0]];pos[0]++)
        {
                if(d_common[4*pos[0]+3]!=1)
                        continue;
                i=d_common[4*pos[0]];
                if(d_apply[turn[0]*d_int[7]+i]==0||d_apply[turn[0]*d_int[7]+i]==2)
                        continue;
                for(pos[1]=d_sc[turn[1]];pos[1]<d_ec[turn[1]];pos[1]++)
                {
                        if(d_common[4*pos[1]]!=i)
                                continue;
                        if(d_common[4*pos[1]+3]!=1)
                                continue;
                        for(pos[2]=d_sc[turn[2]];pos[2]<d_ec[turn[2]];pos[2]++)
                        {
                                if(d_common[4*pos[2]]!=i)
                                        continue;
                                if(d_common[4*pos[2]+2]!=1)
                                        continue;
                                for(pos[3]=d_sc[turn[3]];pos[3]<d_ec[turn[3]];pos[3]++)
                                {
                                        if(d_common[4*pos[3]]!=i)
                                                continue;
                                        if(d_common[4*pos[3]+3]!=1)
                                                continue;
                                        for(pos[4]=d_sc[turn[4]];pos[4]<d_ec[turn[4]];pos[4]++)
                                        {
                                                if(d_common[4*pos[4]]!=i)
                                                        continue;
                                                if(d_common[4*pos[4]+2]!=1)
                                                        continue;
                                                for(pos[5]=d_sc[turn[5]];pos[5]<d_ec[turn[5]];pos[5]++)
                                                {
                                                        if(d_common[4*pos[5]]!=i)
                                                                continue;
                                                        if(d_common[4*pos[5]+2]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_common[4*pos[0]+1]-(d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=(d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]-1)-(d_common[4*pos[2]+1]+d_primer[4*turn[2]+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_common[4*pos[2]+1]-(d_common[4*pos[3]+1]+d_primer[4*turn[3]+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=d_common[4*pos[3]+1]-d_common[4*pos[4]+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_common[4*pos[1]+1]+d_primer[4*turn[1]+1]-1-d_common[4*pos[4]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_common[4*pos[4]+1]-(d_common[4*pos[5]+1]+d_primer[4*turn[5]+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //LF
                                                        if(LF!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_sc[LF];pos[6]<d_ec[LF];pos[6]++)
                                                                {
                                                                        if(d_common[4*pos[6]]!=i)
                                                                                continue;
                                                                        if(d_common[4*pos[6]+2]!=1)
                                                                                continue;
                                                                        if(d_common[4*pos[2]+1]+d_primer[4*turn[2]+1]>d_common[4*pos[6]+1])
                                                                                continue;
                                                                        if(d_common[4*pos[6]+1]+d_primer[4*LF+1]>d_common[4*pos[1]+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                //LB
                                                        if(LB!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_sc[LB];pos[6]=d_ec[LB];pos[6]++)
                                                                {
                                                                        if(d_common[4*pos[6]]!=i)
                                                                                continue;
                                                                        if(d_common[4*pos[6]+3]!=1)
                                                                                continue;
                                                                        if(d_common[4*pos[4]+1]+d_primer[4*turn[4]+1]>d_common[4*pos[6]+1])
                                                                                continue;
                                                                        if(d_common[4*pos[6]+1]+d_primer[4*LB+1]>d_common[4*pos[3]+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
							d_apply[turn[0]*d_int[7]+i]=2;
                                                }
                                        }
                                }
                        }
                }
        }
        for(i=0;i<d_int[7];i++)
        {
                if(d_apply[turn[0]*d_int[7]+i]==1)
                        return 0;
        }
        return 1;
}

__device__ int check_structure_loop(char *d_seq,int *d_primer,int turn[],int *d_int,int LF,int LB,double *parameter,double *d_DPT,int id,int *d_ps)
{
        int i;
        double TH;

        if(LF!=-1)
        {
                for(i=0;i<=1;i++)
                {
                        TH=thal(d_seq,d_primer,turn[i],LF,d_int[11+i],1,1,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LF,d_int[11+i],1,2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LF,d_int[11+i],1,3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;

                        TH=thal(d_seq,d_primer,LF,turn[i],0,(1-d_int[11+i]),2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LF,turn[i],0,(1-d_int[11+i]),3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                }

                for(i=2;i<6;i++)
                {
                        TH=thal(d_seq,d_primer,LF,turn[i],1,d_int[11+i],1,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LF,turn[i],1,d_int[11+i],2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LF,turn[i],1,d_int[11+i],3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;

                        TH=thal(d_seq,d_primer,turn[i],LF,(1-d_int[11+i]),0,2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9]) 
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LF,(1-d_int[11+i]),0,3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                }
        }
        if(LB!=-1)
        {
                for(i=0;i<4;i++)
                {
                        TH=thal(d_seq,d_primer,turn[i],LB,d_int[11+i],0,1,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LB,d_int[11+i],0,2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LB,d_int[11+i],0,3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;

                        TH=thal(d_seq,d_primer,LB,turn[i],1,(1-d_int[11+i]),2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LB,turn[i],1,(1-d_int[11+i]),3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                }
                for(i=4;i<6;i++)
                {
                        TH=thal(d_seq,d_primer,LB,turn[i],0,d_int[11+i],1,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LB,turn[i],0,d_int[11+i],2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,LB,turn[i],0,d_int[11+i],3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;

                        TH=thal(d_seq,d_primer,turn[i],LB,(1-d_int[11+i]),1,2,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                        TH=thal(d_seq,d_primer,turn[i],LB,(1-d_int[11+i]),1,3,parameter,d_DPT,id,d_ps);
                        if(TH>44+5*d_int[9])
                                return 0;
                }
        }
        if(LF!=-1&&LB!=-1)
        {
                TH=thal(d_seq,d_primer,LF,LB,1,0,1,parameter,d_DPT,id,d_ps);
                if(TH>44+5*d_int[9])
                        return 0;
                TH=thal(d_seq,d_primer,LF,LB,1,0,2,parameter,d_DPT,id,d_ps);
                if(TH>44+5*d_int[9])
                        return 0;
                TH=thal(d_seq,d_primer,LF,LB,1,0,3,parameter,d_DPT,id,d_ps);
                if(TH>44+5*d_int[9])
                        return 0;

                TH=thal(d_seq,d_primer,LB,LF,1,0,2,parameter,d_DPT,id,d_ps);
                if(TH>44+5*d_int[9])
                        return 0;
                TH=thal(d_seq,d_primer,LB,LF,1,0,3,parameter,d_DPT,id,d_ps);
                if(TH>44+5*d_int[9])
                        return 0;
        }
        return 1;
}

__device__ int design_loop(int *d_primer,int *d_SLp,int *d_LLp,int *d_LpLp,char *d_seq,int *d_common,int *d_sc,int *d_ec,int turn[],int *d_int,int *d_apply,int *d_par,double *parameter,double *d_DPT,int id,int *d_ps)
{
        int success,LF,LB;

//LF and LB 
        success=0;
	LF=d_SLp[turn[1]];
        while(LF<d_int[2]+d_int[0]+d_int[1])
        {
		if(LF==-1)
			break;
		if(d_primer[4*LF+3]!=1)
		{
			LF++;
			continue;
		}
                if(d_primer[4*LF]+18>d_primer[4*turn[2]])
                        break;
                LB=d_LLp[turn[3]-d_int[0]];
		if(LB==-1||d_primer[4*LB]+18>d_primer[4*turn[4]])
			break;
                while(LB<d_int[2]+d_int[0]+d_int[1])
                {
			if(d_primer[4*LB+2]!=1)
			{
				LB++;
				continue;
			}
                        if(d_primer[4*LB]+18>d_primer[4*turn[4]])
                                break;
                //check_common
                        if(d_int[3])
                        {
                                success=check_common_loop(d_primer,d_common,d_sc,d_ec,turn,LF,LB,d_int,d_apply);
                                if(success==0)
                                {
                                        LB++;
                                        continue;
                                }
                        }
                //check_structure
                        if(d_int[6])
                        {
                                success=check_structure_loop(d_seq,d_primer,turn,d_int,LF,LB,parameter,d_DPT,id,d_ps);
                                if(success==0)
                                {
                                        LB++;
                                        continue;
                                }
                        }
                        d_par[16*turn[0]+12]=d_primer[4*LF];
			d_par[16*turn[0]+13]=d_primer[4*LF+1];
			d_par[16*turn[0]+14]=d_primer[4*LB];
			d_par[16*turn[0]+15]=d_primer[4*LB+1];
                        success=1;
                        break;
                }
                if(success==1)
                        break;
                else
                        LF++;
        }
        if(success==1)
                return success;
//only LF
        LF=d_SLp[turn[1]];
        while(LF<d_int[2]+d_int[1]+d_int[0])
        {
		if(LF==-1)
			break;
                if(d_primer[4*LF]+18>d_primer[4*turn[2]])
                        break;
		if(d_primer[4*LF+3]!=1)
		{
			LF++;
			continue;
		}
        //check_common
                if(d_int[3])
                {
                        success=check_common_loop(d_primer,d_common,d_sc,d_ec,turn,LF,-1,d_int,d_apply);
                        if(success==0)
                        {
                                LF++;
                                continue;
                        }
                }
        //check_structure
                if(d_int[6])
                {
                        success=check_structure_loop(d_seq,d_primer,turn,d_int,LF,-1,parameter,d_DPT,id,d_ps);
                        if(success==0)
                        {
                                LF++;
                                continue;
                        }
                }
		d_par[16*turn[0]+12]=d_primer[4*LF];
		d_par[16*turn[0]+13]=d_primer[4*LF+1];
		d_par[16*turn[0]+14]=0;
		d_par[16*turn[0]+15]=0;
                success=1;
                break;
        }
        if(success==1)
                return success;
//only LB
        LB=d_LLp[turn[3]-d_int[0]];
        while(LB<d_int[2]+d_int[0]+d_int[1])
        {
		if(LB==-1)
			break;
                if(d_primer[LB*4]+18>d_primer[4*turn[4]])
                        break;
		if(d_primer[LB*4+2]!=1)
		{
			LB++;
			continue;
		}
        //check_common
                if(d_int[3])
                {
                        success=check_common_loop(d_primer,d_common,d_sc,d_ec,turn,-1,LB,d_int,d_apply);
                        if(success==0)
                        {
                                LB++;
                                continue;
                        }
                }
        //check_structure
                if(d_int[6])
                {
                        success=check_structure_loop(d_seq,d_primer,turn,d_int,-1,LB,parameter,d_DPT,id,d_ps);
                        if(success==0)
                        {
                                LB++;
                                continue;
                        }
                }
		d_par[16*turn[0]+12]=0;
		d_par[16*turn[0]+13]=0;
		d_par[16*turn[0]+14]=d_primer[4*LB];
		d_par[16*turn[0]+15]=d_primer[4*LB+1];
                success=1;
                break;
        }
        return success;
}

//caculate
__global__ void LAMP(char *d_seq,int *d_primer,int *d_common,int *d_special,int *d_sc,int *d_ec,int *d_ss,int *d_es,int *d_SS,int *d_SL,int *d_SLp,int *d_LL,int *d_LS,int *d_LLp,int *d_LpLp,int *d_int,int *d_par,int *d_apply,double *parameter,int *d_pos,double *d_TH,double *d_DPT,int *d_ps)
//d_int: 0:numS,1:numL,2:numLp,3:common_flag,4:special_flag,5:loop_flag,6:secondary_flag,7:common_num,8:this turn common_num,9:high_GC_flag; 10:expect
{
	int id=blockDim.x*blockIdx.x+threadIdx.x;
	int turn[6],flag;

	while(id<d_int[0])
	{
		d_par[16*id+1]=0;//not LAMP, as a flag
//check add by F3'pos
		turn[0]=0;
		for(turn[1]=0;turn[1]<d_int[10];turn[1]++)
		{
			if(d_pos[turn[1]]==-1)
				break;
			if(d_primer[4*id]-d_pos[turn[1]]<300&&d_primer[4*id]-d_pos[turn[1]]>-300)
			{
				turn[0]++;
				break;
			}
		}
		if(turn[0])
		{
			id=id+blockDim.x*gridDim.x;
			continue;
		}
		if(d_primer[4*id+2]!=1)
		{
			id=id+blockDim.x*gridDim.x;	
			continue;
		}
	//combine
		turn[0]=id; //one thread, one F3
		flag=0;
		for(turn[1]=d_SS[turn[0]];turn[1]<d_int[0];turn[1]++) //F2
		{
			if(turn[1]==-1)
				break;
			if(flag!=0)
				break; //have find one LAMP primer
			if(d_primer[4*turn[1]+2]!=1)
				continue;
			if(d_primer[4*turn[1]]-(d_primer[4*turn[0]]+d_primer[4*turn[0]+1])>20)
				break;
		//	d_par[16*id]=1;
		//	d_par[16*id+1]=1;
		//	flag=1;
		//	break;
			for(turn[2]=d_SL[turn[1]];turn[2]<d_int[1]+d_int[0];turn[2]++) //F1c
			{
				if(turn[2]==-1)
					break;
				if(flag!=0)
					break;
				if(d_primer[4*turn[2]+3]!=1)
					continue;
				if(d_primer[4*turn[2]]-d_primer[4*turn[1]]-1<40)
					continue;
                                if(d_primer[4*turn[2]]-d_primer[4*turn[1]]-1>60)
                                	break;
		//	d_par[16*id]=2;
                  //      d_par[16*id+1]=1;
                    //    flag=1;
                      //  break;
                                for(turn[3]=d_LL[turn[2]-d_int[0]];turn[3]<d_int[1]+d_int[0];turn[3]++)   //B1c
                                {
                                        if(turn[3]==-1)
                                        	break;
					if(flag!=0)
						break;
					if(d_primer[4*turn[3]+2]!=1)
						continue;
                                        if(d_primer[4*turn[3]]-d_primer[4*turn[2]]>85)
                                        	break;
	//		d_par[16*id]=3;
          //              d_par[16*id+1]=1;
            //            flag=1;
              //          break;
                                        for(turn[4]=d_LS[turn[3]-d_int[0]];turn[4]<d_int[0];turn[4]++)   //B2
                                        {
                                                if(turn[4]==-1)
                                                	break;
						if(flag!=0)
							break;
						if(d_primer[4*turn[4]+3]!=1)
							continue;
                                                if((d_primer[4*turn[4]]+d_primer[4*turn[4]+1]-1)-(d_primer[4*turn[3]]+d_primer[4*turn[3]+1])<40)
                                                	continue;
                                                if((d_primer[4*turn[4]]+d_primer[4*turn[4]+1]-1)-(d_primer[4*turn[3]]+d_primer[4*turn[3]+1])>60)
                                                	break;
                                                if(d_primer[4*turn[4]]+d_primer[4*turn[4]+1]-1-d_primer[turn[1]*4]-1<120)
                                                	continue;
                                                if(d_primer[4*turn[4]]+d_primer[4*turn[4]+1]-1-d_primer[turn[1]*4]-1>180)
                                                	break;
						if(d_int[5]&&(d_SLp[turn[1]]==-1||(d_primer[4*d_SLp[turn[1]]]+18>d_primer[4*turn[2]]))&&(d_LLp[turn[3]-d_int[0]]==-1||(d_primer[4*d_LLp[turn[3]-d_int[0]]]+18>d_primer[4*turn[4]])))
							continue;
			//		d_par[16*id]=4;
			//		d_par[16*id+1]=id;
			//		flag=1;
			//		break;
                                                for(turn[5]=d_SS[turn[4]];turn[5]<d_int[0];turn[5]++)  //B3
                                                {
                                                        if(turn[5]==-1)
                                                        	break;
							if(d_primer[4*turn[5]+3]!=1)
								continue;
                                                        if(d_primer[turn[5]*4]-(d_primer[4*turn[4]]+d_primer[4*turn[4]+1])>20)
                                                        	break;
							flag=check_gc(d_seq,d_primer[4*turn[0]],(d_primer[4*turn[5]]+d_primer[4*turn[5]+1]),d_int[9]);
							if(flag==0)
								continue;

							if(d_int[4]!=0)
							{
								flag=check_uniq(d_primer,d_special,d_ss,d_es,turn);
								if(flag==0)
									continue;
							}

				//	d_par[16*id]=5;
                                  //      d_par[16*id+1]=id;
                                    //    flag=1;
                                      //  break;
							if(d_int[3])
							{
								flag=check_common(d_primer,d_common,d_sc,d_ec,turn,d_int[7],d_apply);
								if(flag<d_int[8])
								{
									flag=0;
									continue;
								}
							}

							if(d_int[6])
							{
								flag=check_structure(d_seq,d_primer,turn,d_int,parameter,d_TH,id,d_DPT,d_ps);
								if(flag==0)
									continue;
							}
	
							if(d_int[5])
							{
								flag=design_loop(d_primer,d_SLp,d_LLp,d_LpLp,d_seq,d_common,d_sc,d_ec,turn,d_int,d_apply,d_par,parameter,d_DPT,id,d_ps);
								if(flag==0)
									continue;
							}
							d_par[id*16]=d_primer[4*turn[0]];
							d_par[id*16+1]=d_primer[4*turn[0]+1];
							d_par[id*16+2]=d_primer[4*turn[1]];
							d_par[id*16+3]=d_primer[4*turn[1]+1];
							d_par[id*16+4]=d_primer[4*turn[2]];
							d_par[id*16+5]=d_primer[4*turn[2]+1];
							d_par[id*16+6]=d_primer[4*turn[3]];
							d_par[id*16+7]=d_primer[4*turn[3]+1];
							d_par[id*16+8]=d_primer[4*turn[4]];
							d_par[id*16+9]=d_primer[4*turn[4]+1];
							d_par[id*16+10]=d_primer[4*turn[5]];
							d_par[id*16+11]=d_primer[4*turn[5]+1];
							break;
						}
					}
				}
			}
		}
		id=id+blockDim.x*gridDim.x;
	}
	__syncthreads();
}

void usage()
{
        printf("Usage:\n");
        printf("    LAMP_GPU  -in <name>  -out <result>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the name of candidate single primers file\n");
        printf("    -out  <string>:  the result file of LAMP primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers, default is current directory\n");
        printf("    -ref  <string>:  the reference sequence file used in single program, fasta format\n");
        printf("    -expect  <int>:  the number of LAMP primers needed to be design, default is 10\n"); 
        printf("    -common:         design common LAMP primers\n");
        printf("    -special:        design special LAMP primers\n");
        printf("    -check   <int>:  0: don't check tendency of the left primer to bind to the right primer; !=0: check, default is 1\n");
        printf("    -par  <string>:  the directory of storing parameter files used to check the tendency of two primers binding, default is Par/\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region, high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design LAMP primer with loop primers\n");
        printf("    -h/-help:        usage\n");
}

struct INFO *read_list(char *path,int common_num[])  
{
        char *in,name[301];
        int turn,i,size;
        struct INFO *new_primer,*p_primer,*head;
        FILE *fp;

        i=strlen(path);
        in=(char *)malloc(i+20);              
        memset(in,'\0',i+20);
        strcpy(in,path);
        strcat(in,"-common_list.txt");
        if(access(in,0)==-1)  
        {
                printf("Error! Don't have the %s file!\n",in);
                exit(1);           
        }
        fp=fopen(in,"r");
        if(fp==NULL)
        {          
                printf("Error: can't open the %s file!\n",in);
                exit(1);
        }

        size=sizeof(struct INFO);
        i=0;
        memset(name,'\0',301);
        while(fscanf(fp,"%s\t%d\n",name,&turn)!=EOF)
        {
                new_primer=(struct INFO *)malloc(size);
                new_primer->turn=turn;
                strcpy(new_primer->name,name);
                new_primer->next=NULL;

                if(i==0)
                {
                        head=new_primer;
                        p_primer=new_primer;
                        i++;
                }
                else
                {
                        p_primer->next=new_primer;
                        p_primer=new_primer;
                }
                memset(name,'\0',301);
        }
        fclose(fp);
        common_num[0]=turn;
        free(in);
        return head;
}

main(int argc,char **argv)
{
	int i,j,flag[12],expect,circle,have,common_num[1],num[11],max_loop,min_loop,count[3],block,thread;
	char *output,*prefix,*store_path,*path_fa,*inner,*outer,*loop,*par_path,*temp,*seq,*d_seq,primer[26];
	FILE *fp;
	struct Primer *headL,*headS,*headLoop,*tempL,*tempS,*tempLoop,*storeL,*storeS,*storeLoop; 
	struct Node *p_node,*p_temp;
	struct INFO *headList,*p_list;
	time_t start,end;
	double *H_parameter,*parameter;	
	long int memory;
	hipDeviceProp_t prop;
	int *d_primer,*d_common,*d_special,*d_sc,*d_ec,*d_ss,*d_es,*d_SS,*d_SL,*d_SLp,*d_LL,*d_LS,*d_LLp,*d_LpLp,*d_apply,*d_par,*d_int,*d_pos;
	int *h_primer,*h_common,*h_special,*h_sc,*h_ec,*h_ss,*h_es,*h_apply,*h_par,h_int[17],*h_pos,*d_ps;
	double *h_TH,*d_TH,*d_DPT;
	
	expect=10; //default output max 10 LAMP primers
	start=time(NULL);
/////read the parameters
        for(i=0;i<=11;i++)
                flag[i]=0;
	for(i=0;i<10;i++)
		h_int[i]=0;
        flag[7]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        prefix=(char *)malloc(j+1);
                        memset(prefix,'\0',j+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        output=(char *)malloc(j+1);
                        memset(output,'\0',j+1);
                        strcpy(output,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        if(argv[i+1][j-1]=='/')
                        {
                                store_path=(char *)malloc(j+1);
                                memset(store_path,'\0',j+1);
                                strcpy(store_path,argv[i+1]);
                        }
                        else
                        {
                                store_path=(char *)malloc(j+2);
                                memset(store_path,'\0',j+2);
                                strcpy(store_path,argv[i+1]);
                                store_path[j]='/';
                        }
                        i=i+2;
                }
                else if(strcmp(argv[i],"-ref")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-ref\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        path_fa=(char *)malloc(j+1);
                        memset(path_fa,'\0',j+1);
                        strcpy(path_fa,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-expect")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        expect=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[8]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[9]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[10]=1;
			h_int[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
                else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[7]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[11]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        if(argv[i+1][j-1]=='/')
                        {
                                par_path=(char *)malloc(j+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[j]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(j+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[j]='/';
                                par_path[j+1]='\0';
                        }
                        i=i+2;
                }
                else if(strcmp(argv[i],"-common")==0)
                {
                        flag[5]=1;
			h_int[3]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-special")==0)
                {
                        flag[6]=1;
			h_int[4]=1;
                        i++;
                }
                else
                {
                        printf("Warning! The parameter of %s is invalid.\n\n",argv[i]);
                        i++;
                }
        }
//check parameters
        if(flag[0]==0)
        {
                printf("Error! Users must supply the name of candidate single primers file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the name of output file with -out!\n");
                usage();
                exit(1);
        }
        if(flag[3]==0)
        {
                printf("Error! Users must supply the reference sequence file with -ref!\n");
                usage();
                exit(1);
        }
        if(flag[8]+flag[9]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
//prepare
	if(flag[7]!=0)
		h_int[6]=1;
	else
		h_int[6]=0;
	h_int[9]=flag[8];
	h_int[10]=expect;
        if(flag[2]==0)
        {
                temp=(char *)malloc(4096);
                memset(temp,'\0',4096);
                getcwd(temp,4096);
                j=strlen(temp);
                store_path=(char *)malloc(j+2);
                memset(store_path,'\0',j+2);
                strcpy(store_path,temp);
                free(temp);
                store_path[j]='/';
        }
//secondary
        if(flag[7]&&flag[11]==0)
        {
                temp=(char *)malloc(4096);
                memset(temp,'\0',4096);
                getcwd(temp,4096);
                j=strlen(temp);
                par_path=(char *)malloc(j+10);
                memset(par_path,'\0',j+10);
                strcpy(par_path,temp);
                free(temp);

                j--;
                while(par_path[j]!='/'&&j>=0)
                {
                        par_path[j]='\0';
                        j--;
                }
                strcat(par_path,"Par/");
        }
        if(flag[7])
        {
                H_parameter=(double *)malloc(5730*sizeof(double));
                memset(H_parameter,'\0',5730*sizeof(double));
                hipMalloc((void **)&parameter,5730*sizeof(double));
                hipMemset(parameter,'\0',5730*sizeof(double));

                getStack(par_path,H_parameter);
                getStackint2(par_path,H_parameter);
                getDangle(par_path,H_parameter);
                getLoop(par_path,H_parameter);
                getTstack(par_path,H_parameter);
                getTstack2(par_path,H_parameter);
                tableStartATS(6.9,H_parameter);
                tableStartATH(2200.0,H_parameter);
		hipMemcpy(parameter,H_parameter,5730*sizeof(double),hipMemcpyHostToDevice);

		h_int[11]=0; //F3,plus
		h_int[12]=0;
		h_int[13]=1; //F1c,minus
		h_int[14]=0;
		h_int[15]=1;
		h_int[16]=1;
        }
//F3's pos 
	h_pos=(int *)malloc(expect*sizeof(int));
	for(i=0;i<expect;i++)
		h_pos[i]=-1;
	hipMalloc((void **)&d_pos,expect*sizeof(int));
//d_int
	hipMalloc((void **)&d_int,17*sizeof(int));
//directory for single primers
        j=strlen(store_path)+strlen(prefix)+12;
        outer=(char *)malloc(j);
        memset(outer,'\0',j);
        strcpy(outer,store_path);

        inner=(char *)malloc(j);
        memset(inner,'\0',j);
        strcpy(inner,store_path);

        if(flag[10]==1)
        {
                loop=(char *)malloc(j);
                memset(loop,'\0',j);
                strcpy(loop,store_path);
        }
        if(flag[8]==1)
        {                       
                strcat(outer,"high-outer/");
                strcat(outer,prefix);
                strcat(inner,"high-inner/");
                strcat(inner,prefix);
                if(flag[10]==1)
                {
                        strcat(loop,"high-loop/");
                        strcat(loop,prefix);
                }
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(outer,prefix);
                strcat(inner,"low-inner/");
                strcat(inner,prefix);
                if(flag[10]==1)
                {
                        strcat(loop,"low-loop/");
                        strcat(loop,prefix);
                }
        }

//reference sequence fa
        if(access(path_fa,0)==-1)
        {
                printf("Error! Don't have the %s file!\n",path_fa);
                exit(1);
        }
        i=file_size2(path_fa);
        i=i+100;
        temp=(char *)malloc(i*sizeof(char));
        memset(temp,'\0',i*sizeof(char));
        fp=fopen(path_fa,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the sequence file %s\n",path_fa);
                exit(1);
        }

        fread(temp,i*sizeof(char),1,fp);
        fclose(fp); 
        seq=(char *)malloc(i*sizeof(char));
        memset(seq,'\0',i*sizeof(char));
        
        j=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
                if(temp[i]=='a'||temp[i]=='A')
                        seq[j]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[j]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[j]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[j]='G';
                else
                        seq[j]='N';
                i++;
                j++;
        }
        free(temp);
	num[0]=j; //the length of genome
	hipMalloc((void **)&d_seq,num[0]);
	hipMemset(d_seq,'\0',num[0]);
	hipMemcpy(d_seq,seq,num[0],hipMemcpyHostToDevice);

//common-list
        if(flag[5])
        {
                headList=read_list(inner,common_num);
                common_num[0]++;
        }
        else
                common_num[0]=1;
	h_int[7]=common_num[0];
//read parameters
        headS=read_par(outer,flag[5],flag[6]);
        headL=read_par(inner,flag[5],flag[6]);
        if(flag[10])
        {
                headLoop=read_par(loop,flag[5],0);
                tempLoop=headLoop;
                while(tempLoop->next!=NULL)
                        tempLoop=tempLoop->next;
                max_loop=tempLoop->pos;
		min_loop=headLoop->pos;
        }

//common statistics
        if(flag[5])
        {
                how_many(headL,common_num[0]);
		how_many(headS,common_num[0]);
                if(flag[10])
                        how_many(headLoop,common_num[0]);
        }

	hipGetDeviceProperties(&prop,0); //read parameters
	thread=1;
	fp=fopen(output,"w");
        if(fp==NULL)
        {
                printf("Error: can't create the %s file!\n",output);
                exit(1);
        }
	have=1;
        end=time(NULL);
        printf("The prepare time is %0.1f seconds!\n",difftime(end,start));
        start=time(NULL);

//LAMP-GPU
	for(circle=common_num[0];circle>=1;circle--)
	{
		if(have>expect)
			break;
		storeL=headL;
		storeS=headS;
		while((storeL->pos<=storeS->pos+18)&&storeL!=NULL)
			storeL=storeL->next;
		if(flag[10])
		{
                        storeLoop=headLoop;
			while((storeLoop->pos<=storeS->pos+18)&&storeLoop!=NULL)
				storeLoop=storeLoop->next;
		}
		num[10]=0;
		while(storeS)
		{
			if(have>expect)
				break;
			if(num[10]==1)  //don't have enough primers
				break;
			for(i=1;i<9;i++)
			{
				num[i]=0;
			}
			memory=num[0]/2;
		//statistics	
			tempL=storeL;
			tempS=storeS;
			if(flag[10])
				tempLoop=storeLoop;
			while(tempS&&(memory<prop.totalGlobalMem/6)&&num[2]<20000)
			{
				if(flag[10]&&(tempS->pos+200)<min_loop)
					continue;
				if(flag[10]&&(tempS->pos-200)>max_loop)
					break;
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
				while(tempL&&(tempL->pos<tempS->pos))
				{
					if(tempL->total<circle)
					{
						tempL=tempL->next;
						continue;
					}
					num[1]++;
					if(flag[5])
					{
						num[3]=num[3]+tempL->total_common;
						memory=memory+2+4*tempL->total_common; //2 is common_start and common_end
					}
					if(flag[6])
					{
						num[4]=num[4]+tempL->total_special;
						memory=memory+2+4*tempL->total_special;
					}
					memory=memory+6+flag[10]; //6=4(primers)+2(next_to),flag[10]: next_to_loop
					tempL=tempL->next;
				}
				
				while(flag[10]&&tempLoop&&(tempLoop->pos<tempS->pos))
                                {
                                        if(tempLoop->total<circle)
                                        {
                                                tempLoop=tempLoop->next;
                                                continue;
                                        }
                                        num[7]++;
                                        if(flag[5])
                                        {
                                                num[8]=num[8]+tempLoop->total_common;
                                                memory=memory+2+4*tempLoop->total_common; //2 is common_start and common_end
                                        }
                                        memory=memory+5; //5=4(primers)+1(next_to_self)
                                        tempLoop=tempLoop->next;
                                }

				num[2]++;
				if(flag[5])
				{
					num[5]=num[5]+tempS->total_common;
					memory=memory+2+4*tempS->total_common;
				}
				if(flag[6])
				{
					num[6]=num[6]+tempS->total_special;
					memory=memory+2+4*tempS->total_special;
				}
				memory=memory+22+flag[10]+common_num[0];//14=4(primers)+16(result_par,include loop)+2(next_to)
				if(flag[7])
					memory=memory+5000+50; //one double=4 int, DPT; 50: ps1+ps2
				tempS=tempS->next;
			}
			if(num[2]<4||num[1]<2||(flag[10]&&num[7]<1)) //don't have enough primers
			{
				num[10]=1;
				break;
			}
			if(tempS==NULL)  //check all primers
				num[10]=1;	

			printf("memory is %ld\n",2*memory);
		//malloc
			h_primer=(int *)malloc(4*(num[2]+num[1]+num[7])*sizeof(int));
			hipMalloc((void **)&d_primer,4*(num[2]+num[1]+num[7])*sizeof(int));
			if(flag[5])
			{
				h_common=(int *)malloc(4*(num[5]+num[3]+num[8])*sizeof(int));
				hipMalloc((void **)&d_common,4*(num[5]+num[3]+num[8])*sizeof(int));
				h_sc=(int *)malloc((num[2]+num[1]+num[7])*sizeof(int));
				hipMalloc((void **)&d_sc,(num[2]+num[1]+num[7])*sizeof(int));
	                        h_ec=(int *)malloc((num[2]+num[1]+num[7])*sizeof(int));
	                        hipMalloc((void **)&d_ec,(num[2]+num[1]+num[7])*sizeof(int));
			}
			if(flag[6])
			{
				h_special=(int *)malloc(4*(num[6]+num[4])*sizeof(int));
				hipMalloc((void **)&d_special,4*(num[6]+num[4])*sizeof(int));
				h_ss=(int *)malloc((num[2]+num[1])*sizeof(int));
	                        hipMalloc((void **)&d_ss,(num[2]+num[1])*sizeof(int));
	                        h_es=(int *)malloc((num[2]+num[1])*sizeof(int));
	                        hipMalloc((void **)&d_es,(num[2]+num[1])*sizeof(int));
			}
		//small
			hipMalloc((void **)&d_SS,num[2]*sizeof(int));
			hipMalloc((void **)&d_SL,num[2]*sizeof(int));
			if(flag[10])
				hipMalloc((void **)&d_SLp,num[2]*sizeof(int));
		
			tempS=storeS;
			for(i=0;i<3;i++)
				count[i]=0;
			while(count[0]<num[2])
			{
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
		//primer info
				h_primer[4*count[0]]=tempS->pos;
				h_primer[4*count[0]+1]=tempS->len;
				h_primer[4*count[0]+2]=tempS->plus;
				h_primer[4*count[0]+3]=tempS->minus;
			//common
				if(flag[5])
				{
					h_sc[count[0]]=count[1];
					if(tempS->total_common==0)
						h_ec[count[0]]=-1;
					else
					{
						p_node=tempS->common;
						while(p_node)
						{
							h_common[4*count[1]]=p_node->gi;
							h_common[4*count[1]+1]=p_node->pos;
							h_common[4*count[1]+2]=p_node->plus; 
                                		        h_common[4*count[1]+3]=p_node->minus;
							count[1]++;
							p_node=p_node->next;
						}
						h_ec[count[0]]=count[1];
					}
				}
			//special
				if(flag[6])
				{
					h_ss[count[0]]=count[2];
                	        	if(tempS->total_special==0)
                	        	        h_es[count[0]]=-1;
                	        	else
                	        	{
                	        	        p_node=tempS->special;
                	        	        while(p_node)
                	        	        {
                	        	                h_special[4*count[2]]=p_node->gi;
                	        	                h_special[4*count[2]+1]=p_node->pos;
                	        	                h_special[4*count[2]+2]=p_node->plus;
                	        	                h_special[4*count[2]+3]=p_node->minus;
                	        	                count[2]++;
                	        	                p_node=p_node->next;
                	        	        }
                	        	        h_es[count[0]]=count[2];
                	        	}
				}
				count[0]++;
				tempS=tempS->next;
			}
			h_int[0]=num[2];

		//large primer
        	        hipMalloc((void **)&d_LL,num[1]*sizeof(int));
        	        hipMalloc((void **)&d_LS,num[1]*sizeof(int));
			if(flag[10])
				hipMalloc((void **)&d_LLp,num[1]*sizeof(int));

        	        tempL=storeL;
        	        for(i=0;i<3;i++)
        	                count[i]=0;
        	        while(count[0]<num[1])
        	        {
				if(tempL->total<circle)
				{
					tempL=tempL->next;
					continue;
				}
                	//primer info
                	        h_primer[4*num[2]+4*count[0]]=tempL->pos;
                	        h_primer[4*num[2]+4*count[0]+1]=tempL->len;
                	        h_primer[4*num[2]+4*count[0]+2]=tempL->plus;
                	        h_primer[4*num[2]+4*count[0]+3]=tempL->minus;
                	//common
				if(flag[5])
				{
                	        	h_sc[num[2]+count[0]]=count[1];
                	        	if(tempL->total_common==0)
                	        	        h_ec[num[2]+count[0]]=-1;
                	        	else
                	        	{
                	        	        p_node=tempL->common;
                	        	        while(p_node)
                	        	        {
                	        	                h_common[4*num[5]+4*count[1]]=p_node->gi;
                	        	                h_common[4*num[5]+4*count[1]+1]=p_node->pos;
                	        	                h_common[4*num[5]+4*count[1]+2]=p_node->plus; 
                        		                h_common[4*num[5]+4*count[1]+3]=p_node->minus;
                        		                count[1]++;
                        		                p_node=p_node->next;
                        		        }
                        		        h_ec[num[2]+count[0]]=count[1];
					}
                        	}
                	//special
				if(flag[6])
				{
                        		h_ss[num[2]+count[0]]=count[2];
                        		if(tempL->total_special==0)
                        		        h_es[num[2]+count[0]]=-1;
                        		else
                        		{
                        		        p_node=tempL->special;
                        		        while(p_node)
                        		        {
                        		                h_special[4*num[6]+4*count[2]]=p_node->gi;
                        		                h_special[4*num[6]+4*count[2]+1]=p_node->pos;
                        		                h_special[4*num[6]+4*count[2]+2]=p_node->plus;
                        		                h_special[4*num[6]+4*count[2]+3]=p_node->minus;
                        		                count[2]++;
                        		                p_node=p_node->next;
                        		        }
                        		        h_es[num[2]+count[0]]=count[2];
                        		}
				}
                        	count[0]++;
                        	tempL=tempL->next;
                	}
			h_int[1]=num[1];

                //loop primer
			if(flag[10])
			{
	                        hipMalloc((void **)&d_LpLp,num[7]*sizeof(int));

	                        tempLoop=storeLoop;
                        	for(i=0;i<2;i++)
                        	        count[i]=0;
	                        while(count[0]<num[7])
                        	{
                                	if(tempLoop->total<circle)
                                	{
                                        	tempLoop=tempLoop->next;
                                        	continue;
                                	}
                        	//primer info
                                	h_primer[4*(num[1]+num[2])+4*count[0]]=tempLoop->pos;
                                	h_primer[4*(num[1]+num[2])+4*count[0]+1]=tempLoop->len;
                                	h_primer[4*(num[1]+num[2])+4*count[0]+2]=tempLoop->plus;
                                	h_primer[4*(num[1]+num[2])+4*count[0]+3]=tempLoop->minus;
                        	//common
                                	if(flag[5])
                                	{
                                	        h_sc[num[1]+num[2]+count[0]]=count[1];
                                	        if(tempLoop->total_common==0)
                                	                h_ec[num[1]+num[2]+count[0]]=-1;
                                	        else
                                	        {
                                	                p_node=tempLoop->common;
                                	                while(p_node)
                                	                {
                                	                        h_common[4*(num[5]+num[3])+4*count[1]]=p_node->gi;
                                	                        h_common[4*(num[5]+num[3])+4*count[1]+1]=p_node->pos;
                                	                        h_common[4*(num[5]+num[3])+4*count[1]+2]=p_node->plus; 
                                	                        h_common[4*(num[5]+num[3])+4*count[1]+3]=p_node->minus;
                                	                        count[1]++;
                                	                        p_node=p_node->next;
                                	                }
                                	                h_ec[num[1]+num[2]+count[0]]=count[1];
                                	        }
                                	}
	                                count[0]++;
	                                tempLoop=tempLoop->next;
	                        }
				h_int[2]=num[7];
                        }
	//run
			if(num[2]%thread==0)
                                block=num[2]/thread;
                        else
                                block=(num[2]-num[2]%thread)/thread+1;

			if(block>prop.maxGridSize[0]/2)
				block=prop.maxGridSize[0]/2;

			hipMemcpy(d_primer,h_primer,4*(num[1]+num[2]+num[7])*sizeof(int),hipMemcpyHostToDevice);
			free(h_primer);
			if(flag[5])
			{
				hipMemcpy(d_common,h_common,4*(num[3]+num[5]+num[8])*sizeof(int),hipMemcpyHostToDevice);
				free(h_common);
				hipMemcpy(d_sc,h_sc,(num[1]+num[2]+num[7])*sizeof(int),hipMemcpyHostToDevice);
				free(h_sc);
				hipMemcpy(d_ec,h_ec,(num[1]+num[2]+num[7])*sizeof(int),hipMemcpyHostToDevice);
				free(h_ec);
			}
			if(flag[6])
			{
				hipMemcpy(d_special,h_special,4*(num[4]+num[6])*sizeof(int),hipMemcpyHostToDevice);
				free(h_special);
				hipMemcpy(d_ss,h_ss,(num[1]+num[2])*sizeof(int),hipMemcpyHostToDevice);
				free(h_ss);
				hipMemcpy(d_es,h_es,(num[1]+num[2])*sizeof(int),hipMemcpyHostToDevice);
				free(h_es);
			}
		//next primer
		printf("block is %d,thread is %d\n",block,thread);
			next_one<<<block,thread>>>(d_primer,d_SS,0,num[2],0,num[2]);
			next_one<<<block,thread>>>(d_primer,d_LL,num[2],(num[1]+num[2]),num[2],(num[1]+num[2]));
			next_one<<<block,thread>>>(d_primer,d_SL,0,num[2],num[2],(num[1]+num[2]));
        	        next_one<<<block,thread>>>(d_primer,d_LS,num[2],(num[1]+num[2]),0,num[2]);
			if(flag[10])
			{
				next_one<<<block,thread>>>(d_primer,d_LpLp,(num[1]+num[2]),(num[1]+num[2]+num[7]),(num[1]+num[2]),(num[1]+num[2]+num[7]));
                        	next_one<<<block,thread>>>(d_primer,d_SLp,0,num[2],(num[1]+num[2]),(num[1]+num[2]+num[7]));
                        	next_one<<<block,thread>>>(d_primer,d_LLp,num[2],(num[1]+num[2]),(num[1]+num[2]),(num[1]+num[2]+num[7]));
			}
		//calculate
			if(flag[5])
				hipMalloc((void **)&d_apply,common_num[0]*num[2]*sizeof(int));
			hipMemcpy(d_pos,h_pos,expect*sizeof(int),hipMemcpyHostToDevice);
			h_int[8]=circle;
			hipMemcpy(d_int,h_int,17*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(d_pos,h_pos,expect*sizeof(int),hipMemcpyHostToDevice);
			hipMalloc((void **)&d_par,num[2]*16*sizeof(int));
		hipMalloc((void **)&d_TH,2*num[2]*sizeof(double));
		h_TH=(double *)malloc(2*num[2]*sizeof(double));
			hipMalloc((void **)&d_DPT,num[2]*1250*sizeof(double));
			hipMalloc((void **)&d_ps,num[2]*50*sizeof(int));
			LAMP<<<block,thread>>>(d_seq,d_primer,d_common,d_special,d_sc,d_ec,d_ss,d_es,d_SS,d_SL,d_SLp,d_LL,d_LS,d_LLp,d_LpLp,d_int,d_par,d_apply,parameter,d_pos,d_TH,d_DPT,d_ps);
			hipFree(d_DPT);
			hipFree(d_ps);
		hipMemcpy(h_TH,d_TH,2*num[2]*sizeof(double),hipMemcpyDeviceToHost);
		hipFree(d_TH);
		printf("%lf\t%lf\n",h_TH[0],h_TH[1]);
		free(h_TH);
			h_par=(int *)malloc(16*num[2]*sizeof(int));
			memset(h_par,'\0',16*num[2]*sizeof(int));
			hipMemcpy(h_par,d_par,16*num[2]*sizeof(int),hipMemcpyDeviceToHost);
			hipFree(d_par);
			if(flag[5])
			{
				h_apply=(int *)malloc(common_num[0]*num[2]*sizeof(int));
				hipMemcpy(h_apply,d_apply,common_num[0]*num[2]*sizeof(int),hipMemcpyDeviceToHost);
				hipFree(d_apply);
			}
	//free
			hipFree(d_primer);
			hipFree(d_SS);
			hipFree(d_LL);
			hipFree(d_SL);
			hipFree(d_LS);
			if(flag[10])
			{
				hipFree(d_LpLp);
				hipFree(d_SLp);
				hipFree(d_LLp);
			}
			if(flag[5])
			{
				hipFree(d_common);
				hipFree(d_sc);
				hipFree(d_ec);
			}
			if(flag[6])
			{
                                hipFree(d_special);
                                hipFree(d_ss);
                                hipFree(d_es);
                        }
		//LAMP primers, output
			for(i=0;i<num[2];i++)
			{
				if(have>expect)
					break;
				if(h_par[i*16+1]==0)
					continue;
				if(check_add(h_par[i*16],h_pos,have)==0)
					continue;
				fprintf(fp,"The %d LAMP primers:\n",have);
		                generate_primer(seq,primer,h_par[i*16],h_par[i*16+1],0);
		                fprintf(fp,"  F3: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16],h_par[i*16+1],primer);
		                generate_primer(seq,primer,h_par[i*16+2],h_par[i*16+3],0);
		                fprintf(fp,"  F2: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+2],h_par[i*16+3],primer);
		                generate_primer(seq,primer,h_par[i*16+4],h_par[i*16+5],1);
		                fprintf(fp,"  F1c: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+4],h_par[i*16+5],primer);
		                generate_primer(seq,primer,h_par[i*16+6],h_par[i*16+7],0);
		                fprintf(fp,"  B1c: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+6],h_par[i*16+7],primer);
		                generate_primer(seq,primer,h_par[i*16+8],h_par[i*16+9],1);
		                fprintf(fp,"  B2: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+8],h_par[i*16+9],primer);
		                generate_primer(seq,primer,h_par[i*16+10],h_par[i*16+11],1);
		                fprintf(fp,"  B3: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+10],h_par[i*16+11],primer);
                		if(flag[10])
                		{
                        		if(h_par[i*16+13]==0)
                        		        fprintf(fp,"  LF: NULL\n");
                        		else
                        		{
                        		        generate_primer(seq,primer,h_par[i*16+12],h_par[i*16+13],1);
                        		        fprintf(fp,"  LF: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+12],h_par[i*16+13],primer);
                        		}
		                        if(h_par[i*16+15]==0)
		                                fprintf(fp,"  LB: NULL\n");
		                        else
		                        {
		                                generate_primer(seq,primer,h_par[i*16+14],h_par[i*16+15],0);
		                                fprintf(fp,"  LB: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_par[i*16+14],h_par[i*16+15],primer);
		                        }
		                }
		                if(flag[5])
		                {
					j=0;
		                        fprintf(fp,"  This set of LAMP primers could be used in %d genomes, there are: ",circle);
		                        p_list=headList;
                        		for(j=0;j<common_num[0];j++)
                        		{
                                		if(h_apply[i*common_num[0]+j]==0)
                                		        continue;
                                		while(p_list)
                                		{
                                		        if(p_list->turn==j)
                                		                break;
                                		        else
                                		                p_list=p_list->next;
                                		}
                                		if(j==0)
                                		        fprintf(fp,"%s",p_list->name);
                                		else
                                		        fprintf(fp,", %s",p_list->name);
                                		j++;
	       	                 	}
		                        fprintf(fp,"\n");
				}
				h_pos[have-1]=h_par[i*16];
				have++;
			}
			if(flag[5])
				free(h_apply);
			free(h_par);
			if(have>expect)
				break;
		//new primer start
			if(tempS==NULL)
				storeS=tempS;
			else
			{
				while(tempS->pos-storeS->pos>300)
				{
					storeS=storeS->next;
				}
			}
		}//one circle
	}
	hipFree(d_int);
	hipFree(d_pos);
	hipFree(d_seq);
	free(h_pos);
	free(seq);
	free(output);
        free(prefix);
        free(store_path);
        free(inner);
        free(outer);
        free(path_fa);
	fclose(fp);
//free struct list
        while(headL)
        {
                p_node=headL->common;
                while(p_node)
                {
                        p_temp=p_node->next;
                        free(p_node);
                        p_node=p_temp;
                }
                p_node=headL->special;
                while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
                
                storeL=headL->next;
                free(headL);
                headL=storeL;
        }
        while(headS)
        {
                p_node=headS->common;  
                while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
                p_node=headS->special;
                while(p_node)
                {               
                        p_temp=p_node->next;
                        free(p_node);
                        p_node=p_temp;
                }

                storeS=headS->next;
                free(headS);
                headS=storeS;
        }

        if(flag[5])
        {
                while(headList)
                {
                        p_list=headList->next;
                        free(headList);
                        headList=p_list;
                }
        }

        if(flag[7])
        {
                free(H_parameter);
                hipFree(parameter);
        }
        if(flag[7]||flag[11])
                free(par_path);

        if(flag[10])
        {
                free(loop);
                while(headLoop)
                {
                        p_node=headLoop->common;
                        while(p_node)
                        {
                                p_temp=p_node->next;
                                free(p_node);
                                p_node=p_temp;
                        }
                        p_node=headLoop->special;
                        while(p_node)
                        {
                                p_temp=p_node->next;
                                free(p_node);
                                p_node=p_temp;
                        }

                        storeLoop=headLoop->next;
                        free(headLoop);
                        headLoop=storeLoop;
                }
        }	
	end=time(NULL);
        printf("the time for design is %0.1f seconds!\n",difftime(end,start));
}
