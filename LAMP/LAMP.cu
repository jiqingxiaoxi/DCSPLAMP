#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<sys/stat.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__constant__ int const_int[19];
__constant__ double parameter[5730];

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ char str2int(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;
                case 'T':
                        return 3;
        }
        return 4;
}

__device__ char str2int_rev(char c)
{
        switch (c)
        {
                case 'T':
                        return 0;
                case 'G':
                        return 1;
                case 'C':
                        return 2;                 
                case 'A':               
                        return 3;         
        }
        return 4;
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ double Ss(int i,int j,int k,int *d_ps,char *d_numSeq,int id,double parameter[])
{
	if(k==2)
	{
		if(i>=j)
			return -1.0;
		if(i==d_ps[id*62+50]||j==d_ps[id*62+51]+1)
			return -1.0;

		if(i>d_ps[id*62+50])
			i-=d_ps[id*62+50];
		if(j>d_ps[id*62+51])
			j-=d_ps[id*62+51];
		return parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
	}
	else
		return parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ double Hs(int i,int j,int k,int *d_ps,char *d_numSeq,int id,double parameter[])
{
	if(k==2)
	{
		if(i>= j)
			return 1.0*INFINITY;
		if(i==d_ps[id*62+50]||j==d_ps[id*62+51]+1)
			return 1.0*INFINITY;

		if(i>d_ps[id*62+50])
			i-=d_ps[id*62+50];
		if(j>d_ps[id*62+51])
			j-=d_ps[id*62+51];
		if(fabs(parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]])<999999999)
			return parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
		else
			return 1.0*INFINITY;
	}
	else
		return parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ int equal(double a,double b)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
		return 0;
	return fabs(a-b)<1e-5;
}

__device__ void initMatrix(int *d_ps,double *d_DPT,int id,char *d_numSeq)
{
	for(d_ps[id*62+54]=1;d_ps[id*62+54]<=d_ps[id*62+50];++d_ps[id*62+54])
	{
		for(d_ps[id*62+55]=1;d_ps[id*62+55]<=d_ps[id*62+51];++d_ps[id*62+55])
		{
			if(d_numSeq[id*54+d_ps[id*62+54]]+d_numSeq[id*54+27+d_ps[id*62+55]]!=3)
			{
				d_DPT[id*1263+(d_ps[id*62+54]-1)*d_ps[id*62+51]+d_ps[id*62+55]-1]=1.0*INFINITY;
				d_DPT[id*1263+625+(d_ps[id*62+54]-1)*d_ps[id*62+51]+d_ps[id*62+55]-1]=-1.0;
			}
			else
			{
				d_DPT[id*1263+(d_ps[id*62+54]-1)*d_ps[id*62+51]+d_ps[id*62+55]-1]=0.0;
				d_DPT[id*1263+625+(d_ps[id*62+54]-1)*d_ps[id*62+51]+d_ps[id*62+55]-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,int *d_ps,double *d_DPT,int id,char *d_numSeq,double parameter[])
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1]=-1.0;
		d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1]=1.0*INFINITY;
		return;
	}

	d_DPT[id*1263+1257]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	d_DPT[id*1263+1258]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	if(fabs(d_DPT[id*1263+1258])>999999999)
	{
		d_DPT[id*1263+1258]=1.0*INFINITY;
		d_DPT[id*1263+1257]=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1263+1260]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1263+1261]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1263+1261])>999999999)
		{
			d_DPT[id*1263+1261]=1.0*INFINITY;
			d_DPT[id*1263+1260]=-1.0;
		}
		d_DPT[id*1263+1262]=(d_DPT[id*1263+1261]+200)/(d_DPT[id*1263+1260]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1258])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1257]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1262])
			{
				d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
				d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
			}
		}
		else
		{
			d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
			d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
		}
	}
	else if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999)
	{
		d_DPT[id*1263+1260]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		d_DPT[id*1263+1261]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		if(fabs(d_DPT[id*1263+1261])>999999999)
		{
			d_DPT[id*1263+1261]=1.0*INFINITY;
			d_DPT[id*1263+1260]=-1.0;
		}
		d_DPT[id*1263+1262]=(d_DPT[id*1263+1261]+200)/(d_DPT[id*1263+1260]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1258])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1257]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1262])
			{
				d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
				d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
			}
		}
		else
		{
			d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
			d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
		}
	}
	else if(fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1263+1260]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1263+1261]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1263+1261])>999999999)
		{
			d_DPT[id*1263+1261]=1.0*INFINITY;
			d_DPT[id*1263+1260]=-1.0;
		}
		d_DPT[id*1263+1262]=(d_DPT[id*1263+1261]+200)/(d_DPT[id*1263+1260]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1258])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1257]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1262])
			{
				d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
				d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
			}
		}
		else
		{
			d_DPT[id*1263+1257]=d_DPT[id*1263+1260];
			d_DPT[id*1263+1258]=d_DPT[id*1263+1261];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1262];
		}
	}

	d_DPT[id*1263+1260]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1263+1261]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1263+1262]=(d_DPT[id*1263+1261]+200)/(d_DPT[id*1263+1260]-5.7+d_DPT[id*1263+1252]);
	if(fabs(d_DPT[id*1263+1258])<999999999)
	{
		if(d_DPT[id*1263+1259]<d_DPT[id*1263+1262])
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1260];
			d_DPT[id*1263+1256]=d_DPT[id*1263+1261];
		}
		else
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1257];
			d_DPT[id*1263+1256]=d_DPT[id*1263+1258];
		}
	}
	else
	{
		d_DPT[id*1263+1255]=d_DPT[id*1263+1260];
		d_DPT[id*1263+1256]=d_DPT[id*1263+1261];
	}
	return;
}

__device__ void maxTM(int i,int j,int *d_ps,double *d_DPT,int id,char *d_numSeq,double parameter[])
{
	d_DPT[id*1263+1259]=d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1];
	d_DPT[id*1263+1261]=d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1];
	d_DPT[id*1263+1257]=(d_DPT[id*1263+1261]+200)/(d_DPT[id*1263+1259]-5.7+d_DPT[id*1263+1252]); // at current position 
	if(fabs(d_DPT[id*1263+(i-2)*d_ps[id*62+51]+j-2])<999999999&&fabs(Hs(i-1,j-1,1,d_ps,d_numSeq,id,parameter))<999999999)
	{
		d_DPT[id*1263+1260]=(d_DPT[id*1263+625+(i-2)*d_ps[id*62+51]+j-2]+Ss(i-1,j-1,1,d_ps,d_numSeq,id,parameter));
		d_DPT[id*1263+1262]=(d_DPT[id*1263+(i-2)*d_ps[id*62+51]+j-2]+Hs(i-1,j-1,1,d_ps,d_numSeq,id,parameter));
	}
	else
	{
		d_DPT[id*1263+1260]=-1.0;
		d_DPT[id*1263+1262]=1.0*INFINITY;
	}
	d_DPT[id*1263+1258]=(d_DPT[id*1263+1262]+200)/(d_DPT[id*1263+1260]-5.7+d_DPT[id*1263+1252]);

	if(d_DPT[id*1263+1260]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1263+1260]=-3224.0;
		d_DPT[id*1263+1262]=0.0;
	}
	if(d_DPT[id*1263+1259]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1263+1259]=-3224.0;
		d_DPT[id*1263+1261]=0.0;
	}
	if((d_DPT[id*1263+1258]>d_DPT[id*1263+1257])||(d_DPT[id*1263+1259]>0&&d_DPT[id*1263+1261]>0)) // T1 on suurem 
	{
		d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1]=d_DPT[id*1263+1260];
		d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1]=d_DPT[id*1263+1262];
	}
	else if(d_DPT[id*1263+1257]>=d_DPT[id*1263+1258])
	{
		d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1]=d_DPT[id*1263+1259];
		d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1]=d_DPT[id*1263+1261];
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,int traceback,int *d_ps,double *d_DPT,int id,char *d_numSeq,double parameter[])
{
	d_DPT[id*1263+1259]=-3224.0;
	d_DPT[id*1263+1260]=0;
	d_ps[id*62+59]=ii-i-1;
	d_ps[id*62+60]=jj-j-1;
	d_ps[id*62+61]=d_ps[id*62+59]+d_ps[id*62+60]-1;
	if((d_ps[id*62+59]==0&&d_ps[id*62+60]>0)||(d_ps[id*62+60]==0&&d_ps[id*62+59]>0))// only bulges have to be considered
	{
		if(d_ps[id*62+60]==1||d_ps[id*62+59]==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((d_ps[id*62+60]==1&&d_ps[id*62+59]==0)||(d_ps[id*62+60]==0&&d_ps[id*62+59]==1))
			{
				d_DPT[id*1263+1260]=parameter[3150+d_ps[id*62+61]]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				d_DPT[id*1263+1259]=parameter[3060+d_ps[id*62+61]]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
			}
			d_DPT[id*1263+1260]+=d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1];
			d_DPT[id*1263+1259]+=d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1];
			if(fabs(d_DPT[id*1263+1260])>999999999)
			{
				d_DPT[id*1263+1260]=1.0*INFINITY;
				d_DPT[id*1263+1259]=-1.0;
			}

			d_DPT[id*1263+1257]=(d_DPT[id*1263+1260]+200)/((d_DPT[id*1263+1259]-5.7)+d_DPT[id*1263+1252]);
			d_DPT[id*1263+1258]=(d_DPT[id*1263+(ii-1)*d_ps[id*62+51]+jj-1]+200)/((d_DPT[id*1263+625+(ii-1)*d_ps[id*62+51]+jj-1])-5.7+d_DPT[id*1263+1252]);
			if((d_DPT[id*1263+1257]>d_DPT[id*1263+1258])||((traceback&&d_DPT[id*1263+1257]>=d_DPT[id*1263+1258])||(traceback==1)))
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1259];
				d_DPT[id*1263+1256]=d_DPT[id*1263+1260];
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			d_DPT[id*1263+1260]=parameter[3150+d_ps[id*62+61]]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1263+1260]+=d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1];

			d_DPT[id*1263+1259]=parameter[3060+d_ps[id*62+61]]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1263+1259]+=d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1];
			if(fabs(d_DPT[id*1263+1260])>999999999)
			{
				d_DPT[id*1263+1260]=1.0*INFINITY;
				d_DPT[id*1263+1259]=-1.0;
			}
			d_DPT[id*1263+1257]=(d_DPT[id*1263+1260]+200)/((d_DPT[id*1263+1259]-5.7)+d_DPT[id*1263+1252]);
			d_DPT[id*1263+1258]=(d_DPT[id*1263+(ii-1)*d_ps[id*62+51]+jj-1]+200)/(d_DPT[id*1263+625+(ii-1)*d_ps[id*62+51]+jj-1]-5.7+d_DPT[id*1263+1252]);
			if((d_DPT[id*1263+1257]>d_DPT[id*1263+1258])||((traceback&&d_DPT[id*1263+1257]>=d_DPT[id*1263+1258])||(traceback==1)))
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1259];
				d_DPT[id*1263+1256]=d_DPT[id*1263+1260];
			}
		}
	}
	else if(d_ps[id*62+59]==1&&d_ps[id*62+60]==1)
	{
		d_DPT[id*1263+1259]=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1263+1259]+=d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1];

		d_DPT[id*1263+1260]=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1263+1260]+=d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1];
		if(fabs(d_DPT[id*1263+1260])>999999999)
		{
			d_DPT[id*1263+1260]=1.0*INFINITY;
			d_DPT[id*1263+1259]=-1.0;
		}
		d_DPT[id*1263+1257]=(d_DPT[id*1263+1260]+200)/((d_DPT[id*1263+1259]-5.7)+d_DPT[id*1263+1252]);
		d_DPT[id*1263+1258]=(d_DPT[id*1263+(ii-1)*d_ps[id*62+51]+jj-1]+200)/(d_DPT[id*1263+625+(ii-1)*d_ps[id*62+51]+jj-1]-5.7+d_DPT[id*1263+1252]);
		if((d_DPT[id*1263+1257]-d_DPT[id*1263+1258]>=0.000001)||traceback==1)
		{
			if((d_DPT[id*1263+1257]>d_DPT[id*1263+1258])||(traceback&&d_DPT[id*1263+1257]>=d_DPT[id*1263+1258]))
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1259];
				d_DPT[id*1263+1256]=d_DPT[id*1263+1260];
			}
		}
		return;
	}
	else // only internal loops
	{
		d_DPT[id*1263+1260]=parameter[3120+d_ps[id*62+61]]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1263+1260]+=d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1];

		d_DPT[id*1263+1259]=parameter[3030+d_ps[id*62+61]]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(d_ps[id*62+59]-d_ps[id*62+60]));
		d_DPT[id*1263+1259]+=d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1];
		if(fabs(d_DPT[id*1263+1260])>999999999)
		{
			d_DPT[id*1263+1260]=1.0*INFINITY;
			d_DPT[id*1263+1259]=-1.0;
		}
		d_DPT[id*1263+1257]=(d_DPT[id*1263+1260]+200)/((d_DPT[id*1263+1259]-5.7)+d_DPT[id*1263+1252]);
		d_DPT[id*1263+1258]=(d_DPT[id*1263+(ii-1)*d_ps[id*62+51]+jj-1]+200)/((d_DPT[id*1263+625+(ii-1)*d_ps[id*62+51]+jj-1])-5.7+d_DPT[id*1263+1252]);
		if((d_DPT[id*1263+1257]>d_DPT[id*1263+1258])||((traceback&&d_DPT[id*1263+1257]>=d_DPT[id*1263+1258])||(traceback==1)))
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1259];
			d_DPT[id*1263+1256]=d_DPT[id*1263+1260];
		}
	}
	return;
}

__device__ void fillMatrix(int *d_ps,double *d_DPT,int id,char *d_numSeq)
{
	int i;

	for(i=1;i<=d_ps[id*62+50];i++)
	{
		for(d_ps[id*62+58]=1;d_ps[id*62+58]<=d_ps[id*62+51];d_ps[id*62+58]++)
		{
			if(fabs(d_DPT[id*1263+(i-1)*d_ps[id*62+51]+d_ps[id*62+58]-1])<999999999)
			{
				d_DPT[id*1263+1255]=-1.0;
				d_DPT[id*1263+1256]=1.0*INFINITY;
				LSH(i,d_ps[id*62+58],d_ps,d_DPT,id,d_numSeq,parameter);

				if(fabs(d_DPT[id*1263+1256])<999999999)
				{
					d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+d_ps[id*62+58]-1]=d_DPT[id*1263+1255];
					d_DPT[id*1263+(i-1)*d_ps[id*62+51]+d_ps[id*62+58]-1]=d_DPT[id*1263+1256];
				}
				if(i>1&&d_ps[id*62+58]>1)
				{
					maxTM(i,d_ps[id*62+58],d_ps,d_DPT,id,d_numSeq,parameter);
					for(d_ps[id*62+54]=3;d_ps[id*62+54]<=32;d_ps[id*62+54]++)
					{
						d_ps[id*62+55]=i-1;
						d_ps[id*62+56]=0-d_ps[id*62+55]-d_ps[id*62+54]+d_ps[id*62+58]+i;
						if(d_ps[id*62+56]<1)
						{
							d_ps[id*62+55]-=abs(d_ps[id*62+56]-1);
							d_ps[id*62+56]=1;
						}
						for(;d_ps[id*62+55]>0&&d_ps[id*62+56]<d_ps[id*62+58];d_ps[id*62+55]--,d_ps[id*62+56]++)
						{
							if(fabs(d_DPT[id*1263+(d_ps[id*62+55]-1)*d_ps[id*62+51]+d_ps[id*62+56]-1])<999999999)
							{
								d_DPT[id*1263+1255]=-1.0;
								d_DPT[id*1263+1256]=1.0*INFINITY;
								calc_bulge_internal(d_ps[id*62+55],d_ps[id*62+56],i,d_ps[id*62+58],0,d_ps,d_DPT,id,d_numSeq,parameter);

								if(d_DPT[id*1263+1255]<-2500.0)
								{
									d_DPT[id*1263+1255] =-3224.0;
									d_DPT[id*1263+1256] = 0.0;
								}
								if(fabs(d_DPT[id*1263+1256])<999999999)
								{
									d_DPT[id*1263+(i-1)*d_ps[id*62+51]+d_ps[id*62+58]-1]=d_DPT[id*1263+1256];
									d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+d_ps[id*62+58]-1]=d_DPT[id*1263+1255];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double *d_DPT,char *d_numSeq,int id)
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1263+1250]=-1.0;
		d_DPT[id*1263+1251]=1.0*INFINITY;
		return;
	}
	d_DPT[id*1263+1255]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	d_DPT[id*1263+1257]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	if(fabs(d_DPT[id*1263+1257])>999999999)
	{
		d_DPT[id*1263+1257]=1.0*INFINITY;
		d_DPT[id*1263+1255]=-1.0;
	}
	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1263+1256]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1263+1258]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1263+1258])>999999999)
		{
			d_DPT[id*1263+1258]=1.0*INFINITY;
			d_DPT[id*1263+1256]=-1.0;
		}
		d_DPT[id*1263+1260]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1256]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1257])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1257]+200)/(d_DPT[id*1263+1255]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1260])
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
				d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
			}
		}
		else
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
			d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
		}
	}

	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999)
	{
		d_DPT[id*1263+1256]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		d_DPT[id*1263+1258]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		if(fabs(d_DPT[id*1263+1258])>999999999)
		{
			d_DPT[id*1263+1258]=1.0*INFINITY;
			d_DPT[id*1263+1256]=-1.0;
		}
		d_DPT[id*1263+1260]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1256]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1257])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1257]+200)/(d_DPT[id*1263+1255]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1260])
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
				d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
			}
		}
		else
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
			d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
		}
	}

	if(fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1263+1256]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1263+1258]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1263+1258])>999999999)
		{
			d_DPT[id*1263+1258]=1.0*INFINITY;
			d_DPT[id*1263+1256]=-1.0;
		}
		d_DPT[id*1263+1260]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1256]-5.7+d_DPT[id*1263+1252]);
		if(fabs(d_DPT[id*1263+1257])<999999999)
		{
			d_DPT[id*1263+1259]=(d_DPT[id*1263+1257]+200)/(d_DPT[id*1263+1255]-5.7+d_DPT[id*1263+1252]);
			if(d_DPT[id*1263+1259]<d_DPT[id*1263+1260])
			{
				d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
				d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
				d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
			}
		}
		else
		{
			d_DPT[id*1263+1255]=d_DPT[id*1263+1256];
			d_DPT[id*1263+1257]=d_DPT[id*1263+1258];
			d_DPT[id*1263+1259]=d_DPT[id*1263+1260];
		}
	}
	d_DPT[id*1263+1256]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1263+1258]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1263+1260]=(d_DPT[id*1263+1258]+200)/(d_DPT[id*1263+1256]-5.7+d_DPT[id*1263+1252]);
	if(fabs(d_DPT[id*1263+1257])<999999999)
	{
		if(d_DPT[id*1263+1259]<d_DPT[id*1263+1260])
		{
			d_DPT[id*1263+1250]=d_DPT[id*1263+1256];
			d_DPT[id*1263+1251]=d_DPT[id*1263+1258];
		}
		else
		{
			d_DPT[id*1263+1250]=d_DPT[id*1263+1255];
			d_DPT[id*1263+1251]=d_DPT[id*1263+1257];
		}
	}
	else
	{
		d_DPT[id*1263+1250]=d_DPT[id*1263+1256];
		d_DPT[id*1263+1251]=d_DPT[id*1263+1258];
	}
	return;
}

__device__ void traceback(int i,int j,int *d_ps,double *d_DPT,int id,char *d_numSeq)
{
	d_ps[id*62+i-1]=j;
	d_ps[id*62+25+j-1]=i;
	while(1)
	{
		d_DPT[id*1263+1255]=-1.0;
		d_DPT[id*1263+1256]=1.0*INFINITY;
		LSH(i,j,d_ps,d_DPT,id,d_numSeq,parameter);
		if(equal(d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1],d_DPT[id*1263+1255])&&equal(d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1],d_DPT[id*1263+1256]))
			break;

		d_ps[id*62+57]=0;
		if(i>1&&j>1&&equal(d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1],Ss(i-1,j-1,1,d_ps,d_numSeq,id,parameter)+d_DPT[id*1263+625+(i-2)*d_ps[id*62+51]+j-2]))
		{
			i=i-1;
			j=j-1;
			d_ps[id*62+i-1]=j;
			d_ps[id*62+25+j-1]=i;
			d_ps[id*62+57]=1;
		}
		for(d_ps[id*62+54]=3;!d_ps[id*62+57]&&d_ps[id*62+54]<=32;++d_ps[id*62+54])
		{
			d_ps[id*62+55]=i-1;
			d_ps[id*62+56]=-d_ps[id*62+55]-d_ps[id*62+54]+(j+i);
			if(d_ps[id*62+56]<1)
			{
				d_ps[id*62+55]-=abs(d_ps[id*62+56]-1);
				d_ps[id*62+56]=1;
			}
			for(;!d_ps[id*62+57]&&d_ps[id*62+55]>0&&d_ps[id*62+56]<j;--d_ps[id*62+55],++d_ps[id*62+56])
			{
				d_DPT[id*1263+1255]=-1.0;
				d_DPT[id*1263+1256]=1.0*INFINITY;
				calc_bulge_internal(d_ps[id*62+55],d_ps[id*62+56],i,j,1,d_ps,d_DPT,id,d_numSeq,parameter);
				if(equal(d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1],d_DPT[id*1263+1255])&&equal(d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1],d_DPT[id*1263+1256]))
				{
					i=d_ps[id*62+55];
					j=d_ps[id*62+56];
					d_ps[id*62+i-1]=j;
					d_ps[id*62+25+j-1]=i;
					d_ps[id*62+57]=1;
					break;
				}
			}
		}
	}
}

__device__ double drawDimer(int *d_ps,int id,double H,double S,double *d_DPT)
{
        if(fabs(d_DPT[id*1263+1253])>999999999)
                return (double)0.0;
        else
        {
                d_ps[id*62+55]=0;
                for(d_ps[id*62+54]=0;d_ps[id*62+54]<d_ps[id*62+50];d_ps[id*62+54]++)
                {
                        if(d_ps[id*62+d_ps[id*62+54]]>0)
                                ++d_ps[id*62+55];
                }
                for(d_ps[id*62+54]=0;d_ps[id*62+54]<d_ps[id*62+51];d_ps[id*62+54]++)
                {
                        if(d_ps[id*62+25+d_ps[id*62+54]]>0)
                                ++d_ps[id*62+55];
                }
                d_ps[id*62+55]=(d_ps[id*62+55]/2)-1;
                return (double)(H/(S+(d_ps[id*62+55]*-0.51986)+d_DPT[id*1263+1252])-273.15);
        }
}

__device__ int symmetry_thermo(char *d_seq,int start,int length,int *d_ps,int id)
{
	d_ps[id*62+54]=0;
	if(length%2==1)
		return 0;

	while(d_ps[id*62+54]<length/2)
	{
		if((d_seq[d_ps[id*62+54]+start]=='A'&&d_seq[start+length-1-d_ps[id*62+54]]!='T')||(d_seq[d_ps[id*62+54]+start]=='T'&&d_seq[start+length-1-d_ps[id*62+54]]!='A')||(d_seq[start+length-1-d_ps[id*62+54]]=='A'&&d_seq[d_ps[id*62+54]+start]!='T')||(d_seq[start+length-1-d_ps[id*62+54]]=='T'&&d_seq[d_ps[id*62+54]+start]!='A'))
			return 0;
		if((d_seq[d_ps[id*62+54]+start]=='C'&&d_seq[start+length-1-d_ps[id*62+54]]!='G')||(d_seq[d_ps[id*62+54]+start]=='G'&&d_seq[start+length-1-d_ps[id*62+54]]!='C')||(d_seq[start+length-1-d_ps[id*62+54]]=='C'&&d_seq[d_ps[id*62+54]+start]!='G')||(d_seq[start+length-1-d_ps[id*62+54]]=='G'&&d_seq[d_ps[id*62+54]+start]!='C'))
			return 0;
		d_ps[id*62+54]++;
	}
	return 1;
}

__device__ double thal(char *d_seq,int *d_primer,int one_turn,int two_turn,int one_flag,int two_flag,int type,double *d_DPT,int id,int *d_ps,char *d_numSeq)
{
	int i, j;

/*** INIT values for unimolecular and bimolecular structures ***/
	if(symmetry_thermo(d_seq,d_primer[one_turn*10],d_primer[one_turn*10+1],d_ps,id)&&symmetry_thermo(d_seq,d_primer[two_turn*10],d_primer[two_turn*10+1],d_ps,id))
		d_DPT[id*1263+1252]=1.9872* log(38/1000000000.0);
	else
		d_DPT[id*1263+1252]=1.9872* log(38/4000000000.0);
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		d_ps[id*62+50]=d_primer[one_turn*10+1];
		d_ps[id*62+51]=d_primer[two_turn*10+1];
		if(one_flag==0) //plus
		{
	 		for(i=1;i<=d_ps[id*62+50];++i)
				d_numSeq[id*54+i]=str2int(d_seq[d_primer[one_turn*10]+i-1]);
		}
		else
		{
			for(i=1;i<=d_ps[id*62+50];++i)
				d_numSeq[id*54+i]=str2int_rev(d_seq[d_primer[one_turn*10]+d_primer[one_turn*10+1]-i]);
		}

		if(two_flag==0)
		{
			for(i=1;i<=d_ps[id*62+51];++i)
				d_numSeq[id*54+27+i]=str2int(d_seq[d_primer[two_turn*10]+d_primer[two_turn*10+1]-i]);
		}
		else
		{
			for(i=1;i<=d_ps[id*62+51];++i)
				d_numSeq[id*54+27+i]=str2int_rev(d_seq[d_primer[two_turn*10]+i-1]);
		}
	}
	else if(type==3)
	{
		d_ps[id*62+50]=d_primer[two_turn*10+1];
		d_ps[id*62+51]=d_primer[one_turn*10+1];
		if(two_flag==0)
		{
			for(i=1;i<=d_ps[id*62+50];++i)
				d_numSeq[id*54+i]=str2int(d_seq[d_primer[two_turn*10]+i-1]);
		}
		else
		{
			for(i=1;i<=d_ps[id*62+50];++i)
				d_numSeq[id*54+i]=str2int_rev(d_seq[d_primer[two_turn*10]+d_primer[two_turn*10+1]-i]);
		}
		if(one_flag==0)
		{
			for(i=1;i<=d_ps[id*62+51];++i)
				d_numSeq[id*54+27+i]=str2int(d_seq[d_primer[one_turn*10]+d_primer[one_turn*10+1]-i]);
		}
		else
		{
			for(i=1;i<=d_ps[id*62+51];++i)
				d_numSeq[id*54+27+i]=str2int_rev(d_seq[d_primer[one_turn*10]+i-1]);
		}
	}
	d_numSeq[id*54+0]=d_numSeq[id*54+d_ps[id*62+50]+1]=d_numSeq[id*54+27+0]=d_numSeq[id*54+27+d_ps[id*62+51]+1]=4; /* mark as N-s */

	initMatrix(d_ps,d_DPT,id,d_numSeq);
	fillMatrix(d_ps,d_DPT,id,d_numSeq);

	d_DPT[id*1263+1253]=-1.0*INFINITY;
/* calculate terminal basepairs */
	d_ps[id*62+52]=d_ps[id*62+53]=0;
	if(type==1)
		for (i=1;i<=d_ps[id*62+50];i++)
		{
			for (j=1;j<=d_ps[id*62+51];j++)
			{
				RSH(i,j,d_DPT,d_numSeq,id);
				d_DPT[id*1263+1250]=d_DPT[id*1263+1250]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
				d_DPT[id*1263+1251]=d_DPT[id*1263+1251]+0.000001;
				d_DPT[id*1263+1254]=((d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1]+d_DPT[id*1263+1251]+200)/((d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1])+d_DPT[id*1263+1250]-5.7+d_DPT[id*1263+1252]))-273.15;
				if(d_DPT[id*1263+1254]>d_DPT[id*1263+1253]&&((d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1]+d_DPT[id*1263+1250])<0&&(d_DPT[id*1263+1251]+d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1])<0))
				{
					d_DPT[id*1263+1253]=d_DPT[id*1263+1254];
					d_ps[id*62+52]=i;
					d_ps[id*62+53]=j;
				}
			}
		}
	if(type==2||type==3)
	{
	 //THAL_END1
		d_ps[id*62+53]=0;
		d_ps[id*62+52]=d_ps[id*62+50];
		i=d_ps[id*62+50];
		d_DPT[id*1263+1253]=-1.0*INFINITY;
		for (j=1;j<=d_ps[id*62+51];++j)
		{
			RSH(i,j,d_DPT,d_numSeq,id);
			d_DPT[id*1263+1250]=d_DPT[id*1263+1250]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
			d_DPT[id*1263+1251]=d_DPT[id*1263+1251]+0.000001;
			d_DPT[id*1263+1254]=((d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1]+d_DPT[id*1263+1251]+200)/((d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1])+d_DPT[id*1263+1250]-5.7+d_DPT[id*1263+1252]))-273.15;
			if (d_DPT[id*1263+1254]>d_DPT[id*1263+1253]&&((d_DPT[id*1263+1250]+d_DPT[id*1263+625+(i-1)*d_ps[id*62+51]+j-1])<0&&(d_DPT[id*1263+1251]+d_DPT[id*1263+(i-1)*d_ps[id*62+51]+j-1])<0))
			{
				d_DPT[id*1263+1253]=d_DPT[id*1263+1254];
				d_ps[id*62+53]=j;
			}
		}
	}
	if(fabs(d_DPT[id*1263+1253])>999999999)
		d_ps[id*62+52]=d_ps[id*62+53]=1;
	RSH(d_ps[id*62+52],d_ps[id*62+53],d_DPT,d_numSeq,id);
 // tracebacking 
	for (i=0;i<d_ps[id*62+50];++i)
		d_ps[id*62+i]=0;
	for (j=0;j<d_ps[id*62+51];++j)
		d_ps[id*62+25+j] = 0;
	if(fabs(d_DPT[id*1263+(d_ps[id*62+52]-1)*d_ps[id*62+51]+d_ps[id*62+53]-1])<999999999)
	{
		traceback(d_ps[id*62+52],d_ps[id*62+53],d_ps,d_DPT,id,d_numSeq);
		d_DPT[id*1263+1254]=drawDimer(d_ps,id,(d_DPT[id*1263+(d_ps[id*62+52]-1)*d_ps[id*62+51]+d_ps[id*62+53]-1]+d_DPT[id*1263+1251]+200),(d_DPT[id*1263+625+(d_ps[id*62+52]-1)*d_ps[id*62+51]+d_ps[id*62+53]-1]+d_DPT[id*1263+1250]-5.7),d_DPT);
		d_DPT[id*1263+1254]=(int)(100*d_DPT[id*1263+1254]+0.5)/100.0;
		return d_DPT[id*1263+1254];
	}
        return 0.0;
}

struct Node
{
	int pos;
	int gi;
	int strand;  //1:plus,2:minus,3:all ok
	struct Node *next;
};

struct Primer
{
	int pos;
	int len;
	int strand;
	int total_common;
	int total_special;
	int total; //common number
	struct Primer *next;
	struct Node *common;
	struct Node *special;
};

struct INFO
{
        char name[301];
        int turn;
        struct INFO *next;
};

int check_add(int F3_pos,int *par,int have)
{
        int i,dis;

        for(i=0;i<have;i++)
        {
		if(par[i]==-1)
			return 1;
                dis=par[i]-F3_pos;
                if(abs(dis)<300)              
                        return 0;
        }
        return 1;        
}

void generate_primer(char *seq,char primer[],int start,int length,int flag)
{
        int i;
        if(flag==0)
        {
                for(i=0;i<length;i++)
                	primer[i]=seq[start+i];
        }
        else
        {
                for(i=0;i<length;i++)
                {
                        if(seq[start+length-1-i]=='A')
                                primer[i]='T';
                        else if(seq[start+length-1-i]=='T')
                                primer[i]='A';
                        else if(seq[start+length-1-i]=='C')
                                primer[i]='G';
                        else
                                primer[i]='C';
                }
        }
	primer[length]='\0';
}

__device__ int check_structure(char *d_seq,int *d_primer,int turn[],int ID_thread,double *d_TH,int id,double *d_DPT,int *d_ps,char *d_numSeq)
{
	double TH;
	int i,j;

	for(i=0;i<7;i++)
	{
		for(j=i+1;j<8;j++)
		{
			if((i==2||i==5||j==2||j==5)&&(turn[ID_thread*8+2]==-1&&turn[ID_thread*8+5]==-1))
				continue;  //without-loop
			if(turn[ID_thread*8+i]==-1||turn[ID_thread*8+j]==-1)
				continue; //when loop, don't have loop
		if(i!=3||j!=4)
			continue;
			TH=thal(d_seq,d_primer,turn[ID_thread*8+i],turn[ID_thread*8+j],const_int[11+i],const_int[11+j],1,d_DPT,id,d_ps,d_numSeq);
			if(TH>44+5*const_int[9])
                                return 0;
		d_TH[id*2]=TH;
			TH=thal(d_seq,d_primer,turn[ID_thread*8+i],turn[ID_thread*8+j],const_int[11+i],const_int[11+j],2,d_DPT,id,d_ps,d_numSeq);
                        if(TH>44+5*const_int[9])
                                return 0;
		d_TH[id*2+1]=TH;
			TH=thal(d_seq,d_primer,turn[ID_thread*8+i],turn[ID_thread*8+j],const_int[11+i],const_int[11+j],3,d_DPT,id,d_ps,d_numSeq);
                        if(TH>44+5*const_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
			d_TH[id*2+1]=TH;
			TH=thal(d_seq,d_primer,turn[ID_thread*8+j],turn[ID_thread*8+i],(1-const_int[11+j]),(1-const_int[11+i]),2,d_DPT,id,d_ps,d_numSeq);
                        if(TH>44+5*const_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
                        d_TH[id*2+1]=TH;
                        TH=thal(d_seq,d_primer,turn[ID_thread*8+j],turn[ID_thread*8+i],(1-const_int[11+j]),(1-const_int[11+i]),3,d_DPT,id,d_ps,d_numSeq);
                        if(TH>44+5*const_int[9])
                                return 0;
		if(TH>d_TH[id*2+1])
                        d_TH[id*2+1]=TH;
		}
	}
	return 1;
}

void how_many(struct Primer *head,int common)
{
        struct Primer *p_primer;
        struct Node *p_node;
        int i,num,*list;

	list=(int *)malloc(common*sizeof(int));
        p_primer=head;
        while(p_primer)
        {
                p_node=p_primer->common;
		for(i=0;i<common;i++)
                {
                        list[i]=0;
                }
                i=0;
                while(p_node)
                {
                        i++;
			list[p_node->gi]=1;
                        p_node=p_node->next;
                }
		p_primer->total_common=i;

        //special
                p_node=p_primer->special;
                i=0;
                while(p_node)       
                {
                        i++;
                        p_node=p_node->next;
                }
		p_primer->total_special=i;

		num=0;
                for(i=0;i<common;i++)
                {
                        num=num+list[i];
                }
		p_primer->total=num;
                p_primer=p_primer->next;
        }
	free(list);
}

//get the file size
int file_size2(char* filename)
{
        struct stat statbuf;
        stat(filename,&statbuf);
        int size=statbuf.st_size;
        return size;
}

////function read primer informatin and align information 
struct Primer *read_par(char *path,int common_flag,int special_flag)
{
        char *in;
        int pos,len,gi,position,plus,minus,size,i,flag;
        struct Primer *new_primer,*p_primer,*head;
        struct Node *new_node,*p_node;
        FILE *fp;

///read the  primer file
        if(access(path,0)==-1)
        {
                printf("Error! Don't have the %s file!\n",path);
                exit(1);
        }
        fp=fopen(path,"r");
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",path);
                exit(1);
        }
        
        size=sizeof(struct Primer);
        i=0;
        while(fscanf(fp,"pos:%d\tlength:%d\t+:%d\t-:%d\n",&pos,&len,&plus,&minus)!=EOF)
        {
                new_primer=(struct Primer *)malloc(size);
                new_primer->pos=pos;
                new_primer->len=len;
                new_primer->total=1;
		new_primer->total_common=0;
		new_primer->total_special=0;
		if(plus)
		{
			if(minus)
				new_primer->strand=3;
			else
				new_primer->strand=1;
		}
		else
	                new_primer->strand=2;
                new_primer->next=NULL;
                new_primer->common=NULL;
                new_primer->special=NULL;

                if(i==0)
                {
                        head=new_primer;
                        p_primer=new_primer;
                        i++;
                }
                else
                {
                        p_primer->next=new_primer;
                        p_primer=new_primer;
                }
        }
        fclose(fp);
        if(i==0)
        {
                printf("Sorry! Don't have any candidate single primers in %s!\n",path);
                exit(1);
        }

//parameter of common
        if(common_flag==1)
        {
                i=strlen(path);
                in=(char *)malloc(i+20);
                memset(in,'\0',i+20);
                strcpy(in,path);
                strcat(in,"-common.txt"); //suffix of parameter
                if(access(in,0)==-1)
                {
                        printf("Error! Don't have the %s file!\n",in);
                        exit(1);
                }

                fp=fopen(in,"r");
                if(fp==NULL)
                {
                        printf("Error: can't open the %s file!\n",in);
                        exit(1);
                }

                p_primer=head;
                size=sizeof(struct Node);
                while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
                {
                        new_node=(struct Node *)malloc(size);
                        new_node->pos=position;
                        new_node->gi=gi;
			if(plus)
			{
				if(minus)
					new_node->strand=3;
				else
					new_node->strand=1;
			}
			else
	                        new_node->strand=2;

        //find the primer
                        flag=0;
                        while((p_primer->pos!=pos||p_primer->len!=len)&&flag<2)
                        {
                                if((p_primer->next==NULL)||(p_primer->pos>pos))
                                {
                                        flag++;
                                        p_primer=head;
                                }
                                else
                                {
                                        p_primer=p_primer->next;
                                }
                        }
                        if(flag==2)
                        {
                                printf("Warning: the single primer(pos is %d, length is %d) is not in %s!\n",pos,len,path);
                                free(new_node);
                                continue;
                        } 
                        p_node=p_primer->common;
                        p_primer->common=new_node;
			p_primer->total_common++;
                        new_node->next=p_node;
                }
                fclose(fp);
                free(in);
        }
//paramter for special
        if(special_flag==1)
        {
                i=strlen(path);
                in=(char *)malloc(i+20);
                memset(in,'\0',i+20);
                strcpy(in,path);
                strcat(in,"-special.txt"); //suffix of parameter
                if(access(in,0)==-1)
                {
                        printf("Error! Don't have the %s file!\n",in);
                        exit(1);
                }

                fp=fopen(in,"r");
                if(fp==NULL)
                {
                        printf("Error: can't open the %s file!\n",in);
                        exit(1);
                }
        
                p_primer=head;
                while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
                {
                        new_node=(struct Node *)malloc(size);
                        new_node->pos=position;
                        new_node->gi=gi;
			if(plus)
			{
				if(minus)
					new_node->strand=3;
				else
					new_node->strand=1;
			}
			else
	                        new_node->strand=2;
        
                        //find the primer
                        flag=0;
                        while((p_primer->pos!=pos||p_primer->len!=len)&&flag<2)
                        {
                                if((p_primer->next==NULL)||(p_primer->pos>pos))
                                {
                                        flag++;
                                        p_primer=head;
                                }
                                else
                                        p_primer=p_primer->next;
                        }
                        if(flag==2)
                        {
                                printf("Warning: the single primer(pos is %d, length is %d) is not in %s!\n",pos,len,path);
                                free(new_node);
                                continue;
                        }
                        p_node=p_primer->special;
                        p_primer->special=new_node;
			p_primer->total_special++;
                        new_node->next=p_node;
                }
                fclose(fp);
                free(in);
        }
        return head;
}

//check this LAMP primers are uniq or not
//return=0: stop and return=1: go on
__device__ int check_uniq(int *d_primer,int *d_info,int turn[],int ID_thread)
{
        int pos[6],gi;

//plus
        for(pos[0]=d_primer[turn[ID_thread*8]*10+5];pos[0]<d_primer[turn[ID_thread*8]*10+6];pos[0]++)
        {
                if((d_info[pos[0]*3+2]&1)!=1)
                        continue;
		gi=d_info[pos[0]*3];
                for(pos[1]=d_primer[turn[ID_thread*8+1]*10+5];pos[1]<d_primer[turn[ID_thread*8+1]*10+6];pos[1]++)
                {
			if(d_info[pos[1]*3]!=gi)
                                continue;
                        if((d_info[pos[1]*3+2]&1)!=1)
				continue;
                        for(pos[2]=d_primer[turn[ID_thread*8+3]*10+5];pos[2]<d_primer[turn[ID_thread*8+3]*10+6];pos[2]++) //F1c
                        {
                                if(d_info[pos[2]*3]!=gi)
                                        continue;
                                if((d_info[pos[2]*3+2]&2)!=2)
                                        continue;
                                for(pos[3]=d_primer[turn[ID_thread*8+3]*10+5];pos[3]<d_primer[turn[ID_thread*8+3]*10+6];pos[3]++) //B1c
                                {
                                        if(d_info[pos[3]*3]!=gi)
                                                continue;
                                        if((d_info[pos[3]*3+2]&1)!=1)
                                                continue;
                                        for(pos[4]=d_primer[turn[ID_thread*8+6]*10+5];pos[4]<d_primer[turn[ID_thread*8+6]*10+6];pos[4]++) //B2
                                        {
                                                if(d_info[pos[4]*3]!=gi)
                                                        continue;
                                                if((d_info[pos[4]*3+2]&2)!=2)
                                                        continue;
                                                for(pos[5]=d_primer[turn[ID_thread*8+7]*10+5];pos[5]<d_primer[turn[ID_thread*8+7]*10+6];pos[5]++)
                                                {
                                                        if(d_info[pos[5]*3]!=gi)
                                                                continue;
                                                        if((d_info[pos[5]*3+2]&2)!=2)
                                                                continue;
                                                //F3-F2 
                                                        if(d_info[pos[1]*3+1]<d_info[pos[0]*3+1])
                                                                continue;
                                                //F2-F1c
                                                        if(d_info[pos[2]*3+1]<d_info[pos[1]*3+1]+d_primer[turn[ID_thread*8+1]*10+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(d_info[pos[3]*3+1]<d_info[pos[2]*3+1]+d_primer[turn[ID_thread*8+3]*10+1])
                                                                continue;
                                                //B1c-B2
                                                        if(d_info[pos[4]*3+1]<d_info[pos[3]*3+1]+d_primer[turn[ID_thread*8+3]*10+1])
                                                                continue;
                                                //B2-B3
                                                        if(d_info[pos[5]*3+1]<d_info[pos[4]*3+1])
                                                                continue;
                                                //whole
                                                        if(d_info[pos[5]*3+1]-d_info[pos[0]*3+1]>1000)
                                                                continue;
                                                        return 0;
                                                }//B3
                                        }
                                }//B1c
                        }
                }//F2
        }

//minus
        for(pos[0]=d_primer[turn[ID_thread*8]*10+5];pos[0]<d_primer[turn[ID_thread*8]*10+6];pos[0]++)
        {
                if((d_info[pos[0]*3+2]&2)!=2)
                        continue;
		gi=d_info[pos[0]*3];
                for(pos[1]=d_primer[turn[ID_thread*8+1]*10+5];pos[1]<d_primer[turn[ID_thread*8+1]*10+6];pos[1]++)
                {
                        if(d_info[pos[1]*3]!=gi)
                                continue;
                        if((d_info[pos[1]*3+2]&2)!=2)
                                continue;
                        for(pos[2]=d_primer[turn[ID_thread*8+3]*10+5];pos[2]<d_primer[turn[ID_thread*8+3]*10+6];pos[2]++)
                        {
                                if(d_info[pos[2]*3]!=gi)
                                        continue;
                                if((d_info[pos[2]*3+2]&1)!=1)
                                        continue;
                                for(pos[3]=d_primer[turn[ID_thread*8+3]*10+5];pos[3]<d_primer[turn[ID_thread*8+3]*10+6];pos[3]++)
                                {
                                        if(d_info[pos[3]*3]!=gi)
                                                continue;
                                        if((d_info[pos[3]*3+2]&2)!=2)
                                                continue;
                                        for(pos[4]=d_primer[turn[ID_thread*8+6]*10];pos[4]<d_primer[turn[ID_thread*8+6]*10];pos[4]++)
                                        {
                                                if(d_info[pos[4]*3]!=gi)
                                                        continue;
                                                if((d_info[pos[4]*3+2]&1)!=1)
                                                        continue;
                                                for(pos[5]=d_primer[turn[ID_thread*8+7]*10+5];pos[5]<d_primer[turn[ID_thread*8+7]*10+6];pos[5]++)
                                                {
                                                        if(d_info[pos[5]*3]!=gi)
                                                                continue;
                                                        if((d_info[pos[5]*3+2]&1)!=1)
                                                                continue;
                                                //F3-F2 
                                                        if(d_info[pos[0]*3+1]<d_info[pos[1]*3+1])
                                                                continue;
                                                //F2-F1c
                                                        if(d_info[pos[1]*3+1]<d_info[pos[2]*3+1]+d_primer[turn[ID_thread*8+3]*10+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(d_info[pos[2]*3+1]<d_info[pos[3]*3+1]+d_primer[turn[ID_thread*8+3]*10+1])
                                                                continue;
                                                //B1c-B2
                                                        if(d_info[pos[3]*3+1]<d_info[pos[4]*3+1]+d_primer[turn[ID_thread*8+6]*10+1])
                                                                continue;
                                                //B2-B3
                                                        if(d_info[pos[4]*3+1]<d_info[pos[5]*3+1])
                                                                continue;
                                                //whole
                                                        if(d_info[pos[0]*3+1]-d_info[pos[5]*3+1]>1000)
                                                                continue;
                                                        return 0;
                                                }
                                        }
                                }
                        }
                }
        }
        return 1;
}

//from first to second
__global__ void next_one(int *d_primer,int one_start,int one_end,int two_start,int two_end,int pos) //7,8,9
{
        int id=blockDim.x*blockIdx.x+threadIdx.x;
	int i;

	while(one_start+id<one_end)
	{
		i=id+two_start;
		if(i>=two_end)
		{
			i=two_end-1;
		}
		if(d_primer[10*i]>=d_primer[(id+one_start)*10]+d_primer[(id+one_start)*10+1])
		{
			while((i>=two_start)&&(d_primer[10*i]>=d_primer[(id+one_start)*10]+d_primer[(id+one_start)*10+1]))
			{
				d_primer[10*(id+one_start)+pos]=i;
				i--;
			}
		}
		else
		{
			while((i<two_end)&&(d_primer[10*i]<d_primer[(id+one_start)*10]+d_primer[(id+one_start)*10+1]))
				i++;
			if(i==two_end)
				d_primer[10*(id+one_start)+pos]=-1;
			else
				d_primer[10*(id+one_start)+pos]=i;
		}
		id=id+blockDim.x*gridDim.x;
	}
	__syncthreads();
}

__device__ int check_gc(char *d_seq,int start,int end,int flag)
{
        int i,total=0;
        float gc;

        for(i=start;i<end;i++)
        {
                if(d_seq[i]=='C'||d_seq[i]=='G')
                        total++;
        }
        gc=total*100.0/(end-start);
        if(flag==1&&gc>=45)
                return 1;
        if(flag==0&&gc<=45)
                return 1;
        return 0;
}

__device__ int check_common(int *d_primer,int *d_info,int turn[],int ID_thread,int *d_result)
{
        int dis,i,pos[7];

	for(i=0;i<const_int[7];i++)
        {
                d_result[(8+const_int[7])*turn[ID_thread*8]+8+i]=0;
        }
//plus
        for(pos[0]=d_primer[turn[ID_thread*8]*10+3];pos[0]<d_primer[turn[ID_thread*8]*10+4];pos[0]++)
        {
                if((d_info[pos[0]*3+2]&1)!=1)
                        continue;
		i=d_info[pos[0]*3];
                if(d_result[(8+const_int[7])*turn[ID_thread*8]+8+i]!=0)
                        continue;
                for(pos[1]=d_primer[turn[ID_thread*8+1]*10+3];pos[1]<d_primer[turn[ID_thread*8+1]*10+4];pos[1]++)
                {
                        if(d_info[pos[1]*3]!=i)
                                continue;
                        if((d_info[pos[1]*3+2]&1)!=1)
                                continue;
                        for(pos[2]=d_primer[turn[ID_thread*8+3]*10+3];pos[2]<d_primer[turn[ID_thread*8+3]*10+4];pos[2]++)
                        {
                                if(d_info[pos[2]*3]!=i)
                                        continue;
                                if((d_info[pos[2]*3+2]&2)!=2)
                                        continue;
                                for(pos[3]=d_primer[turn[ID_thread*8+4]*10+3];pos[3]<d_primer[turn[ID_thread*8+4]*10+4];pos[3]++)
                                {
                                        if(d_info[pos[3]*3]!=i)
                                                continue;
                                        if((d_info[pos[3]*3+2]&1)!=1)
                                                continue;
                                        for(pos[4]=d_primer[turn[ID_thread*8+6]*10+3];pos[4]<d_primer[turn[ID_thread*8+6]*10+4];pos[4]++)
                                        {
                                                if(d_info[pos[4]*3]!=i)
                                                        continue;
                                                if((d_info[pos[4]*3+2]&2)!=2)
                                                        continue;
                                                for(pos[5]=d_primer[turn[ID_thread*8+7]*10+3];pos[5]<d_primer[turn[ID_thread*8+7]*10+4];pos[5]++)
                                                {
                                                        if(d_info[pos[5]*3]!=i)
                                                                continue;
                                                        if((d_info[pos[5]*3+2]&2)!=2)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_info[pos[1]*3+1]-(d_info[pos[0]*3+1]+d_primer[turn[ID_thread*8]*10+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=d_info[pos[2]*3+1]-d_info[pos[1]*3+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_info[pos[3]*3+1]-(d_info[pos[2]*3+1]+d_primer[turn[ID_thread*8+3]*10+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=(d_info[pos[4]*3+1]+d_primer[turn[ID_thread*8+6]*10+1]-1)-(d_info[pos[3]*3+1]+d_primer[turn[ID_thread*8+4]*10+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_info[pos[4]*3+1]+d_primer[turn[ID_thread*8+6]*10+1]-1-d_info[pos[1]*3+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_info[pos[5]*3+1]-(d_info[pos[4]*3+1]+d_primer[turn[ID_thread*8+6]*10+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //LF
                                                        if(turn[ID_thread*8+2]!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_primer[turn[ID_thread*8+2]*10+3];pos[6]<d_primer[turn[ID_thread*8+2]*10+4];pos[6]++)
                                                                {
                                                                        if(d_info[pos[6]*3]!=i)
                                                                                continue;
                                                                        if((d_info[pos[6]*3+2]&2)!=2)
                                                                                continue;
                                                                        if(d_info[pos[1]*3+1]+d_primer[turn[ID_thread*8+1]*10+1]>d_info[pos[6]*3+1])
                                                                                continue;
                                                                        if(d_info[pos[6]*3+1]+d_primer[turn[ID_thread*8+2]*10+1]>d_info[pos[2]*3+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                //LB
                                                        if(turn[ID_thread*8+5]!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_primer[turn[ID_thread*8+5]*10+3];pos[6]=d_primer[turn[ID_thread*8+5]*10+4];pos[6]++)
                                                                {
                                                                        if(d_info[pos[6]*3]!=i)
                                                                                continue;
                                                                        if((d_info[pos[6]*3+2]&1)!=1)
                                                                                continue;
                                                                        if(d_info[pos[3]*3+1]+d_primer[turn[ID_thread*8+4]*10+1]>d_info[pos[6]*3+1])
                                                                                continue;
                                                                        if(d_info[pos[6]*3+1]+d_primer[turn[ID_thread*8+5]*10+1]>d_info[pos[4]*3+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                        d_result[(8+const_int[7])*turn[ID_thread*8]+8+i]=1;
                                                }
                                        }
                                }
                        }
                }
        }
//minus
	for(pos[0]=d_primer[turn[ID_thread*8]*10+3];pos[0]<d_primer[turn[ID_thread*8]*10+4];pos[0]++)
        {
                if((d_info[pos[0]*3+2]&2)!=2)
                        continue;
                i=d_info[pos[0]*3];
                if(d_result[(8+const_int[7])*turn[ID_thread*8]+8+i]!=0)
                        continue;
                for(pos[1]=d_primer[turn[ID_thread*8+1]*10+3];pos[1]<d_primer[turn[ID_thread*8+1]*10+4];pos[1]++)
                {
                        if(d_info[pos[1]*3]!=i)
                                continue;
                        if((d_info[pos[1]*3+2]&2)!=2)
                                continue;
                        for(pos[2]=d_primer[turn[ID_thread*8+3]*10+3];pos[2]<d_primer[turn[ID_thread*8+3]*10+4];pos[2]++)
                        {
                                if(d_info[pos[2]*3]!=i)
                                        continue;
                                if((d_info[pos[2]*3+2]&1)!=1)
                                        continue;
                                for(pos[3]=d_primer[turn[ID_thread*8+4]*10+3];pos[3]<d_primer[turn[ID_thread*8+4]*10+4];pos[3]++)
                                {
                                        if(d_info[pos[3]*3]!=i)
                                                continue;
                                        if((d_info[pos[3]*3+2]&2)!=2)
                                                continue;
                                        for(pos[4]=d_primer[turn[ID_thread*8+6]*10+3];pos[4]<d_primer[turn[ID_thread*8+6]*10+4];pos[4]++)
                                        {
                                                if(d_info[pos[4]*3]!=i)
                                                        continue;
                                                if((d_info[pos[4]*3+2]&1)!=1)
                                                        continue;
                                                for(pos[5]=d_primer[turn[ID_thread*8+7]*10+3];pos[5]<d_primer[turn[ID_thread*8+7]*10+4];pos[5]++)
                                                {
                                                        if(d_info[pos[5]*3]!=i)
                                                                continue;
                                                        if((d_info[pos[5]*3+2]&1)!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=d_info[pos[0]*3+1]-(d_info[pos[1]*3+1]+d_primer[turn[ID_thread*8+1]*10+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=(d_info[pos[1]*3+1]+d_primer[turn[ID_thread*8+1]*10+1]-1)-(d_info[pos[2]*3+1]+d_primer[turn[ID_thread*8+3]*10+1]-1)-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=d_info[pos[2]*3+1]-(d_info[pos[3]*3+1]+d_primer[turn[ID_thread*8+4]*10+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=d_info[pos[3]*3+1]-d_info[pos[4]*3+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=d_info[pos[1]*3+1]+d_primer[turn[ID_thread*8+1]*10+1]-1-d_info[pos[4]*3+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=d_info[pos[4]*3+1]-(d_info[pos[5]*3+1]+d_primer[turn[ID_thread*8+7]*10+1]-1)-1;
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //LF
                                                        if(turn[ID_thread*8+2]!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_primer[turn[ID_thread*8+2]*10+3];pos[6]<d_primer[turn[ID_thread*8+2]*10+4];pos[6]++)
                                                                {
                                                                        if(d_info[pos[6]*3]!=i)
                                                                                continue;
                                                                        if((d_info[pos[6]*3+2]&1)!=1)
                                                                                continue;
                                                                        if(d_info[pos[2]*3+1]+d_primer[turn[ID_thread*8+3]*10+1]>d_info[pos[6]*3+1])
                                                                                continue;
                                                                        if(d_info[pos[6]*3+1]+d_primer[turn[ID_thread*8+2]*10+1]>d_info[pos[1]*3+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
                                                //LB
                                                        if(turn[ID_thread*8+5]!=-1)
                                                        {
                                                                dis=0;
                                                                for(pos[6]=d_primer[turn[ID_thread*8+5]*10+3];pos[6]=d_primer[turn[ID_thread*8+5]*10+4];pos[6]++)
                                                                {
                                                                        if(d_info[pos[6]*3]!=i)
                                                                                continue;
                                                                        if((d_info[pos[6]*3+2]&2)!=2)
                                                                                continue;
                                                                        if(d_info[pos[4]*3+1]+d_primer[turn[ID_thread*8+6]*10+1]>d_info[pos[6]*3+1])
                                                                                continue;
                                                                        if(d_info[pos[6]*3+1]+d_primer[turn[ID_thread*8+5]*10+1]>d_info[pos[3]*3+1])
                                                                                continue;
                                                                        dis=1;
                                                                        break;
                                                                }
                                                                if(dis==0)
                                                                        continue;
                                                        }
							d_result[(8+const_int[7])*turn[ID_thread*8]+8+i]=1;
                                                }
                                        }
                                }
                        }
                }
        }
	dis=0;
        for(i=0;i<const_int[7];i++)
        {
                dis=dis+d_result[(8+const_int[7])*turn[ID_thread*8]+8+i];
        }
        return dis;
}

__device__ int design_loop(int *d_primer,char *d_seq,int *d_info,int turn[],int ID_thread,int *d_result,double *d_DPT,int id,int *d_ps,char *d_numSeq,double *d_TH)
{
        int success;

//LF and LB 
        success=0;
	turn[ID_thread*8+2]=d_primer[turn[ID_thread*8+1]*10+9];
        while(turn[ID_thread*8+2]<const_int[2]+const_int[0]+const_int[1])
        {
		if(turn[ID_thread*8+2]==-1)
			break;
		if((d_primer[turn[ID_thread*8+2]*10+2]&2)!=2)
		{
			turn[ID_thread*8+2]++;
			continue;
		}
                if(d_primer[turn[ID_thread*8+2]*10]+18>d_primer[turn[ID_thread*8+3]*10])
                        break;
                turn[ID_thread*8+5]=d_primer[turn[ID_thread*8+3]*10+9];
		if(turn[ID_thread*8+5]==-1||d_primer[turn[ID_thread*8+5]*10]+18>d_primer[turn[ID_thread*8+6]*10])
			break;
                while(turn[ID_thread*8+5]<const_int[2]+const_int[0]+const_int[1])
                {
			if((d_primer[turn[ID_thread*8+5]*10+2]&1)!=1)
			{
				turn[ID_thread*8+5]++;
				continue;
			}
                        if(d_primer[turn[ID_thread*8+5]*10]+18>d_primer[turn[ID_thread*8+6]*10])
                                break;
                //check_common
                        if(const_int[3])
                        {
                                success=check_common(d_primer,d_info,turn,ID_thread,d_result);
                                if(success==0)
                                {
                                        turn[ID_thread*8+5]++;
                                        continue;
                                }
                        }
                //check_structure
                        if(const_int[6])
                        {
                                success=check_structure(d_seq,d_primer,turn,ID_thread,d_TH,id,d_DPT,d_ps,d_numSeq);
                                if(success==0)
                                {
                                        turn[ID_thread*8+5]++;
                                        continue;
                                }
                        }
                        success=1;
                        break;
                }
                if(success==1)
                        break;
                else
                        turn[ID_thread*8+2]++;
        }
        if(success==1)
                return success;
//only LF
        turn[ID_thread*8+2]=d_primer[turn[ID_thread*8+1]*10+9];
	turn[ID_thread*8+5]=-1;
        while(turn[ID_thread*8+2]<const_int[2]+const_int[1]+const_int[0])
        {
		if(turn[ID_thread*8+2]==-1)
			break;
                if(d_primer[turn[ID_thread*8+2]*10]+18>d_primer[turn[ID_thread*8+3]*10])
                        break;
		if((d_primer[4*turn[ID_thread*8+2]+2]&2)!=2)
		{
			turn[ID_thread*8+2]++;
			continue;
		}
        //check_common
                if(const_int[3])
                {
                        success=check_common(d_primer,d_info,turn,ID_thread,d_result);
                        if(success==0)
                        {
                                turn[ID_thread*8+2]++;
                                continue;
                        }
                }
        //check_structure
                if(const_int[6])
                {
                        success=check_structure(d_seq,d_primer,turn,ID_thread,d_TH,id,d_DPT,d_ps,d_numSeq);
                        if(success==0)
                        {
                                turn[ID_thread*8+2]++;
                                continue;
                        }
                }
                success=1;
                break;
        }
        if(success==1)
                return success;
//only LB
        turn[ID_thread*8+5]=d_primer[turn[ID_thread*8+3]*10+9];
	turn[ID_thread*8+2]=-1;
        while(turn[ID_thread*8+5]<const_int[2]+const_int[0]+const_int[1])
        {
		if(turn[ID_thread*8+5]==-1)
			break;
                if(d_primer[turn[ID_thread*8+5]*10]+18>d_primer[turn[ID_thread*8+6]*10])
                        break;
		if((d_primer[turn[ID_thread*8+5]*4+2]&1)!=1)
		{
			turn[ID_thread*8+5]++;
			continue;
		}
        //check_common
                if(const_int[3])
                {
                        success=check_common(d_primer,d_info,turn,ID_thread,d_result);
                        if(success==0)
                        {
                                turn[ID_thread*8+5]++;
                                continue;
                        }
                }
        //check_structure
                if(const_int[6])
                {
                        success=check_structure(d_seq,d_primer,turn,ID_thread,d_TH,id,d_DPT,d_ps,d_numSeq);
                        if(success==0)
                        {
                                turn[ID_thread*8+5]++;
                                continue;
                        }
                }
                success=1;
                break;
        }
        return success;
}

//caculate
__global__ void LAMP(char *d_seq,int *d_primer,int *d_info,int *d_result,double *d_TH,double *d_DPT,int *d_ps,char *d_numSeq)
//const_int: 0:numS,1:numL,2:numLp,3:common_flag,4:special_flag,5:loop_flag,6:secondary_flag,7:common_num,8:this turn common_num,9:high_GC_flag; 10:expect
{
	int id=blockDim.x*blockIdx.x+threadIdx.x;
	int flag;
	__shared__ int turn[8192];

	while(id<const_int[0])
	{
		d_result[id*(8+const_int[7])]=-1;//not LAMP, as a flag
//check add by F3'pos
		if((d_primer[id*10+2]&1)!=1)
		{
			id=id+blockDim.x*gridDim.x;	
			continue;
		}
	//combine
		turn[threadIdx.x*8]=id; //one thread, one F3
		flag=0;
		for(turn[threadIdx.x*8+1]=d_primer[id*10+7];turn[threadIdx.x*8+1]<const_int[0];turn[threadIdx.x*8+1]++) //F2
		{
			if(turn[threadIdx.x*8+1]==-1)
				break;
			if(flag!=0)
				break; //have find one LAMP primer
			if((d_primer[turn[threadIdx.x*8+1]*10+2]&1)!=1)
				continue;
			if(d_primer[turn[threadIdx.x*8+1]*10]-(d_primer[turn[threadIdx.x*8]*10]+d_primer[turn[threadIdx.x*8]*10+1])>20)
				break;
			for(turn[threadIdx.x*8+3]=d_primer[turn[threadIdx.x*8+1]*10+8];turn[threadIdx.x*8+3]<const_int[1]+const_int[0];turn[threadIdx.x*8+3]++) //F1c
			{
				if(turn[threadIdx.x*8+3]==-1)
					break;
				if(flag!=0)
					break;
				if((d_primer[turn[threadIdx.x*8+3]*10+2]&2)!=2)
					continue;
				if(d_primer[turn[threadIdx.x*8+3]*10]-d_primer[turn[threadIdx.x*8+1]*10]-1<40)
					continue;
                                if(d_primer[turn[threadIdx.x*8+3]*10]-d_primer[turn[threadIdx.x*8+1]*10]-1>60)
                                	break;
                                for(turn[threadIdx.x*8+4]=d_primer[turn[threadIdx.x*8+3]*10+7];turn[threadIdx.x*8+4]<const_int[1]+const_int[0];turn[threadIdx.x*8+4]++)   //B1c
                                {
                                        if(turn[threadIdx.x*8+4]==-1)
                                        	break;
					if(flag!=0)
						break;
					if((d_primer[turn[threadIdx.x*8+4]*10+2]&1)!=1)
						continue;
                                        if(d_primer[turn[threadIdx.x*8+4]*10]-d_primer[turn[threadIdx.x*8+3]*10]>85)
                                        	break;
                                        for(turn[threadIdx.x*8+6]=d_primer[turn[threadIdx.x*8+4]*10+8];turn[threadIdx.x*8+6]<const_int[0];turn[threadIdx.x*8+6]++)   //B2
                                        {
                                                if(turn[threadIdx.x*8+6]==-1)
                                                	break;
						if(flag!=0)
							break;
						if((d_primer[turn[threadIdx.x*8+6]*10+2]&2)!=2)
							continue;
                                                if((d_primer[turn[threadIdx.x*8+6]*10]+d_primer[turn[threadIdx.x*8+6]*10+1]-1)-(d_primer[turn[threadIdx.x*8+4]*10]+d_primer[turn[threadIdx.x*8+4]*10+1])<40)
                                                	continue;
                                                if((d_primer[turn[threadIdx.x*8+6]*10]+d_primer[turn[threadIdx.x*8+6]*10+1]-1)-(d_primer[turn[threadIdx.x*8+4]*10]+d_primer[turn[threadIdx.x*8+4]*10+1])>60)
                                                	break;
                                                if(d_primer[turn[threadIdx.x*8+6]*10]+d_primer[turn[threadIdx.x*8+6]*10+1]-1-d_primer[turn[threadIdx.x*8+1]*10]-1<120)
                                                	continue;
                                                if(d_primer[turn[threadIdx.x*8+6]*10]+d_primer[turn[threadIdx.x*8+6]*10+1]-1-d_primer[turn[threadIdx.x*8+1]*10]-1>180)
                                                	break;
						if(const_int[5]&&(d_primer[turn[threadIdx.x*8+1]*10+9]==-1||(d_primer[d_primer[turn[threadIdx.x*8+1]*10+9]*10]+18>d_primer[turn[threadIdx.x*8+3]*10]))&&(d_primer[turn[threadIdx.x*8+4]*10+9]==-1||(d_primer[d_primer[turn[threadIdx.x*8+4]*10+9]*10]+18>d_primer[turn[threadIdx.x*8+6]*10])))
							continue;
                                                for(turn[threadIdx.x*8+7]=d_primer[turn[threadIdx.x*8+6]*10+7];turn[threadIdx.x*8+7]<const_int[0];turn[threadIdx.x*8+7]++)  //B3
                                                {
                                                        if(turn[threadIdx.x*8+7]==-1)
                                                        	break;
							if((d_primer[turn[threadIdx.x*8+7]*10+2]&2)!=2)
								continue;
                                                        if(d_primer[turn[threadIdx.x*8+7]*10]-(d_primer[turn[threadIdx.x*8+6]*10]+d_primer[turn[threadIdx.x*8+6]*10+1])>20)
                                                        	break;
							flag=check_gc(d_seq,d_primer[turn[threadIdx.x*8]*10],(d_primer[turn[threadIdx.x*8+7]*10]+d_primer[turn[threadIdx.x*8+7]*10+1]),const_int[9]);
							if(flag==0)
								continue;
							if(const_int[4]!=0)
							{
								flag=check_uniq(d_primer,d_info,turn,threadIdx.x);
								if(flag==0)
									continue;
							}

							turn[threadIdx.x*8+2]=-1;
							turn[threadIdx.x*8+5]=-1; //loop
							if(const_int[3])
							{
								flag=check_common(d_primer,d_info,turn,threadIdx.x,d_result);
								if(flag<const_int[8])
								{
									flag=0;
									continue;
								}
							}
							if(const_int[6])
							{
								flag=check_structure(d_seq,d_primer,turn,threadIdx.x,d_TH,id,d_DPT,d_ps,d_numSeq);
								if(flag==0)
									continue;
							}
							if(const_int[5])
							{
								flag=design_loop(d_primer,d_seq,d_info,turn,threadIdx.x,d_result,d_DPT,id,d_ps,d_numSeq,d_TH);
								if(flag==0)
									continue;
							}
							d_result[id*(8+const_int[7])]=turn[threadIdx.x*8];
							d_result[id*(8+const_int[7])+1]=turn[threadIdx.x*8+1];
							d_result[id*(8+const_int[7])+2]=turn[threadIdx.x*8+2];
							d_result[id*(8+const_int[7])+3]=turn[threadIdx.x*8+3];
							d_result[id*(8+const_int[7])+4]=turn[threadIdx.x*8+4];
							d_result[id*(8+const_int[7])+5]=turn[threadIdx.x*8+5];
							d_result[id*(8+const_int[7])+6]=turn[threadIdx.x*8+6];
							d_result[id*(8+const_int[7])+7]=turn[threadIdx.x*8+7];
							break;
						}
					}
				}
			}
		}
		id=id+blockDim.x*gridDim.x;
	}
	__syncthreads();
}

void usage()
{
        printf("Usage:\n");
        printf("    LAMP_GPU  -in <name>  -out <result>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the name of candidate single primers file\n");
        printf("    -out  <string>:  the result file of LAMP primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers, default is current directory\n");
        printf("    -ref  <string>:  the reference sequence file used in single program, fasta format\n");
        printf("    -expect  <int>:  the number of LAMP primers needed to be design, default is 10\n"); 
        printf("    -common:         design common LAMP primers\n");
        printf("    -special:        design special LAMP primers\n");
        printf("    -check   <int>:  0: don't check tendency of the left primer to bind to the right primer; !=0: check, default is 1\n");
        printf("    -par  <string>:  the directory of storing parameter files used to check the tendency of two primers binding, default is Par/\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region, high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design LAMP primer with loop primers\n");
        printf("    -h/-help:        usage\n");
}

struct INFO *read_list(char *path,int common_num[])  
{
        char *in,name[301];
        int turn,i,size;
        struct INFO *new_primer,*p_primer,*head;
        FILE *fp;

        i=strlen(path);
        in=(char *)malloc(i+20);              
        memset(in,'\0',i+20);
        strcpy(in,path);
        strcat(in,"-common_list.txt");
        if(access(in,0)==-1)  
        {
                printf("Error! Don't have the %s file!\n",in);
                exit(1);           
        }
        fp=fopen(in,"r");
        if(fp==NULL)
        {          
                printf("Error: can't open the %s file!\n",in);
                exit(1);
        }

        size=sizeof(struct INFO);
        i=0;
        memset(name,'\0',301);
        while(fscanf(fp,"%s\t%d\n",name,&turn)!=EOF)
        {
                new_primer=(struct INFO *)malloc(size);
                new_primer->turn=turn;
                strcpy(new_primer->name,name);
                new_primer->next=NULL;

                if(i==0)
                {
                        head=new_primer;
                        p_primer=new_primer;
                        i++;
                }
                else
                {
                        p_primer->next=new_primer;
                        p_primer=new_primer;
                }
                memset(name,'\0',301);
        }
        fclose(fp);
        common_num[0]=turn;
        free(in);
        return head;
}

main(int argc,char **argv)
{
	int i,j,flag[12],expect,circle,have,common_num[1],num[11],max_loop,min_loop,count[3],block,thread;
	char *output,*prefix,*store_path,*path_fa,*inner,*outer,*loop,*par_path,*temp,*seq,*d_seq,primer[26],*d_numSeq;
	FILE *fp;
	struct Primer *headL,*headS,*headLoop,*tempL,*tempS,*tempLoop,*storeL,*storeS,*storeLoop; 
	struct Node *p_node,*p_temp;
	struct INFO *headList,*p_list;
	time_t start,end;
	double *H_parameter;	
	long int memory;
	hipDeviceProp_t prop;
	int *d_primer,*d_info,*d_result;
	int *h_primer,*h_info,*h_result,h_int[19],*h_pos,*d_ps;
	double *h_TH,*d_TH,*d_DPT;
	
	expect=10; //default output max 10 LAMP primers
	start=time(NULL);
/////read the parameters
        for(i=0;i<=11;i++)
                flag[i]=0;
	for(i=0;i<10;i++)
		h_int[i]=0;
        flag[7]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        prefix=(char *)malloc(j+1);
                        memset(prefix,'\0',j+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        output=(char *)malloc(j+1);
                        memset(output,'\0',j+1);
                        strcpy(output,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        if(argv[i+1][j-1]=='/')
                        {
                                store_path=(char *)malloc(j+1);
                                memset(store_path,'\0',j+1);
                                strcpy(store_path,argv[i+1]);
                        }
                        else
                        {
                                store_path=(char *)malloc(j+2);
                                memset(store_path,'\0',j+2);
                                strcpy(store_path,argv[i+1]);
                                store_path[j]='/';
                        }
                        i=i+2;
                }
                else if(strcmp(argv[i],"-ref")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-ref\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        path_fa=(char *)malloc(j+1);
                        memset(path_fa,'\0',j+1);
                        strcpy(path_fa,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-expect")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        expect=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[8]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[9]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[10]=1;
			h_int[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
                else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[7]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[11]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        j=strlen(argv[i+1]);
                        if(argv[i+1][j-1]=='/')
                        {
                                par_path=(char *)malloc(j+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[j]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(j+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[j]='/';
                                par_path[j+1]='\0';
                        }
                        i=i+2;
                }
                else if(strcmp(argv[i],"-common")==0)
                {
                        flag[5]=1;
			h_int[3]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-special")==0)
                {
                        flag[6]=1;
			h_int[4]=1;
                        i++;
                }
                else
                {
                        printf("Warning! The parameter of %s is invalid.\n\n",argv[i]);
                        i++;
                }
        }
//check parameters
        if(flag[0]==0)
        {
                printf("Error! Users must supply the name of candidate single primers file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the name of output file with -out!\n");
                usage();
                exit(1);
        }
        if(flag[3]==0)
        {
                printf("Error! Users must supply the reference sequence file with -ref!\n");
                usage();
                exit(1);
        }
        if(flag[8]+flag[9]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
//prepare
	if(flag[7]!=0)
		h_int[6]=1;
	else
		h_int[6]=0;
	h_int[9]=flag[8];
	h_int[10]=expect;
        if(flag[2]==0)
        {
                temp=(char *)malloc(4096);
                memset(temp,'\0',4096);
                getcwd(temp,4096);
                j=strlen(temp);
                store_path=(char *)malloc(j+2);
                memset(store_path,'\0',j+2);
                strcpy(store_path,temp);
                free(temp);
                store_path[j]='/';
        }
//secondary
        if(flag[7]&&flag[11]==0)
        {
                temp=(char *)malloc(4096);
                memset(temp,'\0',4096);
                getcwd(temp,4096);
                j=strlen(temp);
                par_path=(char *)malloc(j+10);
                memset(par_path,'\0',j+10);
                strcpy(par_path,temp);
                free(temp);

                j--;
                while(par_path[j]!='/'&&j>=0)
                {
                        par_path[j]='\0';
                        j--;
                }
                strcat(par_path,"Par/");
        }
        if(flag[7])
        {
                H_parameter=(double *)malloc(5730*sizeof(double));
                memset(H_parameter,'\0',5730*sizeof(double));

                getStack(par_path,H_parameter);
                getStackint2(par_path,H_parameter);
                getDangle(par_path,H_parameter);
                getLoop(par_path,H_parameter);
                getTstack(par_path,H_parameter);
                getTstack2(par_path,H_parameter);
                tableStartATS(6.9,H_parameter);
                tableStartATH(2200.0,H_parameter);
		hipMemcpyToSymbol(HIP_SYMBOL(parameter),H_parameter,5730*sizeof(double));
		free(H_parameter);

		h_int[11]=0; //F3,plus
		h_int[12]=0;
		h_int[13]=1; //LF,minus
		h_int[14]=1;//F1c
		h_int[15]=0;//B1c
		h_int[16]=0;
		h_int[17]=1;
		h_int[18]=1;
        }
//F3's pos 
	h_pos=(int *)malloc(expect*sizeof(int));
	for(i=0;i<expect;i++)
		h_pos[i]=-1;
//directory for single primers
        j=strlen(store_path)+strlen(prefix)+12;
        outer=(char *)malloc(j);
        memset(outer,'\0',j);
        strcpy(outer,store_path);

        inner=(char *)malloc(j);
        memset(inner,'\0',j);
        strcpy(inner,store_path);

        if(flag[10]==1)
        {
                loop=(char *)malloc(j);
                memset(loop,'\0',j);
                strcpy(loop,store_path);
        }
        if(flag[8]==1)
        {                       
                strcat(outer,"high-outer/");
                strcat(outer,prefix);
                strcat(inner,"high-inner/");
                strcat(inner,prefix);
                if(flag[10]==1)
                {
                        strcat(loop,"high-loop/");
                        strcat(loop,prefix);
                }
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(outer,prefix);
                strcat(inner,"low-inner/");
                strcat(inner,prefix);
                if(flag[10]==1)
                {
                        strcat(loop,"low-loop/");
                        strcat(loop,prefix);
                }
        }

//reference sequence fa
        if(access(path_fa,0)==-1)
        {
                printf("Error! Don't have the %s file!\n",path_fa);
                exit(1);
        }
        i=file_size2(path_fa);
        i=i+100;
        temp=(char *)malloc(i*sizeof(char));
        memset(temp,'\0',i*sizeof(char));
        fp=fopen(path_fa,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the sequence file %s\n",path_fa);
                exit(1);
        }

        fread(temp,i*sizeof(char),1,fp);
        fclose(fp); 
        seq=(char *)malloc(i*sizeof(char));
        memset(seq,'\0',i*sizeof(char));
        
        j=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
                if(temp[i]=='a'||temp[i]=='A')
                        seq[j]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[j]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[j]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[j]='G';
                else
                        seq[j]='N';
                i++;
                j++;
        }
        free(temp);
	num[0]=j; //the length of genome
	hipMalloc((void **)&d_seq,num[0]);
	hipMemset(d_seq,'\0',num[0]);
	hipMemcpy(d_seq,seq,num[0],hipMemcpyHostToDevice);

//common-list
        if(flag[5])
        {
                headList=read_list(inner,common_num);
                common_num[0]++;
        }
        else
                common_num[0]=1;
	h_int[7]=common_num[0];
//read parameters
        headS=read_par(outer,flag[5],flag[6]);
        headL=read_par(inner,flag[5],flag[6]);
        if(flag[10])
        {
                headLoop=read_par(loop,flag[5],0);
                tempLoop=headLoop;
                while(tempLoop->next!=NULL)
                        tempLoop=tempLoop->next;
                max_loop=tempLoop->pos;
		min_loop=headLoop->pos;
        }

//common statistics
        if(flag[5])
        {
                how_many(headL,common_num[0]);
		how_many(headS,common_num[0]);
                if(flag[10])
                        how_many(headLoop,common_num[0]);
        }

	hipGetDeviceProperties(&prop,0); //read parameters
	thread=200;
	fp=fopen(output,"w");
        if(fp==NULL)
        {
                printf("Error: can't create the %s file!\n",output);
                exit(1);
        }
	have=1;
        end=time(NULL);
        printf("The prepare time is %0.1f seconds!\n",difftime(end,start));
        start=time(NULL);

//LAMP-GPU
	for(circle=common_num[0];circle>=1;circle--)
	{
		if(have>expect)
			break;
		storeL=headL;
		storeS=headS;
		while((storeL->pos<=storeS->pos+18)&&storeL!=NULL)
			storeL=storeL->next;
		if(flag[10])
		{
                        storeLoop=headLoop;
			while((storeLoop->pos<=storeS->pos+18)&&storeLoop!=NULL)
				storeLoop=storeLoop->next;
		}
		num[10]=0;
		while(storeS)
		{
			if(have>expect)
				break;
			if(num[10]==1)  //don't have enough primers
				break;
			for(i=1;i<9;i++)
			{
				num[i]=0;
			}
			memory=num[0]/2;
		//statistics	
			tempL=storeL;
			tempS=storeS;
			if(flag[10])
				tempLoop=storeLoop;
			while(tempS&&(memory<prop.totalGlobalMem/6)&&num[2]<20000)
			{
				if(flag[10]&&(tempS->pos+200)<min_loop)
					continue;
				if(flag[10]&&(tempS->pos-200)>max_loop)
					break;
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
				while(tempL&&(tempL->pos<tempS->pos))
				{
					if(tempL->total<circle)
					{
						tempL=tempL->next;
						continue;
					}
					num[1]++;
					if(flag[5])
					{
						num[3]=num[3]+tempL->total_common;
						memory=memory+3*tempL->total_common; 
					}
					if(flag[6])
					{
						num[4]=num[4]+tempL->total_special;
						memory=memory+3*tempL->total_special;
					}
					memory=memory+10; //one primer, 10 int
					tempL=tempL->next;
				}
				
				while(flag[10]&&tempLoop&&(tempLoop->pos<tempS->pos))
                                {
                                        if(tempLoop->total<circle)
                                        {
                                                tempLoop=tempLoop->next;
                                                continue;
                                        }
                                        num[7]++;
                                        if(flag[5])
                                        {
                                                num[8]=num[8]+tempLoop->total_common;
                                                memory=memory+3*tempLoop->total_common; 
                                        }
                                        memory=memory+10;
                                        tempLoop=tempLoop->next;
                                }

				num[2]++;
				if(flag[5])
				{
					num[5]=num[5]+tempS->total_common;
					memory=memory+3*tempS->total_common;
				}
				if(flag[6])
				{
					num[6]=num[6]+tempS->total_special;
					memory=memory+3*tempS->total_special;
				}
				memory=memory+18+common_num[0];//18=10(primers)+8(result_turn)
				if(flag[7])
					memory=memory+5000+50+27+12+13*4; //one double=4 int, DPT; 50: ps1+ps2; 27: numSeq1+numSeq2, char
				tempS=tempS->next;
			}
			if(num[2]<4||num[1]<2||(flag[10]&&num[7]<1)) //don't have enough primers
			{
				num[10]=1;
				break;
			}
			if(tempS==NULL)  //check all primers
				num[10]=1;	

			printf("memory is %ld\n",2*memory);
		//malloc
			h_primer=(int *)malloc(10*(num[2]+num[1]+num[7])*sizeof(int));
			hipMalloc((void **)&d_primer,10*(num[2]+num[1]+num[7])*sizeof(int));
			if(flag[5]||flag[6])
			{
				h_info=(int *)malloc(3*(num[5]+num[3]+num[8]+num[6]+num[4])*sizeof(int));
				hipMalloc((void **)&d_info,3*(num[5]+num[3]+num[8]+num[6]+num[4])*sizeof(int));
			}
		
			tempS=storeS;
			for(i=0;i<2;i++)
				count[i]=0;
			while(count[0]<num[2])
			{
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
		//primer info
				h_primer[10*count[0]]=tempS->pos;
				h_primer[10*count[0]+1]=tempS->len;
				h_primer[10*count[0]+2]=tempS->strand;
		//common
				if(flag[5])
				{
					h_primer[10*count[0]+3]=count[1];
					if(tempS->total_common==0)
						h_primer[10*count[0]+4]=-1;
					else
					{
						p_node=tempS->common;
						while(p_node)
						{
							h_info[3*count[1]]=p_node->gi;
							h_info[3*count[1]+1]=p_node->pos;
							h_info[3*count[1]+2]=p_node->strand; 
							count[1]++;
							p_node=p_node->next;
						}
						h_primer[10*count[0]+4]=count[1];
					}
				}
			//special
				if(flag[6])
				{
					h_primer[10*count[0]+5]=count[1];
                	        	if(tempS->total_special==0)
                	        	        h_primer[10*count[0]+6]=-1;
                	        	else
                	        	{
                	        	        p_node=tempS->special;
                	        	        while(p_node)
                	        	        {
                	        	                h_info[3*count[1]]=p_node->gi;
                	        	                h_info[3*count[1]+1]=p_node->pos;
                	        	                h_info[3*count[1]+2]=p_node->strand;
                	        	                count[1]++;
                	        	                p_node=p_node->next;
                	        	        }
                	        	        h_primer[10*count[0]+6]=count[1];
                	        	}
				}
				count[0]++;
				tempS=tempS->next;
			}
			h_int[0]=num[2];

		//large primer
        	        tempL=storeL;
        	        while(count[0]<num[1]+num[2])
        	        {
				if(tempL->total<circle)
				{
					tempL=tempL->next;
					continue;
				}
                	//primer info
                	        h_primer[10*count[0]]=tempL->pos;
                	        h_primer[10*count[0]+1]=tempL->len;
                	        h_primer[10*count[0]+2]=tempL->strand;
                	//common
				if(flag[5])
				{
                	        	h_primer[10*count[0]+3]=count[1];
                	        	if(tempL->total_common==0)
                	        	        h_primer[10*count[0]+4]=-1;
                	        	else
                	        	{
                	        	        p_node=tempL->common;
                	        	        while(p_node)
                	        	        {
                	        	                h_info[3*count[1]]=p_node->gi;
                	        	                h_info[3*count[1]+1]=p_node->pos;
                	        	                h_info[3*count[1]+2]=p_node->strand; 
                        		                count[1]++;
                        		                p_node=p_node->next;
                        		        }
                        		        h_primer[10*count[0]+4]=count[1];
					}
                        	}
                	//special
				if(flag[6])
				{
                        		h_primer[10*count[0]+5]=count[1];
                        		if(tempL->total_special==0)
                        		        h_primer[10*count[0]+6]=-1;
                        		else
                        		{
                        		        p_node=tempL->special;
                        		        while(p_node)
                        		        {
                        		                h_info[3*count[1]]=p_node->gi;
                        		                h_info[3*count[1]+1]=p_node->pos;
                        		                h_info[3*count[1]+2]=p_node->strand;
                        		                count[1]++;
                        		                p_node=p_node->next;
                        		        }
                        		        h_primer[10*count[0]+6]=count[1];
                        		}
				}
                        	count[0]++;
                        	tempL=tempL->next;
                	}
			h_int[1]=num[1];

                //loop primer
			if(flag[10])
			{
	                        tempLoop=storeLoop;
	                        while(count[0]<num[7]+num[1]+num[2])
                        	{
                                	if(tempLoop->total<circle)
                                	{
                                        	tempLoop=tempLoop->next;
                                        	continue;
                                	}
                        	//primer info
                                	h_primer[10*count[0]]=tempLoop->pos;
                                	h_primer[10*count[0]+1]=tempLoop->len;
                                	h_primer[10*count[0]+2]=tempLoop->strand;
                        	//common
                                	if(flag[5])
                                	{
                                	        h_primer[10*count[0]+3]=count[1];
                                	        if(tempLoop->total_common==0)
                                	                h_primer[10*count[0]+4]=-1;
                                	        else
                                	        {
                                	                p_node=tempLoop->common;
                                	                while(p_node)
                                	                {
                                	                        h_info[3*count[1]]=p_node->gi;
                                	                        h_info[3*count[1]+1]=p_node->pos;
                                	                        h_info[3*count[1]+2]=p_node->strand; 
                                	                        count[1]++;
                                	                        p_node=p_node->next;
                                	                }
                                	                h_primer[10*count[0]+4]=count[1];
                                	        }
                                	}
	                                count[0]++;
	                                tempLoop=tempLoop->next;
	                        }
				h_int[2]=num[7];
                        }
	//run
			if(num[2]%thread==0)
                                block=num[2]/thread;
                        else
                                block=(num[2]-num[2]%thread)/thread+1;

			if(block>prop.maxGridSize[0]/2)
				block=prop.maxGridSize[0]/2;

			hipMemcpy(d_primer,h_primer,10*count[0]*sizeof(int),hipMemcpyHostToDevice);
			if(flag[5]||flag[6])
			{
				hipMemcpy(d_info,h_info,3*count[1]*sizeof(int),hipMemcpyHostToDevice);
				free(h_info);
			}
		//next primer
		printf("block is %d,thread is %d\n",block,thread);
			next_one<<<block,thread>>>(d_primer,0,num[2],0,num[2],7);//outer-self
			next_one<<<block,thread>>>(d_primer,num[2],(num[1]+num[2]),num[2],(num[1]+num[2]),7);//inner-self
			next_one<<<block,thread>>>(d_primer,0,num[2],num[2],(num[1]+num[2]),8);//outer_to_inner
        	        next_one<<<block,thread>>>(d_primer,num[2],(num[1]+num[2]),0,num[2],8);//inner_to_outer
			if(flag[10])
			{
				next_one<<<block,thread>>>(d_primer,(num[1]+num[2]),(num[1]+num[2]+num[7]),(num[1]+num[2]),(num[1]+num[2]+num[7]),7);//loop-self
                        	next_one<<<block,thread>>>(d_primer,0,num[2],(num[1]+num[2]),(num[1]+num[2]+num[7]),9);//outer_to_loop
                        	next_one<<<block,thread>>>(d_primer,num[2],(num[1]+num[2]),(num[1]+num[2]),(num[1]+num[2]+num[7]),9);//inner_to_loop
			}
		//calculate
			h_int[8]=circle;
			hipMemcpyToSymbol(HIP_SYMBOL(const_int),h_int,19*sizeof(int));
			hipMalloc((void **)&d_result,num[2]*(8+common_num[0])*sizeof(int));
		hipMalloc((void **)&d_TH,2*num[2]*sizeof(double));
		h_TH=(double *)malloc(2*num[2]*sizeof(double));
			hipMalloc((void **)&d_DPT,num[2]*1263*sizeof(double));
			hipMalloc((void **)&d_ps,num[2]*62*sizeof(int));
			hipMalloc((void **)&d_numSeq,num[2]*54*sizeof(char));
			LAMP<<<block,thread>>>(d_seq,d_primer,d_info,d_result,d_TH,d_DPT,d_ps,d_numSeq);
			hipFree(d_DPT);
			hipFree(d_ps);
			hipFree(d_numSeq);
		hipMemcpy(h_TH,d_TH,2*num[2]*sizeof(double),hipMemcpyDeviceToHost);
		hipFree(d_TH);
		printf("%lf\t%lf\n",h_TH[0],h_TH[1]);
		free(h_TH);
			h_result=(int *)malloc((8+common_num[0])*num[2]*sizeof(int));
			memset(h_result,'\0',(8+common_num[0])*num[2]*sizeof(int));
			hipMemcpy(h_result,d_result,(8+common_num[0])*num[2]*sizeof(int),hipMemcpyDeviceToHost);
			hipFree(d_result);
	//free
			hipFree(d_primer);
			if(flag[5]||flag[6])
				hipFree(d_info);
		//LAMP primers, output
			for(i=0;i<num[2];i++)
			{
				if(have>expect)
					break;
				j=h_result[i*(8+common_num[0])];
				if(j==-1)
					continue;
				if(check_add(h_primer[j*10],h_pos,have)==0)
					continue;
				fprintf(fp,"The %d LAMP primers:\n",have);
		                generate_primer(seq,primer,h_primer[10*j],h_primer[j*10+1],0);
		                fprintf(fp,"  F3: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[10*j],h_primer[j*10+1],primer);
				j=h_result[i*(8+common_num[0])+1];
		                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],0);
		                fprintf(fp,"  F2: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[10*j],h_primer[j*10+1],primer);
				j=h_result[i*(8+common_num[0])+3];
		                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],1);
		                fprintf(fp,"  F1c: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
				j=h_result[i*(8+common_num[0])+4];
		                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],0);
		                fprintf(fp,"  B1c: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
				j=h_result[i*(8+common_num[0])+6];
		                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],1);
		                fprintf(fp,"  B2: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
				j=h_result[i*(8+common_num[0])+7];
		                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],1);
		                fprintf(fp,"  B3: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
                		if(flag[10])
                		{
					j=h_result[i*(8+common_num[0])+2];
                        		if(j==-1)
                        		        fprintf(fp,"  LF: NULL\n");
                        		else
                        		{
                        		        generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],1);
                        		        fprintf(fp,"  LF: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
                        		}

					j=h_result[i*(8+common_num[0])+5];
		                        if(j==-1)
		                                fprintf(fp,"  LB: NULL\n");
		                        else
		                        {
		                                generate_primer(seq,primer,h_primer[j*10],h_primer[j*10+1],0);
		                                fprintf(fp,"  LB: pos:%d,length:%d bp, primer(5'-3'):%s\n",h_primer[j*10],h_primer[j*10+1],primer);
		                        }
		                }
		                if(flag[5])
		                {
					j=0;
		                        fprintf(fp,"  This set of LAMP primers could be used in %d genomes, there are: ",circle);
		                        p_list=headList;
                        		for(j=0;j<common_num[0];j++)
                        		{
                                		if(h_result[i*(8+common_num[0])+8+j]==0)
                                		        continue;
                                		while(p_list)
                                		{
                                		        if(p_list->turn==j)
                                		                break;
                                		        else
                                		                p_list=p_list->next;
                                		}
                                		if(j==0)
                                		        fprintf(fp,"%s",p_list->name);
                                		else
                                		        fprintf(fp,", %s",p_list->name);
                                		j++;
	       	                 	}
		                        fprintf(fp,"\n");
				}
				h_pos[have-1]=h_primer[h_result[i*(8+common_num[0])]*10];
				have++;
			}
			free(h_result);
			free(h_primer);
			if(have>expect)
				break;
		//new primer start
			if(tempS==NULL)
				storeS=tempS;
			else
			{
				while(tempS->pos-storeS->pos>300)
				{
					storeS=storeS->next;
				}
			}
		}//one circle
	}
	hipFree(d_seq);
	free(h_pos);
	free(seq);
	free(output);
        free(prefix);
        free(store_path);
        free(inner);
        free(outer);
        free(path_fa);
	fclose(fp);
//free struct list
        while(headL)
        {
                p_node=headL->common;
                while(p_node)
                {
                        p_temp=p_node->next;
                        free(p_node);
                        p_node=p_temp;
                }
                p_node=headL->special;
                while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
                
                storeL=headL->next;
                free(headL);
                headL=storeL;
        }
        while(headS)
        {
                p_node=headS->common;  
                while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
                p_node=headS->special;
                while(p_node)
                {               
                        p_temp=p_node->next;
                        free(p_node);
                        p_node=p_temp;
                }

                storeS=headS->next;
                free(headS);
                headS=storeS;
        }

        if(flag[5])
        {
                while(headList)
                {
                        p_list=headList->next;
                        free(headList);
                        headList=p_list;
                }
        }

        if(flag[7]||flag[11])
                free(par_path);

        if(flag[10])
        {
                free(loop);
                while(headLoop)
                {
                        p_node=headLoop->common;
                        while(p_node)
                        {
                                p_temp=p_node->next;
                                free(p_node);
                                p_node=p_temp;
                        }
                        p_node=headLoop->special;
                        while(p_node)
                        {
                                p_temp=p_node->next;
                                free(p_node);
                                p_node=p_temp;
                        }

                        storeLoop=headLoop->next;
                        free(headLoop);
                        headLoop=storeLoop;
                }
        }	
	end=time(NULL);
        printf("the time for design is %0.1f seconds!\n",difftime(end,start));
}
