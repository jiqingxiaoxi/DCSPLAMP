#include "hip/hip_runtime.h"
//gpu version
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<regex.h>
#include<sys/types.h>
#include<sys/stat.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

struct Node
{
	int pos;
	int gi;
	int plus;  //as a flag, 1 is OK, 0 is no
	int minus; //as a flag
	struct Node *next;
};

struct Primer
{
	int pos;
	int len;
	int plus;
	int minus;
	int total_common;
	int total_special;
	int total; //common number
	struct Primer *next;
	struct Node *common;
	struct Node *special;
};

void take_regulate(regmatch_t pmatch[],int which,char *out,char *input)
{
        int i,j=0;
        for(i=pmatch[which].rm_so;i<pmatch[which].rm_eo;i++)
        {
                out[j]=input[i];
                j++;
        }
        out[j]='\0';
}

int check_add(int F3_pos,int *par,int have)
{
        int i,dis;

        for(i=0;i<have;i++)
        {
                dis=par[i]-F3_pos;
                if(abs(dis)<300)              
                        return 0;
        }
        return 1;        
}

void primer(char *seq,char primer[],int start,int length,int flag)
{
        char read[270];
        int i;
        
        for(i=start;i<start+length;i++)
                read[i-start]=seq[i];


        if(flag==0)
        {
                for(i=0;i<length;i++)
                {
                        if(read[i]=='A'||read[i]=='a')
                                primer[i]='A';
                        else if(read[i]=='T'||read[i]=='t')
                                primer[i]='T';
                        else if(read[i]=='C'||read[i]=='c')
                                primer[i]='C';    
                        else if(read[i]=='G'||read[i]=='g')
                                primer[i]='G';
                        else
                                primer[i]='N';
                }
                primer[i]='\0';
        }
        else
        {
                for(i=0;i<length;i++)
                {
                        if(read[length-1-i]=='A'||read[length-1-i]=='a')
                                primer[i]='T';
                        else if(read[length-1-i]=='T'||read[length-1-i]=='t')
                                primer[i]='A';
                        else if(read[length-1-i]=='C'||read[length-1-i]=='c')
                                primer[i]='G';
                        else if(read[length-1-i]=='G'||read[length-1-i]=='g')
                                primer[i]='C';
                        else 
                                primer[i]='N';
                }
                primer[length]='\0';
        }
}

int secondary_check(int expect,int have,int circle,int *par,char *file,FILE *LAMP,int *h_result,int *apply,char *seq,int common,int *list)
{
	FILE *fp;       
        char line[1000],result[20],pattern1[100],pattern2[100],pattern3[50],F3[30],F2[30],F1c[30],B1c[30],B2[30],B3[30];
        int flag,cflags,value[3],count,i,j;
        regex_t reg1,reg2,reg3;
        regmatch_t pmatch[4];
        float TH,max_TH;
        
        strcpy(pattern1,"SEQUENCE.+\\=(\\w+)\\-(\\w+)\\-(\\w+)");
        cflags=REG_EXTENDED;
        regcomp(&reg1,pattern1,cflags);
        strcpy(pattern2,"PRIMER\\_LEFT\\_0\\_TM\\=(.+)$");
        regcomp(&reg2,pattern2,cflags);
        strcpy(pattern3,"PRIMER.PAIR.+COMPL.+\\=(.+)$");
        regcomp(&reg3,pattern3,cflags);

        fp=fopen(file,"r");
        if(fp==NULL)
        {
                printf("can't open temp-result.txt file\n");
                exit(1);
        }
	count=have;
        while(fgets(line,1000,fp)!=NULL)
        {
                if(regexec(&reg1,line,4,pmatch,0)==0)  //begin
                {
                        take_regulate(pmatch,3,result,line);
                        value[2]=atoi(result); //Primer_turn in LAMP
                        if(value[2]==1) //a new LAMP primers
                        {
                                flag=1;
                        //check F3 pos
                                take_regulate(pmatch,2,result,line);
                                value[1]=atoi(result); //F3_pos
                                flag=check_add(value[1],par,count);
				if(flag==0)
					continue;
                        }

                        if(value[2]==45&&flag)
                        {
                                take_regulate(pmatch,1,result,line);
                                value[0]=atoi(result);
                        }
                        continue;
                }

                if(regexec(&reg2,line,2,pmatch,0)==0&&value[2]==1&&flag) //the max TH
                {
                        take_regulate(pmatch,1,result,line);
                        max_TH=atof(result);
                        continue;
                }
                if(regexec(&reg3,line,2,pmatch,0)==0)
                {
                        take_regulate(pmatch,1,result,line);
                        TH=atof(result);
                        if(TH>max_TH-10)
                                flag=0;
                        continue;                       
                }
                if(line[0]=='='&&value[2]==45&&flag==1)
                {
		//add
			primer(seq,F3,h_result[12*value[0]],h_result[12*value[0]+1],0);
			primer(seq,F2,h_result[12*value[0]+2],h_result[12*value[0]+3],0);
			primer(seq,F1c,h_result[12*value[0]+4],h_result[12*value[0]+5],1);
                        primer(seq,B1c,h_result[12*value[0]+6],h_result[12*value[0]+7],0);
                        primer(seq,B2,h_result[12*value[0]+8],h_result[12*value[0]+9],1);
                        primer(seq,B3,h_result[12*value[0]+10],h_result[12*value[0]+11],1);

			fprintf(LAMP,"The %d LAMP primer can be used in %d genomes:\n",(count+1),circle);
			fprintf(LAMP,"this LAMP primer can be used in:");
			i=0;
			for(j=0;j<common;j++)
			{
				if(apply[common*value[0]+j]==0)
					continue;
				i++;
				if(i==circle)
					fprintf(LAMP,"%d\n",list[j]);
				else
					fprintf(LAMP,"%d,",list[j]);
			}
			if(circle==common)
				fprintf(LAMP,"this LAMP primer can't be use in:None\n");
			else
			{
				i=0;
				for(j=0;j<common;j++)
				{
					if(apply[common*value[0]+j]==1)
						continue;
					i++;
					if(i==(common-circle))
						fprintf(LAMP,"%d\n",list[j]);
					else
						fprintf(LAMP,"%d,",list[j]);
				}
			}
			fprintf(LAMP,"  F3 start pos:%d, length:%d bp, sequence:%s\n",h_result[12*value[0]],h_result[12*value[0]+1],F3);
			fprintf(LAMP,"  F2 start pos:%d, length:%d bp, sequence:%s\n",h_result[12*value[0]+2],h_result[12*value[0]+3],F2);
			fprintf(LAMP,"  F1c start pos:%d, length:%d bp, sequence:%s\n",h_result[12*value[0]+4],h_result[12*value[0]+5],F1c);
			fprintf(LAMP," FIP sequence:%s-%s\n",F1c,F2);
			fprintf(LAMP," BIP sequence:%s-%s\n",B1c,B2);
			fprintf(LAMP,"  B1c start pos:%d, length:%d bp, sequence:%s\n",h_result[12*value[0]+6],h_result[12*value[0]+7],B1c);
			fprintf(LAMP,"  B2 start pos:%d, length:%d bp, sequence:%s\n",h_result[12*value[0]+8],h_result[12*value[0]+9],B2);
			fprintf(LAMP,"  B3 start pos:%d, length:%d bp, sequence:%s\n\n",h_result[12*value[0]+10],h_result[12*value[0]+11],B3);

			par[count]=h_result[12*value[0]];
			count++;
			if(count==expect)
				return count;
                }
        }
        fclose(fp);
        regfree(&reg1);
        regfree(&reg2);
        regfree(&reg3);
	return count;
}
void how_many(struct Primer *head,int common)
{
        struct Primer *p_primer;
        struct Node *p_node;
        int i,num,*list;

	list=(int *)malloc(common*sizeof(int));
        p_primer=head;
        while(p_primer)
        {
                p_node=p_primer->common;
		for(i=0;i<common;i++)
                {
                        list[i]=0;
                }
                i=0;
                while(p_node)
                {
                        i++;
			list[p_node->gi]=1;
                        p_node=p_node->next;
                }
		p_primer->total_common=i;

        //special
                p_node=p_primer->special;
                i=0;
                while(p_node)       
                {
                        i++;
                        p_node=p_node->next;
                }
		p_primer->total_special=i;

		num=0;
                for(i=0;i<common;i++)
                {
                        num=num+list[i];
                }
		p_primer->total=num;
                p_primer=p_primer->next;
        }
	free(list);
}

//get the file size
int file_size2(char* filename)
{
        struct stat statbuf;
        stat(filename,&statbuf);
        int size=statbuf.st_size;
        return size;
}

////function read primer informatin and align information 
struct Primer *read_par(char path[],int common)
{
        char in[2000];
        int pos,len,gi,position,plus,minus,size,i;
        struct Primer *new_primer,*p_primer,*head;
        struct Node *new_node,*p_node;
        FILE *fp;

///read the  primer file
        memset(in,'\0',2000*sizeof(char));       
        strcpy(in,path);
        strcat(in,".txt");  //suffix of primer candidate file
        fp=fopen(in,"r");
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",in);
                exit(1);
        }
        
        size=sizeof(struct Primer);
        i=0;
        while(fscanf(fp,"pos:%d\tlength:%d\t+:%d\t-:%d\n",&pos,&len,&plus,&minus)!=EOF)
        {
                new_primer=(struct Primer *)malloc(size);
                new_primer->pos=pos;
                new_primer->len=len;
                new_primer->total_common=0;
		new_primer->total_special=0;
		new_primer->total=0;
                new_primer->plus=plus;
                new_primer->minus=minus;
                new_primer->next=NULL;
                new_primer->common=NULL;
                new_primer->special=NULL;

                if(i==0)
                {
                        head=new_primer;
                        p_primer=new_primer;
                        i++;
                }
                else
                {
                        p_primer->next=new_primer;
                        p_primer=new_primer;
                }
                
        }
        fclose(fp);
        if(i==0)
        {
                printf("Sorry! Don't have candidate primer!\n");
                exit(1);
        }

//parameter of common
        memset(in,'\0',2000*sizeof(char));
        strcpy(in,path);
        strcat(in,"-common.txt"); //suffix of parameter
        fp=fopen(in,"r");
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",in);
                exit(1);
        }

        p_primer=head;
        size=sizeof(struct Node);
        while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
        {
        //check the common
                if(gi>=common)
                {
                        gi++;
                        printf("ERROR!\n The primer(pos is %d and length is %d) from %s can be used in more than %d GIs, but the common parameter is %d! You should check the common parameter!\n",pos,len,in,gi,common);
                        exit(1);
                } 
                new_node=(struct Node *)malloc(size);
                new_node->pos=position;
                new_node->gi=gi;
                new_node->plus=plus;
                new_node->minus=minus;

        //find the primer
                while(p_primer->pos!=pos||p_primer->len!=len)
                {
                        if(p_primer->next==NULL)
                        {
                                p_primer=head;
                        }
                        else
                        {
                                p_primer=p_primer->next;
                        }
                }
                p_node=p_primer->common;
                p_primer->common=new_node;
                new_node->next=p_node;
        }
        fclose(fp);

//paramter for special
        memset(in,'\0',2000*sizeof(char));
        strcpy(in,path);
        strcat(in,"-special.txt"); //suffix of parameter
        fp=fopen(in,"r");
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",in);
                exit(1);
        }

        p_primer=head;
        while(fscanf(fp,"%d\t%d\t%d\t%d\t%d\t%d\n",&pos,&len,&gi,&position,&plus,&minus)!=EOF)
        {
                new_node=(struct Node *)malloc(size);
                new_node->pos=position;
                new_node->gi=gi;
                new_node->plus=plus;
                new_node->minus=minus;

                //find the primer
                while(p_primer->pos!=pos||p_primer->len!=len)
                {
                        if(p_primer->next==NULL)
                        {
                                p_primer=head;
                        }
                        else
                        {
                                p_primer=p_primer->next;
                        }
                }
                
                p_node=p_primer->special;
                p_primer->special=new_node;
                new_node->next=p_node;
        }
        fclose(fp);
        return head;
}

//function: check how many GIs this primer can be used for
__device__ int check_common(int *primerL,int *primerS,int *commonL,int *commonS,int *start_commonL,int *start_commonS,int *end_commonL,int *end_commonS,int turn[],int common,int *apply)
{
        int pos[6],i,dis,num,start[6],end[6],gi;

        for(i=0;i<common;i++)
        {
                apply[common*turn[0]+i]=0;
        }
//plus
	start[0]=start_commonS[turn[0]];
	end[0]=end_commonS[turn[0]];
	start[1]=start_commonS[turn[1]];
        end[1]=end_commonS[turn[1]];
	start[2]=start_commonL[turn[2]];
        end[2]=end_commonL[turn[2]];
        start[3]=start_commonL[turn[3]];
        end[3]=end_commonL[turn[3]];
	start[4]=start_commonS[turn[4]];
        end[4]=end_commonS[turn[4]];
        start[5]=start_commonS[turn[5]];
        end[5]=end_commonS[turn[5]];
        for(pos[0]=start[0];pos[0]<end[0];pos[0]++)
        {
                if(commonS[4*pos[0]+2]!=1)
                        continue;
		gi=commonS[4*pos[0]];
                if(apply[common*turn[0]+gi]==1)
                        continue;
                for(pos[1]=start[1];pos[1]<end[1];pos[1]++)
                {
                        if(commonS[4*pos[1]]!=gi)
                                continue;
                        if(commonS[4*pos[1]+2]!=1)
                                continue;
                        for(pos[2]=start[2];pos[2]<end[2];pos[2]++)
                        {
                                if(commonL[4*pos[2]]!=gi)
                                        continue;
                                if(commonL[4*pos[2]+3]!=1)
                                        continue;
                                for(pos[3]=start[3];pos[3]<end[3];pos[3]++)
                                {
                                        if(commonL[4*pos[3]]!=gi)
                                                continue;
                                        if(commonL[4*pos[3]+2]!=1)
                                                continue;
                                        for(pos[4]=start[4];pos[4]<end[4];pos[4]++)
                                        {
                                                if(commonS[4*pos[4]]!=gi)
                                                        continue;
                                                if(commonS[4*pos[4]+3]!=1)
                                                        continue;
                                                for(pos[5]=start[5];pos[5]<end[5];pos[5]++)
                                                {
                                                        if(commonS[4*pos[5]]!=gi)
                                                                continue;
                                                        if(commonS[4*pos[5]+3]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=commonS[4*pos[1]+1]-(commonS[4*pos[0]+1]+primerS[4*turn[0]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=commonL[4*pos[2]+1]-commonS[4*pos[1]+1];
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=commonL[4*pos[3]+1]-(commonL[4*pos[2]+1]+primerL[4*turn[2]+1]);
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=(commonS[4*pos[4]+1]+primerS[4*turn[4]+1])-(commonL[4*pos[3]+1]+primerL[4*turn[3]+1]);
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=commonS[4*pos[4]+1]+primerS[4*turn[4]+1]-1-commonS[4*pos[1]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=commonS[4*pos[5]+1]-(commonS[4*pos[4]+1]+primerS[4*turn[4]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                        apply[common*turn[0]+gi]=1;
                                                }
                                        }
                                }
                        }
                }
        }
//minus
        for(pos[0]=start[0];pos[0]<end[0];pos[0]++)
        {
                if(commonS[4*pos[0]+3]!=1)
                        continue;
		gi=commonS[4*pos[0]];
                if(apply[turn[0]*common+gi]==1)
                        continue;  //this GI can common

                for(pos[1]=start[1];pos[1]<end[1];pos[1]++)
                {
                        if(commonS[4*pos[1]]!=gi)
                                continue;
                        if(commonS[4*pos[1]+3]!=1)
                                continue;
                        for(pos[2]=start[2];pos[2]<end[2];pos[2]++)
                        {
                                if(commonL[4*pos[2]]!=gi)
                                        continue;
                                if(commonL[4*pos[2]+2]!=1)
                                        continue;
                                for(pos[3]=start[3];pos[3]<end[3];pos[3]++)
                                {
                                        if(commonL[4*pos[3]]!=gi)
                                                continue;
                                        if(commonL[4*pos[3]+3]!=1)
                                                continue;
                                        for(pos[4]=start[4];pos[4]<end[4];pos[4]++)
                                        {
                                                if(commonS[4*pos[4]]!=gi)
                                                        continue;
                                                if(commonS[4*pos[4]+2]!=1)
                                                        continue;
                                                for(pos[5]=start[5];pos[5]<end[5];pos[5]++)
                                                {
                                                        if(commonS[4*pos[5]]!=gi)
                                                                continue;
                                                        if(commonS[4*pos[5]+2]!=1)
                                                                continue;
                                                //F3-F2 
                                                        dis=commonS[4*pos[0]+1]-(commonS[4*pos[1]+1]+primerS[4*turn[1]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                //F2-F1c
                                                        dis=(commonS[4*pos[1]+1]+primerS[4*turn[1]+1])-(commonL[4*pos[2]+1]+primerL[4*turn[2]+1]);
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F1c-B1c
                                                        dis=commonL[4*pos[2]+1]-(commonL[4*pos[3]+1]+primerL[4*turn[3]+1]);
                                                        if(dis<0)
                                                                continue;
                                                //B1c-B2
                                                        dis=commonL[4*pos[3]+1]-commonS[4*pos[4]+1]-1;
                                                        if(dis<40)
                                                                continue;
                                                        if(dis>60)
                                                                continue;
                                                //F2-B2
                                                        dis=commonS[4*pos[1]+1]+primerS[4*turn[1]+1]-1-commonS[4*pos[4]+1]-1;
                                                        if(dis<120)
                                                                continue;
                                                        if(dis>180)
                                                                continue;
                                                //B2-B3
                                                        dis=commonS[4*pos[4]+1]-(commonS[4*pos[5]+1]+primerS[4*turn[5]+1]);
                                                        if(dis<0)
                                                                continue;
                                                        if(dis>20)
                                                                continue;
                                                        apply[common*turn[0]+gi]=1;
                                                }
                                        }
                                }
                        }
                }
        }
        num=0;
        for(i=0;i<common;i++)
        {
                num=num+apply[common*turn[0]+i];
        }
        return num;
}

//check this LAMP primers are uniq or not
//return=0: stop and return=1: go on
__device__ int check_uniq(int *primerL,int *primerS,int *specialL,int *specialS,int *start_specialL,int *start_specialS,int *end_specialL,int *end_specialS,int turn[])
{
        int pos[6],start[6],end[6],gi;

	start[0]=start_specialS[turn[0]];
	end[0]=end_specialS[turn[0]];
	start[1]=start_specialS[turn[1]];
	end[1]=end_specialS[turn[1]];
	start[2]=start_specialL[turn[2]];
        end[2]=end_specialL[turn[2]];
        start[3]=start_specialL[turn[3]];
        end[3]=end_specialL[turn[3]];
	start[4]=start_specialS[turn[4]];
        end[4]=end_specialS[turn[4]];
        start[5]=start_specialS[turn[5]];
        end[5]=end_specialS[turn[5]];
//plus
        for(pos[0]=start[0];pos[0]<end[0];pos[0]++)
        {
                if(specialS[4*pos[0]+2]!=1)
                        continue;
		gi=specialS[4*pos[0]];
                for(pos[1]=start[1];pos[1]<end[1];pos[1]++)
                {
			if(specialS[4*pos[1]]!=gi)
                                continue;
                        if(specialS[4*pos[1]+2]!=1)
				continue;
                        for(pos[2]=start[2];pos[2]<end[2];pos[2]++) //F1c
                        {
                                if(specialL[4*pos[2]]!=gi)
                                        continue;
                                if(specialL[4*pos[2]+3]!=1)
                                        continue;
                                for(pos[3]=start[3];pos[3]<end[3];pos[3]++) //B1c
                                {
                                        if(specialL[pos[3]*4]!=gi)
                                                continue;
                                        if(specialL[4*pos[3]+2]!=1)
                                                continue;
                                        for(pos[4]=start[4];pos[4]<end[4];pos[4]++) //B2
                                        {
                                                if(specialS[4*pos[4]]!=gi)
                                                        continue;
                                                if(specialS[4*pos[4]+3]!=1)
                                                        continue;
                                                for(pos[5]=start[5];pos[5]<end[5];pos[5]++)
                                                {
                                                        if(specialS[4*pos[5]]!=gi)
                                                                continue;
                                                        if(specialS[4*pos[5]+3]!=1)
                                                                continue;
                                                //F3-F2 
                                                        if(specialS[4*pos[1]+1]<specialS[4*pos[0]+1])
                                                                continue;
                                                //F2-F1c
                                                        if(specialL[4*pos[2]+1]<specialS[4*pos[1]+1]+primerS[4*turn[1]+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(specialL[4*pos[3]+1]<specialL[4*pos[2]+1])
                                                                continue;
                                                //B1c-B2
                                                        if(specialS[4*pos[4]+1]<specialL[4*pos[3]+1]+primerL[4*turn[3]+1])
                                                                continue;
                                                //B2-B3
                                                        if(specialS[4*pos[5]+1]<specialS[4*pos[4]+1])
                                                                continue;
                                                //whole
                                                        if(specialS[4*pos[5]+1]-specialS[4*pos[0]+1]>1000)
                                                                continue;
                                                        return 0;
                                                }//B3
                                        }
                                }//B1c
                        }
                }//F2
        }

//minus
        for(pos[0]=start[0];pos[0]<end[0];pos[0]++)
        {
                if(specialS[4*pos[0]+3]!=1)
                        continue;
		gi=specialS[4*pos[0]];
                for(pos[1]=start[1];pos[1]<end[1];pos[1]++)
                {
                        if(specialS[4*pos[1]]!=gi)
                                continue;
                        if(specialS[4*pos[1]+3]!=1)
                                continue;
                        for(pos[2]=start[2];pos[2]<end[2];pos[2]++)
                        {
                                if(specialL[4*pos[2]]!=gi)
                                        continue;
                                if(specialL[4*pos[2]+2]!=1)
                                        continue;
                                for(pos[3]=start[3];pos[3]<end[3];pos[3]++)
                                {
                                        if(specialL[4*pos[3]]!=gi)
                                                continue;
                                        if(specialL[4*pos[3]+3]!=1)
                                                continue;
                                        for(pos[4]=start[4];pos[4]<end[4];pos[4]++)
                                        {
                                                if(specialS[4*pos[4]]!=gi)
                                                        continue;
                                                if(specialS[4*pos[4]+2]!=1)
                                                        continue;
                                                for(pos[5]=start[5];pos[5]<end[5];pos[5]++)
                                                {
                                                        if(specialS[4*pos[5]]!=gi)
                                                                continue;
                                                        if(specialS[4*pos[5]+2]!=1)
                                                                continue;
                                                //F3-F2 
                                                        if(specialS[4*pos[0]+1]<specialS[4*pos[1]+1])
                                                                continue;
                                                //F2-F1c
                                                        if(specialS[4*pos[1]+1]<specialL[4*pos[2]+1]+primerL[4*turn[2]+1])
                                                                continue;
                                                //F1c-B1c
                                                        if(specialL[4*pos[2]+1]<specialL[4*pos[3]+1])
                                                                continue;
                                                //B1c-B2
                                                        if(specialL[4*pos[3]+1]<specialS[4*pos[4]+1]+primerS[4*turn[4]+1])
                                                                continue;
                                                //B2-B3
                                                        if(specialS[4*pos[4]+1]<specialS[4*pos[5]+1])
                                                                continue;
                                                //whole
                                                        if(specialS[4*pos[0]+1]-specialS[4*pos[5]+1]>1000)
                                                                continue;
                                                        return 0;
                                                }
                                        }
                                }
                        }
                }
        }
        return 1;
}

//from first to second
__global__ void next_one(int *first,int *second,int *next,int num_first,int num_second)
{
        int id=blockDim.x*blockIdx.x+threadIdx.x;
	int i;

	if(id<num_first)
	{
		next[id]=-1;
		i=id;
		if(i>=num_second)
		{
			i=num_second-1;
		}
		if(second[4*i]>first[id*4]+first[id*4+1])
		{
			while((i>=0)&&(second[4*i]>first[id*4]+first[id*4+1]))
			{
				next[id]=i;
				i--;
			}
		}
		else
		{
			while((i<num_second)&&(second[i*4]<=first[id*4]+first[id*4+1]))
			{
				next[id]=i;
				i++;
			}
		}
	}
	__syncthreads();
}

__device__ float cal_GC(char *d_seq,int begin,int length)
{
	int i,total=0;
	float result;

	for(i=0;i<length;i++)
	{
		if(d_seq[begin+i]=='C'||d_seq[begin+i]=='c'||d_seq[begin+i]=='G'||d_seq[begin+i]=='g')
			total++;
	}
	result=(float)total/length;
	return result;
}
//caculate
__global__ void find_primer(int *primerL,int *primerS,int *specialL,int *specialS,int *commonL,int *commonS,int *start_specialL,int *start_specialS,int *start_commonL,int *start_commonS,int *end_specialL,int *end_specialS,int *end_commonL,int *end_commonS,int *nextS,int *nextL,int *other_toL,int *other_toS,int *result,int numL,int numS,int common,int *apply,int *best_par,int circle,int expect,char *d_seq,int begin) //apply is temp
{
	int id=blockDim.x*blockIdx.x+threadIdx.x;
	int turn[6],flag,i;
	float gc[2];

	if(id<numS)
	{
		result[12*id+1]=0;//not LAMP, as a flag
	//check overlap
		for(i=0;i<expect;i++)
		{
			if(best_par[i]==-1)
				break;
			flag=primerS[4*id]-best_par[i];
			if(flag<300&&flag>-300)
				return;
		}
	//combine
		turn[0]=id; //one thread, one F3
		for(turn[1]=nextS[turn[0]];turn[1]<numS;turn[1]++) //F2
		{
			if(turn[1]==-1)
				break;
			if(primerS[turn[1]*4]<primerS[turn[0]*4]+primerS[turn[0]*4+1])
				continue;
			if(primerS[4*turn[1]]-(primerS[4*turn[0]]+primerS[4*turn[0]+1])>20)
				break;
			for(turn[2]=other_toL[turn[1]];turn[2]<numL;turn[2]++) //F1c
			{
				if(turn[2]==-1)
					break;
				if(primerL[4*turn[2]]-primerS[4*turn[1]]-1<40)
					continue;
                                if(primerL[4*turn[2]]-primerS[4*turn[1]]-1>60)
                                	break;
                                for(turn[3]=nextL[turn[2]];turn[3]<numL;turn[3]++)   //B1c
                                {
                                        if(turn[3]==-1)
                                        	break;
                                        if(primerL[4*turn[3]]<primerL[4*turn[2]]+primerL[4*turn[2]+1])
                                        	continue;
                                        if(primerL[4*turn[3]]-primerL[4*turn[2]]>85)
                                        	break;
                                        for(turn[4]=other_toS[turn[3]];turn[4]<numS;turn[4]++)   //B2
                                        {
                                                if(turn[4]==-1)
                                                	break;
                                                if((primerS[4*turn[4]]+primerS[4*turn[4]+1]-1)-(primerL[4*turn[3]]+primerL[4*turn[3]+1])<40)
                                                	continue;
                                                if((primerS[4*turn[4]]+primerS[4*turn[4]+1]-1)-(primerL[4*turn[3]]+primerL[4*turn[3]+1])>60)
                                                	break;
                                                if(primerS[4*turn[4]]+primerS[4*turn[4]+1]-1-primerS[turn[1]*4]-1<120)
                                                	continue;
                                                if(primerS[4*turn[4]]+primerS[4*turn[4]+1]-1-primerS[turn[1]*4]-1>180)
                                                	break;
                                                for(turn[5]=nextS[turn[4]];turn[5]<numS;turn[5]++)  //B3
                                                {
                                                        if(turn[5]==-1)
                                                        	break;
                                                        if(primerS[turn[5]*4]<primerS[4*turn[4]]+primerS[4*turn[4]+1])
                                                        	continue;
                                                        if(primerS[turn[5]*4]-(primerS[4*turn[4]]+primerS[4*turn[4]+1])>20)
                                                        	break;
						//plus-minus
							flag=primerS[4*turn[0]+2]&&primerS[4*turn[1]+2]&&primerL[4*turn[2]+3]&&primerL[4*turn[3]+2]&&primerS[4*turn[4]+3]&&primerS[4*turn[5]+3];
                                                        if(flag==0)
                                                        	continue;
		
							gc[0]=cal_GC(d_seq,(primerS[4*turn[0]]-begin),primerS[4*turn[0]+1]);
							gc[1]=cal_GC(d_seq,(primerS[4*turn[0]]-begin),(primerS[4*turn[5]]+primerS[4*turn[5]+1]-primerS[4*turn[0]]));
							if(gc[1]<0.45&&gc[0]>0.5)
								continue;
							if(gc[1]>0.45&&gc[0]<0.5)
								continue;
							flag=check_uniq(primerL,primerS,specialL,specialS,start_specialL,start_specialS,end_specialL,end_specialS,turn);
							if(flag==0)
								continue;
							flag=check_common(primerL,primerS,commonL,commonS,start_commonL,start_commonS,end_commonL,end_commonS,turn,common,apply);

							if(flag<circle)
								continue;
							result[id*12]=primerS[4*turn[0]];
							result[id*12+1]=primerS[4*turn[0]+1];
							result[id*12+2]=primerS[4*turn[1]];
							result[id*12+3]=primerS[4*turn[1]+1];
							result[id*12+4]=primerL[4*turn[2]];
							result[id*12+5]=primerL[4*turn[2]+1];
							result[id*12+6]=primerL[4*turn[3]];
							result[id*12+7]=primerL[4*turn[3]+1];
							result[id*12+8]=primerS[4*turn[4]];
							result[id*12+9]=primerS[4*turn[4]+1];
							result[id*12+10]=primerS[4*turn[5]];
							result[id*12+11]=primerS[4*turn[5]+1];
							return;
						}
					}
				}
			}
		}
	}
	__syncthreads();
}

main(int argc,char **argv)
{
	int i,j,k,common,flag,expect,circle,have,*list,begin,stop;
	char out[2000],path_small[2000],path_large[2000],path_fa[2000],script[6000],directory[2000],file1[2000],file2[2000],path_common[2000],path_primer3[2000],check[2000];
	char *temp,*seq,F3[26],F2[26],F1c[26],B1c[26],B2[26],B3[26],*h_seq,*d_seq;
	FILE *fp,*LAMP;
	struct Primer *headL,*headS,*tempL,*tempS,*storeL,*storeS; 
	struct Node *p_node,*p_temp;
	char usage[200]="./a.out -small small_primers_path -large large_primer3_path -out out_file -fa fna_path -common common_GI_file -primer3 primer3_path -expect(optional,default=10) expect_output_number\n";
	time_t start,end;
	
	long int memory;
	hipDeviceProp_t prop;
	int *h_primerL,*d_primerL,*h_primerS,*d_primerS; //primer info
	int *h_commonL,*d_commonL,*h_commonS,*d_commonS,*h_specialL,*d_specialL,*h_specialS,*d_specialS; //common and special info
	int *h_result,*d_result,*d_apply,*h_apply,*h_par,*d_par;
	int *d_other_to_L,*d_other_to_S,*d_nextL,*d_nextS;
	int *h_start_commonL,*d_start_commonL,*h_end_commonL,*d_end_commonL,*h_start_specialL,*d_start_specialL,*h_end_specialL,*d_end_specialL,*h_start_commonS,*d_start_commonS,*h_end_commonS,*d_end_commonS,*h_start_specialS,*d_start_specialS,*h_end_specialS,*d_end_specialS;
	int num[7],count[3],block,m,n;
	char *p_primer[6];
	
	expect=10; //default output max 10 LAMP primers
	start=time(NULL);
/////read the parameters
	if(argc<11)
	{
		printf("Error!\n%s\n",usage);
		exit(1);
	}

	j=0;
	for(i=1;i<argc-1;i=i+2)
	{
		if(strcmp(argv[i],"-small")==0)
		{
			strcpy(path_small,argv[i+1]);
			j++;
		}
                else if(strcmp(argv[i],"-large")==0)
                {
                        strcpy(path_large,argv[i+1]);
                        j++;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        strcpy(out,argv[i+1]);
                        j++;
                }
		else if(strcmp(argv[i],"-fa")==0)
		{
			strcpy(path_fa,argv[i+1]);
			j++;
		}
		else if(strcmp(argv[i],"-common")==0)
		{
			strcpy(path_common,argv[i+1]);
			j++;
		}
		else if(strcmp(argv[i],"-primer3")==0)
		{
			strcpy(path_primer3,argv[i+1]);
			j++;
		}
		else if(strcmp(argv[i],"-expect")==0)
		{
			expect=atoi(argv[i+1]);
		}
		else
		{
			printf("Error!\n%s\n",usage);
			exit(1);
		}
	}

	if(j!=6)
	{
		printf("Error!\n%s\n",usage);
		exit(1);
	}
//the directory of program
	strcpy(directory,out);
	j=strlen(directory);
	j--;
	while(directory[j]!='/'&&j>=0)
	{
		directory[j]='\0';
		j--;
	}
	strcpy(file1,directory);
	strcat(file1,"temp-par.txt");
	strcpy(file2,directory);
	strcat(file2,"temp-result.txt");
//common_GI-list
	fp=fopen(path_common,"r");                 
        if(fp==NULL)
        {          
                printf("Error: can't open the %s file!\n",path_common);
                exit(1);                  
        }

	i=0;
        while(fscanf(fp,"%d\n",&j)!=EOF)
        {       
                i++;
        }
        list=(int *)malloc(i*sizeof(int));    
        rewind(fp);
        common=0;
        while(fscanf(fp,"%d\n",&j)!=EOF)
        {
                list[common]=j;
                common++;
        }
        fclose(fp);	

//primer3 path
	i=strlen(path_primer3);
	i--;
	if(path_primer3[i]=='/')
	{
		strcpy(check,path_primer3);
		strcat(check,"primer3_core");
		if(access(check,0)!=0)
		{
			printf("Can't fine primer3_core program in %s directory!\n",path_primer3);
			exit(1);
		}
	}
	else
	{
		for(j=0;j<12;j++)
			check[j]=path_primer3[i-j];
		check[j]='\0';
		if(strcmp(check,"eroc_3remirp")==0)
		{
			if(access(path_primer3,0)==0)
			{
				j=i;
				while(j>=0&&path_primer3[j]!='/')
				{
					path_primer3[j]='\0';
					j--;
				}
			}
			else
			{
				printf("Can't fine primer3_core program in %s directory!\n",path_primer3);
				exit(1);
			}
		}
		else
		{
			strcpy(check,path_primer3);
			strcat(check,"/primer3_core");
			if(access(check,0)==0)
			{
				path_primer3[i+1]='/';
				path_primer3[i+2]='\0';
			}
			else
			{
				printf("Can't fine primer3_core program in %s directory!\n",path_primer3);
				exit(1);
			}
		}
	}

	hipGetDeviceProperties(&prop,0); //read parameters
	headS=read_par(path_small,common);
	headL=read_par(path_large,common);

//common statistics
	how_many(headL,common);
	how_many(headS,common);

//the genome sequence
	i=file_size2(path_fa);
	i=i+100;
        temp=(char *)malloc(i*sizeof(char));
        memset(temp,'\0',i*sizeof(char));
        fp=fopen(path_fa,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the sequence file %s\n",path_fa);
                exit(1);
        }

        fread(temp,i*sizeof(char),1,fp);
        fclose(fp); 
        seq=(char *)malloc(i*sizeof(char));
        memset(seq,'\0',i*sizeof(char));

        flag=0;
        j=0;
        for(i=0;temp[i]!='\0';i++)
        {
                if(temp[i]=='\n')
                {
                        flag+=1;
                        continue;
                }
                else
                {
                        if(flag!=0)
                        {
                                seq[j]=temp[i];
                                j++;
                        }
                }
        }
        free(temp);

	end=time(NULL);
	printf("The prepare time is %0.1f seconds!\n",difftime(end,start));
	start=time(NULL);

//LAMP-GPU
	LAMP=fopen(out,"w");
        if(LAMP==NULL)
        {
                printf("Error: can't create the %s file!\n",out);
                exit(1);
        }
	have=0;
	prop.maxThreadsPerBlock=prop.maxThreadsPerBlock/4;
	k=prop.maxThreadsPerBlock*60000; //max primers, max threads
	h_par=(int *)malloc(expect*sizeof(int));
	for(i=0;i<expect;i++)
	{
		h_par[i]=-1;
	}
	hipMalloc((void **)&d_par,expect*sizeof(int));
	for(circle=common;circle>=1;circle--)
	{
		if(have>=expect)
			break;
		storeL=headL;
		storeS=headS;
		while(storeL->pos<=storeS->pos+storeS->len)
		{
			storeL=storeL->next;
		}
		flag=0;
		while(storeS)
		{
			if(have>=expect)
				break;
			if(flag==1)  //don't have enough primers
				break;
			for(i=0;i<7;i++)
			{
				num[i]=0;
			}
			memory=0;
		//statistics	
			tempL=storeL;
			tempS=storeS;
			while(tempS&&(memory<prop.totalGlobalMem/4)&&(num[0]<k))
			{
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
				while(tempL&&(tempL->pos<tempS->pos))
				{
					if(tempL->total<circle)
					{
						tempL=tempL->next;
						continue;
					}
					num[0]++;
					num[1]++;
					num[3]=num[3]+tempL->total_common;
					num[4]=num[4]+tempL->total_special;
					memory=memory+10+4*(tempL->total_common+tempL->total_special);
					tempL=tempL->next;
				}
				if(num[0]==0)
				{
					begin=tempS->pos;
					stop=tempS->pos+tempS->len-1;
					memory=memory+tempS->len;
				}
				else
				{
					if(tempS->pos+tempS->len-1>stop)
					{
						memory=memory+(tempS->pos+tempS->len-1-stop);
						stop=tempS->pos+tempS->len-1;
					}
				}
				num[0]++;
				num[2]++;
				num[5]=num[5]+tempS->total_common;
				num[6]=num[6]+tempS->total_special;
				memory=memory+22+4*(tempS->total_common+tempS->total_special)+common;
				tempS=tempS->next;
			}
			if(num[2]<4||num[1]<2) //don't have enough primers
			{
				flag=1;
				break;
			}
			if(tempS==NULL)  //check all primers
				flag=1;	
			h_seq=(char *)malloc((stop-begin+1)*sizeof(char));
			hipMalloc((void **)&d_seq,(stop-begin+1)*sizeof(char));
			for(i=begin;i<=stop;i++)
				h_seq[i-begin]=seq[i];

		//malloc small primer
			h_primerS=(int *)malloc(4*num[2]*sizeof(int));
			hipMalloc((void **)&d_primerS,4*num[2]*sizeof(int));
			h_commonS=(int *)malloc(4*num[5]*sizeof(int));
			hipMalloc((void **)&d_commonS,4*num[5]*sizeof(int));
			h_specialS=(int *)malloc(4*num[6]*sizeof(int));
			hipMalloc((void **)&d_specialS,4*num[6]*sizeof(int));
			hipMalloc((void **)&d_nextS,num[2]*sizeof(int));
			hipMalloc((void **)&d_other_to_L,num[2]*sizeof(int));

			h_start_commonS=(int *)malloc(num[2]*sizeof(int));
			hipMalloc((void **)&d_start_commonS,num[2]*sizeof(int));
			h_end_commonS=(int *)malloc(num[2]*sizeof(int));
        	        hipMalloc((void **)&d_end_commonS,num[2]*sizeof(int));
			h_start_specialS=(int *)malloc(num[2]*sizeof(int));
        	        hipMalloc((void **)&d_start_specialS,num[2]*sizeof(int));
        	        h_end_specialS=(int *)malloc(num[2]*sizeof(int));
        	        hipMalloc((void **)&d_end_specialS,num[2]*sizeof(int));
		
			tempS=storeS;
			for(i=0;i<3;i++)
			{
				count[i]=0;
			}
			while(count[0]<num[2])
			{
				if(tempS->total<circle)
				{
					tempS=tempS->next;
					continue;
				}
		//primer info
				h_primerS[4*count[0]]=tempS->pos;
				h_primerS[4*count[0]+1]=tempS->len;
				h_primerS[4*count[0]+2]=tempS->plus;
				h_primerS[4*count[0]+3]=tempS->minus;
			//common
				h_start_commonS[count[0]]=count[1];
				if(tempS->total_common==0)
					h_end_commonS[count[0]]=-1;
				else
				{
					p_node=tempS->common;
					while(p_node)
					{
						h_commonS[4*count[1]]=p_node->gi;
						h_commonS[4*count[1]+1]=p_node->pos;
						h_commonS[4*count[1]+2]=p_node->plus; 
                                	        h_commonS[4*count[1]+3]=p_node->minus;
						count[1]++;
						p_node=p_node->next;
					}
					h_end_commonS[count[0]]=count[1];
				}
			//special
				h_start_specialS[count[0]]=count[2];
                	        if(tempS->total_special==0)
                	                h_end_specialS[count[0]]=-1;
                	        else
                	        {
                	                p_node=tempS->special;
                	                while(p_node)
                	                {
                	                        h_specialS[4*count[2]]=p_node->gi;
                	                        h_specialS[4*count[2]+1]=p_node->pos;
                	                        h_specialS[4*count[2]+2]=p_node->plus;
                	                        h_specialS[4*count[2]+3]=p_node->minus;
                	                        count[2]++;
                	                        p_node=p_node->next;
                	                }
                	                h_end_specialS[count[0]]=count[2];
                	        }
				count[0]++;
				tempS=tempS->next;
			}
		//copy
			hipMemcpy(d_primerS,h_primerS,num[2]*4*sizeof(int),hipMemcpyHostToDevice);
			free(h_primerS);
			hipMemcpy(d_commonS,h_commonS,num[5]*4*sizeof(int),hipMemcpyHostToDevice);
			free(h_commonS);
			hipMemcpy(d_specialS,h_specialS,num[6]*4*sizeof(int),hipMemcpyHostToDevice);
        	        free(h_specialS);
			hipMemcpy(d_start_commonS,h_start_commonS,num[2]*sizeof(int),hipMemcpyHostToDevice);
			free(h_start_commonS);
			hipMemcpy(d_end_commonS,h_end_commonS,num[2]*sizeof(int),hipMemcpyHostToDevice);
        	        free(h_end_commonS);
			hipMemcpy(d_start_specialS,h_start_specialS,num[2]*sizeof(int),hipMemcpyHostToDevice);
        	        free(h_start_specialS);
        	        hipMemcpy(d_end_specialS,h_end_specialS,num[2]*sizeof(int),hipMemcpyHostToDevice);    
        	        free(h_end_specialS);
		//large primer
			h_primerL=(int *)malloc(4*num[1]*sizeof(int));
        	        hipMalloc((void **)&d_primerL,4*num[1]*sizeof(int));
        	        h_commonL=(int *)malloc(4*num[3]*sizeof(int));
        	        hipMalloc((void **)&d_commonL,4*num[3]*sizeof(int));
        	        h_specialL=(int *)malloc(4*num[4]*sizeof(int));
        	        hipMalloc((void **)&d_specialL,4*num[4]*sizeof(int));
        	        hipMalloc((void **)&d_nextL,num[1]*sizeof(int));
        	        hipMalloc((void **)&d_other_to_S,num[1]*sizeof(int));

        	        h_start_commonL=(int *)malloc(num[1]*sizeof(int));
        	        hipMalloc((void **)&d_start_commonL,num[1]*sizeof(int));
        	        h_end_commonL=(int *)malloc(num[1]*sizeof(int));
        	        hipMalloc((void **)&d_end_commonL,num[1]*sizeof(int));
        	        h_start_specialL=(int *)malloc(num[1]*sizeof(int));
        	        hipMalloc((void **)&d_start_specialL,num[1]*sizeof(int));
        	        h_end_specialL=(int *)malloc(num[1]*sizeof(int));
        	        hipMalloc((void **)&d_end_specialL,num[1]*sizeof(int));
                
        	        tempL=storeL;
        	        for(i=0;i<3;i++)
        	        {
        	                count[i]=0;
        	        }
        	        while(count[0]<num[1])
        	        {
				if(tempL->total<circle)
				{
					tempL=tempL->next;
					continue;
				}
                	//primer info
                	        h_primerL[4*count[0]]=tempL->pos;
                	        h_primerL[4*count[0]+1]=tempL->len;
                	        h_primerL[4*count[0]+2]=tempL->plus;
                	        h_primerL[4*count[0]+3]=tempL->minus;
                	//common
                	        h_start_commonL[count[0]]=count[1];
                	        if(tempL->total_common==0)
                	                h_end_commonL[count[0]]=-1;
                	        else
                	        {
                	                p_node=tempL->common;
                	                while(p_node)
                	                {
                	                        h_commonL[4*count[1]]=p_node->gi;
                	                        h_commonL[4*count[1]+1]=p_node->pos;
                	                        h_commonL[4*count[1]+2]=p_node->plus; 
                        	                h_commonL[4*count[1]+3]=p_node->minus;
                        	                count[1]++;
                        	                p_node=p_node->next;
                        	        }
                        	        h_end_commonL[count[0]]=count[1];
                        	}
                	//special
                        	h_start_specialL[count[0]]=count[2];
                        	if(tempL->total_special==0)
                        	        h_end_specialL[count[0]]=-1;
                        	else
                        	{
                        	        p_node=tempL->special;
                        	        while(p_node)
                        	        {
                        	                h_specialL[4*count[2]]=p_node->gi;
                        	                h_specialL[4*count[2]+1]=p_node->pos;
                        	                h_specialL[4*count[2]+2]=p_node->plus;
                        	                h_specialL[4*count[2]+3]=p_node->minus;
                        	                count[2]++;
                        	                p_node=p_node->next;
                        	        }
                        	        h_end_specialL[count[0]]=count[2];
                        	}
                        	count[0]++;
                        	tempL=tempL->next;
                	}
        	//copy
        	        hipMemcpy(d_primerL,h_primerL,num[1]*4*sizeof(int),hipMemcpyHostToDevice);
			free(h_primerL);
                	hipMemcpy(d_commonL,h_commonL,num[3]*4*sizeof(int),hipMemcpyHostToDevice);
                	free(h_commonL);
                	hipMemcpy(d_specialL,h_specialL,num[4]*4*sizeof(int),hipMemcpyHostToDevice);
                	free(h_specialL);
                	hipMemcpy(d_start_commonL,h_start_commonL,num[1]*sizeof(int),hipMemcpyHostToDevice);
                	free(h_start_commonL);
                	hipMemcpy(d_end_commonL,h_end_commonL,num[1]*sizeof(int),hipMemcpyHostToDevice);
                	free(h_end_commonL);
                	hipMemcpy(d_start_specialL,h_start_specialL,num[1]*sizeof(int),hipMemcpyHostToDevice);
                	free(h_start_specialL);
                	hipMemcpy(d_end_specialL,h_end_specialL,num[1]*sizeof(int),hipMemcpyHostToDevice);    
                	free(h_end_specialL);
	//run
			if(num[2]%prop.maxThreadsPerBlock==0)
				block=num[2]/prop.maxThreadsPerBlock;
			else
				block=(num[2]-num[2]%prop.maxThreadsPerBlock)/prop.maxThreadsPerBlock+1;

		//next primer	
			next_one<<<block,prop.maxThreadsPerBlock>>>(d_primerS,d_primerS,d_nextS,num[2],num[2]);
			next_one<<<block,prop.maxThreadsPerBlock>>>(d_primerL,d_primerL,d_nextL,num[1],num[1]);
			next_one<<<block,prop.maxThreadsPerBlock>>>(d_primerS,d_primerL,d_other_to_L,num[2],num[1]);
        	        next_one<<<block,prop.maxThreadsPerBlock>>>(d_primerL,d_primerS,d_other_to_S,num[1],num[2]);

		//calculate
			hipMalloc((void **)&d_apply,common*num[2]*sizeof(int));
			hipMalloc((void **)&d_result,12*num[2]*sizeof(int));
			hipMemset(d_result,'\0',12*num[2]*sizeof(int));
			hipMemcpy(d_par,h_par,expect*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(d_seq,h_seq,(stop-begin+1)*sizeof(char),hipMemcpyHostToDevice);
			free(h_seq);
			find_primer<<<block,prop.maxThreadsPerBlock>>>(d_primerL,d_primerS,d_specialL,d_specialS,d_commonL,d_commonS,d_start_specialL,d_start_specialS,d_start_commonL,d_start_commonS,d_end_specialL,d_end_specialS,d_end_commonL,d_end_commonS,d_nextS,d_nextL,d_other_to_L,d_other_to_S,d_result,num[1],num[2],common,d_apply,d_par,circle,expect,d_seq,begin);
//	exit(0);

	//	hipError_t result0 = hipGetLastError();
	//	printf("%s\n",result0);
			h_result=(int *)malloc(12*num[2]*sizeof(int));
			hipMemcpy(h_result,d_result,12*num[2]*sizeof(int),hipMemcpyDeviceToHost);
			h_apply=(int *)malloc(common*num[2]*sizeof(int));

		//check secondary structure
			fp=fopen(file1,"w");
			if(fp==NULL)
			{
				printf("Can't create the temp-par.txt file!\n");
				exit(1);
			}
			j=0; //as a flag, how many primers
			for(i=0;i<num[2];i++)
			{
				if(h_result[12*i+1]==0)
					continue;
				primer(seq,F3,h_result[12*i],h_result[12*i+1],0);
				primer(seq,F2,h_result[12*i+2],h_result[12*i+3],0);
				primer(seq,F1c,h_result[12*i+4],h_result[12*i+5],1);
				primer(seq,B1c,h_result[12*i+6],h_result[12*i+7],0);
                                primer(seq,B2,h_result[12*i+8],h_result[12*i+9],1);
                                primer(seq,B3,h_result[12*i+10],h_result[12*i+11],1);
				p_primer[0]=F3;
				p_primer[1]=F2;
				p_primer[2]=F1c;
				p_primer[3]=B1c;
				p_primer[4]=B2;
				p_primer[5]=B3;
			//output
				for(m=0;m<5;m++)
				{
					for(n=m+1;n<6;n++)
					{
						fprintf(fp,"SEQUENCE_ID=%d-%d-%d\n",i,h_result[12*i],(m*10+n));
						fprintf(fp,"PRIMER_TASK=check_primers\nPRIMER_PICK_ANYWAY=1\nPRIMER_SALT_DIVALENT=4\nPRIMER_DNTP_CONC=1.4\nPRIMER_DNA_CONC=38\nPRIMER_THERMODYNAMIC_PARAMETERS_PATH=%sprimer3_config/\n",path_primer3);
						fprintf(fp,"SEQUENCE_PRIMER=%s\n",p_primer[m]);
						fprintf(fp,"SEQUENCE_PRIMER_REVCOMP=%s\n=\n",p_primer[n]);
					}
				}
				j++;
			}
			fclose(fp);
			if(j==0) //don't have any candidate LAMP primers
			{
				remove(file1);
				continue;
			}
			memset(script,'\0',6000*sizeof(char));
			sprintf(script,"%sprimer3_core -strict_tags %s > %s",path_primer3,file1,file2);
			system(script);
			remove(file1);

			hipMemcpy(h_apply,d_apply,common*num[2]*sizeof(int),hipMemcpyDeviceToHost);
			have=secondary_check(expect,have,circle,h_par,file2,LAMP,h_result,h_apply,seq,common,list);
			remove(file2);
	//free
			hipFree(d_seq);
			hipFree(d_primerS);
			hipFree(d_primerL);
			hipFree(d_commonS);
			hipFree(d_commonL);
			hipFree(d_specialS);
			hipFree(d_specialL);
			hipFree(d_nextS);
			hipFree(d_nextL);
			hipFree(d_other_to_S);
			hipFree(d_other_to_L);
			hipFree(d_start_commonS);
			hipFree(d_start_commonL);
			hipFree(d_start_specialS);
			hipFree(d_start_specialL);
			hipFree(d_end_commonS);
			hipFree(d_end_specialS);
			hipFree(d_end_commonL);
			hipFree(d_end_specialL);
			hipFree(d_apply);
			hipFree(d_result);
			free(h_result);
			free(h_apply);
		//new primer start
			if(tempS==NULL)
				storeS=tempS;
			else
			{
				while(tempS->pos-storeS->pos>300)
				{
					storeS=storeS->next;
				}
				while(storeL&&(storeL->pos<=storeS->pos+storeS->len))
				{
					storeL=storeL->next;
				}
			}
		}//one circle
	}
	hipFree(d_par);
	free(h_par);
	end=time(NULL);
	printf("the time for design is %0.1f seconds!\n",difftime(end,start));
	fclose(LAMP);
	free(seq);
	free(list);
//free struct list
	while(headL)
	{
		p_node=headL->common;
		while(p_node)
		{
			p_temp=p_node->next;
			free(p_node);
			p_node=p_temp;
		}
		p_node=headL->special;
		while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
		
		tempL=headL->next;
		free(headL);
		headL=tempL;
	}
	while(headS)
        {
                p_node=headS->common;  
                while(p_node)  
                {
                        p_temp=p_node->next;  
                        free(p_node);  
                        p_node=p_temp;  
                }
                p_node=headS->special;
                while(p_node)
                {               
                        p_temp=p_node->next;
                        free(p_node);
                        p_node=p_temp;
                }

                tempS=headS->next;
                free(headS);
                headS=tempS;
        }
}
