#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<sys/stat.h>

__constant__ int d_NumL[2];
__constant__ char d_Pchar[1084];
__constant__ double parameter[5916];
__constant__ float d_stab[4096];
__constant__ float d_deltah[16];
__constant__ float d_deltas[16];

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ void str2int(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'A':
                        d_numSeq[id]=0;
                        break;
                case 'C':
                        d_numSeq[id]=1;
                        break;
                case 'G':
                        d_numSeq[id]=2;
                        break;
                case 'T':
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

__device__ void str2int_rev(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'T':
                        d_numSeq[id]=0;
                        break;
                case 'G':
                        d_numSeq[id]=1;
                        break;
                case 'C':
                        d_numSeq[id]=2;
                        break;                 
                case 'A':               
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

int get_num_line(char *path,int flag)
{
	FILE *fp;
	int i,size;
	char *line;

	i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
	if(flag==0)
	        strcat(line,"triloop.ds");
	else
		strcat(line,"tetraloop.ds");

        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        fp=fopen(line,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	size=0;
	while(fgets(line,i,fp)!=NULL)
		size++;
	return size;
}

void getTriloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i,turn;
        char *line,seq[10],value[10];
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
	
	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for (i=0;i<5;i++)
			Pchar[5*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+turn]=1.0*INFINITY;
		else
			parameter[5730+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

	i=strlen(path)+20;
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<5;i++)
			Pchar[5*NumL[0]+turn*5+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void getTetraloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i, turn;
        char *line,seq[10],value[10];

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+turn*6+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);
        
	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+6*NumL[1]+6*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+NumL[1]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+NumL[1]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ void initMatrix2(int length,double *d_DPT,char *d_numSeq,int id)
{
	int i,j;
	for(i=1;i<=length;++i)
		for(j=i;j<=length;++j)
			if(j-i<4 || (d_numSeq[id*54+i]+d_numSeq[id*54+j]!=3))
			{
				d_DPT[id*1340+(i-1)*(length-1)+j-1]=1.0*INFINITY;
				d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=-1.0;
			}
			else
			{
				d_DPT[id*1340+(i-1)*(length-1)+j-1]=0.0;
				d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=-3224.0;
			}
}

__device__ double Ss(int i,int j,int k,int length,char *d_numSeq,int id)
{
	if(k==2)
	{
		if(i>=j)
			return -1.0;
		if(i==length||j==length+1)
			return -1.0;

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		return parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
	}
	else
		return parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ double Hs(int i,int j,int k,int length,char *d_numSeq,int id)
{
	if(k==2)
	{
		if(i>= j)
			return 1.0*INFINITY;
		if(i==length||j==length+1)
			return 1.0*INFINITY;

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		if(fabs(parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]])<999999999)
			return parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
		else
			return 1.0*INFINITY;
	}
	else
		return parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ void maxTM2(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	double T0,T1,S0,S1,H0,H1;

	S0=d_DPT[id*1340+625+(i-1)*(length-1)+j-1];
	H0=d_DPT[id*1340+(i-1)*(length-1)+j-1];
	T0=(H0+d_DPT[id*1340+1302])/(S0+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
	if(fabs(d_DPT[id*1340+(i-1)*(length-1)+j-1])<999999999)
	{
		S1=(d_DPT[id*1340+625+i*(length-1)+j-2]+Ss(i,j,2,length,d_numSeq,id));
		H1=(d_DPT[id*1340+i*(length-1)+j-2]+Hs(i,j,2,length,d_numSeq,id));
	}
	else
	{
		S1=-1.0;
		H1=1.0*INFINITY;
	}
	T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
	if(S1<-2500.0)
	{
		S1=-3224.0;
		H1=0.0;
	}
	if(S0<-2500.0)
	{
		S0=-3224.0;
		H0=0.0;
 	}

	if(T1>T0)
	{
		d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=S1;
		d_DPT[id*1340+(i-1)*(length-1)+j-1]= H1;
	}
	else
	{
		d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=S0;
		d_DPT[id*1340+(i-1)*(length-1)+j-1]=H0;
	}
}

__device__ void calc_bulge_internal2(int i,int j,int ii,int jj,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	int loopSize1,loopSize2,loopSize;
	double T1,T2,S,H;

	S=-3224.0;
	H=0.0;
	loopSize1=ii-i-1;
	loopSize2=j-jj-1;
	if(loopSize1+loopSize2>30)
	{
		d_DPT[id*1340+pos]=-1.0;
		d_DPT[id*1340+pos+1]=1.0*INFINITY;
		return;
	}

	loopSize=loopSize1+loopSize2-1;
	if((loopSize1==0&&loopSize2>0)||(loopSize2==0&&loopSize1>0))
	{
		if(loopSize2==1||loopSize1==1)
		{ 
			if((loopSize2==1&&loopSize1==0)||(loopSize2==0&&loopSize1==1))
			{
				H=parameter[3150+loopSize]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				S=parameter[3060+loopSize]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
 			}
			if(traceback!=1)
			{
				H+=d_DPT[id*1340+(ii-1)*(length-1)+jj-1];
				S+=d_DPT[id*1340+625+(ii-1)*(length-1)+jj-1];
			}

			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
			T2=(d_DPT[id*1340+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1302])/((d_DPT[id*1340+625+(i-1)*(length-1)+j-1])+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if((T1>T2)||((traceback&&T1>=T2)||traceback==1))
			{
				d_DPT[id*1340+pos]=S;
				d_DPT[id*1340+pos+1]=H;
			}
		}
		else
		{
			H=parameter[3150+loopSize]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				H+=d_DPT[id*1340+(ii-1)*(length-1)+jj-1];

			S=parameter[3060+loopSize]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				S+=d_DPT[id*1340+625+(ii-1)*(length-1)+jj-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
			T2=(d_DPT[id*1340+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1302])/(d_DPT[id*1340+625+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				d_DPT[id*1340+pos]=S;
				d_DPT[id*1340+pos+1]=H;
			}
		}
	}
	else if(loopSize1==1&&loopSize2==1)
	{
		S=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			S+=d_DPT[id*1340+625+(ii-1)*(length-1)+jj-1];

		H=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			H+=d_DPT[id*1340+(ii-1)*(length-1)+jj-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
		T2=(d_DPT[id*1340+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1302])/(d_DPT[id*1340+625+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if((T1-T2>=0.000001)||traceback)
		{
			if((T1>T2)||((traceback&&T1>= T2)||traceback==1))
			{
				d_DPT[id*1340+pos]=S;
				d_DPT[id*1340+pos+1]=H;
			}
		}
		return;
	}
	else
	{
		H=parameter[3120+loopSize]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			H+=d_DPT[id*1340+(ii-1)*(length-1)+jj-1];

		S=parameter[3030+loopSize]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(loopSize1-loopSize2));
		if(traceback!=1)
			S+=d_DPT[id*1340+625+(ii-1)*(length-1)+jj-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}

		T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
		T2=(d_DPT[id*1340+(i-1)*(length-1)+j-1]+d_DPT[id*1340+1302])/((d_DPT[id*1340+625+(i-1)*(length-1)+j-1])+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
		{
			d_DPT[id*1340+pos]=S;
			d_DPT[id*1340+pos+1]=H;
		}
	}
	return;
}

__device__ void CBI(int i,int j,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	int d,ii,jj;

	for(d=j-i-3;d>=4&&d>=j-i-32;--d)
		for(ii=i+1;ii<j-d&&ii<=length;++ii)
		{
			jj=d+ii;
			if(traceback==0)
			{
				d_DPT[id*1340+pos]=-1.0;
				d_DPT[id*1340+pos+1]=1.0*INFINITY;
			}
			if(fabs(d_DPT[id*1340+(ii-1)*(length-1)+jj-1])<999999999)
			{
				calc_bulge_internal2(i,j,ii,jj,pos,traceback,length,d_DPT,d_numSeq,id);
				if(fabs(d_DPT[id*1340+pos+1])<999999999)
				{
					if(d_DPT[id*1340+pos] <-2500.0)
					{
						d_DPT[id*1340+pos+1]=-3224.0;
						d_DPT[id*1340+pos+1]=0.0;
					}
					if(traceback==0)
					{
						d_DPT[id*1340+(i-1)*(length-1)+j-1]=d_DPT[id*1340+pos+1];
						d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=d_DPT[id*1340+pos];
					}
				}
			}
		}
	return;
}

__device__ int find_pos(char *ref,int ref_start,int start,int length,int num)
{
	int flag,i,j;

	for(i=0;i<num;i++)
	{
		flag=0;
		for(j=0;j<length;j++)
		{
			if(ref[ref_start+j]!=d_Pchar[start+i*length+j])
			{
				flag++;
				break;
			}
		}
		if(flag==0)
			return i;
	}
	return -1;
}

__device__ void calc_hairpin(int i,int j,int pos_start,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	int pos,loopSize=j-i-1;
	double T1,T2;
	
	if(loopSize < 3)
	{
		d_DPT[id*1340+pos_start]=-1.0;
		d_DPT[id*1340+pos_start+1]=1.0*INFINITY;
		return;
	}
	if(i<=length&&length<j)
	{
		d_DPT[id*1340+pos_start]=-1.0;
		d_DPT[id*1340+pos_start+1]=1.0*INFINITY;
		return;
	}
	else if(i>length)
	{
		i-= length;
		j-= length;
	}
	if(loopSize<=30)
	{
		d_DPT[id*1340+pos_start+1]=parameter[3090+loopSize-1];
		d_DPT[id*1340+pos_start]=parameter[3000+loopSize-1];
	}
	else
	{
		d_DPT[id*1340+pos_start+1]=parameter[3090+29];
		d_DPT[id*1340+pos_start]=parameter[3000+29];
	}

	if(loopSize>3) // for loops 4 bp and more in length, terminal mm are accounted
	{
		d_DPT[id*1340+pos_start+1]+=parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
		d_DPT[id*1340+pos_start]+=parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
	}
	else if(loopSize == 3) // for loops 3 bp in length at-penalty is considered
	{
		d_DPT[id*1340+pos_start+1]+=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
		d_DPT[id*1340+pos_start]+=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
	}

	if(loopSize==3) // closing AT-penalty (+), triloop bonus, hairpin of 3 (+) 
	{
		pos=find_pos(d_numSeq,(id*54+i),5*d_NumL[0],5,d_NumL[0]);
		if(pos!=-1)
			d_DPT[id*1340+pos_start+1]+=parameter[5730+d_NumL[0]+pos];

		pos=find_pos(d_numSeq,(id*54+i),0,5,d_NumL[0]);
		if(pos!=-1)
			d_DPT[id*1340+pos_start]+=parameter[5730+pos];
	}
	else if (loopSize == 4) // terminal mismatch, tetraloop bonus, hairpin of 4
	{
		pos=find_pos(d_numSeq,(id*54+i),10*d_NumL[0]+6*d_NumL[1],6,d_NumL[1]);
		if(pos!=-1)
			d_DPT[id*1340+pos_start+1]+=parameter[5730+2*d_NumL[0]+d_NumL[1]+pos];

		pos=find_pos(d_numSeq,(id*54+i),10*d_NumL[0],6,d_NumL[1]);
		if(pos!=-1)
			d_DPT[id*1340+pos_start]+=parameter[5730+2*d_NumL[0]+pos];
	}
	if(fabs(d_DPT[id*1340+pos_start+1])>999999999)
	{
		d_DPT[id*1340+pos_start+1] =1.0*INFINITY;
		d_DPT[id*1340+pos_start] = -1.0;
	}
	T1 = (d_DPT[id*1340+pos_start+1] +d_DPT[id*1340+1302]) / ((d_DPT[id*1340+pos_start] +d_DPT[id*1340+1303]+ d_DPT[id*1340+1304]));
	T2 = (d_DPT[id*1340+(i-1)*(length-1)+j-1] +d_DPT[id*1340+1302]) / ((d_DPT[id*1340+625+(i-1)*(length-1)+j-1]) +d_DPT[id*1340+1303]+ d_DPT[id*1340+1304]);
	if(T1 < T2 && traceback == 0)
	{
		d_DPT[id*1340+pos_start] =d_DPT[id*1340+625+(i-1)*(length-1)+j-1];
		d_DPT[id*1340+pos_start+1] =d_DPT[id*1340+(i-1)*(length-1)+j-1];
	}
	return;
}

__device__ void fillMatrix2(int length,double *d_DPT,char *d_numSeq,int id)
{
	int i, j;

	for (j = 2; j <= length; ++j)
		for (i = j - 3 - 1; i >= 1; --i)
		{
			if (fabs(d_DPT[id*1340+(i-1)*(length-1)+j-1])<999999999)
			{
				d_DPT[id*1340+1310] = -1.0;
				d_DPT[id*1340+1311] =1.0*INFINITY;
				maxTM2(i,j,length,d_DPT,d_numSeq,id);
				CBI(i,j,1310,0,length,d_DPT,d_numSeq,id);

				d_DPT[id*1340+1310] = -1.0;
				d_DPT[id*1340+1311]=1.0*INFINITY;
				calc_hairpin(i,j,1310,0,length,d_DPT,d_numSeq,id);
				if(fabs(d_DPT[id*1340+1311])<999999999)
				{
					if(d_DPT[id*1340+1310]<-2500.0) /* to not give dH any value if dS is unreasonable */
					{
						d_DPT[id*1340+1310]=-3224.0;
						d_DPT[id*1340+1311]= 0.0;
					}
					d_DPT[id*1340+625+(i-1)*(length-1)+j-1]=d_DPT[id*1340+1310];
					d_DPT[id*1340+(i-1)*(length-1)+j-1]=d_DPT[id*1340+1311];
				}
			}
		}
}

__device__ int max5(double a,double b,double c,double d,double e)
{
	if(a>b&&a>c&&a>d&&a>e)
		return 1;
	else if(b>c&&b>d&&b>e)
		return 2;
	else if(c>d&&c>e)
		return 3;
	else if(d>e)
		return 4;
	else
		return 5;
}

__device__ double Sd5(int i,int j,char *d_numSeq,int id)
{
	return parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
}

__device__ double Hd5(int i,int j,char *d_numSeq,int id)
{
	return parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
}

__device__ double Sd3(int i,int j,char *d_numSeq,int id)
{
	return parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+j]];
}

__device__ double Hd3(int i,int j,char *d_numSeq,int id)
{
	return parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+j]];
}

__device__ double Ststack(int i,int j,char *d_numSeq,int id)
{
	return parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
}

__device__ double Htstack(int i,int j,char *d_numSeq,int id)
{
	return parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
}

__device__ double END5_1(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id)
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	max_tm=-1.0*INFINITY;
	H_max=1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-5;++k)
	{
		T1=(d_DPT[id*1340+1276+k]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1250+k]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		T2=d_DPT[id*1340+1302]/(d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(T1>=T2)
		{
			H=d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+k*(length-1)+i-1];
			S=d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+625+k*(length-1)+i-1];
			if(fabs(H)>999999999||H>0||S>0)  // H and S must be greater than 0 to avoid BS
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}
		else
		{
			H=parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+k*(length-1)+i-1];
			S=parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+625+k*(length-1)+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_2(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id)
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-6;++k)
	{
		T1=(d_DPT[id*1340+1276+k]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1250+k]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		T2=d_DPT[id*1340+1302]/(d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(T1>=T2)
		{
			H=d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Hd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-1];
			S=d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Sd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}
		else
		{
			H=parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Hd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-1];
			S=parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Sd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_3(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id)
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-6;++k)
	{
		T1=(d_DPT[id*1340+1276+k]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1250+k]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		T2=d_DPT[id*1340+1302]/(d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(T1>=T2)
		{
			H=d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Hd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+k*(length-1)+i-2];
			S=d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Sd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+625+k*(length-1)+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}
		else
		{
			H=parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Hd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+k*(length-1)+i-2];
			S=parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Sd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+625+k*(length-1)+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_4(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id)
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-7;++k)
	{
		T1=(d_DPT[id*1340+1276+k]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1250+k]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		T2=d_DPT[id*1340+1302]/(d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(T1>=T2)
		{
			H=d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Htstack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-2];
			S=d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Ststack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		}
		else
		{
			H=parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Htstack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-2];
			S=parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Ststack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/(S+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
 		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ void calc_terminal_bp(double temp,int length,double *d_DPT,char *d_numSeq,int id)
{
	int i,max;
//back here
	
	d_DPT[id*1340+1250+0]=d_DPT[id*1340+1250+1]= -1.0;
	d_DPT[id*1340+1276+0]=d_DPT[id*1340+1276+1]=1.0*INFINITY;

	for(i=2;i<=length;i++)
	{
		d_DPT[id*1340+1250+i]=-3224.0;
		d_DPT[id*1340+1276+i]=0;
	}

// adding terminal penalties to 3' end and to 5' end 
	for(i=2;i<=length;++i)
	{
		max=0;
		d_DPT[id*1340+1310]=(d_DPT[id*1340+1276+i-1]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1250+i-1]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		d_DPT[id*1340+1315]=END5_1(i,1,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1316]=END5_1(i,2,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1311]=(d_DPT[id*1340+1315]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1316]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		d_DPT[id*1340+1317]=END5_2(i,1,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1318]=END5_2(i,2,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1312]=(d_DPT[id*1340+1317]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1318]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		d_DPT[id*1340+1319]=END5_3(i,1,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1320]=END5_3(i,2,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1313]=(d_DPT[id*1340+1319]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1320]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		d_DPT[id*1340+1321]=END5_4(i,1,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1322]=END5_4(i,2,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1314]=(d_DPT[id*1340+1321]+d_DPT[id*1340+1302])/(d_DPT[id*1340+1322]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);

		max=max5(d_DPT[id*1340+1310],d_DPT[id*1340+1311],d_DPT[id*1340+1312],d_DPT[id*1340+1313],d_DPT[id*1340+1314]);
		switch(max)
		{
			case 1:
				d_DPT[id*1340+1250+i]=d_DPT[id*1340+1250+i-1];
				d_DPT[id*1340+1276+i]=d_DPT[id*1340+1276+i-1];
				break;
			case 2:
				if(d_DPT[id*1340+1315]<temp*d_DPT[id*1340+1316])
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1316];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1315];
				}
				else
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1250+i-1];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1276+i-1];
				}
				break;
			case 3:
				if(d_DPT[id*1340+1317]<temp*d_DPT[id*1340+1318])
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1318];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1317];
				}
				else
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1250+i-1];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1276+i-1];
				}
				break;
			case 4:
				if(d_DPT[id*1340+1319]<temp*d_DPT[id*1340+1320])
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1320];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1319];
				}
				else
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1250+i-1];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1276+i-1];
				}
				break;
			case 5:
				if(d_DPT[id*1340+1321]<temp*d_DPT[id*1340+1322])
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1322];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1321];
				}
				else
				{
					d_DPT[id*1340+1250+i]=d_DPT[id*1340+1250+i-1];
					d_DPT[id*1340+1276+i]=d_DPT[id*1340+1276+i-1];
				}
				break;
			default:
				break;
		}
	}
}

__device__ int newpush(int store[],int i,int j,int mtrx,int total,int next)
{
        int k;
        for(k=total-1;k>=next;k--)
        {
                store[(k+1)*3]=store[k*3];
                store[(k+1)*3+1]=store[k*3+1];
                store[(k+1)*3+2]=store[k*3+2];
        }
        store[next*3]=i;                  
        store[next*3+1]=j;
        store[next*3+2]=mtrx;

        return total+1;           
}

__device__ int equal(double a,double b)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
		return 0;
	return fabs(a-b)<1e-5;
}

__device__ void tracebacku(int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
	int i,j,store[50],total,now,ii,jj,k,d,done;

        total=newpush(store,length,0,1,0,0);
        now=0;
        while(now<total)
        {
                i=store[3*now]; // top->i;
                j=store[3*now+1]; // top->j;
                if(store[now*3+2]==1)
                {
                        while(equal(d_DPT[id*1340+1250+i],d_DPT[id*1340+1250+i-1])&&equal(d_DPT[id*1340+1276+i],d_DPT[id*1340+1276+i-1])) // if previous structure is the same as this one
                                --i;
                        if(i==0)
                                continue;
                        if(equal(d_DPT[id*1340+1250+i],END5_1(i,2,length,d_DPT,d_numSeq,id))&&equal(d_DPT[id*1340+1276+i],END5_1(i,1,length,d_DPT,d_numSeq,id)))
                        {
                                for(k=0;k<=i-5;++k)
                                        if(equal(d_DPT[id*1340+1250+i],parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+625+k*(length-1)+i-1])&&equal(d_DPT[id*1340+1276+i],parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+k*(length-1)+i-1]))
                                        {
                                                total=newpush(store,k+1,i,0,total,now+1);                    
                                                break;
                                        }
                                        else if(equal(d_DPT[id*1340+1250+i],d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+625+k*(length-1)+i-1])&&equal(d_DPT[id*1340+1276+i],d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1340+k*(length-1)+i-1]))
                                        {
                                                total=newpush(store,k+1,i,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(d_DPT[id*1340+1250+i],END5_2(i,2,length,d_DPT,d_numSeq,id))&&equal(d_DPT[id*1340+1276+i],END5_2(i,1,length,d_DPT,d_numSeq,id)))
                        {
                                for (k=0;k<=i-6;++k)
                                        if(equal(d_DPT[id*1340+1250+i],parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Sd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-1])&&equal(d_DPT[id*1340+1276+i],parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Hd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-1]))
                                        {
                                                total=newpush(store,k+2,i,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(d_DPT[id*1340+1250+i],d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Sd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-1])&&equal(d_DPT[id*1340+1276+i],d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i]]+Hd5(i,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-1]))
                                        {
                                                total=newpush(store,k+2,i,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(d_DPT[id*1340+1250+i],END5_3(i,2,length,d_DPT,d_numSeq,id))&&equal(d_DPT[id*1340+1276+i],END5_3(i,1,length,d_DPT,d_numSeq,id)))
                        {
                                for (k=0;k<=i-6;++k)
                                        if(equal(d_DPT[id*1340+1250+i],parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Sd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+625+k*(length-1)+i-2])&&equal(d_DPT[id*1340+1276+i],parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Hd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+k*(length-1)+i-2]))
                                        {
                                                total=newpush(store,k+1,i-1,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(d_DPT[id*1340+1250+i],d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Sd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+625+k*(length-1)+i-2])&&equal(d_DPT[id*1340+1276+i],d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+1]*5+d_numSeq[id*54+i-1]]+Hd3(i-1,k+1,d_numSeq,id)+d_DPT[id*1340+k*(length-1)+i-2]))
                                        {
                                                total=newpush(store,k+1,i-1,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(d_DPT[id*1340+1250+i],END5_4(i,2,length,d_DPT,d_numSeq,id))&&equal(d_DPT[id*1340+1276+i],END5_4(i,1,length,d_DPT,d_numSeq,id)))
                        {
                                for (k=0;k<=i-7;++k)
                                        if(equal(d_DPT[id*1340+1250+i],parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Ststack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-2])&&equal(d_DPT[id*1340+1276+i],parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Htstack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-2]))
                                        {
                                                total=newpush(store,k+2,i-1,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(d_DPT[id*1340+1250+i],d_DPT[id*1340+1250+k]+parameter[5680+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Ststack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+625+(k+1)*(length-1)+i-2])&&equal(d_DPT[id*1340+1276+i],d_DPT[id*1340+1276+k]+parameter[5705+d_numSeq[id*54+k+2]*5+d_numSeq[id*54+i-1]]+Htstack(i-1,k+2,d_numSeq,id)+d_DPT[id*1340+(k+1)*(length-1)+i-2]))
                                        {
                                                total=newpush(store,k+2,i-1,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                }
                else if(store[3*now+2]==0)
                {
                        d_ps[id*50+i-1]=j;
                        d_ps[id*50+j-1]=i;
                        d_DPT[id*1340+1310]=-1.0;
                        d_DPT[id*1340+1311]=1.0*INFINITY;
                        calc_hairpin(i,j,1310,1,length,d_DPT,d_numSeq,id);

                        d_DPT[id*1340+1312]=-1.0;
                        d_DPT[id*1340+1313]=1.0*INFINITY;
                        CBI(i,j,1312,2,length,d_DPT,d_numSeq,id);

                        if (equal(d_DPT[id*1340+625+(i-1)*(length-1)+j-1],Ss(i,j,2,length,d_numSeq,id)+d_DPT[id*1340+625+i*(length-1)+j-2])&&equal(d_DPT[id*1340+(i-1)*(length-1)+j-1],Hs(i,j,2,length,d_numSeq,id)+d_DPT[id*1340+i*(length-1)+j-2]))
                                total=newpush(store,i+1,j-1,0,total,now+1);
                        else if(equal(d_DPT[id*1340+625+(i-1)*(length-1)+j-1],d_DPT[id*1340+1312])&&equal(d_DPT[id*1340+(i-1)*(length-1)+j-1],d_DPT[id*1340+1313]))
                        {
                                for (done=0,d=j-i-3;d>=4&&d>=j-i-32&&!done;--d)
                                        for (ii=i+1;ii<j-d;++ii)
                                        {
                                                jj=d+ii;
                                                d_DPT[id*1340+1314]=-1.0;
                                                d_DPT[id*1340+1315]=1.0*INFINITY;
                                                calc_bulge_internal2(i,j,ii,jj,1314,1,length,d_DPT,d_numSeq,id);

                                                if (equal(d_DPT[id*1340+625+(i-1)*(length-1)+j-1],d_DPT[id*1340+1314]+d_DPT[id*1340+625+(ii-1)*(length-1)+jj-1])&&equal(d_DPT[id*1340+(i-1)*(length-1)+j-1],d_DPT[id*1340+1315]+d_DPT[id*1340+(ii-1)*(length-1)+jj-1]))
                                                {
                                                        total=newpush(store,ii,jj,0,total,now+1);
                                                        ++done;
                                                        break;
                                                }
                                        }
                        }
                }
                now++;
        }
}

__device__ double drawHairpin(int *d_ps,int id,double mh,double ms,int length)
{
        int i,N;

        N=0;
        if(fabs(ms)>999999999||fabs(mh)>999999999)
        {
		return 0.0;
        }
        else
        {
		for(i=1;i<length;++i)
		{
			if(d_ps[id*50+i-1]>0)
				N++;
                }
                return mh/(ms+(((N/2)-1)*-0.51986))-273.15;
        }
}

__device__ void initMatrix(int length,double *d_DPT,char *d_numSeq,int id)
{
	int i,j;

	for(i=1;i<=length;++i)
	{
		for(j=1;j<=length;++j)
		{
			if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
			{
				d_DPT[id*1340+(i-1)*length+j-1]=1.0*INFINITY;
				d_DPT[id*1340+625+(i-1)*length+j-1]=-1.0;
			}
			else
			{
				d_DPT[id*1340+(i-1)*length+j-1]=0.0;
				d_DPT[id*1340+625+(i-1)*length+j-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,double *EntropyEnthalpy,int length,double *d_DPT,char *d_numSeq,int id)
{
	double S1,H1,T1,S2,H2,T2;

	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1340+625+(i-1)*length+j-1]=-1.0;
		d_DPT[id*1340+(i-1)*length+j-1]=1.0*INFINITY;
		return;
	}

	S1=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	H1=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			EntropyEnthalpy[0]=S2;
			EntropyEnthalpy[1]=H2;
		}
		else
		{
			EntropyEnthalpy[0]=S1;
			EntropyEnthalpy[1]=H1;
		}
	}
	else
	{
		EntropyEnthalpy[0]=S2;
		EntropyEnthalpy[1]=H2;
	}
	return;
}

__device__ void maxTM(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	double T0,T1,S0,S1,H0,H1;

	S0=d_DPT[id*1340+625+(i-1)*length+j-1];
	H0=d_DPT[id*1340+(i-1)*length+j-1];
	T0=(H0+d_DPT[id*1340+1302])/(S0+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]); // at current position 
	if(fabs(d_DPT[id*1340+(i-2)*length+j-2])<999999999&&fabs(Hs(i-1,j-1,1,length,d_numSeq,id))<999999999)
	{
		S1=(d_DPT[id*1340+625+(i-2)*length+j-2]+Ss(i-1,j-1,1,length,d_numSeq,id));
		H1=(d_DPT[id*1340+(i-2)*length+j-2]+Hs(i-1,j-1,1,length,d_numSeq,id));
	}
	else
	{
		S1=-1.0;
		H1=1.0*INFINITY;
	}
	T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);

	if(S1<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S1=-3224.0;
		H1=0.0;
	}
	if(S0<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S0=-3224.0;
		H0=0.0;
	}
	if((T1>T0)||(S0>0&&H0>0)) // T1 on suurem 
	{
		d_DPT[id*1340+625+(i-1)*length+j-1]=S1;
		d_DPT[id*1340+(i-1)*length+j-1]=H1;
	}
	else if(T0>=T1)
	{
		d_DPT[id*1340+625+(i-1)*length+j-1]=S0;
		d_DPT[id*1340+(i-1)*length+j-1]=H0;
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,double* EntropyEnthalpy,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	int loopSize1,loopSize2,loopSize,N,N_loop;
	double T1,T2,S,H;

	S=-3224.0;
	H=0;
	loopSize1=ii-i-1;
	loopSize2=jj-j-1;
	if(ii<jj)
	{
		N=i;
		N_loop=N;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
	}
	else
	{
		N=j;
		N_loop=2*jj;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
		N_loop=(N_loop/2)-1;
	}

	loopSize=loopSize1+loopSize2-1;
	if((loopSize1==0&&loopSize2>0)||(loopSize2==0&&loopSize1>0))// only bulges have to be considered
	{
		if(loopSize2==1||loopSize1==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((loopSize2==1&&loopSize1==0)||(loopSize2==0&&loopSize1==1))
			{
				H=parameter[3150+loopSize]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				S=parameter[3060+loopSize]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
			}
			H+=d_DPT[id*1340+(i-1)*length+j-1];
			S+=d_DPT[id*1340+625+(i-1)*length+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}

			T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
			T2=(d_DPT[id*1340+(ii-1)*length+jj-1]+d_DPT[id*1340+1302])/((d_DPT[id*1340+625+(ii-1)*length+jj-1])+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			H=parameter[3150+loopSize]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			H+=d_DPT[id*1340+(i-1)*length+j-1];

			S=parameter[3060+loopSize]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			S+=d_DPT[id*1340+625+(i-1)*length+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
			T2=(d_DPT[id*1340+(ii-1)*length+jj-1]+d_DPT[id*1340+1302])/(d_DPT[id*1340+625+(ii-1)*length+jj-1]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
	}
	else if(loopSize1==1&&loopSize2==1)
	{
		S=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		S+=d_DPT[id*1340+625+(i-1)*length+j-1];

		H=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		H+=d_DPT[id*1340+(i-1)*length+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
		T2=(d_DPT[id*1340+(ii-1)*length+jj-1]+d_DPT[id*1340+1302])/(d_DPT[id*1340+625+(ii-1)*length+jj-1]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if((T1-T2>=0.000001)||traceback==1)
		{
			if((T1>T2)||(traceback&&T1>=T2))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		return;
	}
	else // only internal loops
	{
		H=parameter[3120+loopSize]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		H+=d_DPT[id*1340+(i-1)*length+j-1];

		S=parameter[3030+loopSize]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(loopSize1-loopSize2));
		S+=d_DPT[id*1340+625+(i-1)*length+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+d_DPT[id*1340+1302])/((S+d_DPT[id*1340+1303])+d_DPT[id*1340+1304]);
		T2=(d_DPT[id*1340+(ii-1)*length+jj-1]+d_DPT[id*1340+1302])/((d_DPT[id*1340+625+(ii-1)*length+jj-1])+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
		{
			EntropyEnthalpy[0]=S;
			EntropyEnthalpy[1]=H;
		}
	}
	return;
}

__device__ void fillMatrix(int length,double *d_DPT,char *d_numSeq,int id)
{
	int d,i,j,ii,jj;
	double SH[2];

	for(i=1;i<=length;++i)
	{
		for(j=1;j<=length;++j)
		{
			if(fabs(d_DPT[id*1340+(i-1)*length+j-1])<999999999)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				LSH(i,j,SH,length,d_DPT,d_numSeq,id);

				if(fabs(SH[1])<999999999)
				{
					d_DPT[id*1340+625+(i-1)*length+j-1]=SH[0];
					d_DPT[id*1340+(i-1)*length+j-1]=SH[1];
				}
				if(i>1&&j>1)
				{
					maxTM(i,j,length,d_DPT,d_numSeq,id);
					for(d=3;d<=32;d++)
					{
						ii=i-1;
						jj=-ii-d+(j+i);
						if(jj<1)
						{
							ii-=abs(jj-1);
							jj=1;
						}
						for(;ii>0&&jj<j;--ii,++jj)
						{
							if(fabs(d_DPT[id*1340+(ii-1)*length+jj-1])<999999999)
							{
								SH[0]=-1.0;
								SH[1]=1.0*INFINITY;
								calc_bulge_internal(ii,jj,i,j,SH,0,length,d_DPT,d_numSeq,id);

								if(SH[0]<-2500.0)
								{
									SH[0] =-3224.0;
									SH[1] = 0.0;
								}
								if(fabs(SH[1])<999999999)
								{
									d_DPT[id*1340+(i-1)*length+j-1]=SH[1];
									d_DPT[id*1340+625+(i-1)*length+j-1]=SH[0];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double *d_DPT,char *d_numSeq,int id)
{
	double S1,S2,H1,H2,T1,T2;

	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1340+1306]=-1.0;
		d_DPT[id*1340+1307]=1.0*INFINITY;
		return;
	}
	S1=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	H1=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+d_DPT[id*1340+1302])/(S1+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	S2=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	H2=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	T2=(H2+d_DPT[id*1340+1302])/(S2+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			d_DPT[id*1340+1306]=S2;
			d_DPT[id*1340+1307]=H2;
		}
		else
		{
			d_DPT[id*1340+1306]=S1;
			d_DPT[id*1340+1307]=H1;
		}
	}
	else
	{
		d_DPT[id*1340+1306]=S2;
		d_DPT[id*1340+1307]=H2;
	}
	return;
}

__device__ void traceback(int i,int j,int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
	int d,ii,jj,done;
	double SH[2];

	d_ps[id*50+i-1]=j;
	d_ps[id*50+25+j-1]=i;
	while(1)
	{
		SH[0]=-1.0;
		SH[1]=1.0*INFINITY;
		LSH(i,j,SH,length,d_DPT,d_numSeq,id);
		if(equal(d_DPT[id*1340+625+(i-1)*length+j-1],SH[0])&&equal(d_DPT[id*1340+(i-1)*length+j-1],SH[1]))
			break;

		done = 0;
		if(i>1&&j>1&&equal(d_DPT[id*1340+625+(i-1)*length+j-1],Ss(i-1,j-1,1,length,d_numSeq,id)+d_DPT[id*1340+625+(i-2)*length+j-2]))
		{
			i=i-1;
			j=j-1;
			d_ps[id*50+i-1]=j;
			d_ps[id*50+25+j-1]=i;
			done=1;
		}
		for(d=3;!done&&d<=32;++d)
		{
			ii=i-1;
			jj=-ii-d+(j+i);
			if(jj<1)
			{
				ii-=abs(jj-1);
				jj=1;
			}
			for(;!done&&ii>0&&jj<j;--ii,++jj)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				calc_bulge_internal(ii,jj,i,j,SH,1,length,d_DPT,d_numSeq,id);
				if(equal(d_DPT[id*1340+625+(i-1)*length+j-1],SH[0])&&equal(d_DPT[id*1340+(i-1)*length+j-1],SH[1]))
				{
					i=ii;
					j=jj;
					d_ps[id*50+i-1]=j;
					d_ps[id*50+25+j-1]=i;
					done=1;
					break;
				}
			}
		}
	}
}

__device__ double drawDimer(int *d_ps,int id,double H,double S,double *d_DPT,int length)
{
        int i,N;

        if(fabs(d_DPT[id*1340+1305])>999999999)
                return (double)0.0;
        else
        {
                N=0;
                for(i=0;i<length;i++)
                {
                        if(d_ps[id*50+i]>0)
                                ++N;
                }
                for(i=0;i<length;i++)
                {
                        if(d_ps[id*50+25+i]>0)
                                ++N;
                }
                N=(N/2)-1;
                return (double)(H/(S+(N*-0.51986)+d_DPT[id*1340+1304])-273.15);
        }
}

__device__ int symmetry_thermo(char *d_seq,int start,int length)
{
	int i = 0;
	if(length%2==1)
		return 0;

	while(i<length/2)
	{
		if((d_seq[start+i]=='A'&&d_seq[start+length-1-i]!='T')||(d_seq[start+i]=='T'&&d_seq[start+length-1-i]!='A')||(d_seq[start+length-1-i]=='A'&&d_seq[start+i]!='T')||(d_seq[start+length-1-i]=='T'&&d_seq[start+i]!='A'))
			return 0;
		if((d_seq[start+i]=='C'&&d_seq[start+length-1-i]!='G')||(d_seq[start+i]=='G'&&d_seq[start+length-1-i]!='C')||(d_seq[start+length-1-i]=='C'&&d_seq[start+i]!='G')||(d_seq[start+length-1-i]=='G'&&d_seq[start+i]!='C'))
			return 0;
		i++;
	}
	return 1;
}

__device__ void thal(char *d_seq,int start,int length,int strand_flag,int type,char *d_numSeq,int id,double *d_DPT,int *d_ps)
{
	int Initint[5]; //0 is len1, 1 is len2, 2 is len3, 3 is bestI, 4 is bestJ
	int i, j;

/*** INIT values for unimolecular and bimolecular structures ***/
	if (type==4) /* unimolecular folding */
	{
		d_DPT[id*1340+1302]= 0.0;
		d_DPT[id*1340+1303] = -0.00000000001;
		d_DPT[id*1340+1304]=0;
	}
	else /* hybridization of two oligos */
	{
		d_DPT[id*1340+1302]= 200;
		d_DPT[id*1340+1303]= -5.7;
		if(symmetry_thermo(d_seq,start,length))
			d_DPT[id*1340+1304]=1.9872* log(38/1000000000.0);
		else
			d_DPT[id*1340+1304]=1.9872* log(38/4000000000.0);
	}
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		if(strand_flag==0) //plus
		{
	 		for(i=1;i<=length;++i)
			{
				str2int(d_seq[start+i-1],d_numSeq,(id*54+i));
				str2int(d_seq[start+length-i],d_numSeq,(id*54+27+i));
			}
		}
		else
		{
			for(i=1;i<=length;++i)
			{
				str2int_rev(d_seq[start+length-i],d_numSeq,(id*54+i));
				str2int_rev(d_seq[start+i-1],d_numSeq,(id*54+27+i));
			}
		}
	}
	else
	{
		if(strand_flag==0)
		{
                	for(i=1;i<=length;++i)
			{
				str2int(d_seq[start+i-1],d_numSeq,(id*54+i));
				d_numSeq[id*54+27+i]=d_numSeq[id*54+i];
			}
		}
		else
		{
			for(i=1;i<=length;++i)
			{
				str2int_rev(d_seq[start+length-i],d_numSeq,(id*54+i));
				d_numSeq[id*54+27+i]=d_numSeq[id*54+i];
			}
		}
	}
	d_numSeq[id*54+0]=d_numSeq[id*54+length+1]=d_numSeq[id*54+27+0]=d_numSeq[id*54+27+length+1]=4; /* mark as N-s */

	d_DPT[id*1340+1309]=0;
	if (type==4) /* calculate structure of monomer */
	{
		initMatrix2(length,d_DPT,d_numSeq,id);
		fillMatrix2(length,d_DPT,d_numSeq,id);
		calc_terminal_bp(310.15,length,d_DPT,d_numSeq,id);
		d_DPT[id*1340+1306]=d_DPT[id*1340+1276+length];
		d_DPT[id*1340+1307]=d_DPT[id*1340+1250+length];
		for (i=0;i<length;i++)
			d_ps[id*50+i]=0;
		if(fabs(d_DPT[id*1340+1306])<999999999)
		{
			tracebacku(d_ps,length,d_DPT,d_numSeq,id);
			d_DPT[id*1340+1309]=drawHairpin(d_ps,id,d_DPT[id*1340+1306],d_DPT[id*1340+1307],length);
			d_DPT[id*1340+1309]=(int)(d_DPT[id*1340+1309]*100+0.5)/100.0;
		}
	}
	else  /* Hybridization of two moleculs */
	{
		initMatrix(length,d_DPT,d_numSeq,id);
		fillMatrix(length,d_DPT,d_numSeq,id);

		d_DPT[id*1340+1305]=-1.0*INFINITY;
	/* calculate terminal basepairs */
		Initint[3]=Initint[4]=0;
		if(type==1)
			for (i=1;i<=length;i++)
			{
				for (j=1;j<=length;j++)
				{
					RSH(i,j,d_DPT,d_numSeq,id);
					d_DPT[id*1340+1306]=d_DPT[id*1340+1306]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
					d_DPT[id*1340+1307]=d_DPT[id*1340+1307]+0.000001;
					d_DPT[id*1340+1308]=((d_DPT[id*1340+(i-1)*length+j-1]+d_DPT[id*1340+1307]+d_DPT[id*1340+1302]) / ((d_DPT[id*1340+625+(i-1)*length+j-1])+d_DPT[id*1340+1306]+d_DPT[id*1340+1303] + d_DPT[id*1340+1304])) -273.15;
					if(d_DPT[id*1340+1308]>d_DPT[id*1340+1305]&&((d_DPT[id*1340+625+(i-1)*length+j-1]+d_DPT[id*1340+1306])<0&&(d_DPT[id*1340+1307]+d_DPT[id*1340+(i-1)*length+j-1])<0))
					{
						d_DPT[id*1340+1305]=d_DPT[id*1340+1308];
						Initint[3]=i;
						Initint[4]=j;
					}
				}
			}
		if(type==2)
		{
		 //THAL_END1
			Initint[4]=0;
			Initint[3]=length;
			i=length;
			d_DPT[id*1340+1305]=-1.0*INFINITY;
			for (j=1;j<=length;++j)
			{
				RSH(i,j,d_DPT,d_numSeq,id);
				d_DPT[id*1340+1306]=d_DPT[id*1340+1306]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
				d_DPT[id*1340+1307]=d_DPT[id*1340+1307]+0.000001;
				d_DPT[id*1340+1308]=((d_DPT[id*1340+(i-1)*length+j-1]+d_DPT[id*1340+1307]+d_DPT[id*1340+1302])/((d_DPT[id*1340+625+(i-1)*length+j-1])+d_DPT[id*1340+1306]+d_DPT[id*1340+1303]+d_DPT[id*1340+1304]))-273.15;
				if (d_DPT[id*1340+1308]>d_DPT[id*1340+1305]&&((d_DPT[id*1340+1306]+d_DPT[id*1340+625+(i-1)*length+j-1])<0&&(d_DPT[id*1340+1307]+d_DPT[id*1340+(i-1)*length+j-1])<0))
				{
					d_DPT[id*1340+1305]=d_DPT[id*1340+1308];
					Initint[4]=j;
				}
			}
		}
		if(fabs(d_DPT[id*1340+1305])>999999999)
			Initint[3]=Initint[4]=1;
		RSH(Initint[3],Initint[4],d_DPT,d_numSeq,id);
	 // tracebacking 
		for (i=0;i<length;++i)
			d_ps[id*50+i]=0;
		for (j=0;j<length;++j)
			d_ps[id*50+25+j] = 0;
		if(fabs(d_DPT[id*1340+(Initint[3]-1)*length+Initint[4]-1])<999999999)
		{
			traceback(Initint[3],Initint[4],d_ps,length,d_DPT,d_numSeq,id);
			d_DPT[id*1340+1309]=drawDimer(d_ps,id,(d_DPT[id*1340+(Initint[3]-1)*length+Initint[4]-1]+d_DPT[id*1340+1307]+d_DPT[id*1340+1302]),(d_DPT[id*1340+625+(Initint[3]-1)*length+Initint[4]-1]+d_DPT[id*1340+1306]+d_DPT[id*1340+1303]),d_DPT,length);
			d_DPT[id*1340+1309]=(int)(d_DPT[id*1340+1309]*100+0.5)/100.0;
		}
	}
}

///function in gpu, check the GC-content; int length: the length of read
__device__ int gc(char *d_seq,int start,int length)
{
	int i,number;
	float gc;

	number=0;
	for(i=0;i<length;i++)
	{
		if(d_seq[start+i]=='C')
		{
			number++;
			continue;
		}
	
		if(d_seq[start+i]=='G')
		{
			number++;
		}
	}

	gc=1.0*number/length*100;
	if((gc<40)||(gc>65))
	{
		return 0;
	}
	return 1;
}

///function in gpu, translate A...G to int
__device__ int translate(char a)
{
	if(a=='A')
		return 0;
	if(a=='T')
		return 1;
	if(a=='C')
		return 2;
	return 3;
}

__device__ int translate_rev(char a)
{
        if(a=='T')
                return 0;
        if(a=='A')
                return 1;
        if(a=='G')
                return 2;
        return 3;
}
//function in gpu, caculate tm
__device__ int tm(char *d_seq,int start,int length,float max_tm,float min_tm)
{
	int i,pos;
	float deltah,deltas,result;

	deltah=0;
	deltas=0;
	for(i=0;i<length-1;i++)
	{
		pos=translate(d_seq[start+i]);
		pos=pos*4+translate(d_seq[start+i+1]);
		deltah+=d_deltah[pos];
		deltas+=d_deltas[pos];
	}

	deltah=(-1.0)*deltah;
	deltas=(-1.0)*deltas;
	if((d_seq[start]=='A')||(d_seq[start]=='T'))
	{
		deltah+=2.3;
		deltas+=4.1;
	}
	else
	{
		deltah+=0.1;
		deltas-=2.8;
	}
        if((d_seq[start+length-1]=='A')||(d_seq[start+length-1]=='T'))
        {
                deltah+=2.3;
                deltas+=4.1;
        }
        else
        {
                deltah+=0.1;
                deltas-=2.8;
        }
	result=1000.0*deltah/(deltas-0.51986*(length-1)-36.70381)-273.15;
	if((result<min_tm)||(result>max_tm))
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

///function in gpu, caculate stability, int strand: 0 is 5' and 1 is 3'
__device__ int stability(char *d_seq,int start,int flag,int length,int strand)//flag=0: plus
{
	int i,pos;
	
	pos=0;
	for(i=0;i<6;i++)
	{
		if(flag==0)
		{
			if(strand==0)
			{
				pos=pos*4+translate(d_seq[start+i]);
			}
			else
			{
				pos=pos*4+translate(d_seq[start+i+length-6]);
			}
		}
		else
		{
			if(strand==0)
                        {
                                pos=pos*4+translate_rev(d_seq[start+length-1-i]);
                        }
                        else
                        {
                                pos=pos*4+translate_rev(d_seq[start+5-i]);
                        }
		}
	}
	
	if(d_stab[pos]<4)
	{
		return 0;
	}
//the other part
        pos=0;
        for(i=0;i<6;i++)
        {
		if(flag==0)
		{
                	if(strand==1)
                	{
				pos=pos*4+translate(d_seq[start+i]);
                	}
                	else
                	{
				pos=pos*4+translate(d_seq[start+i+length-6]);
                	}
		}
		else //minus
		{
			if(strand==1)
                        {
                                pos=pos*4+translate_rev(d_seq[start+length-1-i]);      
                        }
                        else
                        {
                                pos=pos*4+translate_rev(d_seq[start+5-i]);
                        }
		}
        }

        if(d_stab[pos]<3)
        {
                return 0;
        }

	return 1;
}

//function in gpu: whether species chars in reads
__device__ int words(char *d_seq,int position,int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(d_seq[position+i]=='N')
		{
			return 0;
		}
	}
	return 1;
}

__device__ int check_long_ploy(char *d_seq,int start,int length)
{
        int i,same;
        char ref;

        same=1;
        ref=d_seq[start];
        for(i=1;i<length;i++)
        {
                if(d_seq[start+i]==ref)
                        same++;
                else
                {
                        if(same>=6)
                                return 0;
                        same=1;
                        ref=d_seq[start+i];
                }
        }
        if(same>=6)
                return 0;
        return 1;
}

///function: int length: the length of genome
__global__ void candidate_primer(char *d_seq,int *d_pos,int *d_len,int *d_rev_len,int strand,float max_tm,float min_tm,int length,int check_flag,char *d_numSeq,double *d_DPT,int *d_ps)
{
	int id,i,circle,check,plus,minus;

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(circle=id;circle<length;circle=circle+blockDim.x*gridDim.x)
	{
		for(i=0;i<8;i++)   //primer length is from 18 to 25
		{
			d_len[8*circle+i]=0;
			d_rev_len[8*circle+i]=0;
		}
		d_pos[circle]=0;
	
		for(i=18;i<=25;i++)  //read length is from 18 to 25
		{
			if(circle+i>length)
				break;
			check=words(d_seq,circle,i);
			if(check==0)
                                break;

			check=gc(d_seq,circle,i);
			if(check==0)
				continue;

			check=check_long_ploy(d_seq,circle,i);
			if(check==0)
                                continue;

			check=tm(d_seq,circle,i,max_tm,min_tm);
			if(check==0)
				continue;

                        check=stability(d_seq,circle,0,i,strand);
                        if(check==1)     //+ strand
				plus=1;
			else
				plus=0;
			
		//secondary structure
			if(check_flag&&plus)
			{
				thal(d_seq,circle,i,0,1,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)
					plus=0;	
			}
			if(check_flag&&plus)
                        {
                                thal(d_seq,circle,i,0,2,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)  
                                        plus=0;
                        }
			if(check_flag&&plus)
                        {
                                thal(d_seq,circle,i,0,4,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)
                                        plus=0;         
                        }
			if(plus)
                                d_len[circle*8+i-18]=1;
	//reverse
			check=stability(d_seq,circle,1,i,strand);
			if(check==1)
				minus=1;
			else
				minus=0;
		//secondary structure      
                        if(check_flag&&minus)
                        {                
                                thal(d_seq,circle,i,1,1,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)
                                        minus=0;         
                        }           
                        if(check_flag&&minus)
                        {
                                thal(d_seq,circle,i,1,2,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)
                                        minus=0;
                        }                
                        if(check_flag&&minus)
                        {
                                thal(d_seq,circle,i,1,4,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1340+1309]>min_tm-10)
                                        minus=0;
                        }
                        if(minus)
				d_rev_len[circle*8+i-18]=1;
		}
		
		for(i=0;i<8;i++)
		{
			d_pos[circle]+=(d_len[circle*8+i]+d_rev_len[8*circle+i]);
		}
	}
	__syncthreads();
}

void usage()
{
        printf("Usage:\n");
        printf("    single  -in <fasta_file>  -out <primers_file_name>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the reference sequence file, fasta formate\n");
        printf("    -out  <string>:  the prefix of output files, those files store candidate single primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers. default is current directory\n");
        printf("    -stab <string>:  the parameter file used in calculating the primers' stability. default is stab_parameter.txt in Par/ directory\n");
        printf("    -tm   <string>:  the parameter file used in calcalating Tm and second structure. default is stab_parameter.txt in Par/ directory\n");
	printf("    -check   <int>:  0: don't check primers' secondary structure; !=0: check, default is 1\n");
        printf("    -par  <string>:  the directory of storing parameter files used to check primers' secondary structure, default is Par/\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region. high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design candidate loop single primers\n");
        printf("    -h/-help:        print usage\n");
}

int create_file(char *prefix,char *dir,char *seq,int *pos,int *len,int *rev_len,int length)
{
	char *file;
	int total,i,j;
	FILE *OUT;

	total=0;
	i=strlen(dir)+strlen(prefix)+20;
	file=(char *)malloc(i);
        memset(file,'\0',i);
        strcpy(file,dir);
        strcat(file,prefix);
        OUT=fopen(file,"w");
        if(OUT==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
	
        for(i=0;i<length;i++)
        {
                if(pos[i]==0)
                        continue;
                for(j=0;j<8;j++)
                {
                        if((len[8*i+j]+rev_len[8*i+j])==0)
                                continue;
                       	fprintf(OUT,"pos:%d\tlength:%d\t+:%d\t-:%d\n",i,(j+18),len[8*i+j],rev_len[8*i+j]);
			total++;
                }
        }
	fclose(OUT);
	free(file);
	return total;
}

main(int argc, char **argv)
{
	double *H_parameter,*d_DPT;
	int *pos,*d_pos,*len,*d_len,length,flag[10],i,*rev_len,*d_rev_len,num_outer,num_inner,num_loop,NumL[2],thread,block,*d_ps;
	float deltah[16],deltas[16],stab[4096],temp1,temp2;
	char *seq,*d_seq,*store_path,*prefix,*stab_path,*tm_path,*curren_path,*input,*outer,*inner,*loop,*par_path,*temp,*Pchar,*d_numSeq;
	FILE *fp;
	time_t start,end;
        struct stat statbuf;
//flag: 0:input; 1: out_prefix; 2: dir; 3: stab; 4: tm; 5: high; 6: low; 7: loop; 8: secondary structure; 9: path for secondary structure

	start=time(NULL);
	thread=200;
	block=200;
//get input
        for(i=0;i<10;i++)
        {
                flag[i]=0;
        }
	flag[8]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			input=(char *)malloc(length+1);
			memset(input,'\0',length+1);
                        strcpy(input,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        prefix=(char *)malloc(length+1);
                        memset(prefix,'\0',length+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			if(argv[i+1][length-1]=='/')
			{
                        	store_path=(char *)malloc(length+1);
                        	memset(store_path,'\0',length+1);
                        	strcpy(store_path,argv[i+1]);
			}
			else
			{
				store_path=(char *)malloc(length+2);
				memset(store_path,'\0',length+2);
				strcpy(store_path,argv[i+1]);
				store_path[length]='/';
			}
                        i=i+2;
                }
                else if(strcmp(argv[i],"-stab")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-stab\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        stab_path=(char *)malloc(length+1);
                        memset(stab_path,'\0',length+1);
                        strcpy(stab_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-tm")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        tm_path=(char *)malloc(length+1);
                        memset(tm_path,'\0',length+1);
                        strcpy(tm_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[6]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[7]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
		else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[8]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[9]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        length=strlen(argv[i+1]);
                        if(argv[i+1][length-1]=='/')
                        {
                                par_path=(char *)malloc(length+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(length+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='/';
                                par_path[length+1]='\0';
                        }
                        i=i+2;
                }		
                else
                {
                        printf("Error: don't have the parameter: %s\n",argv[i]);
                        usage();
                        exit(1);
                }
        }
//check paramters
        if(flag[5]+flag[6]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
        if(flag[0]==0)
        {
                printf("Error! Users must input the reference sequence file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the prefix name for output file with -out!\n");
                usage();
                exit(1);
        }
        for(i=0;i<strlen(prefix);i++)
        {
                if(prefix[i]=='/')
                {
                        printf("Error! the -out parameter couldn't contain any directory!\n");
                        usage();
                        exit(1);
                }
        }
//prepare
	inner=(char *)malloc(4096);
        memset(inner,'\0',4096);
        getcwd(inner,4096);
        length=strlen(inner);
        curren_path=(char *)malloc(length+1);
        memset(curren_path,'\0',length+1);
        strcpy(curren_path,inner);
        if(flag[2]==0)
        {
                store_path=(char *)malloc(length+2);
                memset(store_path,'\0',length+2);
                strcpy(store_path,curren_path);
                store_path[length]='/';
        }
        free(inner);

        length=strlen(store_path)+12;
        outer=(char *)malloc(length);
        memset(outer,'\0',length);
        strcpy(outer,store_path);

        inner=(char *)malloc(length);
        memset(inner,'\0',length);
        strcpy(inner,store_path);

        if(flag[7]==1)
        {
                loop=(char *)malloc(length);
                memset(loop,'\0',length);
                strcpy(loop,store_path);
        }
        if(flag[5]==1)
        {
                strcat(outer,"high-outer/");
                strcat(inner,"high-inner/");
                if(flag[7]==1)
                        strcat(loop,"high-loop/");
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(inner,"low-inner/");
                if(flag[7]==1)
                        strcat(loop,"low-loop/");
        }
        mkdir(outer,0755);
        mkdir(inner,0755);        
        if(flag[7]==1)
                mkdir(loop,0755);        

//stability parameter file
        if(flag[3]==0)
        {
		length=strlen(curren_path);
                stab_path=(char *)malloc(length+30);
                memset(stab_path,'\0',length+30);
                strcpy(stab_path,curren_path);
                i=length-1;
                while(stab_path[i]!='/'&&i>=0)
                {
                        stab_path[i]='\0';
                        i--;
                }
                strcat(stab_path,"Par/stab_parameter.txt");
        }
//tm parameter file
        if(flag[4]==0)
        {
		length=strlen(curren_path);
                tm_path=(char *)malloc(length+30);
                memset(tm_path,'\0',length+30);
                strcpy(tm_path,curren_path);
                i=length-1;
                while(tm_path[i]!='/'&&i>=0)
                {
                        tm_path[i]='\0';
                        i--;
                }
                strcat(tm_path,"Par/tm_nn_parameter.txt");
        }
//secondary structure
	if(flag[8]&&flag[9]==0)
        {
                length=strlen(curren_path);
                par_path=(char *)malloc(length+10);
                memset(par_path,'\0',length+10);
                strcpy(par_path,curren_path);
                i=length-1;
                while(par_path[i]!='/'&&i>=0)
                {
                        par_path[i]='\0';
                        i--;
                }
                strcat(par_path,"Par/");
        }
	if(flag[8])
	{
		NumL[0]=get_num_line(par_path,0);
	        NumL[1]=get_num_line(par_path,1);
	        H_parameter=(double *)malloc((5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        memset(H_parameter,'\0',(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        Pchar=(char *)malloc(10*NumL[0]+12*NumL[1]);
	        memset(Pchar,'\0',10*NumL[0]+12*NumL[1]);

		getStack(par_path,H_parameter);
	        getStackint2(par_path,H_parameter);
	        getDangle(par_path,H_parameter);
	        getLoop(par_path,H_parameter);
	        getTstack(par_path,H_parameter);
	        getTstack2(par_path,H_parameter);
	        getTriloop(par_path,H_parameter,Pchar,NumL);
	        getTetraloop(par_path,H_parameter,Pchar,NumL);
	        tableStartATS(6.9,H_parameter);
	        tableStartATH(2200.0,H_parameter);

		hipMemcpyToSymbol(HIP_SYMBOL(d_NumL),NumL,2*sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_Pchar),Pchar,10*NumL[0]+12*NumL[1]);
		hipMemcpyToSymbol(HIP_SYMBOL(parameter),H_parameter,(5730+2*NumL[0]+2*NumL[1])*sizeof(double));

		hipMalloc((void **)&d_numSeq,54*thread*block*sizeof(char));
		hipMalloc((void **)&d_DPT,1340*thread*block*sizeof(double));
		hipMalloc((void **)&d_ps,50*thread*block*sizeof(int));
	}

//input reference sequence
        if(access(input,0)==-1)
        {
                printf("Error! Don't have the %s file.\n",input);
                exit(1);
        }
        stat(input,&statbuf);
        length=statbuf.st_size;
        length=length+100;
        temp=(char *)malloc(length);
        memset(temp,'\0',length);
        seq=(char *)malloc(length*sizeof(char));
        memset(seq,'\0',length*sizeof(char));

        fp=fopen(input,"r");   //open the sequence file
        if(fp==NULL)
        {
                printf("Error! can't open the %s file!\n",input);
                exit(1);
        }
        fread(temp,length*sizeof(char),1,fp);
        fclose(fp); 

        length=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
		if(temp[i]=='a'||temp[i]=='A')
                        seq[length]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[length]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[length]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[length]='G';
                else
                        seq[length]='N';
                i++;
                length++;
        }
        free(temp);
        length=strlen(seq);

//input Tm parameter
        fp=fopen(tm_path,"r");  //read the paramter of deltah and deltas
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",tm_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f\t%f",&i,&temp1,&temp2)!=EOF)
        {
                deltah[i]=temp1;
                deltas[i]=temp2;
        }
        fclose(fp);

//input stability parameter
        fp=fopen(stab_path,"r");  //read the parameters of stability
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",stab_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f",&i,&temp1)!=EOF)
        {
                stab[i]=temp1;
        }
        fclose(fp);

	hipMalloc((void **)&d_seq,length*sizeof(char));
	hipMemset(d_seq,'\0',length*sizeof(char));

	/////from cpu to gpu
	hipMemcpy(d_seq,seq,length*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltah),deltah,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltas),deltas,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_stab),stab,4096*sizeof(float));

	hipMalloc((void **)&d_pos,length*sizeof(int));
	hipMemset(d_pos,'\0',length*sizeof(int));
	hipMalloc((void **)&d_len,8*length*sizeof(int));
	hipMemset(d_len,'\0',8*length*sizeof(int));
	hipMalloc((void **)&d_rev_len,8*length*sizeof(int));
        hipMemset(d_rev_len,'\0',8*length*sizeof(int));
	pos=(int *)malloc(length*sizeof(int));
	memset(pos,'\0',length*sizeof(int));
	len=(int *)malloc(8*length*sizeof(int));
	memset(len,'\0',8*length*sizeof(int));
        rev_len=(int *)malloc(8*length*sizeof(int));
        memset(rev_len,'\0',8*length*sizeof(int));

	end=time(NULL);
	printf("It takes %d seconds to prepare.\n",(int)difftime(end,start));
	start=time(NULL);
	if(flag[5]==1)
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
		hipMemset(d_len,'\0',8*length*sizeof(int));
		hipMemset(d_rev_len,'\0',8*length*sizeof(int));
		candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
		hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,0,66,64,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);

                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,66,64,length,flag[8],d_numSeq,d_DPT,d_ps);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
        else
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,56,54,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,0,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);
                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
	hipFree(d_pos);
	hipFree(d_len);
	hipFree(d_rev_len);
	hipFree(d_seq);
	free(pos);
        free(len);
        free(rev_len);
	free(seq);

	printf("There ara %d candidate primers used as F3/F2/B2/B3.\n",num_outer);
        printf("There are %d candidate primers used as F1c/B1c.\n",num_inner);
        if(flag[7]==1)
                printf("There are %d candidate primers used as LF/LB.\n",num_loop);
        //check
        if(num_outer<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(num_inner<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && num_loop<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
        printf("It takes %d seconds to design candidate single primers.\n",(int)difftime(end,start));

	free(store_path);
	free(prefix);
	free(stab_path);
	free(tm_path);
	free(curren_path);
	free(input);
	free(outer);
	free(inner);
	if(flag[7])
		free(loop);

	if(flag[8])
	{
		free(Pchar);
		free(H_parameter);
		hipFree(d_numSeq);
		hipFree(d_DPT);
		hipFree(d_ps);
	}
	if(flag[8]||flag[9])
		free(par_path);
}
