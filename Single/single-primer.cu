#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<sys/stat.h>

__constant__ int d_NumL[2];
__constant__ char d_Pchar[1084];
__constant__ double parameter[5916];
__constant__ float d_stab[4096];
__constant__ float d_deltah[16];
__constant__ float d_deltas[16];

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ void str2int(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'A':
                        d_numSeq[id]=0;
                        break;
                case 'C':
                        d_numSeq[id]=1;
                        break;
                case 'G':
                        d_numSeq[id]=2;
                        break;
                case 'T':
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

__device__ void str2int_rev(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'T':
                        d_numSeq[id]=0;
                        break;
                case 'G':
                        d_numSeq[id]=1;
                        break;
                case 'C':
                        d_numSeq[id]=2;
                        break;                 
                case 'A':               
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

int get_num_line(char *path,int flag)
{
	FILE *fp;
	int i,size;
	char *line;

	i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
	if(flag==0)
	        strcat(line,"triloop.ds");
	else
		strcat(line,"tetraloop.ds");

        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        fp=fopen(line,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	size=0;
	while(fgets(line,i,fp)!=NULL)
		size++;
	return size;
}

void getTriloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i,turn;
        char *line,seq[10],value[10];
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
	
	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for (i=0;i<5;i++)
			Pchar[5*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+turn]=1.0*INFINITY;
		else
			parameter[5730+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

	i=strlen(path)+20;
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<5;i++)
			Pchar[5*NumL[0]+turn*5+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void getTetraloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i, turn;
        char *line,seq[10],value[10];

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+turn*6+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);
        
	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+6*NumL[1]+6*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+NumL[1]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+NumL[1]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ void initMatrix2(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*117+104]=1;d_ps[id*117+104]<=length;++d_ps[id*117+104])
		for(d_ps[id*117+105]=d_ps[id*117+104];d_ps[id*117+105]<=length;++d_ps[id*117+105])
			if(d_ps[id*117+105]-d_ps[id*117+104]<4 || (d_numSeq[id*54+d_ps[id*117+104]]+d_numSeq[id*54+d_ps[id*117+105]]!=3))
			{
				d_DPT[id*1331+(d_ps[id*117+104]-1)*(length-1)+d_ps[id*117+105]-1]=1.0*INFINITY;
				d_DPT[id*1331+625+(d_ps[id*117+104]-1)*(length-1)+d_ps[id*117+105]-1]=-1.0;
			}
			else
			{
				d_DPT[id*1331+(d_ps[id*117+104]-1)*(length-1)+d_ps[id*117+105]-1]=0.0;
				d_DPT[id*1331+625+(d_ps[id*117+104]-1)*(length-1)+d_ps[id*117+105]-1]=-3224.0;
			}
}

__device__ void Ss(int i,int j,int k,int length,char *d_numSeq,int id,double *d_DPT)
{
	if(k==2)
	{
		if(i>=j)
		{
			d_DPT[id*1331+1330]=-1.0;
			return;
		}
		if(i==length||j==length+1)
		{
			d_DPT[id*1331+1330]=-1.0;
			return;
		}

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		d_DPT[id*1331+1330]=parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
	}
	else
		d_DPT[id*1331+1330]=parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ void Hs(int i,int j,int k,int length,char *d_numSeq,int id,double *d_DPT)
{
	if(k==2)
	{
		if(i>= j)
		{
			d_DPT[id*1331+1330]=1.0*INFINITY;
			return;
		}
		if(i==length||j==length+1)
		{
			d_DPT[id*1331+1330]=1.0*INFINITY;
			return;
		}

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		if(fabs(parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]])<999999999)
			d_DPT[id*1331+1330]=parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
		else
			d_DPT[id*1331+1330]=1.0*INFINITY;
	}
	else
		d_DPT[id*1331+1330]=parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ void maxTM2(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=d_DPT[id*1331+625+(i-1)*(length-1)+j-1];
	d_DPT[id*1331+1316]=d_DPT[id*1331+(i-1)*(length-1)+j-1];
	d_DPT[id*1331+1312]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1314]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+(i-1)*(length-1)+j-1])<999999999)
	{
		Ss(i,j,2,length,d_numSeq,id,d_DPT);
		d_DPT[id*1331+1315]=(d_DPT[id*1331+625+i*(length-1)+j-2]+d_DPT[id*1331+1330]);
		Hs(i,j,2,length,d_numSeq,id,d_DPT);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+i*(length-1)+j-2]+d_DPT[id*1331+1330]);
	}
	else
	{
		d_DPT[id*1331+1315]=-1.0;
		d_DPT[id*1331+1317]=1.0*INFINITY;
	}
	d_DPT[id*1331+1313]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(d_DPT[id*1331+1315]<-2500.0)
	{
		d_DPT[id*1331+1315]=-3224.0;
		d_DPT[id*1331+1317]=0.0;
	}
	if(d_DPT[id*1331+1314]<-2500.0)
	{
		d_DPT[id*1331+1314]=-3224.0;
		d_DPT[id*1331+1316]=0.0;
 	}

	if(d_DPT[id*1331+1313]>d_DPT[id*1331+1312])
	{
		d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1315];
		d_DPT[id*1331+(i-1)*(length-1)+j-1]= d_DPT[id*1331+1317];
	}
	else
	{
		d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1314];
		d_DPT[id*1331+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1316];
	}
}

__device__ void calc_bulge_internal2(int i,int j,int ii,int jj,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{

	d_DPT[id*1331+1318]=-3224.0;
	d_DPT[id*1331+1319]=0.0;

	if(ii-i-2+j-jj>30)
	{
		d_DPT[id*1331+pos]=-1.0;
		d_DPT[id*1331+pos+1]=1.0*INFINITY;
		return;
	}

	if((ii-i-1==0&&j-jj-1>0)||(j-jj-1==0&&ii-i-1>0))
	{
		if(j-jj-1==1||ii-i-1==1)
		{ 
			if((j-jj-1==1&&ii-i-1==0)||(j-jj-1==0&&ii-i-1==1))
			{
				d_DPT[id*1331+1319]=parameter[3150+ii-i+j-jj-3]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				d_DPT[id*1331+1318]=parameter[3060+ii-i+j-jj-3]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
 			}
			if(traceback!=1)
			{
				d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];
				d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
			}

			if(fabs(d_DPT[id*1331+1319])>999999999)
			{
				d_DPT[id*1331+1319]=1.0*INFINITY;
				d_DPT[id*1331+1318]=-1.0;
			}
			d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(i-1)*(length-1)+j-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||traceback==1))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
		else
		{
			d_DPT[id*1331+1319]=parameter[3150+ii-i+j-jj-3]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];

			d_DPT[id*1331+1318]=parameter[3060+ii-i+j-jj-3]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
			if(fabs(d_DPT[id*1331+1319])>999999999)
			{
				d_DPT[id*1331+1319]=1.0*INFINITY;
				d_DPT[id*1331+1318]=-1.0;
			}
			d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||(traceback==1)))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
	}
	else if(ii-i-1==1&&j-jj-1==1)
	{
		d_DPT[id*1331+1318]=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];

		d_DPT[id*1331+1319]=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];
		if(fabs(d_DPT[id*1331+1319])>999999999)
		{
			d_DPT[id*1331+1319]=1.0*INFINITY;
			d_DPT[id*1331+1318]=-1.0;
		}
		d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1316]-d_DPT[id*1331+1317]>=0.000001)||traceback)
		{
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>= d_DPT[id*1331+1317])||traceback==1))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
		return;
	}
	else
	{
		d_DPT[id*1331+1319]=parameter[3120+ii-i+j-jj-3]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];

		d_DPT[id*1331+1318]=parameter[3030+ii-i+j-jj-3]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(ii-i-j+jj));
		if(traceback!=1)
			d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
		if(fabs(d_DPT[id*1331+1319])>999999999)
		{
			d_DPT[id*1331+1319]=1.0*INFINITY;
			d_DPT[id*1331+1318]=-1.0;
		}

		d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(i-1)*(length-1)+j-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||(traceback==1)))
		{
			d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
			d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
		}
	}
	return;
}

__device__ void CBI(int i,int j,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*117+104]=j-i-3;d_ps[id*117+104]>=4&&d_ps[id*117+104]>=j-i-32;--d_ps[id*117+104])
		for(d_ps[id*117+105]=i+1;d_ps[id*117+105]<j-d_ps[id*117+104]&&d_ps[id*117+105]<=length;++d_ps[id*117+105])
		{
			d_ps[id*117+106]=d_ps[id*117+104]+d_ps[id*117+105];
			if(traceback==0)
			{
				d_DPT[id*1331+pos]=-1.0;
				d_DPT[id*1331+pos+1]=1.0*INFINITY;
			}
			if(fabs(d_DPT[id*1331+(d_ps[id*117+105]-1)*(length-1)+d_ps[id*117+106]-1])<999999999)
			{
				calc_bulge_internal2(i,j,d_ps[id*117+105],d_ps[id*117+106],pos,traceback,length,d_DPT,d_numSeq,id);
				if(fabs(d_DPT[id*1331+pos+1])<999999999)
				{
					if(d_DPT[id*1331+pos] <-2500.0)
					{
						d_DPT[id*1331+pos+1]=-3224.0;
						d_DPT[id*1331+pos+1]=0.0;
					}
					if(traceback==0)
					{
						d_DPT[id*1331+(i-1)*(length-1)+j-1]=d_DPT[id*1331+pos+1];
						d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+pos];
					}
				}
			}
		}
	return;
}

__device__ void find_pos(char *ref,int ref_start,int start,int length,int num,int *d_ps,int id)
{
	for(d_ps[id*117+105]=0;d_ps[id*117+105]<num;d_ps[id*117+105]++)
	{
		d_ps[id*117+104]=0;
		for(d_ps[id*117+106]=0;d_ps[id*117+106]<length;d_ps[id*117+106]++)
		{
			if(ref[ref_start+d_ps[id*117+106]]!=d_Pchar[start+d_ps[id*117+105]*length+d_ps[id*117+106]])
			{
				d_ps[id*117+104]++;
				break;
			}
		}
		if(d_ps[id*117+104]==0)
		{
			d_ps[id*117+107]=d_ps[id*117+105];
			return;
		}
	}
	d_ps[id*117+107]=-1;
}

__device__ void calc_hairpin(int i,int j,int pos_start,int traceback,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	if(j-i<4)
	{
		d_DPT[id*1331+pos_start]=-1.0;
		d_DPT[id*1331+pos_start+1]=1.0*INFINITY;
		return;
	}
	if(i<=length&&length<j)
	{
		d_DPT[id*1331+pos_start]=-1.0;
		d_DPT[id*1331+pos_start+1]=1.0*INFINITY;
		return;
	}
	else if(i>length)
	{
		i-= length;
		j-= length;
	}
	if(j-i-1<=30)
	{
		d_DPT[id*1331+pos_start+1]=parameter[3090+j-i-2];
		d_DPT[id*1331+pos_start]=parameter[3000+j-i-2];
	}
	else
	{
		d_DPT[id*1331+pos_start+1]=parameter[3090+29];
		d_DPT[id*1331+pos_start]=parameter[3000+29];
	}

	if(j-i>4) // for loops 4 bp and more in length, terminal mm are accounted
	{
		d_DPT[id*1331+pos_start+1]+=parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
		d_DPT[id*1331+pos_start]+=parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
	}
	else if(j-i==4) // for loops 3 bp in length at-penalty is considered
	{
		d_DPT[id*1331+pos_start+1]+=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
		d_DPT[id*1331+pos_start]+=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
	}

	if(j-i-1==3) // closing AT-penalty (+), triloop bonus, hairpin of 3 (+) 
	{
		find_pos(d_numSeq,(id*54+i),5*d_NumL[0],5,d_NumL[0],d_ps,id);
		if(d_ps[id*117+107]!=-1)
			d_DPT[id*1331+pos_start+1]+=parameter[5730+d_NumL[0]+d_ps[id*117+107]];

		find_pos(d_numSeq,(id*54+i),0,5,d_NumL[0],d_ps,id);
		if(d_ps[id*117+107]!=-1)
			d_DPT[id*1331+pos_start]+=parameter[5730+d_ps[id*117+107]];
	}
	else if (j-i-1== 4) // terminal mismatch, tetraloop bonus, hairpin of 4
	{
		find_pos(d_numSeq,(id*54+i),10*d_NumL[0]+6*d_NumL[1],6,d_NumL[1],d_ps,id);
		if(d_ps[id*117+107]!=-1)
			d_DPT[id*1331+pos_start+1]+=parameter[5730+2*d_NumL[0]+d_NumL[1]+d_ps[id*117+107]];

		find_pos(d_numSeq,(id*54+i),10*d_NumL[0],6,d_NumL[1],d_ps,id);
		if(d_ps[id*117+107]!=-1)
			d_DPT[id*1331+pos_start]+=parameter[5730+2*d_NumL[0]+d_ps[id*117+107]];
	}
	if(fabs(d_DPT[id*1331+pos_start+1])>999999999)
	{
		d_DPT[id*1331+pos_start+1] =1.0*INFINITY;
		d_DPT[id*1331+pos_start] = -1.0;
	}
	d_DPT[id*1331+1316]= (d_DPT[id*1331+pos_start+1] +d_DPT[id*1331+1302]) / ((d_DPT[id*1331+pos_start] +d_DPT[id*1331+1303]+ d_DPT[id*1331+1304]));
	d_DPT[id*1331+1317]= (d_DPT[id*1331+(i-1)*(length-1)+j-1] +d_DPT[id*1331+1302]) / ((d_DPT[id*1331+625+(i-1)*(length-1)+j-1]) +d_DPT[id*1331+1303]+ d_DPT[id*1331+1304]);
	if(d_DPT[id*1331+1316]<d_DPT[id*1331+1317] && traceback == 0)
	{
		d_DPT[id*1331+pos_start] =d_DPT[id*1331+625+(i-1)*(length-1)+j-1];
		d_DPT[id*1331+pos_start+1] =d_DPT[id*1331+(i-1)*(length-1)+j-1];
	}
	return;
}

__device__ void fillMatrix2(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for (d_ps[id*117+109] = 2; d_ps[id*117+109] <= length; ++d_ps[id*117+109])
		for (d_ps[id*117+108] = d_ps[id*117+109] - 3 - 1; d_ps[id*117+108] >= 1; --d_ps[id*117+108])
		{
			if (fabs(d_DPT[id*1331+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1])<999999999)
			{
				d_DPT[id*1331+1310] = -1.0;
				d_DPT[id*1331+1311] =1.0*INFINITY;
				maxTM2(d_ps[id*117+108],d_ps[id*117+109],length,d_DPT,d_numSeq,id);
				CBI(d_ps[id*117+108],d_ps[id*117+109],1310,0,length,d_DPT,d_numSeq,id,d_ps);

				d_DPT[id*1331+1310] = -1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				calc_hairpin(d_ps[id*117+108],d_ps[id*117+109],1310,0,length,d_DPT,d_numSeq,id,d_ps);
				if(fabs(d_DPT[id*1331+1311])<999999999)
				{
					if(d_DPT[id*1331+1310]<-2500.0) /* to not give dH any value if dS is unreasonable */
					{
						d_DPT[id*1331+1310]=-3224.0;
						d_DPT[id*1331+1311]= 0.0;
					}
					d_DPT[id*1331+625+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1]=d_DPT[id*1331+1310];
					d_DPT[id*1331+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1]=d_DPT[id*1331+1311];
				}
			}
		}
}

__device__ void max5(double a,double b,double c,double d,double e,int *d_ps,int id)
{
	if(a>b&&a>c&&a>d&&a>e)
		d_ps[id*117+104]=1;
	else if(b>c&&b>d&&b>e)
		d_ps[id*117+104]=2;
	else if(c>d&&c>e)
		d_ps[id*117+104]=3;
	else if(d>e)
		d_ps[id*117+104]=4;
	else
		d_ps[id*117+104]=5;
}

__device__ void END5_1(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*117+104]=0;d_ps[id*117+104]<=i-5;++d_ps[id*117+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*117+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*117+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*117+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+d_ps[id*117+104]*(length-1)+i-1];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*117+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+625+d_ps[id*117+104]*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)  // H and S must be greater than 0 to avoid BS
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+d_ps[id*117+104]*(length-1)+i-1];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+625+d_ps[id*117+104]*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_2(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*117+104]=0;d_ps[id*117+104]<=i-6;++d_ps[id*117+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*117+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*117+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*117+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+2-1]]+d_DPT[id*1331+(d_ps[id*117+104]+1)*(length-1)+i-1];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*117+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+2-1]]+d_DPT[id*1331+625+(d_ps[id*117+104]+1)*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+2-1]]+d_DPT[id*1331+(d_ps[id*117+104]+1)*(length-1)+i-1];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+2-1]]+d_DPT[id*1331+625+(d_ps[id*117+104]+1)*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_3(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*117+104]=0;d_ps[id*117+104]<=i-6;++d_ps[id*117+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*117+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*117+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*117+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2625+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+d_ps[id*117+104]*(length-1)+i-2];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*117+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2500+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+625+d_ps[id*117+104]*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2625+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+d_ps[id*117+104]*(length-1)+i-2];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2500+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+625+d_ps[id*117+104]*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_4(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*117+104]=0;d_ps[id*117+104]<=i-7;++d_ps[id*117+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*117+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*117+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*117+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[5055+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+(d_ps[id*117+104]+1)*(length-1)+i-2];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*117+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[4430+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+625+(d_ps[id*117+104]+1)*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[5055+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+(d_ps[id*117+104]+1)*(length-1)+i-2];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[4430+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*117+104]+2]*5+d_numSeq[id*54+d_ps[id*117+104]+1]]+d_DPT[id*1331+625+(d_ps[id*117+104]+1)*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
 		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void calc_terminal_bp(double temp,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1250+0]=d_DPT[id*1331+1250+1]= -1.0;
	d_DPT[id*1331+1276+0]=d_DPT[id*1331+1276+1]=1.0*INFINITY;

	for(d_ps[id*117+105]=2;d_ps[id*117+105]<=length;d_ps[id*117+105]++)
	{
		d_DPT[id*1331+1250+d_ps[id*117+105]]=-3224.0;
		d_DPT[id*1331+1276+d_ps[id*117+105]]=0;
	}

// adding terminal penalties to 3' end and to 5' end 
	for(d_ps[id*117+105]=2;d_ps[id*117+105]<=length;++d_ps[id*117+105])
	{
		d_DPT[id*1331+1310]=(d_DPT[id*1331+1276+d_ps[id*117+105]-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*117+105]-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_1(d_ps[id*117+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1315]=d_DPT[id*1331+1330];
		END5_1(d_ps[id*117+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1316]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1311]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1316]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_2(d_ps[id*117+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1317]=d_DPT[id*1331+1330];
		END5_2(d_ps[id*117+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1318]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1318]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_3(d_ps[id*117+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1319]=d_DPT[id*1331+1330];
		END5_3(d_ps[id*117+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1320]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1313]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1320]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_4(d_ps[id*117+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1321]=d_DPT[id*1331+1330];
		END5_4(d_ps[id*117+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1322]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1314]=(d_DPT[id*1331+1321]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1322]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);

		max5(d_DPT[id*1331+1310],d_DPT[id*1331+1311],d_DPT[id*1331+1312],d_DPT[id*1331+1313],d_DPT[id*1331+1314],d_ps,id);
		switch(d_ps[id*117+104])
		{
			case 1:
				d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1250+d_ps[id*117+105]-1];
				d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1276+d_ps[id*117+105]-1];
				break;
			case 2:
				if(d_DPT[id*1331+1315]<temp*d_DPT[id*1331+1316])
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1316];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1315];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1250+d_ps[id*117+105]-1];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1276+d_ps[id*117+105]-1];
				}
				break;
			case 3:
				if(d_DPT[id*1331+1317]<temp*d_DPT[id*1331+1318])
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1318];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1317];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1250+d_ps[id*117+105]-1];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1276+d_ps[id*117+105]-1];
				}
				break;
			case 4:
				if(d_DPT[id*1331+1319]<temp*d_DPT[id*1331+1320])
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1320];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1319];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1250+d_ps[id*117+105]-1];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1276+d_ps[id*117+105]-1];
				}
				break;
			case 5:
				if(d_DPT[id*1331+1321]<temp*d_DPT[id*1331+1322])
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1322];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1321];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*117+105]]=d_DPT[id*1331+1250+d_ps[id*117+105]-1];
					d_DPT[id*1331+1276+d_ps[id*117+105]]=d_DPT[id*1331+1276+d_ps[id*117+105]-1];
				}
				break;
			default:
				break;
		}
	}
}

__device__ void newpush(int *d_ps,int id,int i,int j,int mtrx,int total,int next)
{
        for(d_ps[id*117+104]=total-1;d_ps[id*117+104]>=next;d_ps[id*117+104]--)
        {
                d_ps[id*117+50+(d_ps[id*117+104]+1)*3]=d_ps[id*117+50+d_ps[id*117+104]*3];
                d_ps[id*117+50+(d_ps[id*117+104]+1)*3+1]=d_ps[id*117+50+d_ps[id*117+104]*3+1];
                d_ps[id*117+50+(d_ps[id*117+104]+1)*3+2]=d_ps[id*117+50+d_ps[id*117+104]*3+2];
        }
        d_ps[id*117+50+next*3]=i;                  
        d_ps[id*117+50+next*3+1]=j;
        d_ps[id*117+50+next*3+2]=mtrx;
}

__device__ int equal(double a,double b)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
		return 0;
	return fabs(a-b)<1e-5;
}

__device__ void tracebacku(int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
        newpush(d_ps,id,length,0,1,0,0);
	d_ps[id*117+110]=1;
        d_ps[id*117+111]=0;
        while(d_ps[id*117+111]<d_ps[id*117+110])
        {
                d_ps[id*117+108]=d_ps[id*117+50+3*d_ps[id*117+111]]; // top->i;
                d_ps[id*117+109]=d_ps[id*117+50+3*d_ps[id*117+111]+1]; // top->j;
                if(d_ps[id*117+50+d_ps[id*117+111]*3+2]==1)
                {
                        while(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1250+d_ps[id*117+108]-1])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1276+d_ps[id*117+108]-1])) // if previous structure is the same as this one
                                --d_ps[id*117+108];
                        if(d_ps[id*117+108]==0)
                                continue;
			END5_1(d_ps[id*117+108],2,length,d_DPT,d_numSeq,id,d_ps);
			d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
			END5_1(d_ps[id*117+108],1,length,d_DPT,d_numSeq,id,d_ps);
                        if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1329])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1330]))
                        {
                               	for(d_ps[id*117+114]=0;d_ps[id*117+114]<=d_ps[id*117+108]-5;++d_ps[id*117+114])
                               	        if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]]]+d_DPT[id*1331+625+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-1])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]]]+d_DPT[id*1331+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-1]))
                               	        {
                               	                newpush(d_ps,id,d_ps[id*117+114]+1,d_ps[id*117+108],0,d_ps[id*117+110],d_ps[id*117+111]+1);
						d_ps[id*117+110]++;                    
                               	                break;
                               	        }
                               	        else if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1250+d_ps[id*117+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]]]+d_DPT[id*1331+625+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-1])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1276+d_ps[id*117+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]]]+d_DPT[id*1331+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-1]))
                               	        {
                               	                newpush(d_ps,id,d_ps[id*117+114]+1,d_ps[id*117+108],0,d_ps[id*117+110],d_ps[id*117+111]+1);
						d_ps[id*117+110]++;
                               	                newpush(d_ps,id,d_ps[id*117+114],0,1,d_ps[id*117+110],d_ps[id*117+111]+1);
						d_ps[id*117+110]++;
                               	                break;
                               	        }
                        }
                        else
			{
				END5_2(d_ps[id*117+108],2,length,d_DPT,d_numSeq,id,d_ps);
				d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
				END5_2(d_ps[id*117+108],1,length,d_DPT,d_numSeq,id,d_ps);
				if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1329])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1330]))
                        	{
                                	for (d_ps[id*117+114]=0;d_ps[id*117+114]<=d_ps[id*117+108]-6;++d_ps[id*117+114])
                                	        if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]]]+parameter[2750+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+2-1]]+d_DPT[id*1331+625+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-1])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]]]+parameter[2875+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+2-1]]+d_DPT[id*1331+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-1]))
                                	        {
                                	                newpush(d_ps,id,d_ps[id*117+114]+2,d_ps[id*117+108],0,d_ps[id*117+110],d_ps[id*117+111]+1);
							d_ps[id*117+110]++;
                                	                break;
                                	        }
                                	        else if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1250+d_ps[id*117+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]]]+parameter[2750+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+2-1]]+d_DPT[id*1331+625+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-1])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1276+d_ps[id*117+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]]]+parameter[2875+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-1]))
                                	        {
                                	                newpush(d_ps,id,d_ps[id*117+114]+2,d_ps[id*117+108],0,d_ps[id*117+110],d_ps[id*117+111]+1);
							d_ps[id*117+110]++;
                                	                newpush(d_ps,id,d_ps[id*117+114],0,1,d_ps[id*117+110],d_ps[id*117+111]+1);
							d_ps[id*117+110]++;
                                	                break;
                                	        }
                        	}
				else
				{
					END5_3(d_ps[id*117+108],2,length,d_DPT,d_numSeq,id,d_ps);
					d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
					END5_3(d_ps[id*117+108],1,length,d_DPT,d_numSeq,id,d_ps);
                        		if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1329])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1330]))
		                        {
		                                for (d_ps[id*117+114]=0;d_ps[id*117+114]<=d_ps[id*117+108]-6;++d_ps[id*117+114])
		                                        if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[2500+d_numSeq[id*54+d_ps[id*117+108]-1]*25+d_numSeq[id*54+d_ps[id*117+108]]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+625+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-2])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[2625+d_numSeq[id*54+d_ps[id*117+108]-1]*25+d_numSeq[id*54+d_ps[id*117+108]]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-2]))
		                                        {
		                                                newpush(d_ps,id,d_ps[id*117+114]+1,d_ps[id*117+108]-1,0,d_ps[id*117+110],d_ps[id*117+111]+1);
								d_ps[id*117+110]++;
		                                                break;
		                                        }
		                                        else if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1250+d_ps[id*117+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[2500+d_numSeq[id*54+d_ps[id*117+108]-1]*25+d_numSeq[id*54+d_ps[id*117+108]]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+625+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-2])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1276+d_ps[id*117+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+1]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[2625+d_numSeq[id*54+d_ps[id*117+108]-1]*25+d_numSeq[id*54+d_ps[id*117+108]]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+d_ps[id*117+114]*(length-1)+d_ps[id*117+108]-2]))
        		                                {
                		                                newpush(d_ps,id,d_ps[id*117+114]+1,d_ps[id*117+108]-1,0,d_ps[id*117+110],d_ps[id*117+111]+1);
								d_ps[id*117+110]++;
                		                                newpush(d_ps,id,d_ps[id*117+114],0,1,d_ps[id*117+110],d_ps[id*117+111]+1);
								d_ps[id*117+110]++;
                		                                break;
                		                        }
                		        }
		                        else
					{
						END5_4(d_ps[id*117+108],2,length,d_DPT,d_numSeq,id,d_ps);
						d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
						END5_4(d_ps[id*117+108],1,length,d_DPT,d_numSeq,id,d_ps);
						if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1329])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1330]))
			                        {
			                                for (d_ps[id*117+114]=0;d_ps[id*117+114]<=d_ps[id*117+108]-7;++d_ps[id*117+114])
			                                        if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[4430+d_numSeq[id*54+d_ps[id*117+108]-1]*125+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+625+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-2])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[5055+d_numSeq[id*54+d_ps[id*117+108]-1]*125+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-2]))
			                                        {
			                                                newpush(d_ps,id,d_ps[id*117+114]+2,d_ps[id*117+108]-1,0,d_ps[id*117+110],d_ps[id*117+111]+1);
									d_ps[id*117+110]++;
			                                                break;
			                                        }
			                                        else if(equal(d_DPT[id*1331+1250+d_ps[id*117+108]],d_DPT[id*1331+1250+d_ps[id*117+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[4430+d_numSeq[id*54+d_ps[id*117+108]-1]*125+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+625+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-2])&&equal(d_DPT[id*1331+1276+d_ps[id*117+108]],d_DPT[id*1331+1276+d_ps[id*117+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+108]-1]]+parameter[5055+d_numSeq[id*54+d_ps[id*117+108]-1]*125+d_numSeq[id*54+d_ps[id*117+108]]*25+d_numSeq[id*54+d_ps[id*117+114]+2]*5+d_numSeq[id*54+d_ps[id*117+114]+1]]+d_DPT[id*1331+(d_ps[id*117+114]+1)*(length-1)+d_ps[id*117+108]-2]))
			                                        {
			                                                newpush(d_ps,id,d_ps[id*117+114]+2,d_ps[id*117+108]-1,0,d_ps[id*117+110],d_ps[id*117+111]+1);
									d_ps[id*117+110]++;
			                                                newpush(d_ps,id,d_ps[id*117+114],0,1,d_ps[id*117+110],d_ps[id*117+111]+1);
									d_ps[id*117+110]++;
			                                                break;
			                                        }
						}
		                        }
				}
			}
                }
                else if(d_ps[id*117+50+3*d_ps[id*117+111]+2]==0)
                {
                        d_ps[id*117+d_ps[id*117+108]-1]=d_ps[id*117+109];
                        d_ps[id*117+d_ps[id*117+109]-1]=d_ps[id*117+108];
                        d_DPT[id*1331+1310]=-1.0;
                        d_DPT[id*1331+1311]=1.0*INFINITY;
                        calc_hairpin(d_ps[id*117+108],d_ps[id*117+109],1310,1,length,d_DPT,d_numSeq,id,d_ps);

                        d_DPT[id*1331+1312]=-1.0;
                        d_DPT[id*1331+1313]=1.0*INFINITY;
                        CBI(d_ps[id*117+108],d_ps[id*117+109],1312,2,length,d_DPT,d_numSeq,id,d_ps);

			Ss(d_ps[id*117+108],d_ps[id*117+109],2,length,d_numSeq,id,d_DPT);
                        if (equal(d_DPT[id*1331+625+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1330]+d_DPT[id*1331+625+d_ps[id*117+108]*(length-1)+d_ps[id*117+109]-2]))
			{
				Hs(d_ps[id*117+108],d_ps[id*117+109],2,length,d_numSeq,id,d_DPT);
				if(equal(d_DPT[id*1331+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1330]+d_DPT[id*1331+d_ps[id*117+108]*(length-1)+d_ps[id*117+109]-2]))
                                newpush(d_ps,id,d_ps[id*117+108]+1,d_ps[id*117+109]-1,0,d_ps[id*117+110],d_ps[id*117+111]+1);
				d_ps[id*117+110]++;
			}
                        else if(equal(d_DPT[id*1331+625+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1312])&&equal(d_DPT[id*1331+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1313]))
                        {
                                for (d_ps[id*117+116]=0,d_ps[id*117+115]=d_ps[id*117+109]-d_ps[id*117+108]-3;d_ps[id*117+115]>=4&&d_ps[id*117+115]>=d_ps[id*117+109]-d_ps[id*117+108]-32&&!d_ps[id*117+116];--d_ps[id*117+115])
                                        for (d_ps[id*117+112]=d_ps[id*117+108]+1;d_ps[id*117+112]<d_ps[id*117+109]-d_ps[id*117+115];++d_ps[id*117+112])
                                        {
                                                d_ps[id*117+113]=d_ps[id*117+115]+d_ps[id*117+112];
                                                d_DPT[id*1331+1314]=-1.0;
                                                d_DPT[id*1331+1315]=1.0*INFINITY;
                                                calc_bulge_internal2(d_ps[id*117+108],d_ps[id*117+109],d_ps[id*117+112],d_ps[id*117+113],1314,1,length,d_DPT,d_numSeq,id);

                                                if (equal(d_DPT[id*1331+625+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1314]+d_DPT[id*1331+625+(d_ps[id*117+112]-1)*(length-1)+d_ps[id*117+113]-1])&&equal(d_DPT[id*1331+(d_ps[id*117+108]-1)*(length-1)+d_ps[id*117+109]-1],d_DPT[id*1331+1315]+d_DPT[id*1331+(d_ps[id*117+112]-1)*(length-1)+d_ps[id*117+113]-1]))
                                                {
                                                        newpush(d_ps,id,d_ps[id*117+112],d_ps[id*117+113],0,d_ps[id*117+110],d_ps[id*117+111]+1);
							d_ps[id*117+110]++;
                                                        ++d_ps[id*117+116];
                                                        break;
                                                }
                                        }
                        }
                }
                d_ps[id*117+111]++;
        }
}

__device__ void drawHairpin(int *d_ps,int id,double mh,double ms,int length,double *d_DPT)
{
        d_ps[id*117+105]=0;
        if(fabs(ms)>999999999||fabs(mh)>999999999)
		d_DPT[id*1331+1309]=0.0;
        else
        {
		for(d_ps[id*117+104]=1;d_ps[id*117+104]<length;++d_ps[id*117+104])
		{
			if(d_ps[id*117+d_ps[id*117+104]-1]>0)
				d_ps[id*117+105]++;
                }
                d_DPT[id*1331+1309]=mh/(ms+(((d_ps[id*117+105]/2)-1)*-0.51986))-273.15;
        }
}

__device__ void initMatrix(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*117+104]=1;d_ps[id*117+104]<=length;++d_ps[id*117+104])
	{
		for(d_ps[id*117+105]=1;d_ps[id*117+105]<=length;++d_ps[id*117+105])
		{
			if(d_numSeq[id*54+d_ps[id*117+104]]+d_numSeq[id*54+27+d_ps[id*117+105]]!=3)
			{
				d_DPT[id*1331+(d_ps[id*117+104]-1)*length+d_ps[id*117+105]-1]=1.0*INFINITY;
				d_DPT[id*1331+625+(d_ps[id*117+104]-1)*length+d_ps[id*117+105]-1]=-1.0;
			}
			else
			{
				d_DPT[id*1331+(d_ps[id*117+104]-1)*length+d_ps[id*117+105]-1]=0.0;
				d_DPT[id*1331+625+(d_ps[id*117+104]-1)*length+d_ps[id*117+105]-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=-1.0;
		d_DPT[id*1331+(i-1)*length+j-1]=1.0*INFINITY;
		return;
	}

	d_DPT[id*1331+1312]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	if(fabs(d_DPT[id*1331+1313])>999999999)
	{
		d_DPT[id*1331+1313]=1.0*INFINITY;
		d_DPT[id*1331+1312]=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}
	else if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}
	else if(fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}

	d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+1313])<999999999)
	{
		if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1316];
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1312];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1313];
		}
	}
	else
	{
		d_DPT[id*1331+1310]=d_DPT[id*1331+1315];
		d_DPT[id*1331+1311]=d_DPT[id*1331+1316];
	}
	return;
}

__device__ void maxTM(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=d_DPT[id*1331+625+(i-1)*length+j-1];
	d_DPT[id*1331+1316]=d_DPT[id*1331+(i-1)*length+j-1];
	d_DPT[id*1331+1312]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1314]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]); // at current position 
	if(fabs(d_DPT[id*1331+(i-2)*length+j-2])<999999999)
	{
		Hs(i-1,j-1,1,length,d_numSeq,id,d_DPT);
		if(fabs(d_DPT[id*1331+1330])<999999999)
		{
			Ss(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			d_DPT[id*1331+1315]=(d_DPT[id*1331+625+(i-2)*length+j-2]+d_DPT[id*1331+1330]);
			Hs(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-2)*length+j-2]+d_DPT[id*1331+1330]);
		}
	}
	else
	{
		d_DPT[id*1331+1315]=-1.0;
		d_DPT[id*1331+1317]=1.0*INFINITY;
	}
	d_DPT[id*1331+1313]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);

	if(d_DPT[id*1331+1315]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1331+1315]=-3224.0;
		d_DPT[id*1331+1317]=0.0;
	}
	if(d_DPT[id*1331+1314]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1331+1314]=-3224.0;
		d_DPT[id*1331+1316]=0.0;
	}
	if((d_DPT[id*1331+1313]>d_DPT[id*1331+1312])||(d_DPT[id*1331+1314]>0&&d_DPT[id*1331+1316]>0)) // T1 on suurem 
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=d_DPT[id*1331+1315];
		d_DPT[id*1331+(i-1)*length+j-1]=d_DPT[id*1331+1317];
	}
	else if(d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=d_DPT[id*1331+1314];
		d_DPT[id*1331+(i-1)*length+j-1]=d_DPT[id*1331+1316];
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=-3224.0;
	d_DPT[id*1331+1315]=0;

	if((ii-i==1&&jj-j-1>0)||(jj-j-1==0&&ii-i-1>0))// only bulges have to be considered
	{
		if(jj-j-1==1||ii-i-1==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((jj-j-1==1&&ii-i-1==0)||(jj-j-1==0&&ii-i-1==1))
			{
				d_DPT[id*1331+1315]=parameter[3150+ii-i-3+jj-j]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				d_DPT[id*1331+1314]=parameter[3060+ii-i-3+jj-j]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
			}
			d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];
			d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
			if(fabs(d_DPT[id*1331+1315])>999999999)
			{
				d_DPT[id*1331+1315]=1.0*INFINITY;
				d_DPT[id*1331+1314]=-1.0;
			}

			d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(ii-1)*length+jj-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			d_DPT[id*1331+1315]=parameter[3150+ii-i-3+jj-j]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];

			d_DPT[id*1331+1314]=parameter[3060+ii-i-3+jj-j]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
			if(fabs(d_DPT[id*1331+1315])>999999999)
			{
				d_DPT[id*1331+1315]=1.0*INFINITY;
				d_DPT[id*1331+1314]=-1.0;
			}
			d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(ii-1)*length+jj-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
	}
	else if(ii-i-1==1&&jj-j-1==1)
	{
		d_DPT[id*1331+1314]=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];

		d_DPT[id*1331+1315]=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];
		if(fabs(d_DPT[id*1331+1315])>999999999)
		{
			d_DPT[id*1331+1315]=1.0*INFINITY;
			d_DPT[id*1331+1314]=-1.0;
		}
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(ii-1)*length+jj-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1312]-d_DPT[id*1331+1313]>=0.000001)||traceback==1)
		{
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||(traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313]))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
		return;
	}
	else // only internal loops
	{
		d_DPT[id*1331+1315]=parameter[3120+ii-i-3+jj-j]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];

		d_DPT[id*1331+1314]=parameter[3030+ii-i-3+jj-j]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(ii-i-jj+j));
		d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
		if(fabs(d_DPT[id*1331+1315])>999999999)
		{
			d_DPT[id*1331+1315]=1.0*INFINITY;
			d_DPT[id*1331+1314]=-1.0;
		}
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(ii-1)*length+jj-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
		}
	}
	return;
}

__device__ void fillMatrix(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	int i;

	for(i=1;i<=length;i++)
	{
		for(d_ps[id*117+106]=1;d_ps[id*117+106]<=length;d_ps[id*117+106]++)
		{
			d_ps[id*117+105]=i;
			if(fabs(d_DPT[id*1331+(d_ps[id*117+105]-1)*length+d_ps[id*117+106]-1])<999999999)
			{
				d_DPT[id*1331+1310]=-1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				LSH(d_ps[id*117+105],d_ps[id*117+106],length,d_DPT,d_numSeq,id);

				if(fabs(d_DPT[id*1331+1311])<999999999)
				{
					d_DPT[id*1331+625+(d_ps[id*117+105]-1)*length+d_ps[id*117+106]-1]=d_DPT[id*1331+1310];
					d_DPT[id*1331+(d_ps[id*117+105]-1)*length+d_ps[id*117+106]-1]=d_DPT[id*1331+1311];
				}
				if(d_ps[id*117+105]>1&&d_ps[id*117+106]>1)
				{
					maxTM(d_ps[id*117+105],d_ps[id*117+106],length,d_DPT,d_numSeq,id);
					for(d_ps[id*117+104]=3;d_ps[id*117+104]<=32;d_ps[id*117+104]++)
					{
						d_ps[id*117+108]=d_ps[id*117+106]+1-d_ps[id*117+104];
						if(d_ps[id*117+108]<1)
						{
							d_ps[id*117+107]=d_ps[id*117+105]-1+d_ps[id*117+108]-1;
							d_ps[id*117+108]=1;
						}
						else
						{
							d_ps[id*117+107]=d_ps[id*117+105]-1;
						}
						for(;d_ps[id*117+107]>0&&d_ps[id*117+108]<d_ps[id*117+106];--d_ps[id*117+107],++d_ps[id*117+108])
						{
							if(fabs(d_DPT[id*1331+(d_ps[id*117+107]-1)*length+d_ps[id*117+108]-1])<999999999)
							{
								d_DPT[id*1331+1310]=-1.0;
								d_DPT[id*1331+1311]=1.0*INFINITY;
								calc_bulge_internal(d_ps[id*117+107],d_ps[id*117+108],i,d_ps[id*117+106],0,length,d_DPT,d_numSeq,id);

								if(d_DPT[id*1331+1310]<-2500.0)
								{
									d_DPT[id*1331+1310] =-3224.0;
									d_DPT[id*1331+1311] = 0.0;
								}
								if(fabs(d_DPT[id*1331+1311])<999999999)
								{
									d_DPT[id*1331+(i-1)*length+d_ps[id*117+106]-1]=d_DPT[id*1331+1311];
									d_DPT[id*1331+625+(i-1)*length+d_ps[id*117+106]-1]=d_DPT[id*1331+1310];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double *d_DPT,char *d_numSeq,int id)
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1331+1306]=-1.0;
		d_DPT[id*1331+1307]=1.0*INFINITY;
		return;
	}
	d_DPT[id*1331+1310]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	d_DPT[id*1331+1312]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	if(fabs(d_DPT[id*1331+1312])>999999999)
	{
		d_DPT[id*1331+1312]=1.0*INFINITY;
		d_DPT[id*1331+1310]=-1.0;
	}
	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}

	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}

	if(fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}
	d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+1312])<999999999)
	{
		if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
		{
			d_DPT[id*1331+1306]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1307]=d_DPT[id*1331+1313];
		}
		else
		{
			d_DPT[id*1331+1306]=d_DPT[id*1331+1310];
			d_DPT[id*1331+1307]=d_DPT[id*1331+1312];
		}
	}
	else
	{
		d_DPT[id*1331+1306]=d_DPT[id*1331+1311];
		d_DPT[id*1331+1307]=d_DPT[id*1331+1313];
	}
	return;
}

__device__ void traceback(int i,int j,int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_ps[id*117+i-1]=j;
	d_ps[id*117+25+j-1]=i;
	while(1)
	{
		d_DPT[id*1331+1310]=-1.0;
		d_DPT[id*1331+1311]=1.0*INFINITY;
		LSH(i,j,length,d_DPT,d_numSeq,id);
		if(equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1310])&&equal(d_DPT[id*1331+(i-1)*length+j-1],d_DPT[id*1331+1311]))
			break;

		d_ps[id*117+107] = 0;
		if(i>1&&j>1)
		{
			Ss(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			if(equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1330]+d_DPT[id*1331+625+(i-2)*length+j-2]))
			{
				i=i-1;
				j=j-1;
				d_ps[id*117+i-1]=j;
				d_ps[id*117+25+j-1]=i;
				d_ps[id*117+107]=1;
			}
		}
		for(d_ps[id*117+104]=3;!d_ps[id*117+107]&&d_ps[id*117+104]<=32;++d_ps[id*117+104])
		{
			d_ps[id*117+105]=i-1;
			d_ps[id*117+106]=-d_ps[id*117+105]-d_ps[id*117+104]+(j+i);
			if(d_ps[id*117+106]<1)
			{
				d_ps[id*117+105]-=abs(d_ps[id*117+106]-1);
				d_ps[id*117+106]=1;
			}
			for(;!d_ps[id*117+107]&&d_ps[id*117+105]>0&&d_ps[id*117+106]<j;--d_ps[id*117+105],++d_ps[id*117+106])
			{
				d_DPT[id*1331+1310]=-1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				calc_bulge_internal(d_ps[id*117+105],d_ps[id*117+106],i,j,1,length,d_DPT,d_numSeq,id);
				if(equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1310])&&equal(d_DPT[id*1331+(i-1)*length+j-1],d_DPT[id*1331+1311]))
				{
					i=d_ps[id*117+105];
					j=d_ps[id*117+106];
					d_ps[id*117+i-1]=j;
					d_ps[id*117+25+j-1]=i;
					d_ps[id*117+107]=1;
					break;
				}
			}
		}
	}
}

__device__ void drawDimer(int *d_ps,int id,double H,double S,double *d_DPT,int length)
{
        if(fabs(d_DPT[id*1331+1305])>999999999)
                d_DPT[id*1331+1309]=0.0;
        else
        {
                d_ps[id*117+105]=0;
                for(d_ps[id*117+104]=0;d_ps[id*117+104]<length;d_ps[id*117+104]++)
                {
                        if(d_ps[id*117+d_ps[id*117+104]]>0)
                                ++d_ps[id*117+105];
                }
                for(d_ps[id*117+104]=0;d_ps[id*117+104]<length;d_ps[id*117+104]++)
                {
                        if(d_ps[id*117+25+d_ps[id*117+104]]>0)
                                ++d_ps[id*117+105];
                }
                d_ps[id*117+105]=(d_ps[id*117+105]/2)-1;
                d_DPT[id*1331+1309]=(H/(S+(d_ps[id*117+105]*-0.51986)+d_DPT[id*1331+1304])-273.15);
        }
}

__device__ void symmetry_thermo(char *d_seq,int start,int length,int id, int *d_ps)
{
	if(length%2==1)
	{
		d_ps[id*117+101]=0;
		return;
	}
	d_ps[id*117+100]=0;
	while(d_ps[id*117+100]<length/2)
	{
		if((d_seq[start+d_ps[id*117+100]]=='A'&&d_seq[start+length-1-d_ps[id*117+100]]!='T')||(d_seq[start+d_ps[id*117+100]]=='T'&&d_seq[start+length-1-d_ps[id*117+100]]!='A')||(d_seq[start+length-1-d_ps[id*117+100]]=='A'&&d_seq[start+d_ps[id*117+100]]!='T')||(d_seq[start+length-1-d_ps[id*117+100]]=='T'&&d_seq[start+d_ps[id*117+100]]!='A'))
		{
			d_ps[id*117+101]=0;
			return;
		}
		if((d_seq[start+d_ps[id*117+100]]=='C'&&d_seq[start+length-1-d_ps[id*117+100]]!='G')||(d_seq[start+d_ps[id*117+100]]=='G'&&d_seq[start+length-1-d_ps[id*117+100]]!='C')||(d_seq[start+length-1-d_ps[id*117+100]]=='C'&&d_seq[start+d_ps[id*117+100]]!='G')||(d_seq[start+length-1-d_ps[id*117+100]]=='G'&&d_seq[start+d_ps[id*117+100]]!='C'))
		{
			d_ps[id*117+101]=0;
			return;
		}
		d_ps[id*117+100]++;
	}
	d_ps[id*117+101]=1;
}

__device__ void thal(char *d_seq,int start,int length,int strand_flag,int type,char *d_numSeq,int id,double *d_DPT,int *d_ps)
{
	if (type==4) /* unimolecular folding */
	{
		d_DPT[id*1331+1302]= 0.0;
		d_DPT[id*1331+1303] = -0.00000000001;
		d_DPT[id*1331+1304]=0;
	}
	else /* hybridization of two oligos */
	{
		d_DPT[id*1331+1302]= 200;
		d_DPT[id*1331+1303]= -5.7;
		symmetry_thermo(d_seq,start,length,id,d_ps);
		if(d_ps[id*117+101]==1)
			d_DPT[id*1331+1304]=1.9872* log(38/1000000000.0);
		else
			d_DPT[id*1331+1304]=1.9872* log(38/4000000000.0);
	}
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		if(strand_flag==0) //plus
		{
	 		for(d_ps[id*117+102]=1;d_ps[id*117+102]<=length;++d_ps[id*117+102])
			{
				str2int(d_seq[start+d_ps[id*117+102]-1],d_numSeq,(id*54+d_ps[id*117+102]));
				str2int(d_seq[start+length-d_ps[id*117+102]],d_numSeq,(id*54+27+d_ps[id*117+102]));
			}
		}
		else
		{
			for(d_ps[id*117+102]=1;d_ps[id*117+102]<=length;++d_ps[id*117+102])
			{
				str2int_rev(d_seq[start+length-d_ps[id*117+102]],d_numSeq,(id*54+d_ps[id*117+102]));
				str2int_rev(d_seq[start+d_ps[id*117+102]-1],d_numSeq,(id*54+27+d_ps[id*117+102]));
			}
		}
	}
	else
	{
		if(strand_flag==0)
		{
                	for(d_ps[id*117+102]=1;d_ps[id*117+102]<=length;++d_ps[id*117+102])
			{
				str2int(d_seq[start+d_ps[id*117+102]-1],d_numSeq,(id*54+d_ps[id*117+102]));
				d_numSeq[id*54+27+d_ps[id*117+102]]=d_numSeq[id*54+d_ps[id*117+102]];
			}
		}
		else
		{
			for(d_ps[id*117+102]=1;d_ps[id*117+102]<=length;++d_ps[id*117+102])
			{
				str2int_rev(d_seq[start+length-d_ps[id*117+102]],d_numSeq,(id*54+d_ps[id*117+102]));
				d_numSeq[id*54+27+d_ps[id*117+102]]=d_numSeq[id*54+d_ps[id*117+102]];
			}
		}
	}
	d_numSeq[id*54+0]=d_numSeq[id*54+length+1]=d_numSeq[id*54+27+0]=d_numSeq[id*54+27+length+1]=4; /* mark as N-s */

	d_DPT[id*1331+1309]=0;
	if (type==4) /* calculate structure of monomer */
	{
		initMatrix2(length,d_DPT,d_numSeq,id,d_ps);
		fillMatrix2(length,d_DPT,d_numSeq,id,d_ps);
		calc_terminal_bp(310.15,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1306]=d_DPT[id*1331+1276+length];
		d_DPT[id*1331+1307]=d_DPT[id*1331+1250+length];
		for (d_ps[id*117+102]=0;d_ps[id*117+102]<length;d_ps[id*117+102]++)
			d_ps[id*117+d_ps[id*117+102]]=0;
		if(fabs(d_DPT[id*1331+1306])<999999999)
		{
			tracebacku(d_ps,length,d_DPT,d_numSeq,id);
			drawHairpin(d_ps,id,d_DPT[id*1331+1306],d_DPT[id*1331+1307],length,d_DPT);
			d_DPT[id*1331+1309]=(int)(d_DPT[id*1331+1309]*100+0.5)/100.0;
		}
	}
	else  /* Hybridization of two moleculs */
	{
		initMatrix(length,d_DPT,d_numSeq,id,d_ps);
		fillMatrix(length,d_DPT,d_numSeq,id,d_ps);

		d_DPT[id*1331+1305]=-1.0*INFINITY;
	/* calculate terminal basepairs */
		d_ps[id*117+100]=d_ps[id*117+101]=0;
		if(type==1)
			for (d_ps[id*117+102]=1;d_ps[id*117+102]<=length;d_ps[id*117+102]++)
			{
				for (d_ps[id*117+103]=1;d_ps[id*117+103]<=length;d_ps[id*117+103]++)
				{
					RSH(d_ps[id*117+102],d_ps[id*117+103],d_DPT,d_numSeq,id);
					d_DPT[id*1331+1306]=d_DPT[id*1331+1306]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
					d_DPT[id*1331+1307]=d_DPT[id*1331+1307]+0.000001;
					d_DPT[id*1331+1308]=((d_DPT[id*1331+(d_ps[id*117+102]-1)*length+d_ps[id*117+103]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302]) / ((d_DPT[id*1331+625+(d_ps[id*117+102]-1)*length+d_ps[id*117+103]-1])+d_DPT[id*1331+1306]+d_DPT[id*1331+1303] + d_DPT[id*1331+1304])) -273.15;
					if(d_DPT[id*1331+1308]>d_DPT[id*1331+1305]&&((d_DPT[id*1331+625+(d_ps[id*117+102]-1)*length+d_ps[id*117+103]-1]+d_DPT[id*1331+1306])<0&&(d_DPT[id*1331+1307]+d_DPT[id*1331+(d_ps[id*117+102]-1)*length+d_ps[id*117+103]-1])<0))
					{
						d_DPT[id*1331+1305]=d_DPT[id*1331+1308];
						d_ps[id*117+100]=d_ps[id*117+102];
						d_ps[id*117+101]=d_ps[id*117+103];
					}
				}
			}
		if(type==2)
		{
		 //THAL_END1
			d_ps[id*117+101]=0;
			d_ps[id*117+100]=length;
			d_DPT[id*1331+1305]=-1.0*INFINITY;
			for (d_ps[id*117+103]=1;d_ps[id*117+103]<=length;++d_ps[id*117+103])
			{
				RSH(length,d_ps[id*117+103],d_DPT,d_numSeq,id);
				d_DPT[id*1331+1306]=d_DPT[id*1331+1306]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
				d_DPT[id*1331+1307]=d_DPT[id*1331+1307]+0.000001;
				d_DPT[id*1331+1308]=((d_DPT[id*1331+(length-1)*length+d_ps[id*117+103]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(length-1)*length+d_ps[id*117+103]-1])+d_DPT[id*1331+1306]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]))-273.15;
				if (d_DPT[id*1331+1308]>d_DPT[id*1331+1305]&&((d_DPT[id*1331+1306]+d_DPT[id*1331+625+(length-1)*length+d_ps[id*117+103]-1])<0&&(d_DPT[id*1331+1307]+d_DPT[id*1331+(length-1)*length+d_ps[id*117+103]-1])<0))
				{
					d_DPT[id*1331+1305]=d_DPT[id*1331+1308];
					d_ps[id*117+101]=d_ps[id*117+103];
				}
			}
		}
		if(fabs(d_DPT[id*1331+1305])>999999999)
			d_ps[id*117+100]=d_ps[id*117+101]=1;
		RSH(d_ps[id*117+100],d_ps[id*117+101],d_DPT,d_numSeq,id);
	 // tracebacking
		for (d_ps[id*117+102]=0;d_ps[id*117+102]<length;++d_ps[id*117+102])
			d_ps[id*117+d_ps[id*117+102]]=0;
		for (d_ps[id*117+103]=0;d_ps[id*117+103]<length;++d_ps[id*117+103])
			d_ps[id*117+25+d_ps[id*117+103]] = 0;
		if(fabs(d_DPT[id*1331+(d_ps[id*117+100]-1)*length+d_ps[id*117+101]-1])<999999999)
		{
			traceback(d_ps[id*117+100],d_ps[id*117+101],d_ps,length,d_DPT,d_numSeq,id);
			drawDimer(d_ps,id,(d_DPT[id*1331+(d_ps[id*117+100]-1)*length+d_ps[id*117+101]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302]),(d_DPT[id*1331+625+(d_ps[id*117+100]-1)*length+d_ps[id*117+101]-1]+d_DPT[id*1331+1306]+d_DPT[id*1331+1303]),d_DPT,length);
			d_DPT[id*1331+1309]=(int)(d_DPT[id*1331+1309]*100+0.5)/100.0;
		}
	}
}

///function in gpu, check the GC-content; int length: the length of read
__device__ int gc(char *d_seq,int start,int length)
{
	int i,number;
	float gc;

	number=0;
	for(i=0;i<length;i++)
	{
		if(d_seq[start+i]=='C')
		{
			number++;
			continue;
		}
	
		if(d_seq[start+i]=='G')
		{
			number++;
		}
	}

	gc=1.0*number/length*100;
	if((gc<40)||(gc>65))
	{
		return 0;
	}
	return 1;
}

///function in gpu, translate A...G to int
__device__ int translate(char a)
{
	if(a=='A')
		return 0;
	if(a=='T')
		return 1;
	if(a=='C')
		return 2;
	return 3;
}

__device__ int translate_rev(char a)
{
        if(a=='T')
                return 0;
        if(a=='A')
                return 1;
        if(a=='G')
                return 2;
        return 3;
}
//function in gpu, caculate tm
__device__ int tm(char *d_seq,int start,int length,float max_tm,float min_tm)
{
	int i,pos;
	float deltah,deltas,result;

	deltah=0;
	deltas=0;
	for(i=0;i<length-1;i++)
	{
		pos=translate(d_seq[start+i]);
		pos=pos*4+translate(d_seq[start+i+1]);
		deltah+=d_deltah[pos];
		deltas+=d_deltas[pos];
	}

	deltah=(-1.0)*deltah;
	deltas=(-1.0)*deltas;
	if((d_seq[start]=='A')||(d_seq[start]=='T'))
	{
		deltah+=2.3;
		deltas+=4.1;
	}
	else
	{
		deltah+=0.1;
		deltas-=2.8;
	}
        if((d_seq[start+length-1]=='A')||(d_seq[start+length-1]=='T'))
        {
                deltah+=2.3;
                deltas+=4.1;
        }
        else
        {
                deltah+=0.1;
                deltas-=2.8;
        }
	result=1000.0*deltah/(deltas-0.51986*(length-1)-36.70381)-273.15;
	if((result<min_tm)||(result>max_tm))
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

///function in gpu, caculate stability, int strand: 0 is 5' and 1 is 3'
__device__ int stability(char *d_seq,int start,int flag,int length,int strand)//flag=0: plus
{
	int i,pos;
	
	pos=0;
	for(i=0;i<6;i++)
	{
		if(flag==0)
		{
			if(strand==0)
			{
				pos=pos*4+translate(d_seq[start+i]);
			}
			else
			{
				pos=pos*4+translate(d_seq[start+i+length-6]);
			}
		}
		else
		{
			if(strand==0)
                        {
                                pos=pos*4+translate_rev(d_seq[start+length-1-i]);
                        }
                        else
                        {
                                pos=pos*4+translate_rev(d_seq[start+5-i]);
                        }
		}
	}
	
	if(d_stab[pos]<4)
	{
		return 0;
	}
//the other part
        pos=0;
        for(i=0;i<6;i++)
        {
		if(flag==0)
		{
                	if(strand==1)
                	{
				pos=pos*4+translate(d_seq[start+i]);
                	}
                	else
                	{
				pos=pos*4+translate(d_seq[start+i+length-6]);
                	}
		}
		else //minus
		{
			if(strand==1)
                        {
                                pos=pos*4+translate_rev(d_seq[start+length-1-i]);      
                        }
                        else
                        {
                                pos=pos*4+translate_rev(d_seq[start+5-i]);
                        }
		}
        }

        if(d_stab[pos]<3)
        {
                return 0;
        }

	return 1;
}

//function in gpu: whether species chars in reads
__device__ int words(char *d_seq,int position,int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(d_seq[position+i]=='N')
		{
			return 0;
		}
	}
	return 1;
}

__device__ int check_long_ploy(char *d_seq,int start,int length)
{
        int i,same;
        char ref;

        same=1;
        ref=d_seq[start];
        for(i=1;i<length;i++)
        {
                if(d_seq[start+i]==ref)
                        same++;
                else
                {
                        if(same>=6)
                                return 0;
                        same=1;
                        ref=d_seq[start+i];
                }
        }
        if(same>=6)
                return 0;
        return 1;
}

///function: int length: the length of genome
__global__ void candidate_primer(char *d_seq,int *d_pos,int *d_len,int *d_rev_len,int strand,float max_tm,float min_tm,int length,int check_flag,char *d_numSeq,double *d_DPT,int *d_ps)
{
	int id,i,circle,check,plus,minus;

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(circle=id;circle<length;circle=circle+blockDim.x*gridDim.x)
	{
		for(i=0;i<8;i++)   //primer length is from 18 to 25
		{
			d_len[8*circle+i]=0;
			d_rev_len[8*circle+i]=0;
		}
		d_pos[circle]=0;
	
		for(i=18;i<=25;i++)  //read length is from 18 to 25
		{
			if(circle+i>length)
				break;
			check=words(d_seq,circle,i);
			if(check==0)
                                break;

			check=gc(d_seq,circle,i);
			if(check==0)
				continue;

			check=check_long_ploy(d_seq,circle,i);
			if(check==0)
                                continue;

			check=tm(d_seq,circle,i,max_tm,min_tm);
			if(check==0)
				continue;

                        check=stability(d_seq,circle,0,i,strand);
                        if(check==1)     //+ strand
				plus=1;
			else
				plus=0;
			
		//secondary structure
			if(check_flag&&plus)
			{
				thal(d_seq,circle,i,0,1,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)
					plus=0;	
			}
			if(check_flag&&plus)
                        {
                                thal(d_seq,circle,i,0,2,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)  
                                        plus=0;
                        }
			if(check_flag&&plus)
                        {
                                thal(d_seq,circle,i,0,4,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)
                                        plus=0;         
                        }
			if(plus)
                                d_len[circle*8+i-18]=1;
	//reverse
			check=stability(d_seq,circle,1,i,strand);
			if(check==1)
				minus=1;
			else
				minus=0;
		//secondary structure      
                        if(check_flag&&minus)
                        {                
                                thal(d_seq,circle,i,1,1,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)
                                        minus=0;         
                        }           
                        if(check_flag&&minus)
                        {
                                thal(d_seq,circle,i,1,2,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)
                                        minus=0;
                        }                
                        if(check_flag&&minus)
                        {
                                thal(d_seq,circle,i,1,4,d_numSeq,id,d_DPT,d_ps);
				if(d_DPT[id*1331+1309]>min_tm-10)
                                        minus=0;
                        }
                        if(minus)
				d_rev_len[circle*8+i-18]=1;
		}
		
		for(i=0;i<8;i++)
		{
			d_pos[circle]+=(d_len[circle*8+i]+d_rev_len[8*circle+i]);
		}
	}
	__syncthreads();
}

void usage()
{
        printf("Usage:\n");
        printf("    single  -in <fasta_file>  -out <primers_file_name>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the reference sequence file, fasta formate\n");
        printf("    -out  <string>:  the prefix of output files, those files store candidate single primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers. default is current directory\n");
        printf("    -stab <string>:  the parameter file used in calculating the primers' stability. default is stab_parameter.txt in Par/ directory\n");
        printf("    -tm   <string>:  the parameter file used in calcalating Tm and second structure. default is stab_parameter.txt in Par/ directory\n");
	printf("    -check   <int>:  0: don't check primers' secondary structure; !=0: check, default is 1\n");
        printf("    -par  <string>:  the directory of storing parameter files used to check primers' secondary structure, default is Par/\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region. high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design candidate loop single primers\n");
        printf("    -h/-help:        print usage\n");
}

int create_file(char *prefix,char *dir,char *seq,int *pos,int *len,int *rev_len,int length)
{
	char *file;
	int total,i,j;
	FILE *OUT;

	total=0;
	i=strlen(dir)+strlen(prefix)+20;
	file=(char *)malloc(i);
        memset(file,'\0',i);
        strcpy(file,dir);
        strcat(file,prefix);
        OUT=fopen(file,"w");
        if(OUT==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
	
        for(i=0;i<length;i++)
        {
                if(pos[i]==0)
                        continue;
                for(j=0;j<8;j++)
                {
                        if((len[8*i+j]+rev_len[8*i+j])==0)
                                continue;
                       	fprintf(OUT,"pos:%d\tlength:%d\t+:%d\t-:%d\n",i,(j+18),len[8*i+j],rev_len[8*i+j]);
			total++;
                }
        }
	fclose(OUT);
	free(file);
	return total;
}

main(int argc, char **argv)
{
	double *H_parameter,*d_DPT;
	int *pos,*d_pos,*len,*d_len,length,flag[10],i,*rev_len,*d_rev_len,num_outer,num_inner,num_loop,NumL[2],thread,block,*d_ps;
	float deltah[16],deltas[16],stab[4096],temp1,temp2;
	char *seq,*d_seq,*store_path,*prefix,*stab_path,*tm_path,*curren_path,*input,*outer,*inner,*loop,*par_path,*temp,*Pchar,*d_numSeq;
	FILE *fp;
	time_t start,end;
        struct stat statbuf;
//flag: 0:input; 1: out_prefix; 2: dir; 3: stab; 4: tm; 5: high; 6: low; 7: loop; 8: secondary structure; 9: path for secondary structure

	start=time(NULL);
	thread=200;
	block=200;
//get input
        for(i=0;i<10;i++)
        {
                flag[i]=0;
        }
	flag[8]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			input=(char *)malloc(length+1);
			memset(input,'\0',length+1);
                        strcpy(input,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        prefix=(char *)malloc(length+1);
                        memset(prefix,'\0',length+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			if(argv[i+1][length-1]=='/')
			{
                        	store_path=(char *)malloc(length+1);
                        	memset(store_path,'\0',length+1);
                        	strcpy(store_path,argv[i+1]);
			}
			else
			{
				store_path=(char *)malloc(length+2);
				memset(store_path,'\0',length+2);
				strcpy(store_path,argv[i+1]);
				store_path[length]='/';
			}
                        i=i+2;
                }
                else if(strcmp(argv[i],"-stab")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-stab\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        stab_path=(char *)malloc(length+1);
                        memset(stab_path,'\0',length+1);
                        strcpy(stab_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-tm")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        tm_path=(char *)malloc(length+1);
                        memset(tm_path,'\0',length+1);
                        strcpy(tm_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[6]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[7]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
		else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[8]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[9]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        length=strlen(argv[i+1]);
                        if(argv[i+1][length-1]=='/')
                        {
                                par_path=(char *)malloc(length+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(length+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='/';
                                par_path[length+1]='\0';
                        }
                        i=i+2;
                }		
                else
                {
                        printf("Error: don't have the parameter: %s\n",argv[i]);
                        usage();
                        exit(1);
                }
        }
//check paramters
        if(flag[5]+flag[6]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
        if(flag[0]==0)
        {
                printf("Error! Users must input the reference sequence file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the prefix name for output file with -out!\n");
                usage();
                exit(1);
        }
        for(i=0;i<strlen(prefix);i++)
        {
                if(prefix[i]=='/')
                {
                        printf("Error! the -out parameter couldn't contain any directory!\n");
                        usage();
                        exit(1);
                }
        }
//prepare
	inner=(char *)malloc(4096);
        memset(inner,'\0',4096);
        getcwd(inner,4096);
        length=strlen(inner);
        curren_path=(char *)malloc(length+1);
        memset(curren_path,'\0',length+1);
        strcpy(curren_path,inner);
        if(flag[2]==0)
        {
                store_path=(char *)malloc(length+2);
                memset(store_path,'\0',length+2);
                strcpy(store_path,curren_path);
                store_path[length]='/';
        }
        free(inner);

        length=strlen(store_path)+12;
        outer=(char *)malloc(length);
        memset(outer,'\0',length);
        strcpy(outer,store_path);

        inner=(char *)malloc(length);
        memset(inner,'\0',length);
        strcpy(inner,store_path);

        if(flag[7]==1)
        {
                loop=(char *)malloc(length);
                memset(loop,'\0',length);
                strcpy(loop,store_path);
        }
        if(flag[5]==1)
        {
                strcat(outer,"high-outer/");
                strcat(inner,"high-inner/");
                if(flag[7]==1)
                        strcat(loop,"high-loop/");
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(inner,"low-inner/");
                if(flag[7]==1)
                        strcat(loop,"low-loop/");
        }
        mkdir(outer,0755);
        mkdir(inner,0755);        
        if(flag[7]==1)
                mkdir(loop,0755);        

//stability parameter file
        if(flag[3]==0)
        {
		length=strlen(curren_path);
                stab_path=(char *)malloc(length+30);
                memset(stab_path,'\0',length+30);
                strcpy(stab_path,curren_path);
                i=length-1;
                while(stab_path[i]!='/'&&i>=0)
                {
                        stab_path[i]='\0';
                        i--;
                }
                strcat(stab_path,"Par/stab_parameter.txt");
        }
//tm parameter file
        if(flag[4]==0)
        {
		length=strlen(curren_path);
                tm_path=(char *)malloc(length+30);
                memset(tm_path,'\0',length+30);
                strcpy(tm_path,curren_path);
                i=length-1;
                while(tm_path[i]!='/'&&i>=0)
                {
                        tm_path[i]='\0';
                        i--;
                }
                strcat(tm_path,"Par/tm_nn_parameter.txt");
        }
//secondary structure
	if(flag[8]&&flag[9]==0)
        {
                length=strlen(curren_path);
                par_path=(char *)malloc(length+10);
                memset(par_path,'\0',length+10);
                strcpy(par_path,curren_path);
                i=length-1;
                while(par_path[i]!='/'&&i>=0)
                {
                        par_path[i]='\0';
                        i--;
                }
                strcat(par_path,"Par/");
        }
	if(flag[8])
	{
		NumL[0]=get_num_line(par_path,0);
	        NumL[1]=get_num_line(par_path,1);
	        H_parameter=(double *)malloc((5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        memset(H_parameter,'\0',(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        Pchar=(char *)malloc(10*NumL[0]+12*NumL[1]);
	        memset(Pchar,'\0',10*NumL[0]+12*NumL[1]);

		getStack(par_path,H_parameter);
	        getStackint2(par_path,H_parameter);
	        getDangle(par_path,H_parameter);
	        getLoop(par_path,H_parameter);
	        getTstack(par_path,H_parameter);
	        getTstack2(par_path,H_parameter);
	        getTriloop(par_path,H_parameter,Pchar,NumL);
	        getTetraloop(par_path,H_parameter,Pchar,NumL);
	        tableStartATS(6.9,H_parameter);
	        tableStartATH(2200.0,H_parameter);

		hipMemcpyToSymbol(HIP_SYMBOL(d_NumL),NumL,2*sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_Pchar),Pchar,10*NumL[0]+12*NumL[1]);
		hipMemcpyToSymbol(HIP_SYMBOL(parameter),H_parameter,(5730+2*NumL[0]+2*NumL[1])*sizeof(double));

		hipMalloc((void **)&d_numSeq,54*thread*block*sizeof(char));
		hipMalloc((void **)&d_DPT,1331*thread*block*sizeof(double));
		hipMalloc((void **)&d_ps,117*thread*block*sizeof(int));
	}

//input reference sequence
        if(access(input,0)==-1)
        {
                printf("Error! Don't have the %s file.\n",input);
                exit(1);
        }
        stat(input,&statbuf);
        length=statbuf.st_size;
        length=length+100;
        temp=(char *)malloc(length);
        memset(temp,'\0',length);
        seq=(char *)malloc(length*sizeof(char));
        memset(seq,'\0',length*sizeof(char));

        fp=fopen(input,"r");   //open the sequence file
        if(fp==NULL)
        {
                printf("Error! can't open the %s file!\n",input);
                exit(1);
        }
        fread(temp,length*sizeof(char),1,fp);
        fclose(fp); 

        length=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
		if(temp[i]=='a'||temp[i]=='A')
                        seq[length]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[length]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[length]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[length]='G';
                else
                        seq[length]='N';
                i++;
                length++;
        }
        free(temp);
        length=strlen(seq);

//input Tm parameter
        fp=fopen(tm_path,"r");  //read the paramter of deltah and deltas
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",tm_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f\t%f",&i,&temp1,&temp2)!=EOF)
        {
                deltah[i]=temp1;
                deltas[i]=temp2;
        }
        fclose(fp);

//input stability parameter
        fp=fopen(stab_path,"r");  //read the parameters of stability
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",stab_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f",&i,&temp1)!=EOF)
        {
                stab[i]=temp1;
        }
        fclose(fp);

	hipMalloc((void **)&d_seq,length*sizeof(char));
	hipMemset(d_seq,'\0',length*sizeof(char));

	/////from cpu to gpu
	hipMemcpy(d_seq,seq,length*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltah),deltah,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltas),deltas,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_stab),stab,4096*sizeof(float));

	hipMalloc((void **)&d_pos,length*sizeof(int));
	hipMemset(d_pos,'\0',length*sizeof(int));
	hipMalloc((void **)&d_len,8*length*sizeof(int));
	hipMemset(d_len,'\0',8*length*sizeof(int));
	hipMalloc((void **)&d_rev_len,8*length*sizeof(int));
        hipMemset(d_rev_len,'\0',8*length*sizeof(int));
	pos=(int *)malloc(length*sizeof(int));
	memset(pos,'\0',length*sizeof(int));
	len=(int *)malloc(8*length*sizeof(int));
	memset(len,'\0',8*length*sizeof(int));
        rev_len=(int *)malloc(8*length*sizeof(int));
        memset(rev_len,'\0',8*length*sizeof(int));

	end=time(NULL);
	printf("It takes %d seconds to prepare.\n",(int)difftime(end,start));
	start=time(NULL);
	if(flag[5]==1)
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
		hipMemset(d_len,'\0',8*length*sizeof(int));
		hipMemset(d_rev_len,'\0',8*length*sizeof(int));
		candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
		hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,0,66,64,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);

                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,66,64,length,flag[8],d_numSeq,d_DPT,d_ps);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
        else
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,56,54,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,0,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);
                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,1,61,59,length,flag[8],d_numSeq,d_DPT,d_ps);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
	hipFree(d_pos);
	hipFree(d_len);
	hipFree(d_rev_len);
	hipFree(d_seq);
	free(pos);
        free(len);
        free(rev_len);
	free(seq);

	printf("There ara %d candidate primers used as F3/F2/B2/B3.\n",num_outer);
        printf("There are %d candidate primers used as F1c/B1c.\n",num_inner);
        if(flag[7]==1)
                printf("There are %d candidate primers used as LF/LB.\n",num_loop);
        //check
        if(num_outer<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(num_inner<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && num_loop<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
        printf("It takes %d seconds to design candidate single primers.\n",(int)difftime(end,start));

	free(store_path);
	free(prefix);
	free(stab_path);
	free(tm_path);
	free(curren_path);
	free(input);
	free(outer);
	free(inner);
	if(flag[7])
		free(loop);

	if(flag[8])
	{
		free(Pchar);
		free(H_parameter);
		hipFree(d_numSeq);
		hipFree(d_DPT);
		hipFree(d_ps);
	}
	if(flag[8]||flag[9])
		free(par_path);
}
