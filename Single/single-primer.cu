#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<sys/stat.h>

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ char str2int(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;
                case 'T':
                        return 3;
        }
        return 4;
}

__device__ int seq_length(char seq[])
{
	int i=0;
	while(seq[i]!='\0')
		i++;
	return i;
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

int get_num_line(char *path,int flag)
{
	FILE *fp;
	int i,size;
	char *line;

	i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
	if(flag==0)
	        strcat(line,"triloop.ds");
	else
		strcat(line,"tetraloop.ds");

        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        fp=fopen(line,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	size=0;
	while(fgets(line,i,fp)!=NULL)
		size++;
	return size;
}

void getTriloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i,turn;
        char *line,seq[10],value[10];
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
	
	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for (i=0;i<5;i++)
			Pchar[5*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+turn]=1.0*INFINITY;
		else
			parameter[5730+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

	i=strlen(path)+20;
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<5;i++)
			Pchar[5*NumL[0]+turn*5+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void getTetraloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i, turn;
        char *line,seq[10],value[10];

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+turn*6+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);
        
	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+6*NumL[1]+6*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+NumL[1]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+NumL[1]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ void initMatrix2(int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[])
{
	int i,j;
	for(i=1;i<=Initint[0];++i)
		for(j=i;j<=Initint[1];++j)
			if(j-i<4 || (numSeq1[i]+numSeq1[j]!=3))
			{
				enthalpyDPT[(i-1)*Initint[2]+j-1]=1.0*INFINITY;
				entropyDPT[(i-1)*Initint[2]+j-1]=-1.0;
			}
			else
			{
				enthalpyDPT[(i-1)*Initint[2]+j-1]=0.0;
				entropyDPT[(i-1)*Initint[2]+j-1]=-3224.0;
			}
}

__device__ double Ss(int i,int j,int k,int Initint[],char numSeq1[],char numSeq2[],double parameter[])
{
	if(k==2)
	{
		if(i>=j)
			return -1.0;
		if(i==Initint[0]||j==Initint[1]+1)
			return -1.0;

		if(i>Initint[0])
			i-=Initint[0];
		if(j>Initint[1])
			j-=Initint[1];
		return parameter[numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]];
	}
	else
		return parameter[numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
}

__device__ double Hs(int i,int j,int k,int Initint[],char numSeq1[],char numSeq2[],double parameter[])
{
	if(k==2)
	{
		if(i>= j)
			return 1.0*INFINITY;
		if(i==Initint[0]||j==Initint[1]+1)
			return 1.0*INFINITY;

		if(i>Initint[0])
			i-=Initint[0];
		if(j>Initint[1])
			j-=Initint[1];
		if(fabs(parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]])<999999999)
			return parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]];
		else
			return 1.0*INFINITY;
	}
	else
		return parameter[625+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
}

__device__ void maxTM2(int i,int j,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	double T0,T1,S0,S1,H0,H1;

	S0=entropyDPT[(i-1)*Initint[2]+j-1];
	H0=enthalpyDPT[(i-1)*Initint[2]+j-1];
	T0=(H0+Initdouble[0])/(S0+Initdouble[1]+Initdouble[2]);
	if(fabs(enthalpyDPT[(i-1)*Initint[2]+j-1])<999999999)
	{
		S1=(entropyDPT[i*Initint[2]+j-2]+Ss(i,j,2,Initint,numSeq1,numSeq2,parameter));
		H1=(enthalpyDPT[i*Initint[2]+j-2]+Hs(i,j,2,Initint,numSeq1,numSeq2,parameter));
	}
	else
	{
		S1=-1.0;
		H1=1.0*INFINITY;
	}
	T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
	if(S1<-2500.0)
	{
		S1=-3224.0;
		H1=0.0;
	}
	if(S0<-2500.0)
	{
		S0=-3224.0;
		H0=0.0;
 	}

	if(T1>T0)
	{
		entropyDPT[(i-1)*Initint[2]+j-1]=S1;
		enthalpyDPT[(i-1)*Initint[2]+j-1]= H1;
	}
	else
	{
		entropyDPT[(i-1)*Initint[2]+j-1]=S0;
		enthalpyDPT[(i-1)*Initint[2]+j-1]=H0;
	}
}

__device__ void calc_bulge_internal2(int i,int j,int ii,int jj,double *EntropyEnthalpy,int traceback,double Initdouble[0],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	int loopSize1,loopSize2,loopSize;
	double T1,T2,S,H;

	S=-3224.0;
	H=0.0;
	loopSize1=ii-i-1;
	loopSize2=j-jj-1;
	if(loopSize1+loopSize2>30)
	{
		EntropyEnthalpy[0]=-1.0;
		EntropyEnthalpy[1]=1.0*INFINITY;
		return;
	}

	loopSize=loopSize1+loopSize2-1;
	if((loopSize1==0&&loopSize2>0)||(loopSize2==0&&loopSize1>0))
	{
		if(loopSize2==1||loopSize1==1)
		{ 
			if((loopSize2==1&&loopSize1==0)||(loopSize2==0&&loopSize1==1))
			{
				H=parameter[3150+loopSize]+parameter[625+numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
				S=parameter[3060+loopSize]+parameter[numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
 			}
			if(traceback!=1)
			{
				H+=enthalpyDPT[(ii-1)*Initint[2]+jj-1];
				S+=entropyDPT[(ii-1)*Initint[2]+jj-1];
			}

			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(enthalpyDPT[(i-1)*Initint[2]+j-1]+Initdouble[0])/((entropyDPT[(i-1)*Initint[2]+j-1])+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||traceback==1))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		else
		{
			H=parameter[3150+loopSize]+parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5705+numSeq1[ii]*5+numSeq2[jj]];
			if(traceback!=1)
				H+=enthalpyDPT[(ii-1)*Initint[2]+jj-1];

			S=parameter[3060+loopSize]+parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[5680+numSeq1[ii]*5+numSeq2[jj]];
			if(traceback!=1)
				S+=entropyDPT[(ii-1)*Initint[2]+jj-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(enthalpyDPT[(i-1)*Initint[2]+j-1]+Initdouble[0])/(entropyDPT[(i-1)*Initint[2]+j-1]+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
	}
	else if(loopSize1==1&&loopSize2==1)
	{
		S=parameter[1250+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]]+parameter[1250+numSeq2[jj]*125+numSeq2[jj+1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		if(traceback!=1)
			S+=entropyDPT[(ii-1)*Initint[2]+jj-1];

		H=parameter[1875+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]]+parameter[1875+numSeq2[jj]*125+numSeq2[jj+1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		if(traceback!=1)
			H+=enthalpyDPT[(ii-1)*Initint[2]+jj-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(enthalpyDPT[(i-1)*Initint[2]+j-1]+Initdouble[0])/(entropyDPT[(i-1)*Initint[2]+j-1]+Initdouble[1]+Initdouble[2]);
		if((T1-T2>=0.000001)||traceback)
		{
			if((T1>T2)||((traceback&&T1>= T2)||traceback==1))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		return;
	}
	else
	{
		H=parameter[3120+loopSize]+parameter[3805+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]]+parameter[3805+numSeq2[jj]*125+numSeq2[jj+1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		if(traceback!=1)
			H+=enthalpyDPT[(ii-1)*Initint[2]+jj-1];

		S=parameter[3030+loopSize]+parameter[3180+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j-1]]+parameter[3180+numSeq2[jj]*125+numSeq2[jj+1]*25+numSeq1[ii]*5+numSeq1[ii-1]]+(-300/310.15*abs(loopSize1-loopSize2));
		if(traceback!=1)
			S+=entropyDPT[(ii-1)*Initint[2]+jj-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}

		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(enthalpyDPT[(i-1)*Initint[2]+j-1]+Initdouble[0])/((entropyDPT[(i-1)*Initint[2]+j-1])+Initdouble[1]+Initdouble[2]);
		if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
		{
			EntropyEnthalpy[0]=S;
			EntropyEnthalpy[1]=H;
		}
	}
	return;
}

__device__ void CBI(int i,int j,double* EntropyEnthalpy,int traceback,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	int d,ii,jj;

	for(d=j-i-3;d>=4&&d>=j-i-32;--d)
		for(ii=i+1;ii<j-d&&ii<=Initint[0];++ii)
		{
			jj=d+ii;
			if(traceback==0)
			{
				EntropyEnthalpy[0]=-1.0;
				EntropyEnthalpy[1]=1.0*INFINITY;
			}
			if(fabs(enthalpyDPT[(ii-1)*Initint[2]+jj-1])<999999999)
			{
				calc_bulge_internal2(i,j,ii,jj,EntropyEnthalpy,traceback,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
				if(fabs(EntropyEnthalpy[1])<999999999)
				{
					if(EntropyEnthalpy[0] <-2500.0)
					{
						EntropyEnthalpy[0]=-3224.0;
						EntropyEnthalpy[1]=0.0;
					}
					if(traceback==0)
					{
						enthalpyDPT[(i-1)*Initint[2]+j-1]=EntropyEnthalpy[1];
						entropyDPT[(i-1)*Initint[2]+j-1]=EntropyEnthalpy[0];
					}
				}
			}
		}
	return;
}

__device__ int find_pos(char *ref,int ref_start,char *source,int start,int length,int num)
{
	int flag,i,j;

	for(i=0;i<num;i++)
	{
		flag=0;
		for(j=0;j<length;j++)
		{
			if(ref[ref_start+j]!=source[start+i*length+j])
			{
				flag++;
				break;
			}
		}
		if(flag==0)
			return i;
	}
	return -1;
}

__device__ void calc_hairpin(int i,int j,double *EntropyEnthalpy,int traceback,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],double parameter[],char *d_Pchar,int *d_NumL)
{
	int pos,loopSize=j-i-1;
	double T1,T2;
	
	if(loopSize < 3)
	{
		EntropyEnthalpy[0]=-1.0;
		EntropyEnthalpy[1]=1.0*INFINITY;
		return;
	}
	if(i<=Initint[0]&&Initint[1]<j)
	{
		EntropyEnthalpy[0]=-1.0;
		EntropyEnthalpy[1]=1.0*INFINITY;
		return;
	}
	else if(i>Initint[1])
	{
		i-= Initint[0];
		j-= Initint[1];
	}
	if(loopSize<=30)
	{
		EntropyEnthalpy[1]=parameter[3090+loopSize-1];
		EntropyEnthalpy[0]=parameter[3000+loopSize-1];
	}
	else
	{
		EntropyEnthalpy[1]=parameter[3090+29];
		EntropyEnthalpy[0]=parameter[3000+29];
	}

	if(loopSize>3) // for loops 4 bp and more in length, terminal mm are accounted
	{
		EntropyEnthalpy[1]+=parameter[5055+numSeq1[i]*125+numSeq1[i+1]*25+numSeq1[j]*5+numSeq1[j-1]];
		EntropyEnthalpy[0]+=parameter[4430+numSeq1[i]*125+numSeq1[i+1]*25+numSeq1[j]*5+numSeq1[j-1]];
	}
	else if(loopSize == 3) // for loops 3 bp in length at-penalty is considered
	{
		EntropyEnthalpy[1]+=parameter[5705+numSeq1[i]*5+numSeq1[j]];
		EntropyEnthalpy[0]+=parameter[5680+numSeq1[i]*5+numSeq1[j]];
	}

	if(loopSize==3) // closing AT-penalty (+), triloop bonus, hairpin of 3 (+) 
	{
		pos=find_pos(numSeq1,i,d_Pchar,5*d_NumL[0],5,d_NumL[0]);
		if(pos!=-1)
			EntropyEnthalpy[1]+=parameter[5730+d_NumL[0]+pos];

		pos=find_pos(numSeq1,i,d_Pchar,0,5,d_NumL[0]);
		if(pos!=-1)
			EntropyEnthalpy[0]+=parameter[5730+pos];
	}
	else if (loopSize == 4) // terminal mismatch, tetraloop bonus, hairpin of 4
	{
		pos=find_pos(numSeq1,i,d_Pchar,10*d_NumL[0]+6*d_NumL[1],6,d_NumL[1]);
		if(pos!=-1)
			EntropyEnthalpy[1]+=parameter[5730+2*d_NumL[0]+d_NumL[1]+pos];

		pos=find_pos(numSeq1,i,d_Pchar,10*d_NumL[0],6,d_NumL[1]);
		if(pos!=-1)
			EntropyEnthalpy[0]+=parameter[5730+2*d_NumL[0]+pos];
	}
	if(fabs(EntropyEnthalpy[1])>999999999)
	{
		EntropyEnthalpy[1] =1.0*INFINITY;
		EntropyEnthalpy[0] = -1.0;
	}
	T1 = (EntropyEnthalpy[1] +Initdouble[0]) / ((EntropyEnthalpy[0] +Initdouble[1]+ Initdouble[2]));
	T2 = (enthalpyDPT[(i-1)*Initint[2]+j-1] +Initdouble[0]) / ((entropyDPT[(i-1)*Initint[2]+j-1]) +Initdouble[1]+ Initdouble[2]);
	if(T1 < T2 && traceback == 0)
	{
		EntropyEnthalpy[0] =entropyDPT[(i-1)*Initint[2]+j-1];
		EntropyEnthalpy[1] =enthalpyDPT[(i-1)*Initint[2]+j-1];
	}
	return;
}

__device__ void fillMatrix2(double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double *parameter,char *d_Pchar,int *d_NumL)
{
	int i, j;
	double SH[2];

	for (j = 2; j <= Initint[1]; ++j)
		for (i = j - 3 - 1; i >= 1; --i)
		{
			if (fabs(enthalpyDPT[(i-1)*Initint[2]+j-1])<999999999)
			{
				SH[0] = -1.0;
				SH[1] =1.0*INFINITY;
				maxTM2(i,j,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
				CBI(i,j,SH,0,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

				SH[0] = -1.0;
				SH[1] =1.0*INFINITY;
				calc_hairpin(i, j, SH, 0,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,parameter,d_Pchar,d_NumL);
				if(fabs(SH[1])<999999999)
				{
					if(SH[0] <-2500.0) /* to not give dH any value if dS is unreasonable */
					{
						SH[0] =-3224.0;
						SH[1] = 0.0;
					}
					entropyDPT[(i-1)*Initint[2]+j-1]= SH[0];
					enthalpyDPT[(i-1)*Initint[2]+j-1]= SH[1];
				}
			}
		}
}

__device__ int max5(double a,double b,double c,double d,double e)
{
	if(a>b&&a>c&&a>d&&a>e)
		return 1;
	else if(b>c&&b>d&&b>e)
		return 2;
	else if(c>d&&c>e)
		return 3;
	else if(d>e)
		return 4;
	else
		return 5;
}

__device__ double Sd5(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[2750+numSeq1[i]*25+numSeq1[j]*5+numSeq1[j-1]];
}

__device__ double Hd5(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[2875+numSeq1[i]*25+numSeq1[j]*5+numSeq1[j-1]];
}

__device__ double Sd3(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[2500+numSeq1[i]*25+numSeq1[i+1]*5+numSeq1[j]];
}

__device__ double Hd3(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq1[j]];
}

__device__ double Ststack(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[4430+numSeq1[i]*125+numSeq1[i+1]*25+numSeq1[j]*5+numSeq1[j-1]];
}

__device__ double Htstack(int i,int j,char numSeq1[],double parameter[])
{
	return parameter[5055+numSeq1[i]*125+numSeq1[i+1]*25+numSeq1[j]*5+numSeq1[j-1]];
}

__device__ double END5_1(int i,int hs,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],double parameter[])
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	max_tm=-1.0*INFINITY;
	H_max=1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-5;++k)
	{
		T1=(hend5[k]+Initdouble[0])/(send5[k]+Initdouble[1]+Initdouble[2]);
		T2=Initdouble[0]/(Initdouble[1]+Initdouble[2]);
		if(T1>=T2)
		{
			H=hend5[k]+parameter[5705+numSeq1[k+1]*5+numSeq1[i]]+enthalpyDPT[k*Initint[2]+i-1];
			S=send5[k]+parameter[5680+numSeq1[k+1]*5+numSeq1[i]]+entropyDPT[k*Initint[2]+i-1];
			if(fabs(H)>999999999||H>0||S>0)  // H and S must be greater than 0 to avoid BS
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}
		else
		{
			H=parameter[5705+numSeq1[k+1]*5+numSeq1[i]]+enthalpyDPT[k*Initint[2]+i-1];
			S=parameter[5680+numSeq1[k+1]*5+numSeq1[i]]+entropyDPT[k*Initint[2]+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_2(int i,int hs,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],double parameter[])
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-6;++k)
	{
		T1=(hend5[k]+Initdouble[0])/(send5[k]+Initdouble[1]+Initdouble[2]);
		T2=Initdouble[0]/(Initdouble[1]+Initdouble[2]);
		if(T1>=T2)
		{
			H=hend5[k]+parameter[5705+numSeq1[k+2]*5+numSeq1[i]]+Hd5(i,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-1];
			S=send5[k]+parameter[5680+numSeq1[k+2]*5+numSeq1[i]]+Sd5(i,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}
		else
		{
			H=parameter[5705+numSeq1[k+2]*5+numSeq1[i]]+Hd5(i,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-1];
			S=parameter[5680+numSeq1[k+2]*5+numSeq1[i]]+Sd5(i,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-1];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_3(int i,int hs,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],double parameter[])
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-6;++k)
	{
		T1=(hend5[k]+Initdouble[0])/(send5[k]+Initdouble[1]+Initdouble[2]);
		T2=Initdouble[0]/(Initdouble[1]+Initdouble[2]);
		if(T1>=T2)
		{
			H=hend5[k]+parameter[5705+numSeq1[k+1]*5+numSeq1[i-1]]+Hd3(i-1,k+1,numSeq1,parameter)+enthalpyDPT[k*Initint[2]+i-2];
			S=send5[k]+parameter[5680+numSeq1[k+1]*5+numSeq1[i-1]]+Sd3(i-1,k+1,numSeq1,parameter)+entropyDPT[k*Initint[2]+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}
		else
		{
			H=parameter[5705+numSeq1[k+1]*5+numSeq1[i-1]]+Hd3(i-1,k+1,numSeq1,parameter)+enthalpyDPT[k*Initint[2]+i-2];
			S=parameter[5680+numSeq1[k+1]*5+numSeq1[i-1]]+Sd3(i-1,k+1,numSeq1,parameter)+entropyDPT[k*Initint[2]+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ double END5_4(int i,int hs,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],double parameter[])
{
	int k;
	double max_tm,T1,T2,H,S,H_max,S_max;

	H_max=1.0*INFINITY;
	max_tm=-1.0*INFINITY;
	S_max=-1.0;
	for(k=0;k<=i-7;++k)
	{
		T1=(hend5[k]+Initdouble[0])/(send5[k]+Initdouble[1]+Initdouble[2]);
		T2=Initdouble[0]/(Initdouble[1]+Initdouble[2]);
		if(T1>=T2)
		{
			H=hend5[k]+parameter[5705+numSeq1[k+2]*5+numSeq1[i-1]]+Htstack(i-1,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-2];
			S=send5[k]+parameter[5680+numSeq1[k+2]*5+numSeq1[i-1]]+Ststack(i-1,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
		}
		else
		{
			H=parameter[5705+numSeq1[k+2]*5+numSeq1[i-1]]+Htstack(i-1,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-2];
			S=parameter[5680+numSeq1[k+2]*5+numSeq1[i-1]]+Ststack(i-1,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-2];
			if(fabs(H)>999999999||H>0||S>0)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/(S+Initdouble[1]+Initdouble[2]);
 		}

		if(max_tm<T1)
		{
			if(S>-2500.0)
			{
				H_max=H;
				S_max=S;
				max_tm=T1;
			}
		}
	}
	if(hs==1)
		return H_max;
	return S_max;
}

__device__ void calc_terminal_bp(double temp,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],double parameter[])
{
	int i,max;
	double T1,T2,T3,T4,T5,G,end5_11,end5_12,end5_21,end5_22,end5_31,end5_32,end5_41,end5_42;
	
	send5[0]=send5[1]= -1.0;
	hend5[0]=hend5[1]=1.0*INFINITY;

	for(i=2;i<=Initint[0];i++)
	{
		send5[i]=-3224.0;
		hend5[i]=0;
	}

// adding terminal penalties to 3' end and to 5' end 
	for(i=2;i<=Initint[0];++i)
	{
		max=0;
		T1=(hend5[i-1]+Initdouble[0])/(send5[i-1]+Initdouble[1]+Initdouble[2]);
		end5_11=END5_1(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		end5_12=END5_1(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		T2=(end5_11+Initdouble[0])/(end5_12+Initdouble[1]+Initdouble[2]);
		end5_21=END5_2(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		end5_22=END5_2(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		T3=(end5_21+Initdouble[0])/(end5_22+Initdouble[1]+Initdouble[2]);
		end5_31=END5_3(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		end5_32=END5_3(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		T4=(end5_31+Initdouble[0])/(end5_32+Initdouble[1]+Initdouble[2]);
		end5_41=END5_4(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		end5_42=END5_4(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		T5=(end5_41+Initdouble[0])/(end5_42+Initdouble[1]+Initdouble[2]);

		max=max5(T1,T2,T3,T4,T5);
		switch(max)
		{
			case 1:
				send5[i]=send5[i-1];
				hend5[i]=hend5[i-1];
				break;
			case 2:
				G=end5_11-temp*end5_12;
				if(G<0.0)
				{
					send5[i]=end5_12;
					hend5[i]=end5_11;
				}
				else
				{
					send5[i]=send5[i-1];
					hend5[i]=hend5[i-1];
				}
				break;
			case 3:
				G=end5_21-temp*end5_22;
				if(G<0.0)
				{
					send5[i]=end5_22;
					hend5[i]=end5_21;
				}
				else
				{
					send5[i]=send5[i-1];
					hend5[i]=hend5[i-1];
				}
				break;
			case 4:
				G=end5_31-temp*end5_32;
				if(G<0.0)
				{
					send5[i]=end5_32;
					hend5[i]=end5_31;
				}
				else
				{
					send5[i]=send5[i-1];
					hend5[i]=hend5[i-1];
				}
				break;
			case 5:
				G=end5_41-temp*end5_42;
				if(G<0.0)
				{
					send5[i]=end5_42;
					hend5[i]=end5_41;
				}
				else
				{
					send5[i]=send5[i-1];
					hend5[i]=hend5[i-1];
				}
				break;
			default:
				break;
		}
	}
}

__device__ int newpush(int store[],int i,int j,int mtrx,int total,int next)
{
        int k;
        for(k=total-1;k>=next;k--)
        {
                store[(k+1)*3]=store[k*3];
                store[(k+1)*3+1]=store[k*3+1];
                store[(k+1)*3+2]=store[k*3+2];
        }
        store[next*3]=i;                  
        store[next*3+1]=j;
        store[next*3+2]=mtrx;

        return total+1;           
}

__device__ int equal(double a,double b)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
		return 0;
	return fabs(a-b)<1e-5;
}

__device__ void tracebacku(int bp[],double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],double send5[],double hend5[],char numSeq1[],char numSeq2[],double *parameter,char *d_Pchar,int *d_NumL)
{
	int i,j,store[50],total,now,ii,jj,k,d,done;
	double SH1[2],SH2[2],EntropyEnthalpy[2];

        total=newpush(store,Initint[0],0,1,0,0);
        now=0;
        while(now<total)
        {
                i=store[3*now]; // top->i;
                j=store[3*now+1]; // top->j;
                if(store[now*3+2]==1)
                {
                        while(equal(send5[i],send5[i-1])&&equal(hend5[i],hend5[i-1])) // if previous structure is the same as this one
                                --i;
                        if(i==0)
                                continue;
                        if(equal(send5[i],END5_1(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter))&&equal(hend5[i],END5_1(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter)))
                        {
                                for(k=0;k<=i-5;++k)
                                        if(equal(send5[i],parameter[5680+numSeq1[k+1]*5+numSeq1[i]]+entropyDPT[k*Initint[2]+i-1])&&equal(hend5[i],parameter[5705+numSeq1[k+1]*5+numSeq1[i]]+enthalpyDPT[k*Initint[2]+i-1]))
                                        {
                                                total=newpush(store,k+1,i,0,total,now+1);                    
                                                break;
                                        }
                                        else if(equal(send5[i],send5[k]+parameter[5680+numSeq1[k+1]*5+numSeq1[i]]+entropyDPT[k*Initint[2]+i-1])&&equal(hend5[i],hend5[k]+parameter[5705+numSeq1[k+1]*5+numSeq1[i]]+enthalpyDPT[k*Initint[2]+i-1]))
                                        {
                                                total=newpush(store,k+1,i,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(send5[i],END5_2(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter))&&equal(hend5[i],END5_2(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter)))
                        {
                                for (k=0;k<=i-6;++k)
                                        if(equal(send5[i],parameter[5680+numSeq1[k+2]*5+numSeq1[i]]+Sd5(i,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-1])&&equal(hend5[i],parameter[5705+numSeq1[k+2]*5+numSeq1[i]]+Hd5(i,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-1]))
                                        {
                                                total=newpush(store,k+2,i,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(send5[i],send5[k]+parameter[5680+numSeq1[k+2]*5+numSeq1[i]]+Sd5(i,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-1])&&equal(hend5[i],hend5[k]+parameter[5705+numSeq1[k+2]*5+numSeq1[i]]+Hd5(i,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-1]))
                                        {
                                                total=newpush(store,k+2,i,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(send5[i],END5_3(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter))&&equal(hend5[i],END5_3(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter)))
                        {
                                for (k=0;k<=i-6;++k)
                                        if(equal(send5[i],parameter[5680+numSeq1[k+1]*5+numSeq1[i-1]]+Sd3(i-1,k+1,numSeq1,parameter)+entropyDPT[k*Initint[2]+i-2])&&equal(hend5[i],parameter[5705+numSeq1[k+1]*5+numSeq1[i-1]]+Hd3(i-1,k+1,numSeq1,parameter)+enthalpyDPT[k*Initint[2]+i-2]))
                                        {
                                                total=newpush(store,k+1,i-1,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(send5[i],send5[k]+parameter[5680+numSeq1[k+1]*5+numSeq1[i-1]]+Sd3(i-1,k+1,numSeq1,parameter)+entropyDPT[k*Initint[2]+i-2])&&equal(hend5[i],hend5[k]+parameter[5705+numSeq1[k+1]*5+numSeq1[i-1]]+Hd3(i-1,k+1,numSeq1,parameter)+enthalpyDPT[k*Initint[2]+i-2]))
                                        {
                                                total=newpush(store,k+1,i-1,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                        else if(equal(send5[i],END5_4(i,2,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter))&&equal(hend5[i],END5_4(i,1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter)))
                        {
                                for (k=0;k<=i-7;++k)
                                        if(equal(send5[i],parameter[5680+numSeq1[k+2]*5+numSeq1[i-1]]+Ststack(i-1,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-2])&&equal(hend5[i],parameter[5705+numSeq1[k+2]*5+numSeq1[i-1]]+Htstack(i-1,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-2]))
                                        {
                                                total=newpush(store,k+2,i-1,0,total,now+1);
                                                break;
                                        }
                                        else if(equal(send5[i],send5[k]+parameter[5680+numSeq1[k+2]*5+numSeq1[i-1]]+Ststack(i-1,k+2,numSeq1,parameter)+entropyDPT[(k+1)*Initint[2]+i-2])&&equal(hend5[i],hend5[k]+parameter[5705+numSeq1[k+2]*5+numSeq1[i-1]]+Htstack(i-1,k+2,numSeq1,parameter)+enthalpyDPT[(k+1)*Initint[2]+i-2]))
                                        {
                                                total=newpush(store,k+2,i-1,0,total,now+1);
                                                total=newpush(store,k,0,1,total,now+1);
                                                break;
                                        }
                        }
                }
                else if(store[3*now+2]==0)
                {
                        bp[i-1]=j;
                        bp[j-1]=i;
                        SH1[0]=-1.0;
                        SH1[1]=1.0*INFINITY;
                        calc_hairpin(i,j,SH1,1,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,parameter,d_Pchar,d_NumL);

                        SH2[0]=-1.0;
                        SH2[1]=1.0*INFINITY;
                        CBI(i,j,SH2,2,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

                        if (equal(entropyDPT[(i-1)*Initint[2]+j-1],Ss(i,j,2,Initint,numSeq1,numSeq2,parameter)+entropyDPT[i*Initint[2]+j-2])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],Hs(i,j,2,Initint,numSeq1,numSeq2,parameter)+enthalpyDPT[i*Initint[2]+j-2]))
                                total=newpush(store,i+1,j-1,0,total,now+1);
                        else if(equal(entropyDPT[(i-1)*Initint[2]+j-1],SH1[0])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],SH1[1]));
                        else if(equal(entropyDPT[(i-1)*Initint[2]+j-1],SH2[0])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],SH2[1]))
                        {
                                for (done=0,d=j-i-3;d>=4&&d>=j-i-32&&!done;--d)
                                        for (ii=i+1;ii<j-d;++ii)
                                        {
                                                jj=d+ii;
                                                EntropyEnthalpy[0]=-1.0;
                                                EntropyEnthalpy[1]=1.0*INFINITY;
                                                calc_bulge_internal2(i,j,ii,jj,EntropyEnthalpy,1,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

                                                if (equal(entropyDPT[(i-1)*Initint[2]+j-1],EntropyEnthalpy[0]+entropyDPT[(ii-1)*Initint[2]+jj-1])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],EntropyEnthalpy[1]+enthalpyDPT[(ii-1)*Initint[2]+jj-1]))
                                                {
                                                        total=newpush(store,ii,jj,0,total,now+1);
                                                        ++done;
                                                        break;
                                                }
                                        }
                        }
                }
                now++;
        }
}

__device__ double drawHairpin(int bp[],double mh,double ms,int Initint[])
{
        int i,N;

        N=0;
        if(fabs(ms)>999999999||fabs(mh)>999999999)
        {
		return 0.0;
        }
        else
        {
		for(i=1;i<Initint[0];++i)
		{
			if(bp[i-1]>0)
				N++;
                }
                return mh/(ms+(((N/2)-1)*-0.51986))-273.15;
        }
}

__device__ void initMatrix(int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[])
{
	int i,j;

	for(i=1;i<=Initint[0];++i)
	{
		for(j=1;j<=Initint[1];++j)
		{
			if(numSeq1[i]+numSeq2[j]!=3)
			{
				enthalpyDPT[(i-1)*Initint[2]+j-1]=1.0*INFINITY;
				entropyDPT[(i-1)*Initint[2]+j-1]=-1.0;
			}
			else
			{
				enthalpyDPT[(i-1)*Initint[2]+j-1]=0.0;
				entropyDPT[(i-1)*Initint[2]+j-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,double *EntropyEnthalpy,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	double S1,H1,T1,S2,H2,T2;

	if(numSeq1[i]+numSeq2[j]!=3)
	{
		entropyDPT[(i-1)*Initint[2]+j-1]=-1.0;
		enthalpyDPT[(i-1)*Initint[2]+j-1]=1.0*INFINITY;
		return;
	}

	S1=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[4430+numSeq2[j]*125+numSeq2[j-1]*25+numSeq1[i]*5+numSeq1[i-1]];
	H1=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5055+numSeq2[j]*125+numSeq2[j-1]*25+numSeq1[i]*5+numSeq1[i-1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]])<999999999&&fabs(parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]]+parameter[2750+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]]+parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq2[j]*25+numSeq2[j-1]*5+numSeq1[i]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	else if(fabs(parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2750+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2875+numSeq2[j]*25+numSeq1[i]*5+numSeq1[i-1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	S2=parameter[5680+numSeq1[i]*5+numSeq2[j]];
	H2=parameter[5705+numSeq1[i]*5+numSeq2[j]];
	T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			EntropyEnthalpy[0]=S2;
			EntropyEnthalpy[1]=H2;
		}
		else
		{
			EntropyEnthalpy[0]=S1;
			EntropyEnthalpy[1]=H1;
		}
	}
	else
	{
		EntropyEnthalpy[0]=S2;
		EntropyEnthalpy[1]=H2;
	}
	return;
}

__device__ void maxTM(int i,int j,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	double T0,T1,S0,S1,H0,H1;

	S0=entropyDPT[(i-1)*Initint[2]+j-1];
	H0=enthalpyDPT[(i-1)*Initint[2]+j-1];
	T0=(H0+Initdouble[0])/(S0+Initdouble[1]+Initdouble[2]); // at current position 
	if(fabs(enthalpyDPT[(i-2)*Initint[2]+j-2])<999999999&&fabs(Hs(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter))<999999999)
	{
		S1=(entropyDPT[(i-2)*Initint[2]+j-2]+Ss(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter));
		H1=(enthalpyDPT[(i-2)*Initint[2]+j-2]+Hs(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter));
	}
	else
	{
		S1=-1.0;
		H1=1.0*INFINITY;
	}
	T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);

	if(S1<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S1=-3224.0;
		H1=0.0;
	}
	if(S0<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		S0=-3224.0;
		H0=0.0;
	}
	if((T1>T0)||(S0>0&&H0>0)) // T1 on suurem 
	{
		entropyDPT[(i-1)*Initint[2]+j-1]=S1;
		enthalpyDPT[(i-1)*Initint[2]+j-1]=H1;
	}
	else if(T0>=T1)
	{
		entropyDPT[(i-1)*Initint[2]+j-1]=S0;
		enthalpyDPT[(i-1)*Initint[2]+j-1]=H0;
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,double* EntropyEnthalpy,int traceback,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double parameter[])
{
	int loopSize1,loopSize2,loopSize,N,N_loop;
	double T1,T2,S,H;

	S=-3224.0;
	H=0;
	loopSize1=ii-i-1;
	loopSize2=jj-j-1;
	if(ii<jj)
	{
		N=i;
		N_loop=N;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
	}
	else
	{
		N=j;
		N_loop=2*jj;
		if(loopSize1>2)
			N_loop-=(loopSize1-2);
		if(loopSize2>2)
			N_loop-=(loopSize2-2);
		N_loop=(N_loop/2)-1;
	}

	loopSize=loopSize1+loopSize2-1;
	if((loopSize1==0&&loopSize2>0)||(loopSize2==0&&loopSize1>0))// only bulges have to be considered
	{
		if(loopSize2==1||loopSize1==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((loopSize2==1&&loopSize1==0)||(loopSize2==0&&loopSize1==1))
			{
				H=parameter[3150+loopSize]+parameter[625+numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
				S=parameter[3060+loopSize]+parameter[numSeq1[i]*125+numSeq1[ii]*25+numSeq2[j]*5+numSeq2[jj]];
			}
			H+=enthalpyDPT[(i-1)*Initint[2]+j-1];
			S+=entropyDPT[(i-1)*Initint[2]+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}

			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(enthalpyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[0])/((entropyDPT[(ii-1)*Initint[2]+jj-1])+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			H=parameter[3150+loopSize]+parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5705+numSeq1[ii]*5+numSeq2[jj]];
			H+=enthalpyDPT[(i-1)*Initint[2]+j-1];

			S=parameter[3060+loopSize]+parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[5680+numSeq1[ii]*5+numSeq2[jj]];
			S+=entropyDPT[(i-1)*Initint[2]+j-1];
			if(fabs(H)>999999999)
			{
				H=1.0*INFINITY;
				S=-1.0;
			}
			T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
			T2=(enthalpyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[0])/(entropyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[1]+Initdouble[2]);
			if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
	}
	else if(loopSize1==1&&loopSize2==1)
	{
		S=parameter[1250+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[1250+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		S+=entropyDPT[(i-1)*Initint[2]+j-1];

		H=parameter[1875+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[1875+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		H+=enthalpyDPT[(i-1)*Initint[2]+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(enthalpyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[0])/(entropyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[1]+Initdouble[2]);
		if((T1-T2>=0.000001)||traceback==1)
		{
			if((T1>T2)||(traceback&&T1>=T2))
			{
				EntropyEnthalpy[0]=S;
				EntropyEnthalpy[1]=H;
			}
		}
		return;
	}
	else // only internal loops
	{
		H=parameter[3120+loopSize]+parameter[3805+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[3805+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]];
		H+=enthalpyDPT[(i-1)*Initint[2]+j-1];

		S=parameter[3030+loopSize]+parameter[3180+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]]+parameter[3180+numSeq2[jj]*125+numSeq2[jj-1]*25+numSeq1[ii]*5+numSeq1[ii-1]]+(-300/310.15*abs(loopSize1-loopSize2));
		S+=entropyDPT[(i-1)*Initint[2]+j-1];
		if(fabs(H)>999999999)
		{
			H=1.0*INFINITY;
			S=-1.0;
		}
		T1=(H+Initdouble[0])/((S+Initdouble[1])+Initdouble[2]);
		T2=(enthalpyDPT[(ii-1)*Initint[2]+jj-1]+Initdouble[0])/((entropyDPT[(ii-1)*Initint[2]+jj-1])+Initdouble[1]+Initdouble[2]);
		if((T1>T2)||((traceback&&T1>=T2)||(traceback==1)))
		{
			EntropyEnthalpy[0]=S;
			EntropyEnthalpy[1]=H;
		}
	}
	return;
}

__device__ void fillMatrix(double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double *parameter)
{
	int d,i,j,ii,jj;
	double SH[2];

	for(i=1;i<=Initint[0];++i)
	{
		for(j=1;j<=Initint[1];++j)
		{
			if(fabs(enthalpyDPT[(i-1)*Initint[2]+j-1])<999999999)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				LSH(i,j,SH,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

				if(fabs(SH[1])<999999999)
				{
					entropyDPT[(i-1)*Initint[2]+j-1]=SH[0];
					enthalpyDPT[(i-1)*Initint[2]+j-1]=SH[1];
				}
				if(i>1&&j>1)
				{
					maxTM(i,j,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
					for(d=3;d<=32;d++)
					{
						ii=i-1;
						jj=-ii-d+(j+i);
						if(jj<1)
						{
							ii-=abs(jj-1);
							jj=1;
						}
						for(;ii>0&&jj<j;--ii,++jj)
						{
							if(fabs(enthalpyDPT[(ii-1)*Initint[2]+jj-1])<999999999)
							{
								SH[0]=-1.0;
								SH[1]=1.0*INFINITY;
								calc_bulge_internal(ii,jj,i,j,SH,0,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

								if(SH[0]<-2500.0)
								{
									SH[0] =-3224.0;
									SH[1] = 0.0;
								}
								if(fabs(SH[1])<999999999)
								{
									enthalpyDPT[(i-1)*Initint[2]+j-1]=SH[1];
									entropyDPT[(i-1)*Initint[2]+j-1]=SH[0];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double EntropyEnthalpy[],double Initdouble[],char numSeq1[],char numSeq2[],double *parameter)
{
	double S1,S2,H1,H2,T1,T2;

	if(numSeq1[i]+numSeq2[j]!=3)
	{
		EntropyEnthalpy[0]=-1.0;
		EntropyEnthalpy[1]=1.0*INFINITY;
		return;
	}
	S1=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[4430+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
	H1=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[5055+numSeq1[i]*125+numSeq1[i+1]*25+numSeq2[j]*5+numSeq2[j+1]];
	if(fabs(H1)>999999999)
	{
		H1=1.0*INFINITY;
		S1=-1.0;
	}
	if(fabs(parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]])<999999999&&fabs(parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]]+parameter[2750+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]]+parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2500+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2625+numSeq1[i]*25+numSeq1[i+1]*5+numSeq2[j]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}

	if(fabs(parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]])<999999999)
	{
		S2=parameter[5680+numSeq1[i]*5+numSeq2[j]]+parameter[2750+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		H2=parameter[5705+numSeq1[i]*5+numSeq2[j]]+parameter[2875+numSeq1[i]*25+numSeq2[j]*5+numSeq2[j+1]];
		if(fabs(H2)>999999999)
		{
			H2=1.0*INFINITY;
			S2=-1.0;
		}
		T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
		if(fabs(H1)<999999999)
		{
			T1=(H1+Initdouble[0])/(S1+Initdouble[1]+Initdouble[2]);
			if(T1<T2)
			{
				S1=S2;
				H1=H2;
				T1=T2;
			}
		}
		else
		{
			S1=S2;
			H1=H2;
			T1=T2;
		}
	}
	S2=parameter[5680+numSeq1[i]*5+numSeq2[j]];
	H2=parameter[5705+numSeq1[i]*5+numSeq2[j]];
	T2=(H2+Initdouble[0])/(S2+Initdouble[1]+Initdouble[2]);
	if(fabs(H1)<999999999)
	{
		if(T1<T2)
		{
			EntropyEnthalpy[0]=S2;
			EntropyEnthalpy[1]=H2;
		}
		else
		{
			EntropyEnthalpy[0]=S1;
			EntropyEnthalpy[1]=H1;
		}
	}
	else
	{
		EntropyEnthalpy[0]=S2;
		EntropyEnthalpy[1]=H2;
	}
	return;
}

__device__ void traceback(int i,int j,int* ps1,int* ps2,double Initdouble[],int Initint[],double enthalpyDPT[],double entropyDPT[],char numSeq1[],char numSeq2[],double *parameter)
{
	int d,ii,jj,done;
	double SH[2];

	ps1[i-1]=j;
	ps2[j-1]=i;
	while(1)
	{
		SH[0]=-1.0;
		SH[1]=1.0*INFINITY;
		LSH(i,j,SH,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
		if(equal(entropyDPT[(i-1)*Initint[2]+j-1],SH[0])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],SH[1]))
			break;

		done = 0;
		if(i>1&&j>1&&equal(entropyDPT[(i-1)*Initint[2]+j-1],Ss(i-1,j-1,1,Initint,numSeq1,numSeq2,parameter)+entropyDPT[(i-2)*Initint[2]+j-2]))
		{
			i=i-1;
			j=j-1;
			ps1[i-1]=j;
			ps2[j-1]=i;
			done=1;
		}
		for(d=3;!done&&d<=32;++d)
		{
			ii=i-1;
			jj=-ii-d+(j+i);
			if(jj<1)
			{
				ii-=abs(jj-1);
				jj=1;
			}
			for(;!done&&ii>0&&jj<j;--ii,++jj)
			{
				SH[0]=-1.0;
				SH[1]=1.0*INFINITY;
				calc_bulge_internal(ii,jj,i,j,SH,1,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
				if(equal(entropyDPT[(i-1)*Initint[2]+j-1],SH[0])&&equal(enthalpyDPT[(i-1)*Initint[2]+j-1],SH[1]))
				{
					i=ii;
					j=jj;
					ps1[i-1]=j;
					ps2[j-1]=i;
					done=1;
					break;
				}
			}
		}
	}
}

__device__ double drawDimer(int *ps1,int *ps2,double H,double S,double Initdouble[],int Initint[])
{
        int i,N;

        if(fabs(Initdouble[3])>999999999)
                return (double)0.0;
        else
        {
                N=0;
                for(i=0;i<Initint[0];i++)
                {
                        if(ps1[i]>0)
                                ++N;
                }
                for(i=0;i<Initint[1];i++)
                {
                        if(ps2[i]>0)
                                ++N;
                }
                N=(N/2)-1;
                return (double)(H/(S+(N*-0.51986)+Initdouble[2])-273.15);
        }
}

__device__ int symmetry_thermo(char seq[])
{
	int i = 0;
	int seq_len=seq_length(seq);
	if(seq_len%2==1)
		return 0;

	while(i<seq_len/2)
	{
		if((seq[i]=='A'&&seq[seq_len-1-i]!='T')||(seq[i]=='T'&&seq[seq_len-1-i]!='A')||(seq[seq_len-1-i]=='A'&&seq[i]!='T')||(seq[seq_len-1-i]=='T'&&seq[i]!='A'))
			return 0;
		if((seq[i]=='C'&&seq[seq_len-1-i]!='G')||(seq[i]=='G'&&seq[seq_len-1-i]!='C')||(seq[seq_len-1-i]=='C'&&seq[i]!='G')||(seq[seq_len-1-i]=='G'&&seq[i]!='C'))
			return 0;
		i++;
	}
	return 1;
}

__device__ double thal(char oligo_f[],char oligo_r[],int type,double *parameter,char *d_Pchar,int *d_NumL)
{
	double SH[2],Initdouble[4];//0 is dplx_init_H, 1 is dplx_init_S, 2 is RC, 3 is SHleft
	int Initint[5]; //0 is len1, 1 is len2, 2 is len3, 3 is bestI, 4 is bestJ
	int i, j;
	double T1,enthalpyDPT[625],entropyDPT[625],send5[26],hend5[26],result_TH;
	int ps1[25],ps2[25];
	char numSeq1[27],numSeq2[27];
	double mh, ms;

/*** INIT values for unimolecular and bimolecular structures ***/
	if (type==4) /* unimolecular folding */
	{
		Initdouble[0]= 0.0;
		Initdouble[1] = -0.00000000001;
		Initdouble[2]=0;
	}
	else /* hybridization of two oligos */
	{
		Initdouble[0]= 200;
		Initdouble[1]= -5.7;
		if(symmetry_thermo(oligo_f) && symmetry_thermo(oligo_r))
			Initdouble[2]=1.9872* log(38/1000000000.0);
		else
			Initdouble[2]=1.9872* log(38/4000000000.0);
	}
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		Initint[0]=seq_length(oligo_f);
		Initint[1]=seq_length(oligo_r);
	 	for(i=1;i<=Initint[0];++i)
			numSeq1[i]=str2int(oligo_f[i-1]);
		for(i=1;i<=Initint[1];++i)
			numSeq2[i]=str2int(oligo_r[Initint[1]-i]);
	}
	else if(type==3)
	{
		Initint[0]=seq_length(oligo_r);
		Initint[1]=seq_length(oligo_f);
		for(i=1;i<=Initint[0];++i)
			numSeq1[i]=str2int(oligo_r[i-1]);
		for(i=1;i<=Initint[1];++i)
			numSeq2[i]=str2int(oligo_f[Initint[1]-i]);
	}
	else
	{
		Initint[0]=seq_length(oligo_f);
                Initint[1]=seq_length(oligo_r);
		Initint[2]=Initint[1]-1;
                for(i=1;i<=Initint[0];++i)      
                        numSeq1[i]=str2int(oligo_f[i-1]);   
                for(i=1;i<=Initint[1];++i)      
                        numSeq2[i]=str2int(oligo_r[i-1]);
	}
	numSeq1[0]=numSeq1[Initint[0]+1]=numSeq2[0]=numSeq2[Initint[1]+1]=4; /* mark as N-s */

	result_TH=0;
	if (type==4) /* calculate structure of monomer */
	{
		initMatrix2(Initint,enthalpyDPT,entropyDPT,numSeq1);
		fillMatrix2(Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter,d_Pchar,d_NumL);
		calc_terminal_bp(310.15,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,parameter);
		mh=hend5[Initint[0]];
		ms=send5[Initint[0]];
		for (i=0;i<Initint[0];i++)
			ps1[i]=0;
		if(fabs(mh)<999999999)
		{
			tracebacku(ps1,Initdouble,Initint,enthalpyDPT,entropyDPT,send5,hend5,numSeq1,numSeq2,parameter,d_Pchar,d_NumL);
			result_TH=drawHairpin(ps1,mh,ms,Initint);
			result_TH=(int)(result_TH*100+0.5)/100.0;
		}
	}
	else  /* Hybridization of two moleculs */
	{
		Initint[2]=Initint[1];
		initMatrix(Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2);
		fillMatrix(Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);

		Initdouble[3]=-1.0*INFINITY;
	/* calculate terminal basepairs */
		Initint[3]=Initint[4]=0;
		if(type==1)
			for (i=1;i<=Initint[0];i++)
			{
				for (j=1;j<=Initint[1];j++)
				{
					RSH(i,j,SH,Initdouble,numSeq1,numSeq2,parameter);
					SH[0]=SH[0]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
					SH[1]=SH[1]+0.000001;
					T1=((enthalpyDPT[(i-1)*Initint[2]+j-1]+ SH[1] +Initdouble[0]) / ((entropyDPT[(i-1)*Initint[2]+j-1]) + SH[0] +Initdouble[1] + Initdouble[2])) -273.15;
					if(T1>Initdouble[3]&&((entropyDPT[(i-1)*Initint[2]+j-1]+SH[0])<0&&(SH[1]+enthalpyDPT[(i-1)*Initint[2]+j-1])<0))
					{
						Initdouble[3]=T1;
						Initint[3]=i;
						Initint[4]=j;
					}
				}
			}
		if(type==2||type==3)
		{
		 //THAL_END1
			Initint[4]=0;
			Initint[3]=Initint[0];
			i=Initint[0];
			Initdouble[3]=-1.0*INFINITY;
			for (j=1;j<=Initint[1];++j)
			{
				RSH(i,j,SH,Initdouble,numSeq1,numSeq2,parameter);
				SH[0]=SH[0]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
				SH[1]=SH[1]+0.000001;
				T1=((enthalpyDPT[(i-1)*Initint[2]+j-1]+SH[1]+Initdouble[0])/((entropyDPT[(i-1)*Initint[2]+j-1])+SH[0]+Initdouble[1]+Initdouble[2]))-273.15;
				if (T1>Initdouble[3]&&((SH[0]+entropyDPT[(i-1)*Initint[2]+j-1])<0&&(SH[1]+enthalpyDPT[(i-1)*Initint[2]+j-1])<0))
				{
					Initdouble[3]=T1;
					Initint[4]=j;
				}
			}
		}
		if(fabs(Initdouble[3])>999999999)
			Initint[3]=Initint[4]=1;
		RSH(Initint[3],Initint[4],SH,Initdouble,numSeq1,numSeq2,parameter);
	 // tracebacking 
		for (i=0;i<Initint[0];++i)
			ps1[i]=0;
		for (j=0;j<Initint[1];++j)
			ps2[j] = 0;
		if(fabs(enthalpyDPT[(Initint[3]-1)*Initint[2]+Initint[4]-1])<999999999)
		{
			traceback(Initint[3],Initint[4],ps1,ps2,Initdouble,Initint,enthalpyDPT,entropyDPT,numSeq1,numSeq2,parameter);
			result_TH=drawDimer(ps1,ps2,(enthalpyDPT[(Initint[3]-1)*Initint[2]+Initint[4]-1]+SH[1]+Initdouble[0]),(entropyDPT[(Initint[3]-1)*Initint[2]+Initint[4]-1]+SH[0]+Initdouble[1]),Initdouble,Initint);
			result_TH=(int)(result_TH*100+0.5)/100.0;
		}
	}
        return result_TH;
}

///function in gpu, generate a read; int length: the length of reads
__device__ void generate(char *d_seq,char seq[],int pos,int length)
{
	int i;
	for(i=0;i<length;i++)
	{
		seq[i]=d_seq[pos+i];
	}
	seq[i]='\0';
}

///function in gpu, check the GC-content; int length: the length of read
__device__ int gc(char seq[],int length)
{
	int i,number;
	float gc;

	number=0;
	for(i=0;i<length;i++)
	{
		if(seq[i]=='C')
		{
			number++;
			continue;
		}
	
		if(seq[i]=='G')
		{
			number++;
		}
	}

	gc=1.0*number/length*100;
	if((gc<40)||(gc>65))
	{
		return 0;
	}
	return 1;
}

///function in gpu, translate A...G to int
__device__ int translate(char a)
{
	if(a=='A')
		return 0;
	if(a=='T')
		return 1;
	if(a=='C')
		return 2;
	return 3;
}

//function in gpu, caculate tm
__device__ int tm(char seq[],float *d_deltah,float *d_deltas,int length,float max_tm,float min_tm)
{
	int i,pos;
	float deltah,deltas,result;

	deltah=0;
	deltas=0;
	for(i=0;i<length-1;i++)
	{
		pos=translate(seq[i]);
		pos=pos*4+translate(seq[i+1]);
		deltah+=d_deltah[pos];
		deltas+=d_deltas[pos];
	}

	deltah=(-1.0)*deltah;
	deltas=(-1.0)*deltas;
	if((seq[0]=='A')||(seq[0]=='T'))
	{
		deltah+=2.3;
		deltas+=4.1;
	}
	else
	{
		deltah+=0.1;
		deltas-=2.8;
	}
        if((seq[length-1]=='A')||(seq[length-1]=='T'))
        {
                deltah+=2.3;
                deltas+=4.1;
        }
        else
        {
                deltah+=0.1;
                deltas-=2.8;
        }
	result=1000.0*deltah/(deltas-0.51986*(length-1)-36.70381)-273.15;
	if((result<min_tm)||(result>max_tm))
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

///function in gpu, caculate stability, int strand: 0 is 5' and 1 is 3'
__device__ int stability(char seq[],float *d_stab,int length,int strand)
{
	int i,pos;
	
	pos=0;
	for(i=0;i<6;i++)
	{
		if(strand==0)
		{
			pos=pos*4+translate(seq[i]);
		}
		else
		{
			pos=pos*4+translate(seq[i+length-6]);
		}
	}
	
	if(d_stab[pos]<4)
	{
		return 0;
	}
//the other part
        pos=0;
        for(i=0;i<6;i++)
        {
                if(strand==1)
                {
                        pos=pos*4+translate(seq[i]);
                }
                else
                {
                        pos=pos*4+translate(seq[i+length-6]);
                }
        }

        if(d_stab[pos]<3)
        {
                return 0;
        }

	return 1;
}

//function in gpu: whether species chars in reads
__device__ int words(char *d_seq,int position,int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(d_seq[position+i]=='N')
		{
			return 0;
		}
	}
	return 1;
}

//function in gpu, reverse the strand,+ to - strand
__device__ void reverse(char seq[],char rev[],int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(seq[length-1-i]=='A')
		{
			rev[i]='T';
			continue;
		}
                if(seq[length-1-i]=='T')
                {
                        rev[i]='A';
                        continue;
                }
                if(seq[length-1-i]=='C')
                {
                        rev[i]='G';
                        continue;
                }
		rev[i]='C';
	}
	rev[i]='\0';
}

__device__ int check_long_ploy(char primer[],int length)
{
        int i,same;
        char ref;

        same=1;
        ref=primer[0];
        for(i=1;i<length;i++)
        {
                if(primer[i]==ref)
                        same++;
                else
                {
                        if(same>=6)
                                return 0;
                        same=1;
                        ref=primer[i];
                }
        }
        if(same>=6)
                return 0;
        return 1;
}

///function: int length: the length of genome
__global__ void candidate_primer(char *d_seq,int *d_pos,int *d_len,int *d_rev_len,float *d_stab,float *d_deltah,float *d_deltas,int strand,float max_tm,float min_tm,int length,int check_flag,double *parameter,char *d_Pchar,int *d_NumL)
{
	int id,i,circle,check,plus,minus;
	char primer[30],rev[30];

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(circle=id;circle<length;circle=circle+blockDim.x*gridDim.x)
	{
		for(i=0;i<8;i++)   //primer length is from 18 to 25
		{
			d_len[8*circle+i]=0;
			d_rev_len[8*circle+i]=0;
		}
		d_pos[circle]=0;
	
		for(i=18;i<=25;i++)  //read length is from 18 to 25
		{
			if(circle+i>length)
				break;
			check=words(d_seq,circle,i);
			if(check==0)
                                break;

			generate(d_seq,primer,circle,i);
			check=gc(primer,i);
			if(check==0)
				continue;

			check=check_long_ploy(primer,i);
			if(check==0)
                                continue;

			check=tm(primer,d_deltah,d_deltas,i,max_tm,min_tm);
			if(check==0)
				continue;

                        check=stability(primer,d_stab,i,strand);
                        if(check==1)     //+ strand
				plus=1;
			else
				plus=0;
			
		//secondary structure
			if(check_flag&&plus)
			{
				if(thal(primer,primer,1,parameter,d_Pchar,d_NumL)>min_tm-10)
					plus=0;	
			}
			if(check_flag&&plus)
                        {
                                if(thal(primer,primer,2,parameter,d_Pchar,d_NumL)>min_tm-10)  
                                        plus=0;
                        }
			if(check_flag&&plus)
                        {                
                                if(thal(primer,primer,4,parameter,d_Pchar,d_NumL)>min_tm-10)
                                        plus=0;         
                        }
			if(plus)
                                d_len[circle*8+i-18]=1;
	
			reverse(primer,rev,i);  //generate - strand
			check=stability(rev,d_stab,i,strand);
			if(check==1)
				minus=1;
			else
				minus=0;
		//secondary structure      
                        if(check_flag&&minus)
                        {                
                                if(thal(rev,rev,1,parameter,d_Pchar,d_NumL)>min_tm-10)
                                        minus=0;         
                        }           
                        if(check_flag&&minus)
                        {
                                if(thal(rev,rev,2,parameter,d_Pchar,d_NumL)>min_tm-10)
                                        minus=0;
                        }                
                        if(check_flag&&minus)
                        {
                                if(thal(rev,rev,4,parameter,d_Pchar,d_NumL)>min_tm-10)
                                        minus=0;
                        }
                        if(minus)
				d_rev_len[circle*8+i-18]=1;
		}
		
		for(i=0;i<8;i++)
		{
			d_pos[circle]+=(d_len[circle*8+i]+d_rev_len[8*circle+i]);
		}
	}
	__syncthreads();
}

void usage()
{
        printf("Usage:\n");
        printf("    single  -in <fasta_file>  -out <primers_file_name>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the reference sequence file, fasta formate\n");
        printf("    -out  <string>:  the prefix of output files, those files store candidate single primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers. default is current directory\n");
        printf("    -stab <string>:  the parameter file used in calculating the primers' stability. default is stab_parameter.txt in Par/ directory\n");
        printf("    -tm   <string>:  the parameter file used in calcalating Tm and second structure. default is stab_parameter.txt in Par/ directory\n");
	printf("    -check   <int>:  0: don't check primers' secondary structure; !=0: check, default is 1\n");
        printf("    -par  <string>:  the directory of storing parameter files used to check primers' secondary structure, default is Par/\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region. high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design candidate loop single primers\n");
        printf("    -h/-help:        print usage\n");
}

int create_file(char *prefix,char *dir,char *seq,int *pos,int *len,int *rev_len,int length)
{
	char *file;
	int total,i,j;
	FILE *OUT;

	total=0;
	i=strlen(dir)+strlen(prefix)+20;
	file=(char *)malloc(i);
        memset(file,'\0',i);
        strcpy(file,dir);
        strcat(file,prefix);
        OUT=fopen(file,"w");
        if(OUT==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
	
        for(i=0;i<length;i++)
        {
                if(pos[i]==0)
                        continue;
                for(j=0;j<8;j++)
                {
                        if((len[8*i+j]+rev_len[8*i+j])==0)
                                continue;
                       	fprintf(OUT,"pos:%d\tlength:%d\t+:%d\t-:%d\n",i,(j+18),len[8*i+j],rev_len[8*i+j]);
			total++;
                }
        }
	fclose(OUT);
	free(file);
	return total;
}

main(int argc, char **argv)
{
	double *H_parameter,*parameter;
	int *pos,*d_pos,*len,*d_len,length,flag[10],i,*rev_len,*d_rev_len,num_outer,num_inner,num_loop,NumL[2],*d_NumL;
	float deltah[16],deltas[16],stab[4096],*d_deltah,*d_deltas,*d_stab,temp1,temp2;
	char *seq,*d_seq,*store_path,*prefix,*stab_path,*tm_path,*curren_path,*input,*outer,*inner,*loop,*par_path,*temp,*Pchar,*d_Pchar;
	FILE *fp;
	time_t start,end;
        struct stat statbuf;
//flag: 0:input; 1: out_prefix; 2: dir; 3: stab; 4: tm; 5: high; 6: low; 7: loop; 8: secondary structure; 9: path for secondary structure

	start=time(NULL);
//get input
        for(i=0;i<10;i++)
        {
                flag[i]=0;
        }
	flag[8]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			input=(char *)malloc(length+1);
			memset(input,'\0',length+1);
                        strcpy(input,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        prefix=(char *)malloc(length+1);
                        memset(prefix,'\0',length+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			if(argv[i+1][length-1]=='/')
			{
                        	store_path=(char *)malloc(length+1);
                        	memset(store_path,'\0',length+1);
                        	strcpy(store_path,argv[i+1]);
			}
			else
			{
				store_path=(char *)malloc(length+2);
				memset(store_path,'\0',length+2);
				strcpy(store_path,argv[i+1]);
				store_path[length]='/';
			}
                        i=i+2;
                }
                else if(strcmp(argv[i],"-stab")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-stab\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        stab_path=(char *)malloc(length+1);
                        memset(stab_path,'\0',length+1);
                        strcpy(stab_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-tm")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        tm_path=(char *)malloc(length+1);
                        memset(tm_path,'\0',length+1);
                        strcpy(tm_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[6]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[7]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
		else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[8]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[9]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        length=strlen(argv[i+1]);
                        if(argv[i+1][length-1]=='/')
                        {
                                par_path=(char *)malloc(length+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(length+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='/';
                                par_path[length+1]='\0';
                        }
                        i=i+2;
                }		
                else
                {
                        printf("Error: don't have the parameter: %s\n",argv[i]);
                        usage();
                        exit(1);
                }
        }
//check paramters
        if(flag[5]+flag[6]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
        if(flag[0]==0)
        {
                printf("Error! Users must input the reference sequence file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the prefix name for output file with -out!\n");
                usage();
                exit(1);
        }
        for(i=0;i<strlen(prefix);i++)
        {
                if(prefix[i]=='/')
                {
                        printf("Error! the -out parameter couldn't contain any directory!\n");
                        usage();
                        exit(1);
                }
        }
//prepare
	inner=(char *)malloc(4096);
        memset(inner,'\0',4096);
        getcwd(inner,4096);
        length=strlen(inner);
        curren_path=(char *)malloc(length+1);
        memset(curren_path,'\0',length+1);
        strcpy(curren_path,inner);
        if(flag[2]==0)
        {
                store_path=(char *)malloc(length+2);
                memset(store_path,'\0',length+2);
                strcpy(store_path,curren_path);
                store_path[length]='/';
        }
        free(inner);

        length=strlen(store_path)+12;
        outer=(char *)malloc(length);
        memset(outer,'\0',length);
        strcpy(outer,store_path);

        inner=(char *)malloc(length);
        memset(inner,'\0',length);
        strcpy(inner,store_path);

        if(flag[7]==1)
        {
                loop=(char *)malloc(length);
                memset(loop,'\0',length);
                strcpy(loop,store_path);
        }
        if(flag[5]==1)
        {
                strcat(outer,"high-outer/");
                strcat(inner,"high-inner/");
                if(flag[7]==1)
                        strcat(loop,"high-loop/");
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(inner,"low-inner/");
                if(flag[7]==1)
                        strcat(loop,"low-loop/");
        }
        mkdir(outer,0755);
        mkdir(inner,0755);        
        if(flag[7]==1)
                mkdir(loop,0755);        

//stability parameter file
        if(flag[3]==0)
        {
		length=strlen(curren_path);
                stab_path=(char *)malloc(length+30);
                memset(stab_path,'\0',length+30);
                strcpy(stab_path,curren_path);
                i=length-1;
                while(stab_path[i]!='/'&&i>=0)
                {
                        stab_path[i]='\0';
                        i--;
                }
                strcat(stab_path,"Par/stab_parameter.txt");
        }
//tm parameter file
        if(flag[4]==0)
        {
		length=strlen(curren_path);
                tm_path=(char *)malloc(length+30);
                memset(tm_path,'\0',length+30);
                strcpy(tm_path,curren_path);
                i=length-1;
                while(tm_path[i]!='/'&&i>=0)
                {
                        tm_path[i]='\0';
                        i--;
                }
                strcat(tm_path,"Par/tm_nn_parameter.txt");
        }
//secondary structure
	if(flag[8]&&flag[9]==0)
        {
                length=strlen(curren_path);
                par_path=(char *)malloc(length+10);
                memset(par_path,'\0',length+10);
                strcpy(par_path,curren_path);
                i=length-1;
                while(par_path[i]!='/'&&i>=0)
                {
                        par_path[i]='\0';
                        i--;
                }
                strcat(par_path,"Par/");
        }
	if(flag[8])
	{
		NumL[0]=get_num_line(par_path,0);
	        NumL[1]=get_num_line(par_path,1);
	        H_parameter=(double *)malloc((5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        memset(H_parameter,'\0',(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        Pchar=(char *)malloc(10*NumL[0]+12*NumL[1]);
	        memset(Pchar,'\0',10*NumL[0]+12*NumL[1]);
	        hipMalloc((void **)&d_Pchar,10*NumL[0]+12*NumL[1]);
	        hipMemset(d_Pchar,'\0',10*NumL[0]+12*NumL[1]);
		hipMalloc((void **)&parameter,(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        hipMemset(parameter,'\0',(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
		hipMalloc((void **)&d_NumL,2*sizeof(int));
	        hipMemset(d_NumL,'\0',2*sizeof(int));

		getStack(par_path,H_parameter);
	        getStackint2(par_path,H_parameter);
	        getDangle(par_path,H_parameter);
	        getLoop(par_path,H_parameter);
	        getTstack(par_path,H_parameter);
	        getTstack2(par_path,H_parameter);
	        getTriloop(par_path,H_parameter,Pchar,NumL);
	        getTetraloop(par_path,H_parameter,Pchar,NumL);
	        tableStartATS(6.9,H_parameter);
	        tableStartATH(2200.0,H_parameter);	
	}

//input reference sequence
        if(access(input,0)==-1)
        {
                printf("Error! Don't have the %s file.\n",input);
                exit(1);
        }
        stat(input,&statbuf);
        length=statbuf.st_size;
        length=length+100;
        temp=(char *)malloc(length);
        memset(temp,'\0',length);
        seq=(char *)malloc(length*sizeof(char));
        memset(seq,'\0',length*sizeof(char));

        fp=fopen(input,"r");   //open the sequence file
        if(fp==NULL)
        {
                printf("Error! can't open the %s file!\n",input);
                exit(1);
        }
        fread(temp,length*sizeof(char),1,fp);
        fclose(fp); 

        length=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
		if(temp[i]=='a'||temp[i]=='A')
                        seq[length]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[length]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[length]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[length]='G';
                else
                        seq[length]='N';
                i++;
                length++;
        }
        free(temp);
        length=strlen(seq);

//input Tm parameter
        fp=fopen(tm_path,"r");  //read the paramter of deltah and deltas
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",tm_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f\t%f",&i,&temp1,&temp2)!=EOF)
        {
                deltah[i]=temp1;
                deltas[i]=temp2;
        }
        fclose(fp);

//input stability parameter
        fp=fopen(stab_path,"r");  //read the parameters of stability
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",stab_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f",&i,&temp1)!=EOF)
        {
                stab[i]=temp1;
        }
        fclose(fp);

	hipMalloc((void **)&d_seq,length*sizeof(char));
	hipMemset(d_seq,'\0',length*sizeof(char));

	hipMalloc((void **)&d_deltah,16*sizeof(float));
	hipMemset(d_deltah,'\0',16*sizeof(float));
	hipMalloc((void **)&d_deltas,16*sizeof(float));
	hipMemset(d_deltas,'\0',16*sizeof(float));
	hipMalloc((void **)&d_stab,4096*sizeof(float));
	hipMemset(d_stab,'\0',4096*sizeof(float));

	/////from cpu to gpu
	hipMemcpy(d_seq,seq,length*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_deltah,deltah,16*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_deltas,deltas,16*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_stab,stab,4096*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void **)&d_pos,length*sizeof(int));
	hipMemset(d_pos,'\0',length*sizeof(int));
	hipMalloc((void **)&d_len,8*length*sizeof(int));
	hipMemset(d_len,'\0',8*length*sizeof(int));
	hipMalloc((void **)&d_rev_len,8*length*sizeof(int));
        hipMemset(d_rev_len,'\0',8*length*sizeof(int));
	pos=(int *)malloc(length*sizeof(int));
	memset(pos,'\0',length*sizeof(int));
	len=(int *)malloc(8*length*sizeof(int));
	memset(len,'\0',8*length*sizeof(int));
        rev_len=(int *)malloc(8*length*sizeof(int));
        memset(rev_len,'\0',8*length*sizeof(int));

//secondary structure
	if(flag[8])
	{
		hipMemcpy(parameter,H_parameter,(5730+2*NumL[0]+2*NumL[1])*sizeof(double),hipMemcpyHostToDevice);
        	hipMemcpy(d_Pchar,Pchar,10*NumL[0]+12*NumL[1],hipMemcpyHostToDevice);
		hipMemcpy(d_NumL,NumL,2*sizeof(int),hipMemcpyHostToDevice);
	}
	end=time(NULL);
	printf("It takes %d seconds to prepare.\n",(int)difftime(end,start));
	start=time(NULL);

	if(flag[5]==1)
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
		hipMemset(d_len,'\0',8*length*sizeof(int));
		hipMemset(d_rev_len,'\0',8*length*sizeof(int));
		candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,61,59,length,flag[8],parameter,d_Pchar,d_NumL);
		hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,0,66,64,length,flag[8],parameter,d_Pchar,d_NumL);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);

                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,66,64,length,flag[8],parameter,d_Pchar,d_NumL);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
        else
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,56,54,length,flag[8],parameter,d_Pchar,d_NumL);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,0,61,59,length,flag[8],parameter,d_Pchar,d_NumL);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length);
                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,61,59,length,flag[8],parameter,d_Pchar,d_NumL);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length);
		}
        }
	hipFree(d_pos);
	hipFree(d_len);
	hipFree(d_rev_len);
	hipFree(d_seq);
	hipFree(d_stab);
	hipFree(d_deltah);
	hipFree(d_deltas);
	free(pos);
        free(len);
        free(rev_len);
	free(seq);

	printf("There ara %d candidate primers used as F3/F2/B2/B3.\n",num_outer);
        printf("There are %d candidate primers used as F1c/B1c.\n",num_inner);
        if(flag[7]==1)
                printf("There are %d candidate primers used as LF/LB.\n",num_loop);
        //check
        if(num_outer<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(num_inner<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && num_loop<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
        printf("It takes %d seconds to design candidate single primers.\n",(int)difftime(end,start));

	free(store_path);
	free(prefix);
	free(stab_path);
	free(tm_path);
	free(curren_path);
	free(input);
	free(outer);
	free(inner);
	if(flag[7])
		free(loop);

	if(flag[8])
	{
		free(Pchar);
		free(H_parameter);
		hipFree(parameter);
        	hipFree(d_Pchar);
        	hipFree(d_NumL);
	}
	if(flag[8]||flag[9])
		free(par_path);
}
