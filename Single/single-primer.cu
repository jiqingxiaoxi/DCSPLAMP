#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<sys/stat.h>
#include<regex.h>
#include<unistd.h>

///function in gpu, generate a read; int length: the length of reads
__device__ void generate(char *d_seq,char seq[],int pos,int length)
{
	int i;
	for(i=0;i<length;i++)
	{
		seq[i]=d_seq[pos+i];
	}
	seq[i]='\0';
}

///function in gpu, check the GC-content; int length: the length of read
__device__ int gc(char seq[],int length)
{
	int i,number;
	float gc;

	number=0;
	for(i=0;i<length;i++)
	{
		if(seq[i]=='C'||seq[i]=='c')
		{
			number++;
			continue;
		}
	
		if(seq[i]=='G'||seq[i]=='g')
		{
			number++;
		}
	}

	gc=1.0*number/length*100;
	if((gc<40)||(gc>60))
	{
		return 0;
	}
	return 1;
}

///function in gpu, translate A...G to int
__device__ int translate(char a)
{
	if(a=='A'||a=='a')
		return 0;
	if(a=='T'||a=='t')
		return 1;
	if(a=='C'||a=='c')
		return 2;
	return 3;
}

//function in gpu, caculate tm
__device__ int tm(char seq[],float *d_deltah,float *d_deltas,int length,float max_tm,float min_tm)
{
	int i,pos;
	float deltah,deltas,result;

	deltah=0;
	deltas=0;
	for(i=0;i<length-1;i++)
	{
		pos=translate(seq[i]);
		pos=pos*4+translate(seq[i+1]);
		deltah+=d_deltah[pos];
		deltas+=d_deltas[pos];
	}

	deltah=(-1.0)*deltah;
	deltas=(-1.0)*deltas;
	if((seq[0]=='A')||(seq[0]=='T')||seq[0]=='a'||seq[0]=='t')
	{
		deltah+=2.3;
		deltas+=4.1;
	}
	else
	{
		deltah+=0.1;
		deltas-=2.8;
	}
        if((seq[length-1]=='A')||(seq[length-1]=='T')||seq[length-1]=='a'||seq[length-1]=='t')
        {
                deltah+=2.3;
                deltas+=4.1;
        }
        else
        {
                deltah+=0.1;
                deltas-=2.8;
        }
	result=1000.0*deltah/(deltas-0.51986*(length-1)-36.70381)-273.15;
	if((result<min_tm)||(result>max_tm))
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

///function in gpu, caculate stability, int strand: 0 is 5' and 1 is 3'
__device__ int stability(char seq[],float *d_stab,int length,int strand)
{
	int i,pos;
	
	pos=0;
	for(i=0;i<6;i++)
	{
		if(strand==0)
		{
			pos=pos*4+translate(seq[i]);
		}
		else
		{
			pos=pos*4+translate(seq[i+length-6]);
		}
	}
	
	if(d_stab[pos]<4)
	{
		return 0;
	}
//the other part
        pos=0;
        for(i=0;i<6;i++)
        {
                if(strand==1)
                {
                        pos=pos*4+translate(seq[i]);
                }
                else
                {
                        pos=pos*4+translate(seq[i+length-6]);
                }
        }

        if(d_stab[pos]<3)
        {
                return 0;
        }

	return 1;
}

//function in gpu: whether species chars in reads
__device__ int words(char *d_seq,int position,int length)
{
	int i,flag;
	
	flag=1;
	for(i=0;i<length;i++)
	{
		if(d_seq[position+i]=='A'||d_seq[position+i]=='a')
		{
			continue;
		}
		if(d_seq[position+i]=='T'||d_seq[position+i]=='t')
		{
			continue;
		}
		if(d_seq[position+i]=='C'||d_seq[position+i]=='c')
                {
                        continue;
                }
                if(d_seq[position+i]=='G'||d_seq[position+i]=='g')
                {
                        continue;
                }
		flag--;
		break;
	}
	return flag;
}

//function in gpu, reverse the strand,+ to - strand
__device__ void reverse(char seq[],char rev[],int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(seq[length-1-i]=='A'||seq[length-1-i]=='a')
		{
			rev[i]='T';
			continue;
		}
                if(seq[length-1-i]=='T'||seq[length-1-i]=='t')
                {
                        rev[i]='A';
                        continue;
                }
                if(seq[length-1-i]=='C'||seq[length-1-i]=='c')
                {
                        rev[i]='G';
                        continue;
                }
		rev[i]='C';
	}
}

///function: int length: the length of genome
__global__ void candidate_primer(char *d_seq,int *d_pos,int *d_len,int *d_rev_len,float *d_stab,float *d_deltah,float *d_deltas,int strand,float max_tm,float min_tm,int length)
{
	int id,i,circle,check;
	char primer[30],rev[30];

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(circle=id;circle<length;circle=circle+blockDim.x*gridDim.x)
	{
		for(i=0;i<8;i++)   //primer length is from 18 to 25
		{
			d_len[8*circle+i]=0;
			d_rev_len[8*circle+i]=0;
		}
		d_pos[circle]=0;
	
		for(i=18;i<=25;i++)  //read length is from 18 to 25
		{
			if(circle+i>length)
				break;
			check=words(d_seq,circle,i);
			if(check==0)
                                continue;

			generate(d_seq,primer,circle,i);
			check=gc(primer,i);
			if(check==0)
				continue;

			check=tm(primer,d_deltah,d_deltas,i,max_tm,min_tm);
			if(check==0)
				continue;

                        check=stability(primer,d_stab,i,strand);
                        if(check==1)     //+ strand
                                d_len[circle*8+i-18]=1;
	
			reverse(primer,rev,i);  //generate - strand
			check=stability(rev,d_stab,i,strand);
			if(check==1)
				d_rev_len[circle*8+i-18]=1;
		}
		
		for(i=0;i<8;i++)
		{
			d_pos[circle]+=(d_len[circle*8+i]+d_rev_len[8*circle+i]);
		}
	}
	__syncthreads();
}

void take_regulate(regmatch_t pmatch[],int which,char *out,char *input)
{
        int i,j=0;
        for(i=pmatch[which].rm_so;i<pmatch[which].rm_eo;i++)
        {
                out[j]=input[i];
                j++;
        }
        out[j]='\0';
}

void cpu_reverse(char seq[],char rev[],int length)
{
        int i;
        
        for(i=0;i<length;i++)
        {
                if(seq[length-1-i]=='A'||seq[length-1-i]=='a')
                {
                        rev[i]='T';
                        continue;
                }
                if(seq[length-1-i]=='T'||seq[length-1-i]=='t')
                {
                        rev[i]='A';
                        continue;
                }
                if(seq[length-1-i]=='C'||seq[length-1-i]=='c')
                {
                        rev[i]='G';
                        continue;
                }
                rev[i]='C';
        }
        rev[i]='\0';
}

int second_structure(char *prefix,char *dir,float tm_threshold,char *primer3)
{
	char *in,*out,*script,*line,result[20],pattern1[50],pattern2[50],pattern3[50];
	FILE *fp1,*fp2;
	int pos,len,plus,minus,total,flag,len_now,pos_now,status,cflags,length;
	regex_t reg1,reg2,reg3;
	regmatch_t pmatch[3];
	float TH;

	len=strlen(prefix)+strlen(dir)+20;
	in=(char *)malloc(len*sizeof(char));
	memset(in,'\0',len*sizeof(char));
	strcpy(in,dir);
	strcat(in,prefix);
	strcat(in,"-primer3.txt");
//run primer3
	out=(char *)malloc(len*sizeof(char));
	memset(out,'\0',len*sizeof(char));
	strcpy(out,dir);
	strcat(out,prefix);
	strcat(out,"-result.txt");

	len=strlen(primer3)+strlen(in)+strlen(out)+50;
	script=(char *)malloc(len*sizeof(char));
	memset(script,'\0',len*sizeof(char));
	sprintf(script,"%s -strict_tags %s > %s",primer3,in,out);
	system(script);
	free(script);
	remove(in);

//check structure
	len=strlen(prefix)+strlen(dir)+20;
	memset(in,'\0',len*sizeof(char));
	strcpy(in,out);
	memset(out,'\0',len*sizeof(char));
	strcpy(out,dir);
	strcat(out,prefix);
	fp1=fopen(in,"r");
	if(fp1==NULL)
	{
		printf("Can't open %s file!\n",in);
		exit(1);
	}
	fp2=fopen(out,"w");
	if(fp2==NULL)
	{
		printf("Can't create %s file!\n",out);
		exit(1);
	}

//prepare regular
	strcpy(pattern1,"ID=(\\w+)-(\\w+)-(.)");
	cflags=REG_EXTENDED;
	regcomp(&reg1,pattern1,cflags);
	strcpy(pattern2,"_TH=(.+)");
	regcomp(&reg2,pattern2,cflags);
	strcpy(pattern3,"Contains too-long poly nucleotide tract");
	regcomp(&reg3,pattern3,cflags);

//read file
	pos=0;
	len=0;
	plus=0;
	minus=0;
	total=0;
	length=200+strlen(primer3);
	line=(char *)malloc(length*sizeof(char));
	memset(line,'\0',length*sizeof(char));
	while(fgets(line,length*sizeof(char),fp1)!=NULL)
	{
		if(regexec(&reg1,line,4,pmatch,0)==0)  //begin
		{
			take_regulate(pmatch,1,result,line);
			pos_now=atoi(result); //pos
			take_regulate(pmatch,2,result,line);
			len_now=atoi(result);

			if(pos_now!=pos||len_now!=len) //new one
			{
				if(plus+minus!=0)
				{
					fprintf(fp2,"pos:%d\tlength:%d\t+:%d\t-:%d\n",pos,len,plus,minus);
					total++;
				}
				pos=pos_now;
				len=len_now;
				plus=0;
				minus=0;
			}
			take_regulate(pmatch,3,result,line);
			if(result[0]=='+')
                        {
                                flag=1;
                        }
                        else
                        {
                                flag=0;
                        }
                        status=1;
			continue;
                }

                if(regexec(&reg2,line,2,pmatch,0)==0&&status==1) //the max TH
                {
                        take_regulate(pmatch,1,result,line);
                        TH=atof(result);
                        if(TH>tm_threshold)
                                status=0;
                        continue;
                }
                if(regexec(&reg3,line,1,pmatch,0)==0)
                {
                        status=0;
                        continue;                       
                }
                if(line[0]=='='&&status==1)
                {
                        if(flag==1)
                                plus=1;
                        else
                                minus=1;
                }
        }
        if(plus+minus!=0)
        {
                fprintf(fp2,"pos:%d\tlength:%d\t+:%d\t-:%d\n",pos,len,plus,minus);
                total++;
        }
        fclose(fp1);
        fclose(fp2);
	remove(in);
	free(out);
	free(in);
	free(line);
	regfree(&reg1);
	regfree(&reg2);
	regfree(&reg3);
        return total;
}
void usage()
{
        printf("Usage:\n");
        printf("    single  -in <fasta_file>  -out <primers_file_name>  -high[-low] [options]*\n\n");
        printf("    -in   <string>:  the reference sequence file, fasta formate\n");
        printf("    -out  <string>:  the prefix of output files, those files store candidate single primers\n");
        printf("    -dir  <string>:  the directory to store candidate single primers. default is current directory\n");
        printf("    -stab <string>:  the parameter file used in calculating the primers' stability. default is stab_parameter.txt in Par/ directory\n");
        printf("    -tm   <string>:  the parameter file used in calcalating Tm and second structure. default is stab_parameter.txt in Par/ directory\n");
        printf("    -P    <string>:  the primer3_core program in Primer3 software. If add this parameter, the second structure would be checked using primer3\n");
        printf("    -high/-low:      design candidate single primers in high/low GC region. high: the GC content>=45%%; low: the GC content <=45%%.\n");
        printf("    -loop:           design candidate loop single primers\n");
        printf("    -h/-help:        print usage\n");
}

void cpu_generate(char *seq,char out[],int pos,int length)
{
        int i;
        for(i=0;i<length;i++)
        {
                out[i]=seq[pos+i];
        }
        out[i]='\0';
}

int create_file(char *prefix,char *dir,char *seq,int *pos,int *len,int *rev_len,int length,int P_flag,char *primer3_config)
{
	char primer[26],rev[26],*file;
	int total,i,j;
	FILE *OUT;

	total=0;
	i=strlen(dir)+strlen(prefix)+20;
	file=(char *)malloc(i);
        memset(file,'\0',i);
        strcpy(file,dir);
        strcat(file,prefix);
	if(P_flag==1)
		strcat(file,"-primer3.txt");
        OUT=fopen(file,"w");
        if(OUT==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
	
	if(P_flag)
	{
		fprintf(OUT,"PRIMER_TASK=check_primers\nPRIMER_PICK_ANYWAY=1\nPRIMER_SALT_DIVALENT=4\nPRIMER_DNTP_CONC=1.4\nPRIMER_DNA_CONC=38\n");
		fprintf(OUT,"PRIMER_THERMODYNAMIC_PARAMETERS_PATH=%s\n",primer3_config);
	}
        for(i=0;i<length;i++)
        {
                if(pos[i]==0)
                        continue;
                for(j=0;j<8;j++)
                {
                        if((len[8*i+j]+rev_len[8*i+j])==0)
                                continue;
			if(P_flag==0)
                        	fprintf(OUT,"pos:%d\tlength:%d\t+:%d\t-:%d\n",i,(j+18),len[8*i+j],rev_len[8*i+j]);
			else
			{
				cpu_generate(seq,primer,i,(j+18));
				if(len[8*i+j]==1)
                		{
                        		fprintf(OUT,"SEQUENCE_ID=%d-%d-+\n",i,(j+18));
                		        fprintf(OUT,"SEQUENCE_PRIMER=%s\n=\n",primer);
                		}
                		if(rev_len[8*i+j]==1)
                		{
                        		cpu_reverse(primer,rev,(j+18));
                        		fprintf(OUT,"SEQUENCE_ID=%d-%d--\n",i,(j+18));
		                        fprintf(OUT,"SEQUENCE_PRIMER=%s\n=\n",rev);
        		        }
			}
			total++;
                }
        }
	fclose(OUT);
	free(file);
	return total;
}

main(int argc, char **argv)
{
	int *pos,*d_pos,*len,*d_len,length,flag[9],i,*rev_len,*d_rev_len,num_outer,num_inner,num_loop;
	float deltah[16],deltas[16],stab[4096],*d_deltah,*d_deltas,*d_stab,temp1,temp2;
	char *seq,*d_seq,*store_path,*prefix,*stab_path,*tm_path,*curren_path,*input,*outer,*inner,*loop,*primer3,*primer3_config,*temp;
	FILE *fp;
	time_t start,end;
        struct stat statbuf;
//flag: 0:input; 1: out_prefix; 2: dir; 3: stab; 4: tm; 5: high; 6: low; 7: loop; 8: primer3

	start=time(NULL);
//get input
        for(i=0;i<9;i++)
        {
                flag[i]=0;
        }
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			input=(char *)malloc(length+1);
			memset(input,'\0',length+1);
                        strcpy(input,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        prefix=(char *)malloc(length+1);
                        memset(prefix,'\0',length+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			if(argv[i+1][length-1]=='/')
			{
                        	store_path=(char *)malloc(length+1);
                        	memset(store_path,'\0',length+1);
                        	strcpy(store_path,argv[i+1]);
			}
			else
			{
				store_path=(char *)malloc(length+2);
				memset(store_path,'\0',length+2);
				strcpy(store_path,argv[i+1]);
				store_path[length]='/';
			}
                        i=i+2;
                }
                else if(strcmp(argv[i],"-stab")==0)
                {
                        flag[3]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-stab\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        stab_path=(char *)malloc(length+1);
                        memset(stab_path,'\0',length+1);
                        strcpy(stab_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-tm")==0)
                {
                        flag[4]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-tm\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        tm_path=(char *)malloc(length+1);
                        memset(tm_path,'\0',length+1);
                        strcpy(tm_path,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-high")==0)
                {
                        flag[5]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-low")==0)
                {
                        flag[6]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[7]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
                else if(strcmp(argv[i],"-P")==0)
                {
                        flag[8]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-P\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        primer3=(char *)malloc(length+1);
                        memset(primer3,'\0',length+1);
                        strcpy(primer3,argv[i+1]);
			if(access(primer3,0)==-1)
                        {
                                printf("Error! Don't have the %s program!\n",primer3);
                                exit(1);
                        }
                        i=i+2;
                }
                else
                {
                        printf("Error: don't have the parameter: %s\n",argv[i]);
                        usage();
                        exit(1);
                }
        }
//check paramters
        if(flag[5]+flag[6]!=1)
        {
                printf("Error! The input parameter must contain one of -high and -low!\n");
                usage();
                exit(1);
        }
        if(flag[0]==0)
        {
                printf("Error! Users must input the reference sequence file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the prefix name for output file with -out!\n");
                usage();
                exit(1);
        }
        for(i=0;i<strlen(prefix);i++)
        {
                if(prefix[i]=='/')
                {
                        printf("Error! the -out parameter couldn't contain any directory!\n");
                        usage();
                        exit(1);
                }
        }
//prepare
	inner=(char *)malloc(4096);
        memset(inner,'\0',4096);
        getcwd(inner,4096);
        length=strlen(inner);
        curren_path=(char *)malloc(length+1);
        memset(curren_path,'\0',length+1);
        strcpy(curren_path,inner);
        if(flag[2]==0)
        {
                store_path=(char *)malloc(length+2);
                memset(store_path,'\0',length+2);
                strcpy(store_path,curren_path);
                store_path[length]='/';
        }
        free(inner);

        length=strlen(store_path)+12;
        outer=(char *)malloc(length);
        memset(outer,'\0',length);
        strcpy(outer,store_path);

        inner=(char *)malloc(length);
        memset(inner,'\0',length);
        strcpy(inner,store_path);

        if(flag[7]==1)
        {
                loop=(char *)malloc(length);
                memset(loop,'\0',length);
                strcpy(loop,store_path);
        }
        if(flag[5]==1)
        {
                strcat(outer,"high-outer/");
                strcat(inner,"high-inner/");
                if(flag[7]==1)
                        strcat(loop,"high-loop/");
        }
        else          
        {                
                strcat(outer,"low-outer/");
                strcat(inner,"low-inner/");
                if(flag[7]==1)
                        strcat(loop,"low-loop/");
        }
        mkdir(outer,0755);
        mkdir(inner,0755);        
        if(flag[7]==1)
                mkdir(loop,0755);        

//stability parameter file
        if(flag[3]==0)
        {
		length=strlen(curren_path);
                stab_path=(char *)malloc(length+30);
                memset(stab_path,'\0',length+30);
                strcpy(stab_path,curren_path);
                i=length-1;
                while(stab_path[i]!='/'&&i>=0)
                {
                        stab_path[i]='\0';
                        i--;
                }
                strcat(stab_path,"Par/stab_parameter.txt");
        }
//tm parameter file
        if(flag[4]==0)
        {
		length=strlen(curren_path);
                tm_path=(char *)malloc(length+30);
                memset(tm_path,'\0',length+30);
                strcpy(tm_path,curren_path);
                i=length-1;
                while(tm_path[i]!='/'&&i>=0)
                {
                        tm_path[i]='\0';
                        i--;
                }
                strcat(tm_path,"Par/tm_nn_parameter.txt");
        }
//primer3 program
	if(flag[8]==1)
	{
		length=strlen(primer3);
                primer3_config=(char *)malloc(length+20);
                memset(primer3_config,'\0',length+20);
                strcpy(primer3_config,primer3);
                i=length-1;
        	while(primer3_config[i]!='/'&&i>=0)
        	{
        	        primer3_config[i]='\0';
        	        i--;
        	}
        	strcat(primer3_config,"primer3_config/");
	}

//input reference sequence
        if(access(input,0)==-1)
        {
                printf("Error! Don't have the %s file.\n",input);
                exit(1);
        }
        stat(input,&statbuf);
        length=statbuf.st_size;
        length=length+100;
        temp=(char *)malloc(length);
        memset(temp,'\0',length);
        seq=(char *)malloc(length*sizeof(char));
        memset(seq,'\0',length*sizeof(char));

        fp=fopen(input,"r");   //open the sequence file
        if(fp==NULL)
        {
                printf("Error! can't open the %s file!\n",input);
                exit(1);
        }
        fread(temp,length*sizeof(char),1,fp);
        fclose(fp); 

        length=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
                seq[length]=temp[i];
                i++;
                length++;
        }
        free(temp);
        length=strlen(seq);

//input Tm parameter
        fp=fopen(tm_path,"r");  //read the paramter of deltah and deltas
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",tm_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f\t%f",&i,&temp1,&temp2)!=EOF)
        {
                deltah[i]=temp1;
                deltas[i]=temp2;
        }
        fclose(fp);

//input stability parameter
        fp=fopen(stab_path,"r");  //read the parameters of stability
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",stab_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f",&i,&temp1)!=EOF)
        {
                stab[i]=temp1;
        }
        fclose(fp);

	hipMalloc((void **)&d_seq,length*sizeof(char));
	hipMemset(d_seq,'\0',length*sizeof(char));

	hipMalloc((void **)&d_deltah,16*sizeof(float));
	hipMemset(d_deltah,'\0',16*sizeof(float));
	hipMalloc((void **)&d_deltas,16*sizeof(float));
	hipMemset(d_deltas,'\0',16*sizeof(float));
	hipMalloc((void **)&d_stab,4096*sizeof(float));
	hipMemset(d_stab,'\0',4096*sizeof(float));

	/////from cpu to gpu
	hipMemcpy(d_seq,seq,length*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_deltah,deltah,16*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_deltas,deltas,16*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_stab,stab,4096*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void **)&d_pos,length*sizeof(int));
	hipMemset(d_pos,'\0',length*sizeof(int));
	hipMalloc((void **)&d_len,8*length*sizeof(int));
	hipMemset(d_len,'\0',8*length*sizeof(int));
	hipMalloc((void **)&d_rev_len,8*length*sizeof(int));
        hipMemset(d_rev_len,'\0',8*length*sizeof(int));
	pos=(int *)malloc(length*sizeof(int));
	memset(pos,'\0',length*sizeof(int));
	len=(int *)malloc(8*length*sizeof(int));
	memset(len,'\0',8*length*sizeof(int));
        rev_len=(int *)malloc(8*length*sizeof(int));
        memset(rev_len,'\0',8*length*sizeof(int));

	end=time(NULL);
	printf("It takes %d seconds to prepare.\n",(int)difftime(end,start));
	start=time(NULL);

	if(flag[5]==1)
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
		hipMemset(d_len,'\0',8*length*sizeof(int));
		hipMemset(d_rev_len,'\0',8*length*sizeof(int));
		candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,61,59,length);
		hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
        	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length,flag[8],primer3_config);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,0,66,64,length);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length,flag[8],primer3_config);

                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,66,64,length);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length,flag[8],primer3_config);
		}
        }
        else
        {
		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,56,54,length);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_outer=create_file(prefix,outer,seq,pos,len,rev_len,length,flag[8],primer3_config);

		hipMemset(d_pos,'\0',length*sizeof(int));
                hipMemset(d_len,'\0',8*length*sizeof(int));
                hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,0,61,59,length);
                hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                num_inner=create_file(prefix,inner,seq,pos,len,rev_len,length,flag[8],primer3_config);
                if(flag[7]==1)
		{
			hipMemset(d_pos,'\0',length*sizeof(int));
                	hipMemset(d_len,'\0',8*length*sizeof(int));
                	hipMemset(d_rev_len,'\0',8*length*sizeof(int));
                	candidate_primer<<<200,200>>>(d_seq,d_pos,d_len,d_rev_len,d_stab,d_deltah,d_deltas,1,61,59,length);
                	hipMemcpy(pos,d_pos,length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(len,d_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	hipMemcpy(rev_len,d_rev_len,8*length*sizeof(int),hipMemcpyDeviceToHost);
                	num_loop=create_file(prefix,loop,seq,pos,len,rev_len,length,flag[8],primer3_config);
		}
        }
	hipFree(d_pos);
	hipFree(d_len);
	hipFree(d_rev_len);
	hipFree(d_seq);
	hipFree(d_stab);
	hipFree(d_deltah);
	hipFree(d_deltas);
	free(pos);
        free(len);
        free(rev_len);
	free(seq);

	printf("There ara %d candidate primers used as F3/F2/B2/B3.\n",num_outer);
        printf("There are %d candidate primers used as F1c/B1c.\n",num_inner);
        if(flag[7]==1)
                printf("There are %d candidate primers used as LF/LB.\n",num_loop);
        //check
        if(num_outer<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(num_inner<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && num_loop<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
        printf("It takes %d seconds to design candidate single primers(without second structure check).\n",(int)difftime(end,start));
        start=time(NULL);

        if(flag[8]==0)
        {
		free(store_path);
                free(prefix);
                free(stab_path);
                free(tm_path);
                free(curren_path);
                free(input);
                free(outer);
                free(inner);
                if(flag[7])
                        free(loop);
                exit(1);
        }
//check the second structure
        if(flag[5]==1)
        {
		num_outer=second_structure(prefix,outer,49.0,primer3);
                num_inner=second_structure(prefix,inner,54.0,primer3);
                if(flag[7]==1)
                        num_loop=second_structure(prefix,loop,54.0,primer3);
        }
        else
        {
                num_outer=second_structure(prefix,outer,44.0,primer3);
                num_inner=second_structure(prefix,inner,49.0,primer3);
                if(flag[7]==1)
                        num_loop=second_structure(prefix,loop,49.0,primer3);
        }

        //check
        printf("After second structure check, there ara %d candidate primers used as F3/F2/B2/B3.\n",num_outer);
        printf("After second structure check, there are %d candidate primers used as F1c/B1c.\n",num_inner);
        if(flag[7]==1)
                printf("After second structure check, there are %d candidate primers used as LF/LB.\n",num_loop);
        //check
        if(num_outer<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(num_inner<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && num_loop<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
	printf("It takes %d seconds to check single primers' second structure.\n",(int)difftime(end,start));

	free(store_path);
        free(prefix);
        free(stab_path);
        free(tm_path);
        free(curren_path);
        free(input);
        free(outer);
        free(inner);
        if(flag[7])
                free(loop);
	free(primer3);
	free(primer3_config);
}
